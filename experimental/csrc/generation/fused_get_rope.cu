#include "hip/hip_runtime.h"
// Copyright (c) 2023 PaddlePaddle Authors. All Rights Reserved.
// 
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
// 
//     http://www.apache.org/licenses/LICENSE-2.0
// 
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "helper.h"

/*
Position_ids: bsz, max_seq_length 
*/

template<typename T, int N>
struct GetPackType {
  using type = typename std::aligned_storage<N * sizeof(T), N * sizeof(T)>::type;
};

template<typename T, int N>
using PackType = typename GetPackType<T, N>::type;

template<typename T, int N>
union Pack {
  static_assert(sizeof(PackType<T, N>) == sizeof(T) * N, "");
  __device__ Pack() {
    // do nothing
  }
  PackType<T, N> storage;
  T elem[N];
};

__global__ __launch_bounds__(kBlockSize) void fused_get_rotary_embedding_neox(const int64_t* position_ids,
                                                                              const int32_t bsz,
                                                                              const int32_t max_seq_length,
                                                                              const int32_t max_position_seq_length,
                                                                              const int32_t head_dim,
                                                                              const int32_t prompt_num,
                                                                              const float inv_head_dim,
                                                                              const int32_t elem_cnt,
                                                                              float* rope_embedding) {
    /*
    In Naive implementation, it will stacks [freqs, freqs]
    And actually, each threads can process 1 values, and store continuous 2 same values.
    So here We construct a Pack to store 2 values.
    */
    constexpr int PackSize = 2;
    // Pack<float, PackSize> SinStorePack{};
    // Pack<float, PackSize> CosStorePack{};

    const int half_head_dim = head_dim / PackSize;
    const int32_t global_thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
    for(int idx = global_thread_idx, step=blockDim.x * gridDim.x; idx < elem_cnt; idx += step){
        const int32_t bsz_seq_idx = idx / half_head_dim;
        const int32_t bsz_idx =  bsz_seq_idx / max_seq_length;
        const int32_t seq_idx = bsz_seq_idx % max_seq_length;
        const int64_t position_offset = bsz_idx * max_position_seq_length + seq_idx + prompt_num;
        const int32_t half_head_idx = (idx % half_head_dim) * PackSize;
        const float exponent_factor = -static_cast<float>(half_head_idx) * inv_head_dim; // * inv_head_dim equals to / head_dim.
        const float inv_freq_val = powf(10000.0f, exponent_factor);
        const float freqs_val = static_cast<float>(position_ids[position_offset]) * inv_freq_val;
        const float cos_embedding_val = cos(freqs_val);
        const float sin_embedding_val = sin(freqs_val);

        const int32_t cos_offset = bsz_seq_idx * head_dim + half_head_idx / PackSize;
        rope_embedding[cos_offset] = cos_embedding_val;
        rope_embedding[cos_offset + half_head_dim] = cos_embedding_val;
        const int32_t sin_offset = bsz * max_seq_length * head_dim + cos_offset;
        rope_embedding[sin_offset] = sin_embedding_val;
        rope_embedding[sin_offset + half_head_dim] = sin_embedding_val;

        // /*
        // Since After stack, the continuous 2 elements value is same.
        // So here each threads store 2 computed embedding value.
        // */
        // #pragma unroll
        // for(int unroll_idx = 0; unroll_idx < PackSize; unroll_idx++){
        //     CosStorePack.elem[unroll_idx] = cos_embedding_val;
        //     SinStorePack.elem[unroll_idx] = sin_embedding_val;
        // }
        //
        // const int32_t cos_offset = bsz_seq_idx * head_dim + half_head_idx;
        // const int32_t sin_offset = bsz * max_seq_length * head_dim + cos_offset;
        // *(reinterpret_cast<PackType<float, PackSize>*>(rope_embedding + cos_offset)) = CosStorePack.storage;
        // *(reinterpret_cast<PackType<float, PackSize>*>(rope_embedding + sin_offset)) = SinStorePack.storage;
    }
}

__global__ __launch_bounds__(kBlockSize) void fused_get_rotary_embedding(const int64_t* position_ids, 
                                                                         const int32_t bsz, 
                                                                         const int32_t max_seq_length, 
                                                                         const int32_t max_position_seq_length,
                                                                         const int32_t head_dim, 
                                                                         const int32_t prompt_num,
                                                                         const float inv_head_dim, 
                                                                         const int32_t elem_cnt, 
                                                                         float* rope_embedding) {
    /*
    In Naive implementation, it will stacks [freqs, freqs]
    And actually, each threads can process 1 values, and store continuous 2 same values. 
    So here We construct a Pack to store 2 values. 
    */
    constexpr int PackSize = 2; 
    Pack<float, PackSize> SinStorePack{}; 
    Pack<float, PackSize> CosStorePack{}; 

    const int half_head_dim = head_dim / PackSize; 
    const int32_t global_thread_idx = blockIdx.x * blockDim.x + threadIdx.x; 
    for(int idx = global_thread_idx, step=blockDim.x * gridDim.x; idx < elem_cnt; idx += step){
        const int32_t bsz_seq_idx = idx / half_head_dim;
        const int32_t bsz_idx =  bsz_seq_idx / max_seq_length;
        const int32_t seq_idx = bsz_seq_idx % max_seq_length;
        const int64_t position_offset = bsz_idx * max_position_seq_length + seq_idx + prompt_num;
        const int32_t half_head_idx = (idx % half_head_dim) * PackSize; 
        const float exponent_factor = -static_cast<float>(half_head_idx) * inv_head_dim; // * inv_head_dim equals to / head_dim. 
        const float inv_freq_val = powf(10000.0f, exponent_factor); 
        const float freqs_val = static_cast<float>(position_ids[position_offset]) * inv_freq_val; 
        const float cos_embedding_val = cos(freqs_val); 
        const float sin_embedding_val = sin(freqs_val); 

        /*
        Since After stack, the continuous 2 elements value is same. 
        So here each threads store 2 computed embedding value. 
        */
        #pragma unroll 
        for(int unroll_idx = 0; unroll_idx < PackSize; unroll_idx++){
            CosStorePack.elem[unroll_idx] = cos_embedding_val; 
            SinStorePack.elem[unroll_idx] = sin_embedding_val; 
        }

        const int32_t cos_offset = bsz_seq_idx * head_dim + half_head_idx; 
        const int32_t sin_offset = bsz * max_seq_length * head_dim + cos_offset; 
        *(reinterpret_cast<PackType<float, PackSize>*>(rope_embedding + cos_offset)) = CosStorePack.storage;
        *(reinterpret_cast<PackType<float, PackSize>*>(rope_embedding + sin_offset)) = SinStorePack.storage;
    }
}

std::vector<paddle::Tensor> GetRoPE(const paddle::Tensor& input_ids, 
                                    const paddle::Tensor& position_ids, 
                                    const paddle::Tensor& head_dim_shape_tensor,
                                    int prompt_num,
                                    bool use_neox) {
    const int64_t batch_size = input_ids.shape()[0]; 
    const int64_t max_seq_length = input_ids.shape()[1]; 
    const int64_t max_position_seq_length = position_ids.shape()[1];
    const int64_t head_dim = head_dim_shape_tensor.shape()[0]; 
    const float inv_head_dim = 1.0f / static_cast<float>(head_dim); 

    auto cu_stream = position_ids.stream();

    auto rotary_embedding = paddle::full({2, batch_size, 1, max_seq_length, head_dim}, -1, paddle::DataType::FLOAT32, position_ids.place());

    assert(head_dim % 2 == 0); 
    const int32_t elem_cnt = batch_size * max_seq_length * head_dim / 2; 
    int32_t grid_size = 1; 
    GetNumBlocks(elem_cnt, &grid_size); 
    if (use_neox) {
      fused_get_rotary_embedding_neox<<<grid_size, kBlockSize, 0, cu_stream>>> (
          position_ids.data<int64_t>(),
          batch_size,
          max_seq_length,
          max_position_seq_length,
          head_dim,
          prompt_num,
          inv_head_dim,
          elem_cnt,
          reinterpret_cast<float*>(rotary_embedding.data<float>()));
    } else {
      fused_get_rotary_embedding<<<grid_size, kBlockSize, 0, cu_stream>>> (
          position_ids.data<int64_t>(),
          batch_size, 
          max_seq_length, 
          max_position_seq_length,
          head_dim, 
          prompt_num,
          inv_head_dim, 
          elem_cnt, 
          reinterpret_cast<float*>(rotary_embedding.data<float>()));
    }
    return {rotary_embedding};
}



std::vector<std::vector<int64_t>> GetRoPEInferShape(const std::vector<int64_t>& input_ids_shape, 
                                                    const std::vector<int64_t>& position_ids_shape, 
                                                    const std::vector<int64_t>& head_dim_shape_tensor_shape) {
    const int64_t batch_size = position_ids_shape[0]; 
    const int64_t max_seq_length = input_ids_shape[1]; 
    const int64_t head_dim = head_dim_shape_tensor_shape[0]; 
    std::vector<int64_t> out_shape = {2, batch_size, 1, max_seq_length, head_dim};                                                          
    return {out_shape};
}

std::vector<paddle::DataType> GetRoPEInferDtype(const paddle::DataType& input_ids_dtype, 
                                                const paddle::DataType& position_ids_dtype, 
                                                const paddle::DataType& head_dim_shape_tensor_dtype) {
    // RoPE output dtype is Float. 
    return {paddle::DataType::FLOAT32};
}

PD_BUILD_OP(fused_get_rotary_embedding)
    .Inputs({"input_ids", "position_ids", "head_dim_shape_tensor"})
    .Outputs({"rotary_embedding"})
    .Attrs({"prompt_num: int",
            "use_neox: bool"})
    .SetKernelFn(PD_KERNEL(GetRoPE))
    .SetInferShapeFn(PD_INFER_SHAPE(GetRoPEInferShape))
    .SetInferDtypeFn(PD_INFER_DTYPE(GetRoPEInferDtype));