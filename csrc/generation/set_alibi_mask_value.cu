#include "hip/hip_runtime.h"
// Copyright (c) 2023 PaddlePaddle Authors. All Rights Reserved.
// 
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
// 
//     http://www.apache.org/licenses/LICENSE-2.0
// 
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "helper.h"

template <typename T>
__global__ void set_value_by_id(const int *seq_lens, 
                               const bool *stop_flags, 
                              const float *alibi_slopes, 
                              const int64_t *tgt_pos, 
                              T *output_data, 
                              int *sequence_lengths, 
                              int bs, 
                              int length,
                              int num_head) {
    int bs_id = blockIdx.x;                          
    int hid = threadIdx.x;
    if (bs_id < bs) {
        T *output_data_now = output_data + bs_id * num_head * length + hid * length;
        float tgt_pos_now = static_cast<float>(tgt_pos[bs_id]);
        output_data_now[seq_lens[bs_id]] = static_cast<T>(tgt_pos_now * alibi_slopes[hid]);
        if (stop_flags[bs_id]) {
            sequence_lengths[bs_id] = 0;
        }
    }
}

template <paddle::DataType D>
std::vector<paddle::Tensor> set_mask_value(const paddle::Tensor& input_data, 
                                           const paddle::Tensor& stop_flags, 
                                          const paddle::Tensor& seq_lens,
                                          const paddle::Tensor& alibi_slopes,
                                          const paddle::Tensor& tgt_pos
                                          ) {
    typedef PDTraits<D> traits_;
    typedef typename traits_::DataType DataType_;
    typedef typename traits_::data_t data_t;

    PD_CHECK(seq_lens.dtype() == paddle::DataType::INT32);
    PD_CHECK(stop_flags.dtype() == paddle::DataType::BOOL);
    auto cu_stream = input_data.stream();
    std::vector<int64_t> input_data_shape = input_data.shape();
    std::vector<int64_t> seq_lens_shape = seq_lens.shape();
    auto sequence_lengths = seq_lens.copy_to(seq_lens.place(), false);

    int input_bs = input_data_shape[0];
    int length = input_data_shape[3];
    int seq_bs = seq_lens_shape[0];
    int num_head = alibi_slopes.shape()[0];

    int grid_size = input_bs;
    int block_size = num_head;
    set_value_by_id<<<grid_size, block_size, 0, cu_stream>>>(seq_lens.data<int>(), 
                                                     stop_flags.data<bool>(), 
                                                     alibi_slopes.data<float>(),
                                                     tgt_pos.data<int64_t>(),
                                                     reinterpret_cast<DataType_*>(const_cast<data_t*>(input_data.data<data_t>())), 
                                                     sequence_lengths.data<int>(), seq_bs, length, num_head);
    return {sequence_lengths};
}

std::vector<paddle::Tensor> SetMaskValue(const paddle::Tensor& input_data, 
                                          const paddle::Tensor& stop_flags, 
                                          const paddle::Tensor& seq_lens,
                                          const paddle::Tensor& alibi_slopes,
                                          const paddle::Tensor& tgt_pos) {
    switch (input_data.type()) {
        case paddle::DataType::BFLOAT16: {
            return set_mask_value<paddle::DataType::BFLOAT16>(
                input_data,
                stop_flags,
                seq_lens,
                alibi_slopes,
                tgt_pos
            );
        }
        case paddle::DataType::FLOAT16: {
            return set_mask_value<paddle::DataType::FLOAT16>(
                input_data,
                stop_flags,
                seq_lens,
                alibi_slopes,
                tgt_pos
            );
        }
        case paddle::DataType::FLOAT32: {
            return set_mask_value<paddle::DataType::FLOAT32>(
                input_data,
                stop_flags,
                seq_lens,
                alibi_slopes,
                tgt_pos
            );
        }
        default: {
            PD_THROW(
                "NOT supported data type. "
                "Only float16, bfloat16 and float32 are supported. ");
            break;
        }
    }
}

std::vector<std::vector<int64_t>> SetMaskValueInferShape(const std::vector<int64_t>& input_data_shape, 
                                                         const std::vector<int64_t>& stop_flags_shape, 
                                                         const std::vector<int64_t>& seq_lens_shape,
                                                         const std::vector<int64_t>& alibi_slopes_shape,
                                                         const std::vector<int64_t>& tgt_pos) {
    return {seq_lens_shape};
}

std::vector<paddle::DataType> SetMaskValueInferDtype(const paddle::DataType& input_data_dtype, 
                                                      const paddle::DataType& stop_flags_dtype, 
                                                      const paddle::DataType& seq_lens_dtype,
                                                      const paddle::DataType& alibi_slopes_dtype,
                                                      const paddle::DataType& tgt_pos_dtype) {
    return {seq_lens_dtype};
}

PD_BUILD_OP(set_alibi_mask_value)
    .Inputs({"input_data", "stop_flags", "seq_lens", "alibi_slopes", "tgt_pos"})
    .Outputs({"sequence_lengths"})
    .SetKernelFn(PD_KERNEL(SetMaskValue))
    .SetInferShapeFn(PD_INFER_SHAPE(SetMaskValueInferShape))
    .SetInferDtypeFn(PD_INFER_DTYPE(SetMaskValueInferDtype));