#include "hip/hip_runtime.h"
// Copyright (c) 2023 PaddlePaddle Authors. All Rights Reserved.
// 
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
// 
//     http://www.apache.org/licenses/LICENSE-2.0
// 
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "helper.h"

__global__ void RemovePadding(int64_t *output_data,
                              const int64_t *input_data,
                              const int *seq_lens,
                              const int *cum_offsets,
                              const int sequence_length) {
  const int bi = blockIdx.x;
  const int tid = threadIdx.x;

  for (int i = tid; i < seq_lens[bi]; i += blockDim.x) {
    const int tgt_seq_id = bi * sequence_length - cum_offsets[bi] + i;
    const int src_seq_id = bi * sequence_length + i;
    output_data[tgt_seq_id] = input_data[src_seq_id];
  }
}

__global__ void GetCumOffsetKernel(int *token_num,
                                   int *enc_token_num,
                                   int *dec_token_num,
                                   int *cum_offsets,
                                   const int *sequence_lengths,
                                   const int *sequence_lengths_encoder,
                                   const int *sequence_lengths_decoder,
                                   const int batch_size,
                                   const int max_seq_len) {
  // get padding offset of each batch
  int total_seq_len = 0;
  int enc_total_seq_len = 0;
  int dec_total_seq_len = 0;
  int cum_offset = 0;
  int index = 0;
  
  for (int i = 0; i < batch_size; i++) {
    cum_offsets[i] = cum_offset;
    int seq_len = sequence_lengths[i];
    int seq_len_enc = sequence_lengths_encoder[i];
    int seq_len_dec = sequence_lengths_decoder[i];

    cum_offset += max_seq_len - seq_len;

    total_seq_len += seq_len;
    enc_total_seq_len += seq_len_enc;
    dec_total_seq_len += seq_len_dec;
  }
  token_num[0] = total_seq_len;
  enc_token_num[0] = enc_total_seq_len;
  dec_token_num[0] = dec_total_seq_len;
}

__global__ void GetPaddingOffsetKernel(int *padding_offset,
                                       int *cum_offsets_out,
                                       const int *cum_offsets,
                                       const int *seq_lens,
                                       const int max_seq_len) {
  // get padding offset of each batch
  const int bi = blockIdx.x;
  const int ti = threadIdx.x;
  if (ti == 0) {
    cum_offsets_out[bi] = bi == 0 ? 0 : cum_offsets[bi - 1];
  }
  int cum_offset = bi == 0 ? 0 : cum_offsets[bi - 1];
  for (int i = ti; i < seq_lens[bi]; i += blockDim.x) {
    padding_offset[bi * max_seq_len - cum_offset + i] = cum_offset;
  }
}


std::vector<paddle::Tensor> GetPaddingOffset(const paddle::Tensor& input_ids,
                                             const paddle::Tensor& cum_offsets,
                                             const paddle::Tensor& token_num,
                                             const paddle::Tensor& seq_len) {
    auto cu_stream = input_ids.stream();
    std::vector<int64_t> input_ids_shape = input_ids.shape();
    const int bsz = input_ids_shape[0];
    const int seq_length = input_ids_shape[1];
    auto cum_offsets_out = cum_offsets.copy_to(cum_offsets.place(), false);
    auto cpu_token_num = token_num.copy_to(paddle::CPUPlace(), false);
    const int token_num_data = cpu_token_num.data<int64_t>()[0];
    auto x_remove_padding = paddle::full({token_num_data}, 0, paddle::DataType::INT64, input_ids.place());
    auto padding_offset = paddle::full({token_num_data}, 0, paddle::DataType::INT32, input_ids.place());
    int blockSize = min((token_num_data + 32 - 1) / 32 * 32, 128);
    GetPaddingOffsetKernel<<<bsz, 128, 0, cu_stream>>>(
      padding_offset.data<int>(), 
      cum_offsets_out.data<int>(),
      cum_offsets.data<int>(),
      seq_len.data<int>(),
      seq_length);
    RemovePadding<<<bsz, blockSize, 0, cu_stream>>>(
      x_remove_padding.data<int64_t>(), 
      input_ids.data<int64_t>(), 
      seq_len.data<int>(),
      cum_offsets_out.data<int>(), 
      seq_length);
    return {x_remove_padding, cum_offsets_out, padding_offset}; // , enc_token_num, dec_token_num};
}

std::vector<std::vector<int64_t>> GetPaddingOffsetInferShape(const std::vector<int64_t>& input_ids_shape,
                                                             const std::vector<int64_t>& cum_offsets_shape,
                                                             const std::vector<int64_t>& token_num_shape,
                                                             const std::vector<int64_t>& seq_len_shape) {
    int64_t bsz = input_ids_shape[0];
    int64_t seq_len = input_ids_shape[1];
    return {{-1}, {bsz}, {-1}};
}

std::vector<paddle::DataType> GetPaddingOffsetInferDtype(const paddle::DataType& input_ids_dtype,
                                                         const paddle::DataType& cum_offsets_dtype,
                                                         const paddle::DataType& token_num_dtype,
                                                         const paddle::DataType& seq_len_dtype) {
    return {input_ids_dtype, seq_len_dtype, seq_len_dtype};
}

PD_BUILD_OP(get_padding_offset)
    .Inputs({"input_ids", "cum_offsets", "token_num", "seq_len"})
    .Outputs({"x_remove_padding", "cum_offsets_out", "padding_offset"})
    .SetKernelFn(PD_KERNEL(GetPaddingOffset))
    .SetInferShapeFn(PD_INFER_SHAPE(GetPaddingOffsetInferShape))
    .SetInferDtypeFn(PD_INFER_DTYPE(GetPaddingOffsetInferDtype));