#include "hip/hip_runtime.h"
// Copyright (c) 2023 PaddlePaddle Authors. All Rights Reserved.
// 
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
// 
//     http://www.apache.org/licenses/LICENSE-2.0
// 
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "helper.h"


template <typename T>
inline __device__ __host__ T div_up(T m, T n) {
  return (m + n - 1) / n;
}

template <typename T>
__global__ void write_cache_k_kernel(T *cache_k,
                                     const T *k,
                                     const int *seq_lens,
                                     const int num_head,
                                     const int dim_head,
                                     const int seq_len,
                                     const int max_seq_len) {
  const int bi = blockIdx.y;
  const int len = seq_lens ? seq_lens[bi] : seq_len;
  if (len == 0) {
    return;
  }

  const int hi = blockIdx.z;
  constexpr int X_ELEMS = VEC_16B / sizeof(T);

  // [bsz, num_head, seq_len, dim_head/x, x]
  auto k_src = reinterpret_cast<const uint4 *>(
      k + bi * num_head * seq_len * dim_head + hi * seq_len * dim_head);
  // [bsz, num_head, dim_head/x, max_seq_len, x]
  auto k_dst = reinterpret_cast<uint4 *>(
      cache_k + bi * num_head * max_seq_len * dim_head +
      hi * max_seq_len * dim_head);

  const int out_idx = blockIdx.x * blockDim.x + threadIdx.x;
  // vec size
  int dim_head_div_x = dim_head / X_ELEMS;

  // FIXME(wangxi): num_head is not need?
  // if (out_idx >= num_head * dim_head_div_x * max_seq_len) return;
  if (out_idx >= dim_head_div_x * max_seq_len) return;

  int idx = out_idx;
  const int k_seq_len_id = idx % max_seq_len;
  // idx = (idx - k_seq_len_id) / max_seq_len;
  idx = idx / max_seq_len;
  const int k_vec_id = idx % dim_head_div_x;

  if (k_seq_len_id < len) {
    k_dst[out_idx] = k_src[k_seq_len_id * dim_head_div_x + k_vec_id];
  }
}

template <typename T>
__global__ void write_cache_v_kernel(T *cache_v,
                                     const T *v,
                                     const int *seq_lens,
                                     const int num_head,
                                     const int dim_head,
                                     const int seq_len,
                                     const int max_seq_len) {
  const int bi = blockIdx.y;
  const int len = seq_lens ? seq_lens[bi] : seq_len;
  if (len == 0) {
    return;
  }

  const int hi = blockIdx.z;

  // [bsz, num_head, seq_len, dim_head/x, x]
  auto v_src = reinterpret_cast<const uint4 *>(
      v + bi * num_head * seq_len * dim_head + hi * seq_len * dim_head);
  // [bsz, num_head, max_seq_len, dim_head/x, x]
  auto v_dst = reinterpret_cast<uint4 *>(
      cache_v + bi * num_head * max_seq_len * dim_head +
      hi * max_seq_len * dim_head);

  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  constexpr int X_ELEMS = VEC_16B / sizeof(T);
  const int dim_head_div_x = dim_head / X_ELEMS;

  if (idx >= dim_head_div_x * len) return;

  v_dst[idx] = v_src[idx];
}

template <paddle::DataType D>
void LaunchWriteCacheKV(const paddle::Tensor& input_k, 
                        const paddle::Tensor& input_v, 
                        const paddle::Tensor& cache_kv,
                        const paddle::Tensor& sequence_lengths) {
    typedef PDTraits<D> traits_;
    typedef typename traits_::DataType DataType_;
    typedef typename traits_::data_t data_t;

    const int64_t bsz = input_k.shape()[0];
    const int64_t seq_len = input_k.shape()[2]; 
    const int64_t cache_bsz = cache_kv.shape()[1]; 
    const int64_t num_head = cache_kv.shape()[2]; 
    const int64_t dim_head = cache_kv.shape()[4]; 
    // printf("bsz: %d, cache_bsz: %d, num_head: %d, seq_len: %d, dim_head: %d.\n", bsz, cache_bsz, num_head, seq_len, dim_head);

    auto cache_kv_out = paddle::full({1}, -1, cache_kv.dtype(), cache_kv.place());

    const DataType_ *k_ptr = reinterpret_cast<const DataType_*>(input_k.data<data_t>());
    const DataType_ *v_ptr = reinterpret_cast<const DataType_*>(input_v.data<data_t>());

    // [2, bsz, num_head, max_seq_len, head_dim]
    int max_seq_len = cache_kv.shape()[3];
    DataType_ *cache_kv_data = reinterpret_cast<DataType_*>(const_cast<data_t*>(cache_kv.data<data_t>()));

    int64_t cache_k_size = cache_bsz * num_head * max_seq_len * dim_head;

    DataType_ *cache_k_ptr = cache_kv_data;
    DataType_ *cache_v_ptr = cache_kv_data + cache_k_size;

    constexpr int block_sz = 128;
    constexpr int x = VEC_16B / sizeof(DataType_);

    assert(dim_head % x == 0);
    // PD_CHECK((dim_head % x) == 0, "PD_CHECK returns ", false, ", dim_head must be divisible by vec_size.");

    int max_size = max_seq_len * dim_head / x;
    int size = seq_len * dim_head / x;
    dim3 grid(div_up(max_size, block_sz), bsz, num_head);
    dim3 grid_v(div_up(size, block_sz), bsz, num_head);

    // transpose [bsz, num_head, seq_len, dim_head/x, x]->
    // [bsz, num_head, dim_head/x, max_seq_len, x]
    write_cache_k_kernel<<<grid, block_sz, 0, input_k.stream()>>>(
        cache_k_ptr, k_ptr, sequence_lengths.data<int>(), num_head, dim_head, seq_len, max_seq_len);

    // copy [bsz, num_head, seq_len, dim_head/x, x]->
    // [bsz, num_head, max_seq_len, dim_head/x, x]
    write_cache_v_kernel<<<grid_v, block_sz, 0, input_k.stream()>>>(
        cache_v_ptr, v_ptr, sequence_lengths.data<int>(), num_head, dim_head, seq_len, max_seq_len);
}

void WriteCacheKV(const paddle::Tensor& input_k,
                  const paddle::Tensor& input_v,
                  const paddle::Tensor& cache_kv,
                  const paddle::Tensor& sequence_lengths_shape) {
    switch (cache_kv.type()) {
        case paddle::DataType::BFLOAT16: {
            return LaunchWriteCacheKV<paddle::DataType::BFLOAT16>(
                input_k, input_v, cache_kv, sequence_lengths_shape
            );
        }
        case paddle::DataType::FLOAT16: {
            return LaunchWriteCacheKV<paddle::DataType::FLOAT16>(
                input_k, input_v, cache_kv, sequence_lengths_shape
            );
        }
        case paddle::DataType::FLOAT32: {
            return LaunchWriteCacheKV<paddle::DataType::FLOAT32>(
                input_k, input_v, cache_kv, sequence_lengths_shape
            );
        }
        default: {
            PD_THROW(
                "NOT supported data type. "
                "Only bfloat16, float16 and float32 are supported. ");
            break;
        }
    }
}

PD_BUILD_OP(write_cache_kv)
    .Inputs({"input_k", "input_v", "cache_kv", "sequence_lengths"})
    .Outputs({"cache_kv_out"})
    .SetInplaceMap({{"cache_kv", "cache_kv_out"}})
    .SetKernelFn(PD_KERNEL(WriteCacheKV));