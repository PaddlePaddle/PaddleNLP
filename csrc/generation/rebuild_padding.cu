#include "hip/hip_runtime.h"
// Copyright (c) 2023 PaddlePaddle Authors. All Rights Reserved.
// 
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
// 
//     http://www.apache.org/licenses/LICENSE-2.0
// 
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "helper.h"

template <typename T, int VecSize>
__global__ void RebuildPaddingKernel(T *output_data,
                                     const T *input_data,
                                     const int *cum_offsets,
                                     const int *seq_lens,
                                     const int max_seq_len,
                                     const int dim_embed,
                                     const int elem_nums) {
  using LoadT = AlignedVector<T, VecSize>;
  LoadT src_vec;
  const int global_idx = blockDim.x * blockIdx.x + threadIdx.x;
  for (int i = global_idx * VecSize; i < elem_nums; i += gridDim.x * blockDim.x * VecSize) {
    const int bi = i / dim_embed;
    const int bias_idx = i % dim_embed;
    int seq_id = seq_lens[bi] - 1;
    const int ori_token_idx = bi * max_seq_len - cum_offsets[bi] + seq_id;
    const int src_offset = ori_token_idx * dim_embed + bias_idx;
    Load<T, VecSize>(&input_data[src_offset], &src_vec);
    Store<T, VecSize>(src_vec, &output_data[i]);
  }
}

template <typename T>
__global__ void RebuildPaddingKernel(T *output_data,
                                    const T *input_data,
                                    const int *padding_offset,
                                    const int dim_embed) {
  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  const int dst_seq_id = bid + padding_offset[bid];
  const int src_seq_id = bid;

  for (int i = tid; i < dim_embed; i += blockDim.x) {
    output_data[dst_seq_id * dim_embed + i] =
        input_data[src_seq_id * dim_embed + i];
  }
}

template <typename T>
void InvokeRebuildPadding(T *output_data,
                          const T *input_data,
                          const int *padding_offset,
                          const int token_num,
                          const int dim_embed,
                          hipStream_t stream) {
  // src: [token_num, dim_embed]
  // dst: [batch_size * max_seq_len, dim_embed]
  RebuildPaddingKernel<<<token_num, 256, 0, stream>>>(
      output_data, input_data, padding_offset, dim_embed);
}

template <paddle::DataType D>
std::vector<paddle::Tensor> rebuild_padding(const paddle::Tensor& tmp_out, // [token_num, dim_embed]
                                            const paddle::Tensor& padding_offset, // [bsz, 1]
                                            const paddle::Tensor& seq_lens,
                                            const paddle::Tensor& input_ids) {
    typedef PDTraits<D> traits_;
    typedef typename traits_::DataType DataType_;
    typedef typename traits_::data_t data_t;

    auto cu_stream = tmp_out.stream();
    std::vector<int64_t> tmp_out_shape = tmp_out.shape();
    const int token_num = tmp_out_shape[0];
    const int dim_embed = tmp_out_shape[1];
    const int bsz = seq_lens.shape()[0];
    auto out = paddle::full({bsz, dim_embed}, 0, tmp_out.dtype(), tmp_out.place());
    constexpr int PackSize = VEC_16B / sizeof(DataType_);
    int elem_nums = out.numel();
    int pack_num = elem_nums / PackSize;
    const int blocksize = 128;
    const int grid_size = (pack_num + blocksize - 1) / blocksize;
    RebuildPaddingKernel<DataType_, PackSize><<<grid_size, blocksize, 0, tmp_out.stream()>>>(
        reinterpret_cast<DataType_*>(out.data<data_t>()), 
        reinterpret_cast<DataType_*>(const_cast<data_t*>(tmp_out.data<data_t>())), 
        padding_offset.data<int>(), 
        seq_lens.data<int>(), 
        input_ids.shape()[1], 
        dim_embed, 
        elem_nums);
    // InvokeRebuildPadding(
    //     reinterpret_cast<DataType_*>(out.data<data_t>()), 
    //     reinterpret_cast<DataType_*>(const_cast<data_t*>(tmp_out.data<data_t>())), 
    //     padding_offset.data<int>(),
    //     token_num,
    //     dim_embed,
    //     tmp_out.stream()
    // );
    return {out};
}

std::vector<paddle::Tensor> RebuildPadding(const paddle::Tensor& tmp_out, 
                                           const paddle::Tensor& padding_offset, 
                                           const paddle::Tensor& seq_lens,
                                           const paddle::Tensor& input_ids) {
    switch (tmp_out.type()) {
        case paddle::DataType::BFLOAT16: {
            return rebuild_padding<paddle::DataType::BFLOAT16>(
                tmp_out,
                padding_offset,
                seq_lens,
                input_ids
            );
        }
        case paddle::DataType::FLOAT16: {
            return rebuild_padding<paddle::DataType::FLOAT16>(
                tmp_out,
                padding_offset,
                seq_lens,
                input_ids
            );
        }
        case paddle::DataType::FLOAT32: {
            return rebuild_padding<paddle::DataType::FLOAT32>(
                tmp_out,
                padding_offset,
                seq_lens,
                input_ids
            );
        }
        default: {
            PD_THROW(
                "NOT supported data type. "
                "Only float16, bfloat16 and float32 are supported. ");
            break;
        }
    }
}

std::vector<std::vector<int64_t>> RebuildPaddingInferShape(const std::vector<int64_t>& tmp_out_shape,
                                                           const std::vector<int64_t>& padding_offset_shape,
                                                           const std::vector<int64_t>& seq_lens_shape,
                                                           const std::vector<int64_t>& input_ids_shape) {
    int64_t bsz = seq_lens_shape[0];
    int64_t dim_embed = tmp_out_shape[1];
    return {{bsz, dim_embed}};
}

std::vector<paddle::DataType> RebuildPaddingInferDtype(const paddle::DataType& tmp_out_dtype,
                                                       const paddle::DataType& padding_offset_dtype,
                                                       const paddle::DataType& seq_lens_dtype,
                                                       const paddle::DataType& input_ids_dtype) {
    return {tmp_out_dtype};
}

PD_BUILD_OP(rebuild_padding)
    .Inputs({"tmp_out", "padding_offset", "seq_lens", "input_ids"})
    .Outputs({"out"})
    .SetKernelFn(PD_KERNEL(RebuildPadding))
    .SetInferShapeFn(PD_INFER_SHAPE(RebuildPaddingInferShape))
    .SetInferDtypeFn(PD_INFER_DTYPE(RebuildPaddingInferDtype));