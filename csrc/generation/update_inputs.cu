#include "hip/hip_runtime.h"
// Copyright (c) 2024 PaddlePaddle Authors. All Rights Reserved.
// 
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
// 
//     http://www.apache.org/licenses/LICENSE-2.0
// 
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "helper.h"

template <int THREADBLOCK_SIZE>
__global__ void update_inputs_kernel(bool *not_need_stop,
                                     int *seq_lens_this_time,
                                     int *seq_lens_encoder,
                                     int *seq_lens_decoder,
                                     int64_t *input_ids,
                                     const int64_t *stop_nums,
                                     const bool *stop_flags,
                                     const bool *is_block_step,
                                     const int64_t *next_tokens,
                                     const int bsz,
                                     const int max_bsz,
                                     const int input_ids_stride) {
  int thread_idx = threadIdx.x;
  typedef hipcub::BlockReduce<int64_t, THREADBLOCK_SIZE> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;

  bool stop_flag_now = false;
  int64_t stop_flag_now_int = 0;
  if (thread_idx < max_bsz) {
    if (thread_idx < bsz) {
      stop_flag_now = stop_flags[thread_idx];
      if (is_block_step[thread_idx]) {
        stop_flag_now_int = 0;
      } else {
        stop_flag_now_int = static_cast<int64_t>(stop_flag_now);
      }
    } else {
      stop_flag_now_int = 1;
    }
  }
  if (thread_idx < bsz) {
    const int seq_len_this_time = seq_lens_this_time[thread_idx];
    const int seq_len_encoder = seq_lens_encoder[thread_idx];
    const int seq_len_decoder = seq_lens_decoder[thread_idx];

    seq_lens_decoder[thread_idx] =
        stop_flag_now
            ? 0
            : (seq_len_decoder == 0 ? seq_len_encoder : seq_len_decoder + 1);

    seq_lens_this_time[thread_idx] = stop_flag_now ? 0 : 1;
    seq_lens_encoder[thread_idx] = 0;
    int64_t *input_ids_now = input_ids + thread_idx * input_ids_stride;
    input_ids_now[0] = next_tokens[thread_idx];
  }
  __syncthreads();
  int64_t stop_sum = BlockReduce(temp_storage).Sum(stop_flag_now_int);
  if (thread_idx == 0) {
    not_need_stop[0] = stop_sum < stop_nums[0];
  }
}

void UpdateInputes(const paddle::Tensor &stop_flags,
                   const paddle::Tensor &not_need_stop,
                   const paddle::Tensor &seq_lens_this_time,
                   const paddle::Tensor &seq_lens_encoder,
                   const paddle::Tensor &seq_lens_decoder,
                   const paddle::Tensor &input_ids,
                   const paddle::Tensor &stop_nums,
                   const paddle::Tensor &next_tokens,
                   const paddle::Tensor &is_block_step) {
  const int max_bsz = stop_flags.shape()[0];
  const int now_bsz = seq_lens_this_time.shape()[0];
  const int input_ids_stride = input_ids.shape()[1];
  update_inputs_kernel<1024><<<1, 1024, 0, input_ids.stream()>>>(
      const_cast<bool *>(not_need_stop.data<bool>()),
      const_cast<int *>(seq_lens_this_time.data<int>()),
      const_cast<int *>(seq_lens_encoder.data<int>()),
      const_cast<int *>(seq_lens_decoder.data<int>()),
      const_cast<int64_t *>(input_ids.data<int64_t>()),
      stop_nums.data<int64_t>(),
      stop_flags.data<bool>(),
      is_block_step.data<bool>(),
      next_tokens.data<int64_t>(),
      now_bsz,
      max_bsz,
      input_ids_stride);
}

PD_BUILD_OP(update_inputs)
    .Inputs({"stop_flags",
             "not_need_stop",
             "seq_lens_this_time",
             "seq_lens_encoder",
             "seq_lens_decoder",
             "input_ids",
             "stop_nums",
             "next_tokens",
             "is_block_step"})
    .Outputs({"not_need_stop_out",
              "seq_lens_this_time_out",
              "seq_lens_encoder_out",
              "seq_lens_decoder_out",
              "input_ids_out"})
    .SetInplaceMap({{"not_need_stop", "not_need_stop_out"},
                    {"seq_lens_this_time", "seq_lens_this_time_out"},
                    {"seq_lens_encoder", "seq_lens_encoder_out"},
                    {"seq_lens_decoder", "seq_lens_decoder_out"},
                    {"input_ids", "input_ids_out"}})
    .SetKernelFn(PD_KERNEL(UpdateInputes));
