#include "hip/hip_runtime.h"
// Copyright (c) 2023 PaddlePaddle Authors. All Rights Reserved.
// 
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
// 
//     http://www.apache.org/licenses/LICENSE-2.0
// 
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "helper.h"

template <typename T, int VecSize>
__global__ void fusedQKV_transpose_split_kernel(
    T *q_buf,
    T *k_buf,
    T *v_buf,
    const T *qkv,
    const int *padding_offset,
    const int *seq_lens,
    const int32_t elem_cnt,
    const int batch_size,
    const int max_len_this_time,
    const int seq_len,
    const int token_num,
    const int head_num,
    const int size_per_head) {
  const int32_t offset = batch_size * max_len_this_time * head_num * size_per_head;
  const int32_t hidden_size = head_num * size_per_head;
  const int32_t fused_hidden_size = 3 * hidden_size;
  int64_t global_thread_idx = blockDim.x * blockIdx.x + threadIdx.x;
  using LoadT = AlignedVector<T, VecSize>;
  LoadT src_vec;
  LoadT bias_vec;

  for (int32_t linear_index = global_thread_idx * VecSize,
               step = gridDim.x * blockDim.x * VecSize;
       linear_index < elem_cnt;
       linear_index += step) {
    Load<T, VecSize>(&qkv[linear_index], &src_vec);
    int32_t bias_idx = linear_index % fused_hidden_size;
    const int32_t token_idx = linear_index / fused_hidden_size;
    const int32_t ori_token_idx =
        token_idx + (padding_offset == nullptr ? 0 : padding_offset[token_idx]);
    const int32_t target_batch_id = ori_token_idx / seq_len;
    if (seq_lens[target_batch_id] == 0) continue;
    const int32_t seq_id = ori_token_idx % seq_len;

    // equal to:
    // const int qkv_id  = (linear_index % fused_hidden_size) / hidden_size;
    const int32_t qkv_id = bias_idx / hidden_size;
    const int32_t head_id = (linear_index % hidden_size) / size_per_head;
    const int32_t size_id = linear_index % size_per_head;

    if (qkv_id == 0) {
      Store<T, VecSize>(
          src_vec,
          &q_buf[target_batch_id * head_num * max_len_this_time * size_per_head +
                 head_id * max_len_this_time * size_per_head + seq_id * size_per_head +
                 size_id]);
    } else if (qkv_id == 1) {
      Store<T, VecSize>(
          src_vec,
          &k_buf[target_batch_id * head_num * max_len_this_time * size_per_head +
                 head_id * max_len_this_time * size_per_head + seq_id * size_per_head +
                 size_id]);
    } else {
      Store<T, VecSize>(
          src_vec,
          &v_buf[target_batch_id * head_num * max_len_this_time * size_per_head +
                 head_id * max_len_this_time * size_per_head + seq_id * size_per_head +
                 size_id]);
    }
  }
}

template <paddle::DataType D>
std::vector<paddle::Tensor> qkv_transpose_split(const paddle::Tensor& qkv, // [token_num, dim_embed]
                                                const paddle::Tensor& padding_offset, // [bsz, 1]
                                                const paddle::Tensor& seq_lens,
                                                const paddle::Tensor& input_ids,
                                                int num_head,
                                                int head_size) {
    typedef PDTraits<D> traits_;
    typedef typename traits_::DataType DataType_;
    typedef typename traits_::data_t data_t;

    auto cu_stream = qkv.stream();
    std::vector<int64_t> qkv_shape = qkv.shape();
    const int token_num = qkv_shape[0];
    const int bsz = seq_lens.shape()[0];
    const int max_seq_len = input_ids.shape()[1]; //max_seq_len_tensor.copy_to(paddle::CPUPlace(), false).data<int>()[0];
    auto q_out = paddle::full({bsz, num_head, max_seq_len, head_size}, 0, qkv.dtype(), qkv.place());
    auto k_out = paddle::full({bsz, num_head, max_seq_len, head_size}, 0, qkv.dtype(), qkv.place());
    auto v_out = paddle::full({bsz, num_head, max_seq_len, head_size}, 0, qkv.dtype(), qkv.place());
    constexpr int PackSize = VEC_16B / sizeof(DataType_);
    const int elem_cnt = token_num * num_head * head_size * 3;
    const int pack_num = elem_cnt / PackSize;
    const int blocksize = 128;
    const int grid_size = (pack_num + blocksize - 1) / blocksize;
    fusedQKV_transpose_split_kernel<DataType_, PackSize>
      <<<grid_size, blocksize, 0, qkv.stream()>>>(
        reinterpret_cast<DataType_*>(q_out.data<data_t>()),
        reinterpret_cast<DataType_*>(k_out.data<data_t>()),
        reinterpret_cast<DataType_*>(v_out.data<data_t>()),
        reinterpret_cast<DataType_*>(const_cast<data_t*>(qkv.data<data_t>())),
        padding_offset.data<int>(),
        seq_lens.data<int>(),
        elem_cnt,
        bsz,
        max_seq_len,
        max_seq_len,
        token_num,
        num_head,
        head_size);
    return {q_out, k_out, v_out};
}

std::vector<paddle::Tensor> QKVTransposeSplit(const paddle::Tensor& qkv, 
                                              const paddle::Tensor& padding_offset, 
                                              const paddle::Tensor& seq_lens,
                                              const paddle::Tensor& input_ids,
                                              int num_head,
                                              int head_size) {
    switch (qkv.type()) {
        case paddle::DataType::BFLOAT16: {
            return qkv_transpose_split<paddle::DataType::BFLOAT16>(
                qkv,
                padding_offset,
                seq_lens,
                input_ids,
                num_head,
                head_size
            );
        }
        case paddle::DataType::FLOAT16: {
            return qkv_transpose_split<paddle::DataType::FLOAT16>(
                qkv,
                padding_offset,
                seq_lens,
                input_ids,
                num_head,
                head_size
            );
        }
        case paddle::DataType::FLOAT32: {
            return qkv_transpose_split<paddle::DataType::FLOAT32>(
                qkv,
                padding_offset,
                seq_lens,
                input_ids,
                num_head,
                head_size
            );
        }
        default: {
            PD_THROW(
                "NOT supported data type. "
                "Only float16, bfloat16 and float32 are supported. ");
            break;
        }
    }
}

std::vector<std::vector<int64_t>> QKVTransposeSplitInferShape(const std::vector<int64_t>& qkv_shape,
                                                              const std::vector<int64_t>& padding_offset_shape,
                                                              const std::vector<int64_t>& seq_lens_shape,
                                                              const std::vector<int64_t>& input_ids_shape,
                                                              int num_head,
                                                              int head_size) {
    int64_t bsz = seq_lens_shape[0];
    return {{bsz, num_head, -1, head_size}, {bsz, num_head, -1, head_size}, {bsz, num_head, -1, head_size}};
}

std::vector<paddle::DataType> QKVTransposeSplitInferDtype(const paddle::DataType& qkv_dtype,
                                                        const paddle::DataType& padding_offset_dtype,
                                                        const paddle::DataType& seq_lens_dtype,
                                                        const paddle::DataType& input_ids_dtype) {
    return {qkv_dtype, qkv_dtype, qkv_dtype};
}

PD_BUILD_OP(qkv_transpose_split)
    .Inputs({"qkv", "padding_offset", "seq_lens", "input_ids"})
    .Outputs({"q_out", "k_out", "v_out"})
    .Attrs({"num_head: int",
            "head_size: int"})
    .SetKernelFn(PD_KERNEL(QKVTransposeSplit))
    .SetInferShapeFn(PD_INFER_SHAPE(QKVTransposeSplitInferShape))
    .SetInferDtypeFn(PD_INFER_DTYPE(QKVTransposeSplitInferDtype));