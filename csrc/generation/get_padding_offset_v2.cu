#include "hip/hip_runtime.h"
// Copyright (c) 2024 PaddlePaddle Authors. All Rights Reserved.
// 
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
// 
//     http://www.apache.org/licenses/LICENSE-2.0
// 
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/extension.h"

__global__ void RemovePaddingV2(int64_t *output_data,
                                const int64_t *input_data,
                                const int *seq_lens,
                                const int *cum_offsets,
                                const int sequence_length) {
  const int bi = blockIdx.x;
  const int tid = threadIdx.x;

  for (int i = tid; i < seq_lens[bi]; i += blockDim.x) {
    const int tgt_seq_id = bi * sequence_length - cum_offsets[bi] + i;
    const int src_seq_id = bi * sequence_length + i;
    output_data[tgt_seq_id] = input_data[src_seq_id];
  }
}

__global__ void GetPaddingOffsetKernelV2(int *padding_offset,
                                         int *cum_offsets_out,
                                         int *cu_seqlens_q,
                                         int *cu_seqlens_k,
                                         const int *cum_offsets,
                                         const int *seq_lens,
                                         const int max_seq_len) {
  // get padding offset of each batch
  const int bi = blockIdx.x;
  const int ti = threadIdx.x;
  int cum_offset = bi == 0 ? 0 : cum_offsets[bi - 1];
  for (int i = ti; i < seq_lens[bi]; i += blockDim.x) {
    padding_offset[bi * max_seq_len - cum_offset + i] = cum_offset;
  }
  if (ti == 0) {
    cum_offsets_out[bi] = cum_offset;
    int cum_seq_len = (bi + 1) * max_seq_len - cum_offsets[bi];
    cu_seqlens_q[bi + 1] = cum_seq_len;
    cu_seqlens_k[bi + 1] = cum_seq_len;
  }
}


std::vector<paddle::Tensor> GetPaddingOffsetV2(const paddle::Tensor& input_ids,
                                               const paddle::Tensor& cum_offsets,
                                               const paddle::Tensor& token_num,
                                               const paddle::Tensor& seq_len) {
    auto cu_stream = input_ids.stream();
    std::vector<int64_t> input_ids_shape = input_ids.shape();
    const int bsz = seq_len.shape()[0];
    const int seq_length = input_ids_shape[1];
    auto cum_offsets_out = cum_offsets.copy_to(cum_offsets.place(), false);
    auto cpu_token_num = token_num.copy_to(paddle::CPUPlace(), false);

    const int token_num_data = cpu_token_num.data<int64_t>()[0];
    auto x_remove_padding = paddle::full({token_num_data}, 0, paddle::DataType::INT64, input_ids.place());
    auto padding_offset = paddle::full({token_num_data}, 0, paddle::DataType::INT32, input_ids.place());
    auto cu_seqlens_q = paddle::full({bsz + 1}, 0, paddle::DataType::INT32, input_ids.place());
    auto cu_seqlens_k = paddle::full({bsz + 1}, 0, paddle::DataType::INT32, input_ids.place());
    int blockSize = min((token_num_data + 32 - 1) / 32 * 32, 128);
    GetPaddingOffsetKernelV2<<<bsz, 128, 0, cu_stream>>>(
      padding_offset.data<int>(), 
      cum_offsets_out.data<int>(),
      cu_seqlens_q.data<int>(),
      cu_seqlens_k.data<int>(),
      cum_offsets.data<int>(),
      seq_len.data<int>(),
      seq_length);
    RemovePaddingV2<<<bsz, blockSize, 0, cu_stream>>>(
      x_remove_padding.data<int64_t>(), 
      input_ids.data<int64_t>(), 
      seq_len.data<int>(),
      cum_offsets_out.data<int>(), 
      seq_length);
    return {x_remove_padding, cum_offsets_out, padding_offset, cu_seqlens_q, cu_seqlens_k}; // , enc_token_num, dec_token_num};
}

std::vector<std::vector<int64_t>> GetPaddingOffsetV2InferShape(const std::vector<int64_t>& input_ids_shape,
                                                             const std::vector<int64_t>& cum_offsets_shape,
                                                             const std::vector<int64_t>& token_num_shape,
                                                             const std::vector<int64_t>& seq_len_shape) {
    int64_t bsz = seq_len_shape[0];
    int64_t seq_len = input_ids_shape[1];
    return {{-1}, {bsz}, {-1}, {bsz + 1}, {bsz + 1}};
}

std::vector<paddle::DataType> GetPaddingOffsetV2InferDtype(const paddle::DataType& input_ids_dtype,
                                                         const paddle::DataType& cum_offsets_dtype,
                                                         const paddle::DataType& token_num_dtype,
                                                         const paddle::DataType& seq_len_dtype) {
    return {input_ids_dtype, seq_len_dtype, seq_len_dtype, seq_len_dtype, seq_len_dtype};
}

PD_BUILD_OP(get_padding_offset_v2)
    .Inputs({"input_ids", "token_num", "cum_offsets", "seq_len"})
    .Outputs({"x_remove_padding", "cum_offsets_out", "padding_offset", "cu_seqlens_q", "cu_seqlens_k"})
    .SetKernelFn(PD_KERNEL(GetPaddingOffsetV2))
    .SetInferShapeFn(PD_INFER_SHAPE(GetPaddingOffsetV2InferShape))
    .SetInferDtypeFn(PD_INFER_DTYPE(GetPaddingOffsetV2InferDtype));