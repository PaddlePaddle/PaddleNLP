#include "hip/hip_runtime.h"
// Copyright (c) 2023 PaddlePaddle Authors. All Rights Reserved.
// 
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
// 
//     http://www.apache.org/licenses/LICENSE-2.0
// 
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "helper.h"

template <typename T, int VecSize>
__global__ void TransposeRemovingPadding(const T* input_data,
                                         const int* seq_lens,
                                         T* output_data,
                                         const int batch_size,
                                         const int num_head,
                                         const int max_len_this_time,
                                         const int seq_len,
                                         const int head_dim,
                                         const int token_num,
                                         const int elem_cnt,
                                         const int* padding_offset) {
  // transpose and remove padding
  // [batch_size, num_head, max_len_this_time, head_dim] -> [token_num, num_head,
  // head_dim]
  int64_t idx = blockDim.x * blockIdx.x + threadIdx.x;
  const int dim_embed = num_head * head_dim;
  using LoadT = AlignedVector<T, VecSize>;
  LoadT src_vec;

  for (int32_t linear_index = idx * VecSize,
               step = gridDim.x * blockDim.x * VecSize;
       linear_index < elem_cnt;
       linear_index += step) {
    const int token_idx = linear_index / dim_embed;
    const int ori_token_idx =
        token_idx + (padding_offset == nullptr ? 0 : padding_offset[token_idx]);
    const int ori_batch_id = ori_token_idx / seq_len;
    if (seq_lens && seq_lens[ori_batch_id] == 0) continue;
    const int ori_seq_id = ori_token_idx % seq_len;
    const int ori_head_id = (linear_index % dim_embed) / head_dim;
    const int ori_head_lane = (linear_index % dim_embed) % head_dim;
    const int ori_idx = ori_batch_id * num_head * max_len_this_time * head_dim +
                        ori_head_id * max_len_this_time * head_dim +
                        ori_seq_id * head_dim + ori_head_lane;
    Load<T, VecSize>(&input_data[ori_idx], &src_vec);
    Store<T, VecSize>(src_vec, &output_data[linear_index]);
  }
}

template <typename T>
void InvokeTransposeRemovePadding(const T* input_data,
                                  const int* seq_lens,
                                  T* output_data,
                                  const int batch_size,
                                  const int num_head,
                                  const int max_len_this_time,
                                  const int seq_len,
                                  const int head_dim,
                                  const int token_num,
                                  const int* padding_offset,
                                  hipStream_t cu_stream) {
  // [batch_size, num_head, max_len_this_time, head_dim] -> [token_num, num_head,
  // head_dim]
  constexpr int VEC_16B = 16;
  const int elem_cnt = token_num * num_head * head_dim;
  constexpr int PackSize = VEC_16B / sizeof(T);
  const int32_t pack_num = elem_cnt / PackSize;
  const int32_t block_size = 128;
  int32_t grid_size = (pack_num + block_size - 1) / block_size;
  TransposeRemovingPadding<T, PackSize>
      <<<grid_size, block_size, 0, cu_stream>>>(input_data,
                                                seq_lens,
                                                output_data,
                                                batch_size,
                                                num_head,
                                                max_len_this_time,
                                                seq_len,
                                                head_dim,
                                                token_num,
                                                elem_cnt,
                                                padding_offset);
}

template <paddle::DataType D>
std::vector<paddle::Tensor> apply_transpose_remove_padding(const paddle::Tensor& input, 
                                                           const paddle::Tensor& seq_lens, 
                                                           const paddle::Tensor& padding_offset) {
    typedef PDTraits<D> traits_;
    typedef typename traits_::DataType DataType_;
    typedef typename traits_::data_t data_t;

    auto cu_stream = input.stream();
    std::vector<int64_t> input_shape = input.shape();
    const int bsz = input_shape[0];
    const int num_head = input_shape[1];
    const int seq_len = input_shape[2];
    const int dim_head = input_shape[3];
    const int token_num = padding_offset.shape()[0];

    auto out = paddle::full({token_num, num_head * dim_head}, 0, input.dtype(), input.place());
    InvokeTransposeRemovePadding(
        reinterpret_cast<DataType_*>(const_cast<data_t*>(input.data<data_t>())),
        seq_lens.data<int>(),
        reinterpret_cast<DataType_*>(out.data<data_t>()),
        bsz,
        num_head,
        seq_len,
        seq_len,
        dim_head,
        token_num,
        padding_offset.data<int>(),
        cu_stream
    );
    return {out};
}

std::vector<paddle::Tensor> ApplyTransposeRemovingPadding(const paddle::Tensor& input, 
                                                          const paddle::Tensor& seq_lens, 
                                                          const paddle::Tensor& padding_offset) {
    switch (input.type()) {
        case paddle::DataType::BFLOAT16: {
            return apply_transpose_remove_padding<paddle::DataType::BFLOAT16>(
                input,
                seq_lens,
                padding_offset
            );
        }
        case paddle::DataType::FLOAT16: {
            return apply_transpose_remove_padding<paddle::DataType::FLOAT16>(
                input,
                seq_lens,
                padding_offset
            );
        }
        case paddle::DataType::FLOAT32: {
            return apply_transpose_remove_padding<paddle::DataType::FLOAT32>(
                input,
                seq_lens,
                padding_offset
            );
        }
        default: {
            PD_THROW(
                "NOT supported data type. "
                "Only float16, bfloat16 and float32 are supported. ");
            break;
        }
    }
}

std::vector<std::vector<int64_t>> ApplyTransposeRemovingPaddingInferShape(
        const std::vector<int64_t>& input_shape, 
        const std::vector<int64_t>& seq_lens_shape,
        const std::vector<int64_t>& padding_offset_shape) {
    return {{padding_offset_shape[0], input_shape[1] * input_shape[3]}};
}

std::vector<paddle::DataType> ApplyTransposeRemovingPaddingInferDtype(
        const paddle::DataType& input_dtype, 
        const paddle::DataType& seq_lens_dtype,
        const paddle::DataType& padding_offset_dtype) {
    return {input_dtype};
}

PD_BUILD_OP(transpose_remove_padding)
    .Inputs({"input", "seq_lens", "padding_offset"})
    .Outputs({"fmha_out"})
    .SetKernelFn(PD_KERNEL(ApplyTransposeRemovingPadding))
    .SetInferShapeFn(PD_INFER_SHAPE(ApplyTransposeRemovingPaddingInferShape))
    .SetInferDtypeFn(PD_INFER_DTYPE(ApplyTransposeRemovingPaddingInferDtype));