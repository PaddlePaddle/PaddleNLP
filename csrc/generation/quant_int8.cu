#include "hip/hip_runtime.h"
// Copyright (c) 2023 PaddlePaddle Authors. All Rights Reserved.
// 
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
// 
//     http://www.apache.org/licenses/LICENSE-2.0
// 
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "helper.h"
#include<stdlib.h>
#include<string.h>
#include<sys/types.h>
#include<sys/stat.h>
#include<unistd.h>
#include<fcntl.h>
#include<sys/mman.h>
#include<stdio.h>
#include<algorithm>
#include<hip/hip_fp16.h>
#include<hip/hip_bf16.h>


constexpr int DequantKernelVecSize = 4;

template <typename data_t>
inline HOSTDEVICE data_t roundWithTiesToEven(data_t x) {
  data_t xLower = floor(x);
  data_t xUpper = ceil(x);
  // x is in interval [xl,xu]. Choose closest of two bounds, breaking ties to
  // even.
  data_t dLower = x - xLower;
  data_t dUpper = xUpper - x;
  return static_cast<data_t>(
      (dLower == dUpper ? fmod(xLower, 2.0F) == 0.0F : dLower < dUpper)
          ? xLower
          : xUpper);
}

template <typename T>
__forceinline__ __device__ T add_mul(T a, T b, T c) {
    return (a + b) * c;
}

template<>
__forceinline__ __device__ half add_mul<half>(half a, half b, half c) {
    return __hmul(__hadd(a, b), c);
}

template<>
__forceinline__ __device__ __hip_bfloat16 add_mul<__hip_bfloat16>(__hip_bfloat16 a, __hip_bfloat16 b, __hip_bfloat16 c) {
    return __hmul(__hadd(a, b), c);
}



template <typename data_t>
__forceinline__ __device__ int8_t quant_helper(const data_t input,
                                               const float scale,
                                               const int round_type,
                                               const float max_bound,
                                               const float min_bound) {
  float quant_value = max_bound * scale * static_cast<float>(input);

  if (round_type == 0) {
    quant_value = static_cast<float>(roundWithTiesToEven(quant_value));
  } else {
    quant_value = static_cast<float>(round(quant_value));
  }
  quant_value = quant_value > max_bound ? max_bound : quant_value;
  quant_value = quant_value < min_bound ? min_bound : quant_value;
  return static_cast<int8_t>(quant_value);
}

template <typename data_t>
__forceinline__ __device__ int8_t quant_helper(const data_t input,
                                               const data_t shift,
                                               const data_t smooth,
                                               const float scale,
                                               const int round_type,
                                               const float max_bound,
                                               const float min_bound) {
  auto smooth_out = add_mul(input, shift, smooth);
  float quant_value = max_bound * scale * static_cast<float>(smooth_out);

  if (round_type == 0) {
    quant_value = static_cast<float>(roundWithTiesToEven(quant_value));
  } else {
    quant_value = static_cast<float>(round(quant_value));
  }
  quant_value = quant_value > max_bound ? max_bound : quant_value;
  quant_value = quant_value < min_bound ? min_bound : quant_value;
  return static_cast<int8_t>(quant_value);
}

template <typename data_t>
__global__ void QuantKernel(const data_t* input,
                            char4* output,
                            const float scale,
                            const int m,
                            const int n,
                            const int round_type,
                            const float max_bound,
                            const float min_bound) {
  int n_id = (blockIdx.x * blockDim.x + threadIdx.x) << 2;
  int m_id = blockIdx.y * blockDim.y + threadIdx.y;
  bool check = ((m_id < m) && (n_id < n));

  if (check) {
    char4 tmp;
    tmp.x = quant_helper(
        input[m_id * n + n_id], scale, round_type, max_bound, min_bound);
    tmp.y = quant_helper(
        input[m_id * n + n_id + 1], scale, round_type, max_bound, min_bound);
    tmp.z = quant_helper(
        input[m_id * n + n_id + 2], scale, round_type, max_bound, min_bound);
    tmp.w = quant_helper(
        input[m_id * n + n_id + 3], scale, round_type, max_bound, min_bound);

    output[(m_id * n + n_id) >> 2] = tmp;
  }
}

template <typename data_t>
__global__ void QuantKernel(const data_t* input,
                            const data_t* shift,
                            const data_t* smooth,
                            char4* output,
                            const float scale,
                            const int m,
                            const int n,
                            const int round_type,
                            const float max_bound,
                            const float min_bound) {
  int n_id = (blockIdx.x * blockDim.x + threadIdx.x) << 2;
  int m_id = blockIdx.y * blockDim.y + threadIdx.y;
  bool check = ((m_id < m) && (n_id < n));

  if (check) {
    char4 tmp;
    tmp.x = quant_helper(
        input[m_id * n + n_id], shift[n_id], smooth[n_id], scale, round_type, max_bound, min_bound);
    tmp.y = quant_helper(
        input[m_id * n + n_id + 1], shift[n_id + 1], smooth[n_id + 1], scale, round_type, max_bound, min_bound);
    tmp.z = quant_helper(
        input[m_id * n + n_id + 2], shift[n_id + 2], smooth[n_id + 2], scale, round_type, max_bound, min_bound);
    tmp.w = quant_helper(
        input[m_id * n + n_id + 3], shift[n_id + 3], smooth[n_id + 3], scale, round_type, max_bound, min_bound);

    output[(m_id * n + n_id) >> 2] = tmp;
  }
}



template <paddle::DataType D>
std::vector<paddle::Tensor> LaunchQuantInt8(const paddle::Tensor& input,
                                      const paddle::optional<paddle::Tensor>& shift,
                                      const paddle::optional<paddle::Tensor>& smooth,
                                      float scale,
                                      int32_t round_type,
                                      float max_bound,
                                      float min_bound) {
    typedef PDTraits<D> traits_;
    typedef typename traits_::DataType DataType_;
    typedef typename traits_::data_t data_t;
    std::vector<int64_t> input_shape = input.shape();
    auto output=paddle::full(input_shape, -1, paddle::DataType::INT8, input.place());
    int m = input_shape[0];
    int n = input_shape[1];
    dim3 grid((n >> 2 + 31) / 32, (m + 31) / 32);
    dim3 block(32, 32);
    auto stream = input.stream();
    if (shift && smooth) {
        QuantKernel<DataType_><<<grid, block, 0, stream>>>(reinterpret_cast<const DataType_*>(input.data<data_t>()),
                                                       reinterpret_cast<const DataType_*>(shift.get().data<data_t>()),
                                                       reinterpret_cast<const DataType_*>(smooth.get().data<data_t>()),
                                                       reinterpret_cast<char4*>(output.data<int8_t>()),  // NOLINT
                                                       scale,
                                                       m,
                                                       n,
                                                       round_type,
                                                       max_bound,
                                                       min_bound);
    } else {
        QuantKernel<DataType_><<<grid, block, 0, stream>>>(reinterpret_cast<const DataType_*>(input.data<data_t>()),
                                                       reinterpret_cast<char4*>(output.data<int8_t>()),  // NOLINT
                                                       scale,
                                                       m,
                                                       n,
                                                       round_type,
                                                       max_bound,
                                                       min_bound);
    }
    return {output};

}

std::vector<paddle::Tensor> QuantInt8(const paddle::Tensor& input,
                                      const paddle::optional<paddle::Tensor>& shift,
                                      const paddle::optional<paddle::Tensor>& smooth,
                                      float scale,
                                      int32_t round_type,
                                      float max_bound,
                                      float min_bound) {
    // printf("#### quant int8 scale:%f \n",scale);
    switch (input.type()) {
        case paddle::DataType::BFLOAT16: {
            return LaunchQuantInt8<paddle::DataType::BFLOAT16>(
                input, shift, smooth, scale, round_type, max_bound, min_bound
            );
        }
        case paddle::DataType::FLOAT16: {
            return LaunchQuantInt8<paddle::DataType::FLOAT16>(
                input, shift, smooth, scale, round_type, max_bound, min_bound
            );
        }
        case paddle::DataType::FLOAT32: {
            return LaunchQuantInt8<paddle::DataType::FLOAT32>(
                input, shift, smooth, scale, round_type, max_bound, min_bound
            );
        }
        default: {
            PD_THROW(
                "NOT supported data type. "
                "Only bfloat16, float16 and float32 are supported. ");
            break;
        }
    }
}



std::vector<std::vector<int64_t>> QuantInt8Shape(const std::vector<int64_t>& input_shape,
                                                const paddle::optional<std::vector<int64_t>>& shift_shape,
                                                const paddle::optional<std::vector<int64_t>>& smooth_shape
                                                ) {
    return {input_shape};
}

std::vector<paddle::DataType> QuantInt8Dtype(const paddle::DataType& input_dtype,
                                            const paddle::optional<paddle::DataType>& shift_dtype,
                                            const paddle::optional<paddle::DataType>& smooth_dtype
                                            ) {
    return {paddle::DataType::INT8};
}

PD_BUILD_OP(quant_int8)
    .Inputs({"intput", paddle::Optional("shift"),paddle::Optional("smooth") })
    .Outputs({"output"})
    .Attrs({"scale: float","round_type: int","max_bound: float", "min_bound: float"})
    .SetKernelFn(PD_KERNEL(QuantInt8))
    .SetInferShapeFn(PD_INFER_SHAPE(QuantInt8Shape))
    .SetInferDtypeFn(PD_INFER_DTYPE(QuantInt8Dtype));