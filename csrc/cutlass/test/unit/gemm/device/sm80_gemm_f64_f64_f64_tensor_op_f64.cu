/***************************************************************************************************
 * Copyright (c) 2017 - 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/*! \file
    \brief Tests for device-wide GEMM interface
*/

#include <iostream>

#include "cutlass/cutlass.h"
#include "cute/tensor.hpp"
#include "cute/atom/mma_atom.hpp"

#include "cutlass/numeric_types.h"

#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "default_gemm_configuration.hpp"

#include "../../common/cutlass_unit_test.h"

#include "gemm_testbed_3x.hpp"

using namespace cute;

//#if defined(CUTLASS_ARCH_MMA_SM80_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(SM80_Device_Gemm_f64n_f64t_f64n_tensor_op_f64, 128x128x64_64x64x64) {

  using Config = cutlass::gemm::device::DefaultGemmConfigurationToCutlass3Types<
    cutlass::arch::OpClassTensorOp, cutlass::arch::Sm80,
    double, cutlass::layout::ColumnMajor,
    double, cutlass::layout::ColumnMajor,
    double, cutlass::layout::ColumnMajor,
    double>;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      Config::CollectiveMainloop,
      Config::CollectiveEpilogue
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  EXPECT_TRUE(test::gemm::device::TestAll<Gemm>());
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(SM80_Device_Gemm_f64t_f64n_f64n_tensor_op_f64, 128x128x64_64x64x64) {

  using Config = cutlass::gemm::device::DefaultGemmConfigurationToCutlass3Types<
    cutlass::arch::OpClassTensorOp, cutlass::arch::Sm80,
    double, cutlass::layout::RowMajor,
    double, cutlass::layout::ColumnMajor,
    double, cutlass::layout::ColumnMajor,
    double>;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      Config::CollectiveMainloop,
      Config::CollectiveEpilogue
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  EXPECT_TRUE(test::gemm::device::TestAll<Gemm>());
}

/////////////////////////////////////////////////////////////////////////////////////////////////

// #endif
