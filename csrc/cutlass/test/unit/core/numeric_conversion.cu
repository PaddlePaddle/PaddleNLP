#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/*! \file
    \brief Unit tests for conversion operators.
*/

#include "../common/cutlass_unit_test.h"

#include "cutlass/numeric_conversion.h"

#include "cutlass/layout/matrix.h"
#include "cutlass/util/host_tensor.h"

/////////////////////////////////////////////////////////////////////////////////////////////////

namespace test {
namespace core {
namespace kernel {

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Simple conversion function
template <typename Destination, typename Source, int Count>
__global__ void convert(
  cutlass::Array<Destination, Count> *destination,
  cutlass::Array<Source, Count> const *source) {

  cutlass::NumericArrayConverter<Destination, Source, Count> convert;

  *destination = convert(*source);
}

/////////////////////////////////////////////////////////////////////////////////////////////////

template <typename Destination, typename Source, int Count>
void run_test() {
  const int kN = Count;

  dim3 grid(1, 1);
  dim3 block(1, 1);

  cutlass::HostTensor<Destination, cutlass::layout::RowMajor> destination({1, kN});
  cutlass::HostTensor<Source, cutlass::layout::RowMajor> source({1, kN});

  for (int i = 0; i < kN; ++i) {
    source.host_data()[i] = Source(i % 4);
  }

  source.sync_device();

  convert<Destination, Source, kN><<< grid, block >>>(
    reinterpret_cast<cutlass::Array<Destination, kN> *>(destination.device_data()),
    reinterpret_cast<cutlass::Array<Source, kN> const *>(source.device_data())
  );

  destination.sync_host();

  for (int i = 0; i < kN; ++i) {
    EXPECT_TRUE(float(destination.host_data()[i]) == float(source.host_data()[i]));
  }
}

} // namespace kernel
} // namespace core
} // namespace test

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(NumericConversion, f32_to_f16_rn) {
  int const kN = 1;
  using Source = float;
  using Destination = cutlass::half_t;
  test::core::kernel::run_test<Destination, Source, kN>();
}

TEST(NumericConversion, f32x8_to_f16x8_rn) {
  int const kN = 8;
  using Source = float;
  using Destination = cutlass::half_t;
  test::core::kernel::run_test<Destination, Source, kN>();
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(NumericConversion, f16_to_f32_rn) {  
  int const kN = 1;
  using Source = cutlass::half_t;
  using Destination = float;
  test::core::kernel::run_test<Destination, Source, kN>();
}

TEST(NumericConversion, f16x8_to_f32x8_rn) {
  int const kN = 8;
  using Source = cutlass::half_t;
  using Destination = float;
  test::core::kernel::run_test<Destination, Source, kN>();
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(NumericConversion, f32_to_fe4m3_rn) {
  int const kN = 1;
  using Source = float;
  using Destination = cutlass::float_e4m3_t;
  test::core::kernel::run_test<Destination, Source, kN>();
}

TEST(NumericConversion, f32_to_fe4m3_rn_array) {
  int const kN = 27;
  using Source = float;
  using Destination = cutlass::float_e4m3_t;

  test::core::kernel::run_test<Destination, Source, kN>();
}

TEST(NumericConversion, f32_to_fe5m2_rn) {
  int const kN = 1;
  using Source = float;
  using Destination = cutlass::float_e5m2_t;
  test::core::kernel::run_test<Destination, Source, kN>();
}

TEST(NumericConversion, f32_to_fe5m2_rn_array) {
  int const kN = 27;
  using Source = float;
  using Destination = cutlass::float_e5m2_t;
  test::core::kernel::run_test<Destination, Source, kN>();
}

TEST(NumericConversion, f16_to_fe4m3_rn) {
  int const kN = 1;
  using Source = cutlass::half_t;
  using Destination = cutlass::float_e4m3_t;
  test::core::kernel::run_test<Destination, Source, kN>();
}

TEST(NumericConversion, f16_to_fe4m3_rn_array) {
  int const kN = 27;
  using Source = cutlass::half_t;
  using Destination = cutlass::float_e4m3_t;
  test::core::kernel::run_test<Destination, Source, kN>();
}

TEST(NumericConversion, f16_to_fe5m2_rn) {
  int const kN = 1;
  using Source = cutlass::half_t;
  using Destination = cutlass::float_e5m2_t;
  test::core::kernel::run_test<Destination, Source, kN>();
}

TEST(NumericConversion, f16_to_fe5m2_rn_array) {
  int const kN = 27;
  using Source = cutlass::half_t;
  using Destination = cutlass::float_e5m2_t;
  test::core::kernel::run_test<Destination, Source, kN>();
}

TEST(NumericConversion, bf16_to_fe4m3_rn) {
  int const kN = 1;
  using Source = cutlass::bfloat16_t;
  using Destination = cutlass::float_e4m3_t;
  test::core::kernel::run_test<Destination, Source, kN>();
}

TEST(NumericConversion, bf16_to_fe4m3_rn_array) {
  int const kN = 27;
  using Source = cutlass::bfloat16_t;
  using Destination = cutlass::float_e4m3_t;
  test::core::kernel::run_test<Destination, Source, kN>();
}

TEST(NumericConversion, bf16_to_fe5m2_rn) {
  int const kN = 1;
  using Source = cutlass::bfloat16_t;
  using Destination = cutlass::float_e5m2_t;
  test::core::kernel::run_test<Destination, Source, kN>();
}

TEST(NumericConversion, bf16_to_fe5m2_rn_array) {
  int const kN = 27;
  using Source = cutlass::bfloat16_t;
  using Destination = cutlass::float_e5m2_t;
  test::core::kernel::run_test<Destination, Source, kN>();
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(NumericConversion, fe4m3_to_fe5m2_rn) {
  int const kN = 1;
  using Source = cutlass::float_e4m3_t;
  using Destination = cutlass::float_e5m2_t;
  test::core::kernel::run_test<Destination, Source, kN>();
}

TEST(NumericConversion, fe4m3_to_fe5m2_array) {
  int const kN = 27;
  using Source = cutlass::float_e4m3_t;
  using Destination = cutlass::float_e5m2_t;
  test::core::kernel::run_test<Destination, Source, kN>();
}

TEST(NumericConversion, fe5m2_to_fe4m3_rn) {
  int const kN = 1;
  using Source = cutlass::float_e5m2_t;
  using Destination = cutlass::float_e4m3_t;
  test::core::kernel::run_test<Destination, Source, kN>();
}

TEST(NumericConversion, fe5m2_to_fe4m3_array) {
  int const kN = 27;
  using Source = cutlass::float_e5m2_t;
  using Destination = cutlass::float_e4m3_t;
  test::core::kernel::run_test<Destination, Source, kN>();
}

TEST(NumericConversion, fe4m3_to_f32_rn) {
  int const kN = 1;
  using Source = cutlass::float_e4m3_t;
  using Destination = float;
  test::core::kernel::run_test<Destination, Source, kN>();
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(NumericConversion, f32x8_to_s8x8_rn) {

  int const kN = 8;
  using Source = float;
  using Destination = int8_t;
  test::core::kernel::run_test<Destination, Source, kN>();
}

TEST(NumericConversion, fe4m3_to_f32_array) {
  int const kN = 27;
  using Source = cutlass::float_e4m3_t;
  using Destination = float;
  test::core::kernel::run_test<Destination, Source, kN>();
}

TEST(NumericConversion, fe5m2_to_f32_array) {
  int const kN = 27;
  using Source = cutlass::float_e5m2_t;
  using Destination = float;
  test::core::kernel::run_test<Destination, Source, kN>();
}

TEST(NumericConversion, fe4m3_to_f16_rn) {
  int const kN = 1;
  using Source = cutlass::float_e4m3_t;
  using Destination = cutlass::half_t;
  test::core::kernel::run_test<Destination, Source, kN>();
}

TEST(NumericConversion, fe4m3_to_f16_array) {
  int const kN = 27;
  using Source = cutlass::float_e4m3_t;
  using Destination = cutlass::half_t;
  test::core::kernel::run_test<Destination, Source, kN>();
}

TEST(NumericConversion, fe5m2_to_f16_rn) {
  int const kN = 1;
  using Source = cutlass::float_e5m2_t;
  using Destination = cutlass::half_t;
  test::core::kernel::run_test<Destination, Source, kN>();
}

TEST(NumericConversion, fe5m2_to_f16_array) {
  int const kN = 27;
  using Source = cutlass::float_e5m2_t;
  using Destination = cutlass::half_t;
  test::core::kernel::run_test<Destination, Source, kN>();
}

TEST(NumericConversion, fe4m3_to_bf16_rn) {
  int const kN = 1;
  using Source = cutlass::float_e4m3_t;
  using Destination = cutlass::bfloat16_t;
  test::core::kernel::run_test<Destination, Source, kN>();
}

TEST(NumericConversion, fe4m3_to_bf16_array) {
  int const kN = 27;
  using Source = cutlass::float_e4m3_t;
  using Destination = cutlass::bfloat16_t;
  test::core::kernel::run_test<Destination, Source, kN>();
}

TEST(NumericConversion, fe5m2_to_bf16_rn) {
  int const kN = 1;
  using Source = cutlass::float_e5m2_t;
  using Destination = cutlass::bfloat16_t;
  test::core::kernel::run_test<Destination, Source, kN>();
}

TEST(NumericConversion, fe5m2_to_bf16_array) {
  int const kN = 27;
  using Source = cutlass::float_e5m2_t;
  using Destination = cutlass::bfloat16_t;
  test::core::kernel::run_test<Destination, Source, kN>();
}

/////////////////////////////////////////////////////////////////////////////////////////////////
