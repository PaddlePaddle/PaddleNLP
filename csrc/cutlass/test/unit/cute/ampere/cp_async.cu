#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

#include "cutlass_unit_test.h"

#include <iostream>
#include <iomanip>
#include <utility>
#include <type_traits>
#include <vector>
#include <numeric>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <cute/tensor.hpp>

using namespace cute;

__global__ void
test(double const* g_in, double* g_out)
{
  extern __shared__ double smem[];

  smem[threadIdx.x] = g_in[threadIdx.x];

  __syncthreads();

  g_out[threadIdx.x] = 2 * smem[threadIdx.x];
}

__global__ void
test2(double const* g_in, double* g_out)
{
  using namespace cute;

  extern __shared__ double smem[];

  auto s_tensor = make_tensor(make_smem_ptr(smem + threadIdx.x), Int<1>{});
  auto g_tensor = make_tensor(make_gmem_ptr(g_in + threadIdx.x), Int<1>{});

  copy(g_tensor, s_tensor);

  cp_async_fence();
  cp_async_wait<0>();
  __syncthreads();

  g_out[threadIdx.x] = 2 * smem[threadIdx.x];
}

TEST(SM80_CuTe_Ampere, CpAsync)
{
  constexpr int count = 32;
  thrust::host_vector<double> h_in(count);
  for (int i = 0; i < count; ++i) {
    h_in[i] = double(i);
  }

  thrust::device_vector<double> d_in(h_in);

  thrust::device_vector<double> d_out(count, -1);
  test<<<1, count, sizeof(double) * count>>>(
    thrust::raw_pointer_cast(d_in.data()),
    thrust::raw_pointer_cast(d_out.data()));
  thrust::host_vector<double> h_result = d_out;

  thrust::device_vector<double> d_out_cp_async(count, -2);
  test2<<<1, count, sizeof(double) * count>>>(
    thrust::raw_pointer_cast(d_in.data()),
    thrust::raw_pointer_cast(d_out_cp_async.data()));
  thrust::host_vector<double> h_result_cp_async = d_out_cp_async;

  for (int i = 0; i < count; ++i) {
    EXPECT_EQ(h_result[i], h_result_cp_async[i]);
  }
}
