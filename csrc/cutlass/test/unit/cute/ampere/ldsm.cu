#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

#include "cutlass_unit_test.h"

#include <iostream>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <cute/tensor.hpp>

#include <cute/atom/copy_traits_sm75.hpp>


using namespace cute;

template <class T>
__global__ void
ldsm_test_device(uint16_t* g_in, uint16_t* g_out)
{
  constexpr int count = sizeof(T) / 4;
  int tid = threadIdx.x;
  int stride = blockDim.x;

  // load input gmem -> smem
  __shared__ uint32_t smem[32 * count];
  for (int i = 0; i < count; ++i) {
    smem[tid + (stride * i)] = reinterpret_cast<uint32_t*>(g_in)[tid + (stride * i)];
  }

  __syncthreads();

  uint32_t reg[count];
  for (int i = 0; i < count; ++i) {
    reg[i] = 0;
  }

  // load smem -> rmem using LDSM
  uint128_t* smem_ptr = reinterpret_cast<uint128_t*>(smem) + tid;
  T*         rmem_ptr = reinterpret_cast<T*>(reg);
  cute::copy_ldsm(smem_ptr, rmem_ptr);

  // store output rmem -> gmem
  for (int i = 0; i < count; ++i) {
    reinterpret_cast<uint32_t*>(g_out)[tid + (stride * i)] = reg[i];
  }
}

template <class TiledCopy, class SmemLayout>
__global__ void
ldsm_test_device_cute(uint16_t* g_in, uint16_t* g_out,
                      TiledCopy tiled_copy, SmemLayout smem_layout)
{
  using namespace cute;

  __shared__ uint16_t smem[size(smem_layout)];

  auto t_g_in  = make_tensor(make_gmem_ptr(g_in),  smem_layout);
  auto t_g_out = make_tensor(make_gmem_ptr(g_out), smem_layout);
  auto t_smem  = make_tensor(make_smem_ptr(smem),  smem_layout);

  int tid = threadIdx.x;

  // Load input gmem -> smem
  for (int i = tid; i < size(t_smem); i += size(tiled_copy)) {
    t_smem(i) = t_g_in(i);
  }

  __syncthreads();

  auto thr_copy = tiled_copy.get_thread_slice(tid);

  auto tXsX = thr_copy.partition_S(t_smem);   // (V,M,N)
  auto tXgX = thr_copy.partition_D(t_g_out);  // (V,M,N)

  auto tXrX = make_tensor<uint16_t>(shape(tXgX)); // (V,M,N)
  clear(tXrX);  // Just to make sure

/*
  if (thread0()) {
    print("tXsX: " ); print(tXsX.layout()); print("\n");
    print("tXgX: " ); print(tXgX.layout()); print("\n");
    print("tXrX: " ); print(tXrX.layout()); print("\n");
  }
*/

  // Copy smem -> rmem via tiled_copy (LDSM, LDS)
  copy(tiled_copy, tXsX, tXrX);

  // Output rmem -> gmem
  copy(tXrX, tXgX);
}


TEST(SM80_CuTe_Ampere, Ldsm)
{
  constexpr int count = 1024;

  thrust::host_vector<uint16_t> h_in(count);
  for (int i = 0; i < count; ++i) {
    h_in[i] = uint16_t(i);
  }
  thrust::device_vector<uint16_t> d_in = h_in;

  //
  // LDSM 1x (32b)
  //

  {
  thrust::device_vector<uint16_t> d_out(count);
  ldsm_test_device<uint32_t><<<1, 32>>>(
    thrust::raw_pointer_cast(d_in.data()),
    thrust::raw_pointer_cast(d_out.data()));
  thrust::host_vector<uint16_t> h_out = d_out;
  for (int i = 0; i < 32; ++i) {
    EXPECT_EQ(h_out[i], h_in[i]);
  }
  CUTLASS_TRACE_HOST("LDSM 1x ldsm_test_device SUCCESS\n");
  }

  //
  // LDSM 2x (64b)
  //

  {
  thrust::device_vector<uint16_t> d_out(count);
  ldsm_test_device<uint64_t><<<1, 32>>>(
    thrust::raw_pointer_cast(d_in.data()),
    thrust::raw_pointer_cast(d_out.data()));
  thrust::host_vector<uint16_t> h_out = d_out;
  for (int i = 0; i < 64; ++i) {
    //printf("%d  %d\n", int(h_in[i]), int(h_out[i]));
    EXPECT_EQ(h_out[i], h_in[i]);
  }
  CUTLASS_TRACE_HOST("LDSM 2x ldsm_test_device SUCCESS\n");
  }

  //
  // LDSM 4x (128b)
  //

  {
  thrust::device_vector<uint16_t> d_out(count);
  ldsm_test_device<uint128_t><<<1, 32>>>(
    thrust::raw_pointer_cast(d_in.data()),
    thrust::raw_pointer_cast(d_out.data()));
  thrust::host_vector<uint16_t> h_out = d_out;
  for (int i = 0; i < 128; ++i) {
    //printf("%d  %d\n", int(h_in[i]), int(h_out[i]));
    EXPECT_EQ(h_out[i], h_in[i]);
  }
  CUTLASS_TRACE_HOST("LDSM 4x ldsm_test_device SUCCESS\n");
  }

  //
  // CuTe LDSM
  //

  {
  thrust::device_vector<uint16_t> d_out(count);

  auto smem_layout = Layout<Shape <_32,Shape <_2, _4>>,
                            Stride< _2,Stride<_1,_64>>>{};
  auto tiled_copy = make_tiled_copy(Copy_Atom<SM75_U32x1_LDSM_N, uint16_t>{},
                                    Layout<Shape<_32,_1>>{},
                                    Layout<Shape< _1,_8>>{});

  ldsm_test_device_cute<<<1, int(size(tiled_copy))>>>(
    thrust::raw_pointer_cast(d_in.data()),
    thrust::raw_pointer_cast(d_out.data()),
    tiled_copy,
    smem_layout);
  thrust::host_vector<uint16_t> h_out = d_out;
  for (int i = 0; i < size(smem_layout); ++i) {
    //printf("%d  %d\n", int(h_in[i]), int(h_out[i]));
    EXPECT_EQ(h_out[i], h_in[i]);
  }
  CUTLASS_TRACE_HOST("CuTe 32x8 interleaved U32x1_LDSM_N SUCCESS\n");
  }

  {
  thrust::device_vector<uint16_t> d_out(count);

  auto smem_layout = Layout<Shape <_32,Shape <_2, _4>>,
                            Stride< _2,Stride<_1,_64>>>{};
  auto tiled_copy = make_tiled_copy(Copy_Atom<SM75_U32x2_LDSM_N, uint16_t>{},
                                    Layout<Shape<_32,_1>>{},
                                    Layout<Shape< _1,_8>>{});

  ldsm_test_device_cute<<<1, int(size(tiled_copy))>>>(
    thrust::raw_pointer_cast(d_in.data()),
    thrust::raw_pointer_cast(d_out.data()),
    tiled_copy,
    smem_layout);
  thrust::host_vector<uint16_t> h_out = d_out;
  for (int i = 0; i < size(smem_layout); ++i) {
    //printf("%d  %d\n", int(h_in[i]), int(h_out[i]));
    EXPECT_EQ(h_out[i], h_in[i]);
  }
  CUTLASS_TRACE_HOST("CuTe 32x8 interleaved U32x2_LDSM_N SUCCESS\n");
  }

  {
  thrust::device_vector<uint16_t> d_out(count);

  auto smem_layout = Layout<Shape <_32,Shape <_2, _4>>,
                            Stride< _2,Stride<_1,_64>>>{};
  auto tiled_copy = make_tiled_copy(Copy_Atom<SM75_U32x4_LDSM_N, uint16_t>{},
                                    Layout<Shape<_32,_1>>{},
                                    Layout<Shape< _1,_8>>{});

  ldsm_test_device_cute<<<1, int(size(tiled_copy))>>>(
    thrust::raw_pointer_cast(d_in.data()),
    thrust::raw_pointer_cast(d_out.data()),
    tiled_copy,
    smem_layout);
  thrust::host_vector<uint16_t> h_out = d_out;
  for (int i = 0; i < size(smem_layout); ++i) {
    //printf("%d  %d\n", int(h_in[i]), int(h_out[i]));
    EXPECT_EQ(h_out[i], h_in[i]);
  }
  CUTLASS_TRACE_HOST("CuTe 32x8 interleaved U32x4_LDSM_N SUCCESS\n");
  }

  {
  thrust::device_vector<uint16_t> d_out(count);

  auto smem_layout = Layout<Shape <_32,Shape <_2, _4>>,
                            Stride< _2,Stride<_1,_64>>>{};
  auto tiled_copy = make_tiled_copy(Copy_Atom<UniversalCopy<uint16_t>, uint16_t>{},
                                    Layout<Shape<_32,_1>>{},
                                    Layout<Shape< _1,_8>>{});

  ldsm_test_device_cute<<<1, int(size(tiled_copy))>>>(
    thrust::raw_pointer_cast(d_in.data()),
    thrust::raw_pointer_cast(d_out.data()),
    tiled_copy,
    smem_layout);
  thrust::host_vector<uint16_t> h_out = d_out;
  for (int i = 0; i < size(smem_layout); ++i) {
    //printf("%d  %d\n", int(h_in[i]), int(h_out[i]));
    EXPECT_EQ(h_out[i] , h_in[i]);
  }
  CUTLASS_TRACE_HOST("CuTe 32x8 interleaved LDS.U16 SUCCESS\n");
  }

  {
  thrust::device_vector<uint16_t> d_out(count);

  auto smem_layout = Layout<Shape <_32,_32>,
                            Stride< _1,_32>>{};
  auto tiled_copy = make_tiled_copy(Copy_Atom<SM75_U32x1_LDSM_N, uint16_t>{},
                                    Layout<Shape<_16,_2>>{},
                                    Layout<Shape< _2,_4>>{});

  ldsm_test_device_cute<<<1, int(size(tiled_copy))>>>(
    thrust::raw_pointer_cast(d_in.data()),
    thrust::raw_pointer_cast(d_out.data()),
    tiled_copy,
    smem_layout);
  thrust::host_vector<uint16_t> h_out = d_out;
  for (int i = 0; i < size(smem_layout); ++i) {
    //printf("%d  %d\n", int(h_in[i]), int(h_out[i]));
    EXPECT_EQ(h_out[i], h_in[i]);
  }
  CUTLASS_TRACE_HOST("CuTe 32x32 U32x1_LDSM_N SUCCESS\n");
  }

  {
  thrust::device_vector<uint16_t> d_out(count);

  auto smem_layout = Layout<Shape <_32,_32>,
                            Stride< _1,_32>>{};
  auto tiled_copy = make_tiled_copy(Copy_Atom<SM75_U32x2_LDSM_N, uint16_t>{},
                                    Layout<Shape<_16,_2>>{},
                                    Layout<Shape< _2,_4>>{});

  ldsm_test_device_cute<<<1, int(size(tiled_copy))>>>(
    thrust::raw_pointer_cast(d_in.data()),
    thrust::raw_pointer_cast(d_out.data()),
    tiled_copy,
    smem_layout);
  thrust::host_vector<uint16_t> h_out = d_out;
  for (int i = 0; i < size(smem_layout); ++i) {
    //printf("%d  %d\n", int(h_in[i]), int(h_out[i]));
    EXPECT_EQ(h_out[i], h_in[i]);
  }
  CUTLASS_TRACE_HOST("CuTe 32x32 U32x2_LDSM_N SUCCESS\n");
  }

  {
  thrust::device_vector<uint16_t> d_out(count);

  auto smem_layout = Layout<Shape <_32,_32>,
                            Stride< _1,_32>>{};
  auto tiled_copy = make_tiled_copy(Copy_Atom<SM75_U32x4_LDSM_N, uint16_t>{},
                                    Layout<Shape<_16,_2>>{},
                                    Layout<Shape< _2,_4>>{});

  ldsm_test_device_cute<<<1, int(size(tiled_copy))>>>(
    thrust::raw_pointer_cast(d_in.data()),
    thrust::raw_pointer_cast(d_out.data()),
    tiled_copy,
    smem_layout);
  thrust::host_vector<uint16_t> h_out = d_out;
  for (int i = 0; i < size(smem_layout); ++i) {
    //printf("%d  %d\n", int(h_in[i]), int(h_out[i]));
    EXPECT_EQ(h_out[i], h_in[i]);
  }
  CUTLASS_TRACE_HOST("CuTe 32x32 U32x4_LDSM_N SUCCESS\n");
  }

  {
  thrust::device_vector<uint16_t> d_out(count);

  auto smem_layout = Layout<Shape <_32,_32>,
                            Stride< _1,_32>>{};
  auto tiled_copy = make_tiled_copy(Copy_Atom<UniversalCopy<uint16_t>, uint16_t>{},
                                    Layout<Shape<_16,_2>>{},
                                    Layout<Shape< _2,_4>>{});

  ldsm_test_device_cute<<<1, int(size(tiled_copy))>>>(
    thrust::raw_pointer_cast(d_in.data()),
    thrust::raw_pointer_cast(d_out.data()),
    tiled_copy,
    smem_layout);
  thrust::host_vector<uint16_t> h_out = d_out;
  for (int i = 0; i < size(smem_layout); ++i) {
    //printf("%d  %d\n", int(h_in[i]), int(h_out[i]));
    EXPECT_EQ(h_out[i], h_in[i]);
  }
  CUTLASS_TRACE_HOST("CuTe 32x32 LDS.U16 SUCCESS\n");
  }

  {
  thrust::device_vector<uint16_t> d_out(count);

  auto smem_layout = Layout<Shape <_32,_32>,
                            Stride<_32, _1>>{};
  auto tiled_copy = make_tiled_copy(Copy_Atom<SM75_U16x2_LDSM_T, uint16_t>{},
                                    Layout<Shape<_4,_8>>{},
                                    Layout<Shape<_2,_1>>{});

  ldsm_test_device_cute<<<1, int(size(tiled_copy))>>>(
    thrust::raw_pointer_cast(d_in.data()),
    thrust::raw_pointer_cast(d_out.data()),
    tiled_copy,
    smem_layout);
  thrust::host_vector<uint16_t> h_out = d_out;
  for (int i = 0; i < size(smem_layout); ++i) {
    //printf("%d  %d\n", int(h_in[i]), int(h_out[i]));
    EXPECT_EQ(h_out[i],  h_in[i]);
  }
  CUTLASS_TRACE_HOST("CuTe 32x32 U16x2_LDSM_T SUCCESS\n");
  }

  {
  thrust::device_vector<uint16_t> d_out(count);

  auto smem_layout = Layout<Shape <_32,_32>,
                            Stride<_32, _1>>{};
  auto tiled_copy = make_tiled_copy(Copy_Atom<SM75_U16x4_LDSM_T, uint16_t>{},
                                    Layout<Shape<_4,_8>>{},
                                    Layout<Shape<_4,_1>>{});

  ldsm_test_device_cute<<<1, int(size(tiled_copy))>>>(
    thrust::raw_pointer_cast(d_in.data()),
    thrust::raw_pointer_cast(d_out.data()),
    tiled_copy,
    smem_layout);
  thrust::host_vector<uint16_t> h_out = d_out;
  for (int i = 0; i < size(smem_layout); ++i) {
    //printf("%d  %d\n", int(h_in[i]), int(h_out[i]));
    EXPECT_EQ(h_out[i],  h_in[i]);
  }
  CUTLASS_TRACE_HOST("CuTe 32x32 U16x4_LDSM_T SUCCESS\n");
  }

  {
  thrust::device_vector<uint16_t> d_out(count);

  auto smem_layout = Layout<Shape <_32,_32>,
                            Stride<_32, _1>>{};
  auto tiled_copy = make_tiled_copy(Copy_Atom<SM75_U16x8_LDSM_T, uint16_t>{},
                                    Layout<Shape<_4,_8>>{},
                                    Layout<Shape<_8,_1>>{});

  ldsm_test_device_cute<<<1, int(size(tiled_copy))>>>(
    thrust::raw_pointer_cast(d_in.data()),
    thrust::raw_pointer_cast(d_out.data()),
    tiled_copy,
    smem_layout);
  thrust::host_vector<uint16_t> h_out = d_out;
  for (int i = 0; i < size(smem_layout); ++i) {
    //printf("%d  %d\n", int(h_in[i]), int(h_out[i]));
    EXPECT_EQ(h_out[i], h_in[i]);
  }
  CUTLASS_TRACE_HOST("CuTe 32x32 U16x8_LDSM_T SUCCESS\n");
  }

  CUTLASS_TRACE_HOST("PASS");
}
