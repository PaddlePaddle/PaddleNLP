#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

#include "cutlass_unit_test.h"

#include <iostream>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <cute/tensor.hpp>
#include <cute/arch/copy_sm90.hpp>

using namespace cute;

template<class T>
__global__ void
stsm_test_device(uint16_t* g_in, uint16_t* g_out)
{
  constexpr int count = sizeof(T) / 4;
  int tid = threadIdx.x;
  int stride = blockDim.x;

  // load input gmem -> rmem
  uint32_t reg[count];
  for (int i = 0; i < (sizeof(T) / 4); i++) {
    reg[i] = reinterpret_cast<uint32_t*>(g_in)[tid + (stride * i)];
  }

  __shared__ uint32_t smem[32 * count];

  // load rmem -> smem using STSM
  uint128_t* smem_ptr = reinterpret_cast<uint128_t*>(smem) + tid;
  T*         rmem_ptr = reinterpret_cast<T*>(reg);
  cute::copy_stsm(rmem_ptr, smem_ptr);

  __syncthreads();

  // store output smem -> gmem
  for (int i = 0; i < (sizeof(T) / 4); i++) {
    reinterpret_cast<uint32_t*>(g_out)[tid + (stride * i)] = smem[tid + (stride * i)];
  }
}

template <class TiledCopy, class SmemLayout>
__global__ void
stsm_test_device_cute(uint16_t* g_in, uint16_t* g_out,
                      TiledCopy tiled_copy, SmemLayout smem_layout)
{
  using namespace cute;

  __shared__ uint16_t smem[size(smem_layout)];

  Tensor t_g_in  = make_tensor(make_gmem_ptr(g_in),  smem_layout);
  Tensor t_g_out = make_tensor(make_gmem_ptr(g_out), smem_layout);
  Tensor t_smem  = make_tensor(make_smem_ptr(smem),  smem_layout);

  int tid = threadIdx.x;

  auto thr_copy = tiled_copy.get_thread_slice(tid);

  Tensor tXgX = thr_copy.partition_S(t_g_in);   // (V,M,N)
  Tensor tXsX = thr_copy.partition_D(t_smem);   // (V,M,N)

  Tensor tXrX = make_tensor<uint16_t>(shape(tXgX)); // (V,M,N)
  clear(tXrX);    // Just to make sure

/*
  if (thread0()) {
    print("tXsX: " ); print(tXsX.layout()); print("\n");
    print("tXgX: " ); print(tXgX.layout()); print("\n");
    print("tXrX: " ); print(tXrX.layout()); print("\n");
  }
*/

  // Load input gmem -> rmem
  copy(tXgX, tXrX);

  // Copy rmem -> smem via tiled_copy (STSM, STS)
  copy(tiled_copy, tXrX, tXsX);

  // Output smem -> gmem
  for (int i = tid; i < size(t_smem); i += size(tiled_copy)) {
    t_g_out(i) = t_smem(i);
  }
}

#if CUDA_12_0_SM90_FEATURES_SUPPORTED
TEST(SM90_CuTe_Hopper, Stsm)
{
  constexpr int count = 1024;

  thrust::host_vector<uint16_t> h_in(count);
  for (int i = 0; i < count; ++i) {
    h_in[i] = uint16_t(i);
  }
  thrust::device_vector<uint16_t> d_in = h_in;

  //
  // STSM 1x (32b)
  //

  {
  thrust::device_vector<uint16_t> d_out(count);
  stsm_test_device<uint32_t><<<1, 32>>>(
    thrust::raw_pointer_cast(d_in.data()),
    thrust::raw_pointer_cast(d_out.data()));
  thrust::host_vector<uint16_t> h_out = d_out;
  for (int i = 0; i < 32; ++i) {
    //printf("%d  %d\n", int(h_in[i]), int(h_out[i]));
    EXPECT_EQ(h_out[i], h_in[i]);
  }
  CUTLASS_TRACE_HOST("STSM 1x stsm_test_device SUCCESS\n");
  }

  //
  // STSM 2x (64b)
  //

  {
  thrust::device_vector<uint16_t> d_out(count);
  stsm_test_device<uint64_t><<<1, 32>>>(
    thrust::raw_pointer_cast(d_in.data()),
    thrust::raw_pointer_cast(d_out.data()));
  thrust::host_vector<uint16_t> h_out = d_out;
  for (int i = 0; i < 64; ++i) {
    //printf("%d  %d\n", int(h_in[i]), int(h_out[i]));
    EXPECT_EQ(h_out[i], h_in[i]);
  }
  CUTLASS_TRACE_HOST("STSM 2x stsm_test_device SUCCESS\n");
  }

  //
  // STSM 4x (128b)
  //

  {
  thrust::device_vector<uint16_t> d_out(count);
  stsm_test_device<uint128_t><<<1, 32>>>(
    thrust::raw_pointer_cast(d_in.data()),
    thrust::raw_pointer_cast(d_out.data()));
  thrust::host_vector<uint16_t> h_out = d_out;
  for (int i = 0; i < 128; ++i) {
    //printf("%d  %d\n", int(h_in[i]), int(h_out[i]));
    EXPECT_EQ(h_out[i], h_in[i]);
  }
  CUTLASS_TRACE_HOST("STSM 4x stsm_test_device SUCCESS\n");
  }

  //
  // CuTe STSM
  //

  {
  thrust::device_vector<uint16_t> d_out(count);

  auto smem_layout = Layout<Shape <_32,Shape <_2, _4>>,
                            Stride< _2,Stride<_1,_64>>>{};
  auto tiled_copy = make_tiled_copy(Copy_Atom<SM90_U32x1_STSM_N, uint16_t>{},
                                    Layout<Shape<_32,_1>>{},
                                    Layout<Shape< _1,_8>>{});

  stsm_test_device_cute<<<1, int(size(tiled_copy))>>>(
    thrust::raw_pointer_cast(d_in.data()),
    thrust::raw_pointer_cast(d_out.data()),
    tiled_copy,
    smem_layout);
  thrust::host_vector<uint16_t> h_out = d_out;
  for (int i = 0; i < size(smem_layout); ++i) {
    //printf("%d  %d\n", int(h_in[i]), int(h_out[i]));
    EXPECT_EQ(h_out[i], h_in[i]);
  }
  CUTLASS_TRACE_HOST("CuTe 32x8 interleaved U32x1_STSM_N SUCCESS\n");
  }

  {
  thrust::device_vector<uint16_t> d_out(count);

  auto smem_layout = Layout<Shape <_32,Shape <_2, _4>>,
                            Stride< _2,Stride<_1,_64>>>{};
  auto tiled_copy = make_tiled_copy(Copy_Atom<SM90_U32x2_STSM_N, uint16_t>{},
                                    Layout<Shape<_32,_1>>{},
                                    Layout<Shape< _1,_8>>{});

  stsm_test_device_cute<<<1, int(size(tiled_copy))>>>(
    thrust::raw_pointer_cast(d_in.data()),
    thrust::raw_pointer_cast(d_out.data()),
    tiled_copy,
    smem_layout);
  thrust::host_vector<uint16_t> h_out = d_out;
  for (int i = 0; i < size(smem_layout); ++i) {
    //printf("%d  %d\n", int(h_in[i]), int(h_out[i]));
    EXPECT_EQ(h_out[i], h_in[i]);
  }
  CUTLASS_TRACE_HOST("CuTe 32x8 interleaved U32x2_STSM_N SUCCESS\n");
  }

  {
  thrust::device_vector<uint16_t> d_out(count);

  auto smem_layout = Layout<Shape <_32,Shape <_2, _4>>,
                            Stride< _2,Stride<_1,_64>>>{};
  auto tiled_copy = make_tiled_copy(Copy_Atom<SM90_U32x4_STSM_N, uint16_t>{},
                                    Layout<Shape<_32,_1>>{},
                                    Layout<Shape< _1,_8>>{});

  stsm_test_device_cute<<<1, int(size(tiled_copy))>>>(
    thrust::raw_pointer_cast(d_in.data()),
    thrust::raw_pointer_cast(d_out.data()),
    tiled_copy,
    smem_layout);
  thrust::host_vector<uint16_t> h_out = d_out;
  for (int i = 0; i < size(smem_layout); ++i) {
    //printf("%d  %d\n", int(h_in[i]), int(h_out[i]));
    EXPECT_EQ(h_out[i], h_in[i]);
  }
  CUTLASS_TRACE_HOST("CuTe 32x8 interleaved U32x4_STSM_N SUCCESS\n");
  }

  {
  thrust::device_vector<uint16_t> d_out(count);

  auto smem_layout = Layout<Shape <_32,Shape <_2, _4>>,
                            Stride< _2,Stride<_1,_64>>>{};
  auto tiled_copy = make_tiled_copy(Copy_Atom<UniversalCopy<uint16_t>, uint16_t>{},
                                    Layout<Shape<_32,_1>>{},
                                    Layout<Shape< _1,_8>>{});

  stsm_test_device_cute<<<1, int(size(tiled_copy))>>>(
    thrust::raw_pointer_cast(d_in.data()),
    thrust::raw_pointer_cast(d_out.data()),
    tiled_copy,
    smem_layout);
  thrust::host_vector<uint16_t> h_out = d_out;
  for (int i = 0; i < size(smem_layout); ++i) {
    //printf("%d  %d\n", int(h_in[i]), int(h_out[i]));
    EXPECT_EQ(h_out[i], h_in[i]);
  }
  CUTLASS_TRACE_HOST("CuTe 32x8 interleaved STS.U16 SUCCESS\n");
  }

  {
  thrust::device_vector<uint16_t> d_out(count);

  auto smem_layout = Layout<Shape <_32,_32>,
                            Stride< _1,_32>>{};
  auto tiled_copy = make_tiled_copy(Copy_Atom<SM90_U32x1_STSM_N, uint16_t>{},
                                    Layout<Shape<_16,_2>>{},
                                    Layout<Shape< _2,_4>>{});

  stsm_test_device_cute<<<1, int(size(tiled_copy))>>>(
    thrust::raw_pointer_cast(d_in.data()),
    thrust::raw_pointer_cast(d_out.data()),
    tiled_copy,
    smem_layout);
  thrust::host_vector<uint16_t> h_out = d_out;
  for (int i = 0; i < size(smem_layout); ++i) {
    //printf("%d  %d\n", int(h_in[i]), int(h_out[i]));
    EXPECT_EQ(h_out[i], h_in[i]);
  }
  CUTLASS_TRACE_HOST("CuTe 32x32 U32x1_STSM_N SUCCESS\n");
  }

  {
  thrust::device_vector<uint16_t> d_out(count);

  auto smem_layout = Layout<Shape <_32,_32>,
                            Stride< _1,_32>>{};
  auto tiled_copy = make_tiled_copy(Copy_Atom<SM90_U32x2_STSM_N, uint16_t>{},
                                    Layout<Shape<_16,_2>>{},
                                    Layout<Shape< _2,_4>>{});

  stsm_test_device_cute<<<1, int(size(tiled_copy))>>>(
    thrust::raw_pointer_cast(d_in.data()),
    thrust::raw_pointer_cast(d_out.data()),
    tiled_copy,
    smem_layout);
  thrust::host_vector<uint16_t> h_out = d_out;
  for (int i = 0; i < size(smem_layout); ++i) {
    //printf("%d  %d\n", int(h_in[i]), int(h_out[i]));
    EXPECT_EQ(h_out[i], h_in[i]);
  }
  CUTLASS_TRACE_HOST("CuTe 32x32 U32x2_STSM_N SUCCESS\n");
  }

  {
  thrust::device_vector<uint16_t> d_out(count);

  auto smem_layout = Layout<Shape <_32,_32>,
                            Stride< _1,_32>>{};
  auto tiled_copy = make_tiled_copy(Copy_Atom<SM90_U32x4_STSM_N, uint16_t>{},
                                    Layout<Shape<_16,_2>>{},
                                    Layout<Shape< _2,_4>>{});

  stsm_test_device_cute<<<1, int(size(tiled_copy))>>>(
    thrust::raw_pointer_cast(d_in.data()),
    thrust::raw_pointer_cast(d_out.data()),
    tiled_copy,
    smem_layout);
  thrust::host_vector<uint16_t> h_out = d_out;
  for (int i = 0; i < size(smem_layout); ++i) {
    //printf("%d  %d\n", int(h_in[i]), int(h_out[i]));
    EXPECT_EQ(h_out[i], h_in[i]);
  }
  CUTLASS_TRACE_HOST("CuTe 32x32 U32x4_STSM_N SUCCESS\n");
  }

  {
  thrust::device_vector<uint16_t> d_out(count);

  auto smem_layout = Layout<Shape <_32,_32>,
                            Stride< _1,_32>>{};
  auto tiled_copy = make_tiled_copy(Copy_Atom<UniversalCopy<uint16_t>, uint16_t>{},
                                    Layout<Shape<_16,_2>>{},
                                    Layout<Shape< _2,_4>>{});

  stsm_test_device_cute<<<1, int(size(tiled_copy))>>>(
    thrust::raw_pointer_cast(d_in.data()),
    thrust::raw_pointer_cast(d_out.data()),
    tiled_copy,
    smem_layout);
  thrust::host_vector<uint16_t> h_out = d_out;
  for (int i = 0; i < size(smem_layout); ++i) {
    //printf("%d  %d\n", int(h_in[i]), int(h_out[i]));
    EXPECT_EQ(h_out[i], h_in[i]);
  }
  CUTLASS_TRACE_HOST("CuTe 32x32 STS.U16 SUCCESS\n");
  }

  {
  thrust::device_vector<uint16_t> d_out(count);

  auto smem_layout = Layout<Shape <_32,_32>,
                            Stride<_32, _1>>{};
  auto tiled_copy = make_tiled_copy(Copy_Atom<SM90_U16x2_STSM_T, uint16_t>{},
                                    Layout<Shape<_4,_8>>{},
                                    Layout<Shape<_2,_1>>{});

  stsm_test_device_cute<<<1, int(size(tiled_copy))>>>(
    thrust::raw_pointer_cast(d_in.data()),
    thrust::raw_pointer_cast(d_out.data()),
    tiled_copy,
    smem_layout);
  thrust::host_vector<uint16_t> h_out = d_out;
  for (int i = 0; i < size(smem_layout); ++i) {
    //printf("%d  %d\n", int(h_in[i]), int(h_out[i]));
    EXPECT_EQ(h_out[i], h_in[i]);
  }
  CUTLASS_TRACE_HOST("CuTe 32x32 U16x2_STSM_T SUCCESS\n");
  }

  {
  thrust::device_vector<uint16_t> d_out(count);

  auto smem_layout = Layout<Shape <_32,_32>,
                            Stride<_32, _1>>{};
  auto tiled_copy = make_tiled_copy(Copy_Atom<SM90_U16x4_STSM_T, uint16_t>{},
                                    Layout<Shape<_4,_8>>{},
                                    Layout<Shape<_4,_1>>{});

  stsm_test_device_cute<<<1, int(size(tiled_copy))>>>(
    thrust::raw_pointer_cast(d_in.data()),
    thrust::raw_pointer_cast(d_out.data()),
    tiled_copy,
    smem_layout);
  thrust::host_vector<uint16_t> h_out = d_out;
  for (int i = 0; i < size(smem_layout); ++i) {
    //printf("%d  %d\n", int(h_in[i]), int(h_out[i]));
    EXPECT_EQ(h_out[i], h_in[i]);
  }
  CUTLASS_TRACE_HOST("CuTe 32x32 U16x4_STSM_T SUCCESS\n");
  }

  {
  thrust::device_vector<uint16_t> d_out(count);

  auto smem_layout = Layout<Shape <_32,_32>,
                            Stride<_32, _1>>{};
  auto tiled_copy = make_tiled_copy(Copy_Atom<SM90_U16x8_STSM_T, uint16_t>{},
                                    Layout<Shape<_4,_8>>{},
                                    Layout<Shape<_8,_1>>{});

  stsm_test_device_cute<<<1, int(size(tiled_copy))>>>(
    thrust::raw_pointer_cast(d_in.data()),
    thrust::raw_pointer_cast(d_out.data()),
    tiled_copy,
    smem_layout);
  thrust::host_vector<uint16_t> h_out = d_out;
  for (int i = 0; i < size(smem_layout); ++i) {
    //printf("%d  %d\n", int(h_in[i]), int(h_out[i]));
    EXPECT_EQ(h_out[i], h_in[i]);
  }
  CUTLASS_TRACE_HOST("CuTe 32x32 U16x8_STSM_T SUCCESS\n");
  }

  CUTLASS_TRACE_HOST("PASS");
}
#endif
