#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*! \file
    \brief Unit test for the PipelineAsync class
*/

#define KERNEL_DBG_TRACE false

#include "../common/cutlass_unit_test.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <cute/tensor.hpp>
#include <cute/arch/cluster_sm90.hpp> 

#include <cutlass/util/reference/host/gemm.h>
#include <cutlass/cluster_launch.hpp>

#include "cutlass/core_io.h"

#include "cutlass/util/print_error.hpp"
#include "cutlass/util/GPU_Clock.hpp"

#include "testbed.h"
#include "cutlass/pipeline.hpp"
#include "cutlass/arch/barrier.h"
#include "cute/arch/cluster_sm90.hpp"

using namespace cute;

//////////////////// KERNEL /////////////////////////

template <uint32_t Stages>
struct SharedStorage
{
  typename cutlass::PipelineAsync<Stages>::SharedStorage storage;
};

// Goal of this kernel is to complete deadlock-free
// Simple 1 producer warp, one consumer warp scenario
template <class ClusterShape, uint32_t NumStages>
__global__ static 
void pipeline_async_basic_device(uint32_t const num_iterations)
{

  extern __shared__ char shared_memory[];
  using MainloopPipeline = typename cutlass::PipelineAsync<NumStages>;
  using PipelineState = typename cutlass::PipelineState<NumStages>;

  using SharedStorage = SharedStorage<NumStages>;
  SharedStorage& shared_storage = *reinterpret_cast<SharedStorage*>(shared_memory);


  auto cta_layout = Layout<ClusterShape>{}; // (m,n) -> cta_id

  int warp_idx = __shfl_sync(0xffffffff, threadIdx.x / 32, 0);
  int lane_predicate = cute::elect_one_sync();
  dim3 block_id_in_cluster = cute::block_id_in_cluster();
  auto cluster_shape = ClusterShape{};
  
  // This example showcases 2 producer 1 consumer example 
  typename MainloopPipeline::Params params;
  params.producer_arv_count = 2;
  params.consumer_arv_count = 1;
  MainloopPipeline pipeline(shared_storage.storage, params);

  // Ensure All CTAs in Cluster have completed init before issuing commits
  cute::cluster_arrive_relaxed();  
  cute::cluster_wait();
  __syncthreads();

  if (lane_predicate) {
    // Producer Warps
    if (warp_idx==0 || warp_idx==1) {

      int prologue_iterations = min(NumStages, num_iterations);
      for ( int i = 0; i < prologue_iterations; ++i) {
        // Can also specify stage to commit directly
        pipeline.producer_commit(i);
      }

      int mainloop_iterations = num_iterations - prologue_iterations;

      // Only the mainloop needs a PipelineState because this is where we start "waiting" (acquiring)
      PipelineState smem_pipe_write;

      for ( ; mainloop_iterations > 0; --mainloop_iterations) {
        pipeline.producer_acquire(smem_pipe_write);
        pipeline.producer_commit(smem_pipe_write);
        ++smem_pipe_write;
      }
    }
    else {
      PipelineState smem_pipe_read;
      for (int iter=0 ; iter < num_iterations; ++iter) {
        pipeline.consumer_wait(smem_pipe_read);
        pipeline.consumer_release(smem_pipe_read.index());
        ++smem_pipe_read;
      }
    }
  }

  // To make sure remote SMEM doesn't get destroyed
  cute::cluster_arrive();  
  cute::cluster_wait();  
}
/////////////////////////////////////////////////////

template<uint32_t Stages_, typename ClusterShape_>
struct PipelineTest {

  //
  // Data members
  //
  static constexpr uint32_t Stages = Stages_;
  static constexpr uint32_t kBlockSize = 96;
  using ClusterShape = ClusterShape_;

  //
  // Methods
  //

  // Ctor
  PipelineTest() = default;


  // Run CuTe GEMM kernel
  hipError_t run(uint32_t const kNumIters,
                  hipStream_t stream = nullptr) {

    // Pipeline (multistage pipeline)
    auto num_stages = Int<Stages>{};

    auto cluster_shape = Shape<Int<ClusterShape::kM>, Int<ClusterShape::kN>, _1>{};

    //
    // Configure and launch
    //
    int iterations = 2;
    hipError_t result;

    for (int iter = 0; iter < iterations; ++iter) {

      // Define the tiled MMA layout (static, 4warps)
      using MainloopPipeline = typename cutlass::PipelineAsync<Stages>;

      int smem_size = int(sizeof(SharedStorage<Stages>));

      result = hipFuncSetAttribute(
        pipeline_async_basic_device<decltype(cluster_shape), Stages>,
        hipFuncAttributeMaxDynamicSharedMemorySize,
        smem_size);

      // Launch a single Cluster, with 128 thread per CTA
      dim3 dimCluster(size<0>(cluster_shape), size<1>(cluster_shape), 1);    
      dim3 dimGrid(size<0>(cluster_shape), size<1>(cluster_shape), 1);    
      dim3 dimBlock(kBlockSize,1,1);

      const void* kernel = (const void*)pipeline_async_basic_device<decltype(cluster_shape), Stages>;
      int iters = kNumIters;
      void* kernel_params[] = {reinterpret_cast<void*>(&iters)};
      cutlass::ClusterLauncher::launch(dimGrid, dimCluster, dimBlock, smem_size, stream, kernel, kernel_params);
  
    } // profiling loop ends

    result = hipDeviceSynchronize();

    if (result != hipSuccess) {
      std::cerr << "Error: hipDeviceSynchronize() failed" << std::endl;
      return result;
    }

    return hipSuccess;
  }

};

#if CUDA_12_0_SM90_FEATURES_SUPPORTED
TEST(SM90_Verify_PipelineAsync, Cluster1x1_Stage2) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<1, 1, 1>;
  static constexpr uint32_t Stages = 2;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineAsync, Cluster1x1_Stage5) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<1, 1, 1>;
  static constexpr uint32_t Stages = 5;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineAsync, Cluster1x1_Stage10) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<1, 1, 1>;
  static constexpr uint32_t Stages = 10;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineAsync, Cluster2x2_Stage2) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<2, 2, 1>;
  static constexpr uint32_t Stages = 2;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineAsync, Cluster2x2_Stage5) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<2, 2, 1>;
  static constexpr uint32_t Stages = 5;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineAsync, Cluster2x2_Stage10) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<2, 2, 1>;
  static constexpr uint32_t Stages = 10;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineAsync, Cluster1x2_Stage2) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<1, 2, 1>;
  static constexpr uint32_t Stages = 2;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineAsync, Cluster1x2_Stage7) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<1, 2, 1>;
  static constexpr uint32_t Stages = 7;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineAsync, Cluster1x2_Stage10) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<1, 2, 1>;
  static constexpr uint32_t Stages = 10;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineAsync, Cluster2x1_Stage2) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<2, 1, 1>;
  static constexpr uint32_t Stages = 2;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineAsync, Cluster2x1_Stage7) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<2, 1, 1>;
  static constexpr uint32_t Stages = 7;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineAsync, Cluster4x1_Stage2) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<4, 1, 1>;
  static constexpr uint32_t Stages = 2;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineAsync, Cluster4x1_Stage7) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<4, 1, 1>;
  static constexpr uint32_t Stages = 7;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineAsync, Cluster1x4_Stage2) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<1, 4, 1>;
  static constexpr uint32_t Stages = 2;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineAsync, Cluster1x4_Stage7) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<1, 4, 1>;
  static constexpr uint32_t Stages = 7;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineAsync, Cluster2x4_Stage2) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<2, 4, 1>;
  static constexpr uint32_t Stages = 2;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineAsync, Cluster2x4_Stage7) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<2, 4, 1>;
  static constexpr uint32_t Stages = 7;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineAsync, Cluster4x2_Stage2) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<4, 2, 1>;
  static constexpr uint32_t Stages = 2;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineAsync, Cluster4x2_Stage7) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<4, 2, 1>;
  static constexpr uint32_t Stages = 7;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineAsync, Cluster4x4_Stage2) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<4, 4, 1>;
  static constexpr uint32_t Stages = 2;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineAsync, Cluster4x4_Stage3) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<4, 4, 1>;
  static constexpr uint32_t Stages = 3;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineAsync, Cluster4x4_Stage4) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<4, 4, 1>;
  static constexpr uint32_t Stages = 4;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineAsync, Cluster4x4_Stage5) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<4, 4, 1>;
  static constexpr uint32_t Stages = 5;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineAsync, Cluster4x4_Stage6) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<4, 4, 1>;
  static constexpr uint32_t Stages = 6;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineAsync, Cluster4x4_Stage7) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<4, 4, 1>;
  static constexpr uint32_t Stages = 7;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineAsync, Cluster4x4_Stage8) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<4, 4, 1>;
  static constexpr uint32_t Stages = 8;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineAsync, Cluster4x4_Stage9) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<4, 4, 1>;
  static constexpr uint32_t Stages = 9;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineAsync, Cluster4x4_Stage10) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<4, 4, 1>;
  static constexpr uint32_t Stages = 10;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineAsync, Cluster4x4_Stage11) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<4, 4, 1>;
  static constexpr uint32_t Stages = 11;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}
#endif
