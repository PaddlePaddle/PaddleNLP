#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*! \file
    \brief Unit test for the PipelineTmaAsync class
*/


#define KERNEL_DBG_TRACE false

#include "../common/cutlass_unit_test.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <cute/tensor.hpp>
#include <cute/arch/cluster_sm90.hpp> 

#include <cutlass/util/reference/host/gemm.h>
#include <cutlass/cluster_launch.hpp>

#include "cutlass/core_io.h"

#include "cutlass/util/print_error.hpp"
#include "cutlass/util/GPU_Clock.hpp"

#include "testbed.h"
#include "cutlass/pipeline.hpp"
#include "cutlass/arch/barrier.h"
#include "cute/arch/cluster_sm90.hpp"

using namespace cute;

//////////////////// KERNEL /////////////////////////

template <uint32_t Stages, typename ClusterShape>
struct SharedStorage
{
  typename cutlass::PipelineTmaAsync<Stages, ClusterShape>::SharedStorage storage;
};

// Goal of this kernel is to complete deadlock-free
template <class ClusterShape, uint32_t NumStages>
__global__ static 
void pipeline_device(uint32_t const NumIterations)
{

  extern __shared__ char shared_memory[];
  using DispatchPolicy = cutlass::gemm::MainloopSm90TmaGmma<NumStages, ClusterShape>;
  using MainloopPipeline = cutlass::PipelineTmaAsync<NumStages, ClusterShape>;
  using PipelineState = cutlass::PipelineState<NumStages>;

  using SharedStorage = SharedStorage<NumStages, ClusterShape>;
  SharedStorage& shared_storage = *reinterpret_cast<SharedStorage*>(shared_memory);

  auto cta_layout = Layout<ClusterShape>{}; // (m,n) -> cta_id
  int warp_idx = __shfl_sync(0xffffffff, threadIdx.x / 32, 0);
  int warp_group_thread_idx = threadIdx.x % 128;
  dim3 block_id_in_cluster = cute::block_id_in_cluster();

  auto cluster_shape = ClusterShape{};
  
  // #Producers = #RowsInCluster + #ColsInCluster - 1 
  uint32_t const NumProducers = cute::size<0>(cluster_shape) + cute::size<1>(cluster_shape) - 1;
  uint32_t const TmaTransactionBytes = sizeof(uint32_t) * NumProducers;
  uint32_t const per_cta_bytes = sizeof(uint32_t);

  // mbarrier.init
  typename MainloopPipeline::Params params;
  params.transaction_bytes = TmaTransactionBytes;
  params.role = MainloopPipeline::ThreadCategory::ProducerConsumer;
  params.is_leader = warp_group_thread_idx == 0;
  params.num_consumers = 128;

  MainloopPipeline pipeline(shared_storage.storage, params);

  __syncthreads();

  // Ensure All CTAs in Cluster have completed init before issuing commits
  cute::cluster_arrive_relaxed();  
  cute::cluster_wait();

  // Total number of gemm_k_iterations
  auto mma_k_iterations  = NumIterations;
  auto tma_k_iterations  = NumIterations;

  PipelineState smem_pipe_read;
  // For the DMA (prologue) - we start with an opposite phase - since we skip all waits
  // i.e., we know that the buffer is indeed empty
  PipelineState smem_pipe_write = cutlass::make_producer_start_state<MainloopPipeline>();
  PipelineState smem_pipe_release;
  int K_TILE_MMAS = 1;

  int lane_predicate = cute::elect_one_sync();
  int k_pipe_tma_prologue = min(NumStages, tma_k_iterations);

  // DMA Prologue (Loads)
  CUTLASS_PRAGMA_UNROLL
  for(int i = 0; i < k_pipe_tma_prologue; ++i) {
    pipeline.producer_acquire(smem_pipe_write);
    // cp.async.bulk.tensor would typically happen here
    pipeline.producer_commit(smem_pipe_write.index(), per_cta_bytes);
    ++smem_pipe_write;
  }
  tma_k_iterations -= k_pipe_tma_prologue;

  // MMA Prologue (Compute) - modeling inflight MMAs
  for (int iter = 0; iter < K_TILE_MMAS; ++iter)
  {
    pipeline.consumer_wait(smem_pipe_read);
    warpgroup_arrive();
    // GMMA would typically happen here

    ++smem_pipe_read;
  }

  mma_k_iterations -= K_TILE_MMAS;

  CUTLASS_PRAGMA_NO_UNROLL
  for (int iter = 0; iter < mma_k_iterations; ++iter)
  {
    pipeline.consumer_wait(smem_pipe_read);

    warpgroup_arrive();
    // GMMA would typically happen here

    pipeline.consumer_release(smem_pipe_release);

    if (lane_predicate && (warp_idx == 0) && (tma_k_iterations > 0)) {
      pipeline.producer_acquire(smem_pipe_write);
      // cp.async.bulk.tensor would typically happen here
      pipeline.producer_commit(smem_pipe_write.index(), per_cta_bytes);
      ++smem_pipe_write;
      --tma_k_iterations;
    }

    // next read stage
    ++smem_pipe_read;
    ++smem_pipe_release;
  }

  // To make sure remote SMEM doesn't get destoryed
  cute::cluster_arrive();  
  cute::cluster_wait();
}
/////////////////////////////////////////////////////

/// Device NT GMMA + TMA specialized
template<uint32_t Stages_, typename ClusterShape_>
struct PipelineTest {

  //
  // Data members
  //
  static constexpr uint32_t Stages = Stages_;
  static constexpr uint32_t kBlockSize = 128;
  using ClusterShape = ClusterShape_;

  //
  // Methods
  //

  // Ctor
  PipelineTest(){};


  // Run CuTe GEMM kernel
  hipError_t run(uint32_t const kNumIters,
                  hipStream_t stream = 0) {

    float elapsed_ms = 0.0f;
    // Pipeline (multistage pipeline)
    auto num_stages = Int<Stages>{};

    auto cluster_shape = Shape<Int<ClusterShape::kM>, Int<ClusterShape::kN>, _1>{};

    //
    // Configure and launch
    //
    int iterations = 1;
    hipEvent_t events[2];
    hipError_t result;

    for (hipEvent_t & event : events) {
      result = hipEventCreate(&event);
      if (result != hipSuccess) {
        std::cerr << "Error: Failed to create event.";
        return result;
      }
    }

    result = hipEventRecord(events[0]);

    if (result != hipSuccess) {
      std::cerr << "Error: Failed to record start event.";
      return result;
    }

    for (int iter = 0; iter < iterations; ++iter) {

      // Define the tiled MMA layout (static, 4warps)
      using DispatchPolicy = cutlass::gemm::MainloopSm90TmaGmma<Stages, decltype(cluster_shape)>;
      using MainloopPipeline = typename cutlass::PipelineTmaAsync<Stages, decltype(cluster_shape)>;

      int smem_size = int(sizeof(SharedStorage<Stages, decltype(cluster_shape)>));

      result = hipFuncSetAttribute(
        pipeline_device<decltype(cluster_shape), Stages>,
        hipFuncAttributeMaxDynamicSharedMemorySize,
        smem_size);

      // Launch a single Cluster, with 128 thread per CTA
      dim3 dimCluster(size<0>(cluster_shape), size<1>(cluster_shape), 1);    
      dim3 dimGrid(size<0>(cluster_shape), size<1>(cluster_shape), 1);    
      dim3 dimBlock(kBlockSize,1,1);

      const void* kernel = (const void*)pipeline_device<decltype(cluster_shape), Stages>;
      int iters = kNumIters;
      void* kernel_params[] = {reinterpret_cast<void*>(&iters)};
      cutlass::ClusterLauncher::launch(dimGrid, dimCluster, dimBlock, smem_size, stream, kernel, kernel_params);
  
    } // profiling loop ends

    result = hipEventRecord(events[1]);

    if (result != hipSuccess) {
      std::cerr << "Error: Failed to record stop event.";
      return result;
    }

    result = hipDeviceSynchronize();

    if (result != hipSuccess) {
      std::cerr << "Error: hipDeviceSynchronize() failed" << std::endl;
      return result;
    }

    result = hipEventElapsedTime(&elapsed_ms, events[0], events[1]);

    if (result != hipSuccess) {
      std::cerr << "Failed to create event.";
      return result;
    }

    for (hipEvent_t & event : events) {
      (void)hipEventDestroy(event);
    }

    return hipSuccess;
  }
};

#if CUDA_12_0_SM90_FEATURES_SUPPORTED
TEST(SM90_Verify_PipelineTmaAsync, Cluster1x1_Stage2) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<1, 1, 1>;
  static constexpr uint32_t Stages = 2;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineTmaAsync, Cluster1x1_Stage5) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<1, 1, 1>;
  static constexpr uint32_t Stages = 5;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineTmaAsync, Cluster1x1_Stage10) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<1, 1, 1>;
  static constexpr uint32_t Stages = 10;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineTmaAsync, Cluster2x2_Stage2) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<2, 2, 1>;
  static constexpr uint32_t Stages = 2;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineTmaAsync, Cluster2x2_Stage5) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<2, 2, 1>;
  static constexpr uint32_t Stages = 5;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineTmaAsync, Cluster2x2_Stage10) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<2, 2, 1>;
  static constexpr uint32_t Stages = 10;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineTmaAsync, Cluster4x4_Stage2) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<4, 4, 1>;
  static constexpr uint32_t Stages = 2;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineTmaAsync, Cluster4x4_Stage10) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<4, 4, 1>;
  static constexpr uint32_t Stages = 10;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineTmaAsync, Cluster1x2_Stage2) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<1, 2, 1>;
  static constexpr uint32_t Stages = 2;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineTmaAsync, Cluster1x2_Stage7) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<1, 2, 1>;
  static constexpr uint32_t Stages = 7;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineTmaAsync, Cluster1x2_Stage10) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<1, 2, 1>;
  static constexpr uint32_t Stages = 10;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineTmaAsync, Cluster2x1_Stage2) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<2, 1, 1>;
  static constexpr uint32_t Stages = 2;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineTmaAsync, Cluster2x1_Stage7) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<2, 1, 1>;
  static constexpr uint32_t Stages = 7;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineTmaAsync, Cluster4x1_Stage2) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<4, 1, 1>;
  static constexpr uint32_t Stages = 2;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineTmaAsync, Cluster4x1_Stage7) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<4, 1, 1>;
  static constexpr uint32_t Stages = 7;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineTmaAsync, Cluster1x4_Stage2) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<1, 4, 1>;
  static constexpr uint32_t Stages = 2;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineTmaAsync, Cluster1x4_Stage7) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<1, 4, 1>;
  static constexpr uint32_t Stages = 7;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineTmaAsync, Cluster2x4_Stage2) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<2, 4, 1>;
  static constexpr uint32_t Stages = 2;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineTmaAsync, Cluster2x4_Stage7) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<2, 4, 1>;
  static constexpr uint32_t Stages = 7;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineTmaAsync, Cluster4x2_Stage2) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<4, 2, 1>;
  static constexpr uint32_t Stages = 2;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineTmaAsync, Cluster4x2_Stage7) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<4, 2, 1>;
  static constexpr uint32_t Stages = 7;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}
#endif
