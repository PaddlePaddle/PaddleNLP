#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*! \file
    \brief Unit test for the PipelineTmaAsync class used in a WarpSpecialized Persistent loop
*/

#define KERNEL_DBG_TRACE false

#include "../common/cutlass_unit_test.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <cute/tensor.hpp>
#include <cute/arch/cluster_sm90.hpp> 

#include <cutlass/util/reference/host/gemm.h>
#include <cutlass/cluster_launch.hpp>

#include "cutlass/core_io.h"
#include "cutlass/util/print_error.hpp"
#include "cutlass/util/GPU_Clock.hpp"

#include "testbed.h"
#include "cutlass/pipeline.hpp"
#include "cutlass/arch/barrier.h"
#include "cute/arch/cluster_sm90.hpp"
#include "cutlass/arch/barrier.h"
#include "cutlass/arch/reg_reconfig.h"


using namespace cute;
using namespace cutlass;

//////////////////// KERNEL /////////////////////////

template <uint32_t Stages, typename ClusterShape, typename PingPongBarrier>
struct SharedStorage
{
  typename cutlass::PipelineTmaAsync<Stages, ClusterShape>::SharedStorage pipeline_storage;
  typename PingPongBarrier::SharedStorage pingpong_storage;
};

template <typename ClusterShape, uint32_t Stages>
struct CollectiveSimulation {
  using MainloopPipeline = typename cutlass::PipelineTmaAsync<Stages, ClusterShape>;
  using PipelineState = typename cutlass::PipelineState<Stages>;

  CUTLASS_DEVICE
  static void
  dma_wg_simulation(MainloopPipeline pipeline, PipelineState tile_start_state_pipe,
                    uint32_t const num_iterations) {
    uint32_t const per_cta_bytes = sizeof(uint32_t);
    int warp_idx_in_warpgroup = __shfl_sync(0xffffffff, (threadIdx.x / 32) % 4, 0);
    int lane_predicate = cute::elect_one_sync();
    if (warp_idx_in_warpgroup==0 && lane_predicate) {

      int tma_k_prologue = min(Stages, num_iterations);

      // Simulating Prologue TMA Loads
      CUTLASS_PRAGMA_UNROLL
      for(int i = 0; i < tma_k_prologue; ++i) {
        pipeline.producer_acquire(tile_start_state_pipe);
        // Simulating cp.async.bulk.tensor behavior
        pipeline.producer_commit(tile_start_state_pipe.index(), per_cta_bytes);
        ++tile_start_state_pipe;
      }
      int tma_k_iter = num_iterations - tma_k_prologue;

      PipelineState wr_pipe = tile_start_state_pipe;
      // Simulating Mainloop TMA Loads
      CUTE_NO_UNROLL
      for ( ; tma_k_iter > 0; --tma_k_iter){

        pipeline.producer_acquire(wr_pipe);

        // Simulating cp.async.bulk.tensor behavior
        pipeline.producer_commit(wr_pipe.index(), per_cta_bytes);

        // Advance write stage
        ++wr_pipe;
      }
    }
  }

  CUTLASS_DEVICE
  static void
  math_wg_simulation(MainloopPipeline pipeline, PipelineState tile_start_state_pipe,
                    uint32_t const num_iterations, int* data_ptr) {
    PipelineState rd_pipe = tile_start_state_pipe;
    PipelineState release_pipe = rd_pipe;

    // simulates accumulators + extra reg. pressure
    int arr[168];

    // Init Shared Memory read stages & PhaseBit
    static constexpr uint32_t K_PIPE_MMAS = 1;
    static_assert( K_PIPE_MMAS < Stages, "ERROR : Too many MMAs in flight");

    // Total number of gemm iterations
    auto gemm_k_iterations  = num_iterations;

    // Simulating Prologue MMAs
    int mma_k_prologue = min(K_PIPE_MMAS, gemm_k_iterations);
    CUTLASS_PRAGMA_UNROLL
    for (int iter = 0; iter < mma_k_prologue; ++iter) {
      pipeline.consumer_wait(rd_pipe);

      warpgroup_arrive();
      // GMMA would typically happen here

      ++rd_pipe;
    }
    gemm_k_iterations -= mma_k_prologue;

    // Simulating Mainloop MMAs
    CUTLASS_PRAGMA_NO_UNROLL
    for ( ; gemm_k_iterations > 0; --gemm_k_iterations) {

      /// Wait on the rd_pipe stage / phase
      pipeline.consumer_wait(rd_pipe);

      warpgroup_arrive();
      // GMMA would typically happen here

      // Dummy op - which will never happen
      // But simulates high register usage.
      CUTE_UNROLL
      for(int i = 0; i < 168; ++i){
        if (threadIdx.x > 384){
          arr[i] += data_ptr[i];
        }
      }

      pipeline.consumer_release(release_pipe);

      // Advance stages
      ++rd_pipe;
      ++release_pipe;
    }

    // Dummy op - which will never happen
    CUTE_UNROLL
    for(int i = 0; i < 168; ++i){
      if (threadIdx.x > 384){
        data_ptr[i] = arr[i];
      }
    }

    // Tail Loop
    for (int i = 0; i < K_PIPE_MMAS; ++i){
      pipeline.consumer_release(release_pipe);
      ++release_pipe;
    }

  }
};

struct KernelParams
{
  uint32_t num_iterations;
  int tiles_per_cluster;
  int* data_ptr;
};

// Goal of this kernel is to complete deadlock-free
template <typename ClusterShape, uint32_t Stages>
__launch_bounds__(384, 1)
__global__ static
void pipeline_device(KernelParams params)
{
  extern __shared__ char shared_memory[];
  using DispatchPolicy = cutlass::gemm::MainloopSm90TmaGmmaWarpSpecialized<Stages,
                          ClusterShape,
                          cutlass::gemm::KernelTmaWarpSpecializedPersistent>;
  using MainloopPipeline = typename cutlass::PipelineTmaAsync<Stages, ClusterShape>;
  using PipelineState = typename cutlass::PipelineState<Stages>;

  /* One for Mainloop and one for Epilogue */
  constexpr int StagesPerMathWarpGroup = 2;
  constexpr int MathWarpGroupCountPersistent = 2;
  using PingPongBarrier = typename cutlass::OrderedSequenceBarrier<StagesPerMathWarpGroup, MathWarpGroupCountPersistent>;

  using SharedStorage = SharedStorage<Stages, ClusterShape, PingPongBarrier>;
  SharedStorage& shared_storage = *reinterpret_cast<SharedStorage*>(shared_memory);

  auto cta_layout = Layout<ClusterShape>{};            // (m,n) -> cta_id
  int warp_group_idx = __shfl_sync(0xffffffff, threadIdx.x / NumThreadsPerWarpGroup, 0);
  int warp_group_thread_idx = threadIdx.x % NumThreadsPerWarpGroup;
  dim3 block_id_in_cluster = cute::block_id_in_cluster();

  auto cluster_shape = ClusterShape{};

  // #Producers = #RowsInCluster + #ColsInCluster - 1 
  uint32_t const NumProducers = cute::size<0>(cluster_shape) + cute::size<1>(cluster_shape) - 1;
  uint32_t const TmaTransactionBytes = static_cast<uint32_t>(sizeof(uint32_t) * NumProducers);

  // mbarrier.init
  typename MainloopPipeline::Params pipeline_params;
  pipeline_params.transaction_bytes = TmaTransactionBytes;
  if (warp_group_idx == 0) {
    pipeline_params.role = MainloopPipeline::ThreadCategory::Producer;
  }
  else {
    pipeline_params.role = MainloopPipeline::ThreadCategory::Consumer;
  }
  pipeline_params.is_leader = warp_group_thread_idx == 0;
  pipeline_params.num_consumers = NumThreadsPerWarpGroup;

  MainloopPipeline pipeline(shared_storage.pipeline_storage, pipeline_params);
  PipelineState tile_start_state_pipe;

  int tiles_per_cluster = params.tiles_per_cluster;

  /* Offset pipeline start state for Math WG 2 */
  if (warp_group_idx == 2) {
    // Update pipeline state for next persistent tile
    tile_start_state_pipe.advance(params.num_iterations);
    tiles_per_cluster--;
  }

  typename PingPongBarrier::Params pingpong_params;
  pingpong_params.group_id = warp_group_idx - 1; // Since DMA Warp Group Idx 0 will not participate
  pingpong_params.group_size = NumThreadsPerWarpGroup; // Number of threads / participants in a group
  PingPongBarrier math_wg_barrier(shared_storage.pingpong_storage, pingpong_params);

  __syncthreads();

  // Ensure All CTAs in Cluster have completed init before issuing commits
  cute::cluster_arrive_relaxed();  
  cute::cluster_wait();

  // Producer/DMA WarpGroup
  if (warp_group_idx == 0) {
    cutlass::arch::warpgroup_reg_dealloc<40>();
    // For the DMA (prologue) - we start with an opposite phase - since we skip all waits
    // i.e., we know that the buffer is indeed empty
    PipelineState tile_prologue_state_pipe = make_producer_start_state<MainloopPipeline>();
    while (tiles_per_cluster > 0) {
      CollectiveSimulation<ClusterShape,Stages>::dma_wg_simulation(pipeline, tile_prologue_state_pipe, params.num_iterations);
      // Update pipeline state for next persistent tile
      tile_prologue_state_pipe.advance(params.num_iterations);
      tiles_per_cluster--;
    }
  }
  // Math WarpGropups
  if(warp_group_idx == 1 || warp_group_idx == 2) {
    cutlass::arch::warpgroup_reg_alloc<232>();
    while (tiles_per_cluster > 0) {
      // MMA
      math_wg_barrier.wait();
      CollectiveSimulation<ClusterShape,Stages>::math_wg_simulation(pipeline, tile_start_state_pipe, params.num_iterations, params.data_ptr);
      math_wg_barrier.arrive();
      // Epilogue
      math_wg_barrier.wait();
      // Simulates long running stage
      #if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 700)
      __nanosleep(100000);
      #endif
      math_wg_barrier.arrive();
      // Update pipeline state for next persistent tile
      tile_start_state_pipe.advance(params.num_iterations * 2);
      tiles_per_cluster -= 2;
    }
  }

  // Makes sure remote SMEM doesn't get destroyed
  cute::cluster_arrive_relaxed();  
  cute::cluster_wait();
}
/////////////////////////////////////////////////////

/// Device NT GMMA + TMA specialized
template<uint32_t Stages_, typename ClusterShape_>
struct PipelineTest {

  //
  // Data members
  //
  static constexpr uint32_t Stages = Stages_;
  static constexpr uint32_t kBlockSize = 128 * 3;
  using ClusterShape = ClusterShape_;

  //
  // Methods
  //

  // Run CuTe GEMM kernel
  hipError_t run(uint32_t const kNumIters,
                  hipStream_t stream = 0) {

    float elapsed_ms = 0.0f;
    // Pipeline (multistage pipeline)
    auto num_stages = Int<Stages>{}; 
    auto cluster_shape = Shape<Int<ClusterShape::kM>, Int<ClusterShape::kN>, _1>{};

    //
    // Configure and launch
    //
    int iterations = 1;
    hipEvent_t events[2];
    hipError_t result;

    for (hipEvent_t & event : events) {
      result = hipEventCreate(&event);
      if (result != hipSuccess) {
        std::cerr << "Error: Failed to create event.";
        return result;
      }
    }

    result = hipEventRecord(events[0]);

    if (result != hipSuccess) {
      std::cerr << "Error: Failed to record start event.";
      return result;
    }

    for (int iter = 0; iter < iterations; ++iter) {
    
      using MainloopPipeline = typename cutlass::PipelineTmaAsync<Stages, decltype(cluster_shape)>;

      constexpr int StagesPerMathWarpGroup = 2;
      constexpr int MathWarpGroupCountPersistent = 2;
      int smem_size = int(sizeof(SharedStorage<Stages, decltype(cluster_shape), 
                          typename cutlass::OrderedSequenceBarrier<StagesPerMathWarpGroup, MathWarpGroupCountPersistent>>));

      result = hipFuncSetAttribute(
        pipeline_device<decltype(cluster_shape), Stages>,
        hipFuncAttributeMaxDynamicSharedMemorySize,
        smem_size);

      // Launch a single Cluster, with kBlockSize threads per CTA
      dim3 dimCluster(size<0>(cluster_shape), size<1>(cluster_shape), 1);    
      dim3 dimGrid(size<0>(cluster_shape), size<1>(cluster_shape), 1);    
      dim3 dimBlock(kBlockSize,1,1);

      int tiles_per_cluster = (kNumIters % 10) + 1;
      printf("Persistent version: Tiles per Cluster = %d\n", tiles_per_cluster);

      const void* kernel = (const void*)pipeline_device<decltype(cluster_shape), Stages>;
      KernelParams params{kNumIters, tiles_per_cluster, nullptr};
      void *kernel_params[] = {&params};
      cutlass::ClusterLauncher::launch(dimGrid, dimCluster, dimBlock, smem_size, stream, kernel, kernel_params);

    }

    result = hipEventRecord(events[1]);

    if (result != hipSuccess) {
      std::cerr << "Error: Failed to record stop event.";
      return result;
    }

    result = hipDeviceSynchronize();

    if (result != hipSuccess) {
      std::cerr << "Error: hipDeviceSynchronize() failed" << std::endl;
      return result;
    }

    result = hipEventElapsedTime(&elapsed_ms, events[0], events[1]);

    if (result != hipSuccess) {
      std::cerr << "Failed to create event.";
      return result;
    }

    for (hipEvent_t & event : events) {
      (void)hipEventDestroy(event);
    }

    return hipSuccess;
  }
};

#if CUDA_12_0_SM90_FEATURES_SUPPORTED
TEST(SM90_Verify_PipelineTmaAsync_WS_Persistent, Cluster1x1_Stage2) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<1, 1, 1>;
  static constexpr uint32_t Stages = 2;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineTmaAsync_WS_Persistent, Cluster1x1_Stage5) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<1, 1, 1>;
  static constexpr uint32_t Stages = 5;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineTmaAsync_WS_Persistent, Cluster1x1_Stage10) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<1, 1, 1>;
  static constexpr uint32_t Stages = 10;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineTmaAsync_WS_Persistent, Cluster2x2_Stage2) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<2, 2, 1>;
  static constexpr uint32_t Stages = 2;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineTmaAsync_WS_Persistent, Cluster2x2_Stage5) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<2, 2, 1>;
  static constexpr uint32_t Stages = 5;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineTmaAsync_WS_Persistent, Cluster2x2_Stage7) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<2, 2, 1>;
  static constexpr uint32_t Stages = 7;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineTmaAsync_WS_Persistent, Cluster4x4_Stage2) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<4, 4, 1>;
  static constexpr uint32_t Stages = 2;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineTmaAsync_WS_Persistent, Cluster4x4_Stage7) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<4, 4, 1>;
  static constexpr uint32_t Stages = 7;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineTmaAsync_WS_Persistent, Cluster2x1_Stage2) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<2, 1, 1>;
  static constexpr uint32_t Stages = 2;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineTmaAsync_WS_Persistent, Cluster2x1_Stage7) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<2, 1, 1>;
  static constexpr uint32_t Stages = 7;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineTmaAsync_WS_Persistent, Cluster1x2_Stage2) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<1, 2, 1>;
  static constexpr uint32_t Stages = 2;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineTmaAsync_WS_Persistent, Cluster1x2_Stage7) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<1, 2, 1>;
  static constexpr uint32_t Stages = 7;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineTmaAsync_WS_Persistent, Cluster4x1_Stage2) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<4, 1, 1>;
  static constexpr uint32_t Stages = 2;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineTmaAsync_WS_Persistent, Cluster4x1_Stage7) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<4, 1, 1>;
  static constexpr uint32_t Stages = 7;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineTmaAsync_WS_Persistent, Cluster1x4_Stage2) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<1, 4, 1>;
  static constexpr uint32_t Stages = 2;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineTmaAsync_WS_Persistent, Cluster1x4_Stage7) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<1, 4, 1>;
  static constexpr uint32_t Stages = 7;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineTmaAsync_WS_Persistent, Cluster2x4_Stage2) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<2, 4, 1>;
  static constexpr uint32_t Stages = 2;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineTmaAsync_WS_Persistent, Cluster2x4_Stage7) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<2, 4, 1>;
  static constexpr uint32_t Stages = 7;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineTmaAsync_WS_Persistent, Cluster4x2_Stage2) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<4, 2, 1>;
  static constexpr uint32_t Stages = 2;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}

TEST(SM90_Verify_PipelineTmaAsync_WS_Persistent, Cluster4x2_Stage7) {
  Options options;
  using ClusterShape = cutlass::gemm::GemmShape<4, 2, 1>;
  static constexpr uint32_t Stages = 7;
  using Test = PipelineTest<Stages, ClusterShape>;
  Testbed<Test> testbed(options);
  EXPECT_TRUE(testbed.verification());
}
#endif
