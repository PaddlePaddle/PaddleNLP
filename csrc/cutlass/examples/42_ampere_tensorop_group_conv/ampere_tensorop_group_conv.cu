/***************************************************************************************************
 * Copyright (c) 2017 - 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/**
This example shows how to run group convolution kernels using functions and data structures
provided by CUTLASS using tensor cores; which we run on a NVIDIA Ampere GPU.

There are 2 group conv mode:
  1. cutlass::conv::GroupMode::kSingleGroup
      This mode is for large K problem size: k_per_group (K/groups) equals or larger than
      threadblock_tile_N. One or multiple threadblocks calculate data of one group.
  2. cutlass::conv::GroupMode::kMultipleGroup
      This mode is for small K problem size: k_per_group (K/groups) is smaller than threadblock_tile_N.
      One threadblock will calculate data from more than one group.

Function profile_convolution_selecter() shows how to choose kernel with different group mode according
to problem size and threadblock_tile size.
*/

#include <iostream>
#include <sstream>

#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm.h"
#include "cutlass/conv/kernel/default_conv2d_group_fprop.h"
#include "cutlass/conv/device/implicit_gemm_convolution.h"

#include "cutlass/util/command_line.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/device/gemm.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/host/convolution.h"
#include "cutlass/util/reference/device/convolution.h"
#include "cutlass/util/tensor_view_io.h"

#include "helper.h"

// The code section below describes datatype for input, output tensors and computation between
// elements 
using ElementAccumulator = float;                  // Data type of accumulator
using ElementComputeEpilogue = float;              // Data type of epilogue computation (alpha, beta)
using ElementInputA = cutlass::half_t;             // Data type of elements in input tensor
using ElementInputB = cutlass::half_t;             // Data type of elements in input tensor
using ElementOutput = float;                       // Data type of elements in output tensor

using LayoutInputA = cutlass::layout::TensorNHWC;
using LayoutInputB = cutlass::layout::TensorNHWC;
using LayoutOutput = cutlass::layout::TensorNHWC;

// This code section describes whether you want to use tensor cores or regular SIMT cores on GPU SM
using MMAOp = cutlass::arch::OpClassTensorOp;

// This code section describes CUDA SM architecture number
using SmArch = cutlass::arch::Sm80;

// This code section describes the tile size a thread block will compute
using ThreadblockShape = cutlass::gemm::GemmShape<64, 64, 64>;   // Threadblock tile shape

// This code section describes tile size a warp will compute
using WarpShape = cutlass::gemm::GemmShape<32, 32, 64>;          // Warp tile shape

// This code section describes the size of MMA op
using InstructionShape = cutlass::gemm::GemmShape<16, 8, 16>;    // TensorCore instruction shape

// This code section describes how threadblocks are scheduled on GPU
using SwizzleThreadBlock = cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>;

// Number of pipelines you want to use
constexpr int NumStages = 3;

// This code section describes the epilogue part of the kernel, we use default value
using EpilogueOp = cutlass::epilogue::thread::LinearCombination<
    ElementOutput,                                     // Data type of output matrix.
    128 / cutlass::sizeof_bits<ElementOutput>::value,  // The number of elements per vectorized.
                                                       // memory access. This becomes the vector width of
                                                       // math instructions in the epilogue too.
    ElementAccumulator,                                // Data type of accumulator
    ElementComputeEpilogue>;                           // Data type for alpha/beta in linear combination

// Analytic kernel and operation for single group problem size
using AnalyticSingleGroupKernel = typename cutlass::conv::kernel::DefaultConv2dGroupFprop<
  ElementInputA, LayoutInputA,
  ElementInputB, LayoutInputB,
  ElementOutput, LayoutOutput,
  ElementAccumulator,
  MMAOp,
  SmArch,
  ThreadblockShape,
  WarpShape,
  InstructionShape,
  EpilogueOp,
  SwizzleThreadBlock,
  NumStages,
  cutlass::arch::OpMultiplyAdd,
  cutlass::conv::GroupMode::kSingleGroup,
  cutlass::conv::IteratorAlgorithm::kAnalytic
>::Kernel;
using AnalyticSingleGroupOperation = cutlass::conv::device::ImplicitGemmConvolution<AnalyticSingleGroupKernel>;

// Analytic kernel and operation for multiple group problem size
using AnalyticMultipleGroupKernel = typename cutlass::conv::kernel::DefaultConv2dGroupFprop<
  ElementInputA, LayoutInputA,
  ElementInputB, LayoutInputB,
  ElementOutput, LayoutOutput,
  ElementAccumulator,
  MMAOp,
  SmArch,
  ThreadblockShape,
  WarpShape,
  InstructionShape,
  EpilogueOp,
  SwizzleThreadBlock,
  NumStages,
  cutlass::arch::OpMultiplyAdd,
  cutlass::conv::GroupMode::kMultipleGroup,
  cutlass::conv::IteratorAlgorithm::kAnalytic
>::Kernel;
using AnalyticMultipleGroupOperation = cutlass::conv::device::ImplicitGemmConvolution<AnalyticMultipleGroupKernel>;

// Optimized kernel and operation for single group problem size
using OptimizedSingleGroupKernel = typename cutlass::conv::kernel::DefaultConv2dGroupFprop<
  ElementInputA, LayoutInputA,
  ElementInputB, LayoutInputB,
  ElementOutput, LayoutOutput,
  ElementAccumulator,
  MMAOp,
  SmArch,
  ThreadblockShape,
  WarpShape,
  InstructionShape,
  EpilogueOp,
  SwizzleThreadBlock,
  NumStages,
  cutlass::arch::OpMultiplyAdd,
  cutlass::conv::GroupMode::kSingleGroup,
  cutlass::conv::IteratorAlgorithm::kOptimized
>::Kernel;
using OptimizedSingleGroupOperation = cutlass::conv::device::ImplicitGemmConvolution<OptimizedSingleGroupKernel>;

/////////////////////////////////////////////////////////////////////////////////////////////////

// Command line options parsing
struct Options {

  bool help;
  cutlass::Tensor4DCoord input_size;
  cutlass::Tensor4DCoord filter_size;
  cutlass::Tensor4DCoord padding;
  cutlass::MatrixCoord conv_stride;
  cutlass::MatrixCoord dilation;
  int groups;
  bool reference_check;
  bool measure_performance;
  int iterations;
  ElementComputeEpilogue alpha;
  ElementComputeEpilogue beta;
  bool optimized;
  std::string tag;

  Options():
    help(false),
    input_size(1, 32, 32, 32),
    filter_size(32, 3, 3, 32),
    padding(1, 1, 1, 1),
    conv_stride(1, 1),
    dilation(1, 1),
    groups(1),
    reference_check(false),
    measure_performance(false),
    iterations(20),
    alpha(1),
    beta(0),
    optimized(false) { }

  // Verify the problem size is compatible with the CUTLASS Convolution implementation.
  bool valid() {

    //
    // CUTLASS attempts to load 128b vectors of cutlass::half_t (F16) elements. Consequently,
    // all pointers, strides, and tensor extents must be divisible by 8 elements.
    //
    int const kAlignment = 8;

    if ((input_size.c() % kAlignment) ||
      (filter_size.n() % kAlignment)) {

      // misaligned tensors
      return false;
    }

    // Invalid padding
    if ((padding.h() != filter_size.h() / 2) ||
      (padding.w() != filter_size.w() / 2)) {

      return false;
    }

    return true;
  }

  /// Updates input and filter sizes
  void update(
    cutlass::Tensor4DCoord input_size,
    cutlass::Tensor4DCoord filter_size) {

    this->input_size = input_size;
    this->filter_size = filter_size;

    padding.n() = filter_size.h() / 2;
    padding.h() = filter_size.h() / 2;
    padding.w() = filter_size.w() / 2;
    padding.c() = filter_size.w() / 2;
  }

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
    }

    if (cmd.check_cmd_line_flag("ref-check")) {
      reference_check = true;
    }

    if (cmd.check_cmd_line_flag("perf-check")) {
      measure_performance = true;
    }

    if (cmd.check_cmd_line_flag("optimized")) {
      optimized = true;
    }

    cmd.get_cmd_line_argument("n", input_size.n());
    cmd.get_cmd_line_argument("h", input_size.h());
    cmd.get_cmd_line_argument("w", input_size.w());
    cmd.get_cmd_line_argument("c", input_size.c());

    cmd.get_cmd_line_argument("k", filter_size.n());
    cmd.get_cmd_line_argument("r", filter_size.h());
    cmd.get_cmd_line_argument("s", filter_size.w());

    cmd.get_cmd_line_argument("g", groups);
    filter_size.c() = input_size.c() / groups;

    cmd.get_cmd_line_argument("u", conv_stride.row());
    cmd.get_cmd_line_argument("v", conv_stride.column());

    cmd.get_cmd_line_argument("alpha", alpha);
    cmd.get_cmd_line_argument("beta", beta);
    
    cmd.get_cmd_line_argument("iterations", iterations);
    cmd.get_cmd_line_argument("tag", tag);

    if (filter_size.h() == 3 && filter_size.w() == 3) {
      padding = {1, 1, 1, 1};
    }
    else {
      filter_size.h() = 1;
      filter_size.w() = 1;
      padding = {0, 0, 0, 0};
    }
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "42_ampere_tensorop_group_conv example\n\n"
      << "  This example uses Ampere's Tensor Core operators on F16 data types to compute\n"
      << "  forward grouped convolution on tensors of layout NHWC.\n\n"
      << "Options:\n\n"
      << "  --help               If specified, displays this usage statement.\n\n"
      << "  --n=<int>            Input tensor extent N\n"
      << "  --h=<int>            Input tensor extent H\n"
      << "  --w=<int>            Input tensor extent W\n"
      << "  --c=<int>            Input tensor extent C\n"
      << "  --k=<int>            Filter extent K\n"
      << "  --r=<int>            Filter extent R\n"
      << "  --s=<int>            Filter extent S\n\n"
      << "  --g=<int>            Conv groups G\n\n"
      << "  --u=<int>            Conv stride_h\n\n"
      << "  --v=<int>            Conv stride_w\n\n"
      << "  --alpha=<float>      Epilogue scalar alpha\n"
      << "  --beta=<float>       Epilogue scalar beta\n\n"
      << "  --ref-check          If set (true), reference check is computed\n"
      << "  --perf-check         If set (true), performance is measured.\n"
      << "  --optimized          If set (true), use optimized kernel, otherwise use analytic kernel.\n"
      << "  --iterations=<int>   Number of profiling iterations to perform.\n"
      << "  --tag=<string>       String to replicate across the first column in the results table\n";

    out << "\n\nExamples:\n\n"
      << "$ ./examples/42_ampere_tensorop_group_conv/42_ampere_tensorop_group_conv  --n=4 --h=16 --w=16 --c=256 --k=128 --r=3 --s=3 --g=8 --ref-check\n\n"
      << "$ ./examples/42_ampere_tensorop_group_conv/42_ampere_tensorop_group_conv  --n=4 --h=16 --w=16 --c=256 --k=128 --r=3 --s=3 --g=2 --ref-check\n\n"
      << "$ ./examples/42_ampere_tensorop_group_conv/42_ampere_tensorop_group_conv  --n=4 --h=16 --w=16 --c=256 --k=128 --r=3 --s=3 --g=2 --ref-check --optimized\n\n";

    return out;
  }
  
  /// Computes the output tensor size (NPQK)
  cutlass::Tensor4DCoord output_size() const {
    return cutlass::Tensor4DCoord(
      input_size.n(),
      (input_size.h() + padding.n() + padding.h() - filter_size.h()) / conv_stride.row() + 1,
      (input_size.w() + padding.w() + padding.c() - filter_size.w()) / conv_stride.column() + 1,
      filter_size.n());
  }

  /// Compute performance in GFLOP/s
  double gflops(double runtime_s) const {

    // Number of multiply-adds = NPQK * CRS
    int64_t fmas = output_size().product() * int64_t(filter_size.h() * filter_size.w() * filter_size.c());
    
    // Two flops per multiply-add
    return 2.0 * double(fmas) / double(1.0e9) / runtime_s;
  }
};

/////////////////////////////////////////////////////////////////////////////////////////////////

struct Result {
  double runtime_ms;
  double gflops;
  cutlass::Status status;
  cutlass::Status reference_check;
  hipError_t error;

  Result(): 
    runtime_ms(0), 
    gflops(0),
    status(cutlass::Status::kSuccess),
    reference_check(cutlass::Status::kInvalid),
    error(hipSuccess) { }

  static std::ostream & print_header(std::ostream &out, Options const &options) {

    if (!options.tag.empty()) {
      out << "Name,";
    }

    out << "Layer,N,H,W,C,K,R,S,G,Runtime,GFLOPs";

    return out;
  }

  std::ostream & print(std::ostream &out, int idx, Options const &options) {

    if (!options.tag.empty()) {
      out << options.tag << ",";
    }

    out
      << "conv_" << idx << ","
      << options.input_size.n() << ","
      << options.input_size.h() << ","
      << options.input_size.w() << ","
      << options.input_size.c() << ","
      << options.filter_size.n() << ","
      << options.filter_size.h() << ","
      << options.filter_size.w() << ","
      << options.groups << ","
      << runtime_ms << ","
      << gflops;

    return out;
  }
};

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Runs one benchmark
template <typename Conv2dOperation>
Result profile_convolution(Options const &options) {

  Result result;

  //
  // Allocate host-device tensors using the CUTLASS Utilities.
  //

  cutlass::HostTensor<ElementInputA, LayoutInputA> tensor_a(options.input_size);
  cutlass::HostTensor<ElementInputB, LayoutInputB> tensor_b(options.filter_size);
  cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_c(options.output_size());
  cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_d(options.output_size());
  cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_ref_d(options.output_size());

  //
  // Initialize tensors
  //

  // Fill tensor A on host with uniform-distribution random data
  cutlass::reference::host::TensorFillRandomUniform(
      tensor_a.host_view(),
      1,
      ElementInputA(7),
      ElementInputA(-8),
      0);

  // Fill tensor B on host with uniform-distribution random data
  cutlass::reference::host::TensorFillRandomUniform(
      tensor_b.host_view(),
      1,
      ElementInputB(7),
      ElementInputB(-8),
      0);

  // Fill tensor C on host with uniform-distribution random data
  cutlass::reference::host::TensorFillRandomUniform(
      tensor_c.host_view(),
      1,
      ElementOutput(7),
      ElementOutput(-8),
      0);

  // Fill tensor D on host with zeros
  cutlass::reference::host::TensorFill(
      tensor_d.host_view());

  // Fill tensor D for reference on host with zeros
  cutlass::reference::host::TensorFill(
      tensor_ref_d.host_view());

  // Copy data from host to GPU
  tensor_a.sync_device();
  tensor_b.sync_device();
  tensor_c.sync_device();
  tensor_d.sync_device();
  tensor_ref_d.sync_device();

  //
  // Define arguments for CUTLASS Convolution
  //

  cutlass::conv::Mode mode = cutlass::conv::Mode::kCrossCorrelation;

  // Split K dimension into 1 partitions
  int split_k_slices = 1;

  // Construct Conv2dProblemSize with user defined output size
  cutlass::conv::Conv2dProblemSize problem_size(
      options.input_size,
      options.filter_size,
      options.padding,
      options.conv_stride,
      options.dilation,
      options.output_size(),
      mode,
      split_k_slices,
      options.groups
  );

  // Construct Conv2dOperation::Argument structure with conv2d 
  // problem size, data pointers, and epilogue values
  typename Conv2dOperation::Arguments arguments{
    problem_size,
    tensor_a.device_ref(),
    tensor_b.device_ref(),
    tensor_c.device_ref(),
    tensor_d.device_ref(),
    {options.alpha, options.beta},
  };

  //
  // Initialize CUTLASS Convolution
  //

  Conv2dOperation implicit_gemm_op;

  size_t workspace_size = implicit_gemm_op.get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  result.status = implicit_gemm_op.can_implement(arguments);
  CUTLASS_CHECK(result.status);

  result.status = implicit_gemm_op.initialize(arguments, workspace.get());
  CUTLASS_CHECK(result.status);

  //
  // Launch initialized CUTLASS kernel
  //
  result.status = implicit_gemm_op();

  CUTLASS_CHECK(result.status);

  //
  // Optional reference check
  //

  if (options.reference_check) {
    std::cout << "Verification on device...\n";

    // Compute with reference implementation
    cutlass::reference::device::Conv2dFprop<
      ElementInputA,
      LayoutInputA,
      ElementInputB,
      LayoutInputB,
      ElementOutput,
      LayoutOutput,
      ElementComputeEpilogue,
      ElementAccumulator,
      cutlass::NumericConverter<ElementOutput, ElementComputeEpilogue>
    >(
      problem_size,
      tensor_a.device_ref(),
      tensor_b.device_ref(),
      tensor_c.device_ref(),
      tensor_ref_d.device_ref(),
      options.alpha,
      options.beta
    );

    tensor_ref_d.sync_host();

    // Check if output from CUTLASS kernel and reference kernel are equal or not
    tensor_d.sync_host();

    bool passed = cutlass::reference::host::TensorEquals(
      tensor_d.host_view(),
      tensor_ref_d.host_view());

    if (!passed) {
      result.reference_check = cutlass::Status::kErrorInternal;
      std::cout << "ERROR - results miscompared.\n";
    } else {
      result.reference_check = cutlass::Status::kSuccess;
      std::cout << "Passed.\n";
    }
  } else {
    result.reference_check = cutlass::Status::kInvalid;
  }

  //
  // Performance measurement
  //

  if (options.measure_performance) {

    hipEvent_t events[2];
    
    for (auto & event : events) {
      result.error = hipEventCreate(&event);
      if (result.error != hipSuccess) {
        std::cerr << "hipEventCreate() failed: " << hipGetErrorString(result.error) << std::endl;
        return result;
      }
    }

    // Record an event at the start of a series of convolution operations.
    result.error = hipEventRecord(events[0]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Launch a sequence of implicit GEMM operations on the device
    for (int iteration = 0; iteration < options.iterations; ++iteration) {
      result.status = implicit_gemm_op();
      CUTLASS_CHECK(result.status);
    }

    // Record an event when the convolutions have been launched.
    result.error = hipEventRecord(events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Wait for work on the device to complete.
    result.error = hipEventSynchronize(events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventSynchronize() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Measure elapsed runtime
    float runtime_ms = 0;
    result.error = hipEventElapsedTime(&runtime_ms, events[0], events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "cudaEventElapsed() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Print average runtime and GFLOPs.
    result.runtime_ms = double(runtime_ms) / double(options.iterations);
    result.gflops = options.gflops(result.runtime_ms / 1000.0);

    // Cleanup
    for (auto event : events) {
      (void)hipEventDestroy(event);
    }
  }

  return result;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

Result profile_convolution_selecter(Options const &options) {
  int k_per_group = options.filter_size.n() / options.groups;

  // In group conv, if k_per_group < threadblock_N, one Threadblock will calculate multiple groups
  if (k_per_group < ThreadblockShape::kN) { // MultipleGroup mode
    if (options.optimized) {
      std::cerr << "Invalid problem: optimized group conv kernel doesn't support MultipleGroup (one CTA calculate multiple groups) mode" << std::endl;
      exit(-1);
    } else {
      std::cout << "Select AnalyticMultipleGroupOperation\n";
      return profile_convolution<AnalyticMultipleGroupOperation>(options);
    }
  } else { // SingleGroup mode
    if (options.optimized) {
      std::cout << "Select OptimizedSingleGroupOperation\n";
      return profile_convolution<OptimizedSingleGroupOperation>(options);
    } else {
      std::cout << "Select AnalyticSingleGroupOperation\n";
      return profile_convolution<AnalyticSingleGroupOperation>(options);
    }
  }
}

/////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {

  bool notSupported = false;

  // Ampere Tensor Core operations exposed with mma.sync are first available in CUDA 11.0.
  //
  // CUTLASS must be compiled with CUDA 11 Toolkit to run Conv2dFprop examples.
  if (!(__CUDACC_VER_MAJOR__ > 11 || (__CUDACC_VER_MAJOR__ == 11 && __CUDACC_VER_MINOR__ >= 0))) {
    std::cerr << "Ampere Tensor Core operations must be compiled with CUDA 11.0 Toolkit or later." << std::endl;
    notSupported = true;
  }

  hipDeviceProp_t props;
  CUDA_CHECK(hipGetDeviceProperties(&props, 0));

  if (!(props.major > 8 || (props.major == 8 && props.minor >= 0))) {
    std::cerr << "Ampere Tensor Ops must be run on a machine with compute capability at least 80."
              << std::endl;
    notSupported = true;
  }

  if (notSupported) {
    return 0;
  }

  Options options;
  
  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  // Execute one problem size
  if (!options.valid()) {
    std::cerr << "Invalid problem." << std::endl;
    return -1;
  }

  Result result = profile_convolution_selecter(options);

  Result::print_header(std::cout, options) << std::endl;
  result.print(std::cout, 1, options) << std::endl;

  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
