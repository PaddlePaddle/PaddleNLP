/***************************************************************************************************
 * Copyright (c) 2017 - 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/**
This example shows how to run depthwise 2d convolution kernels using functions and data structures
provided by CUTLASS using SIMT instruction;

There are 3 types of implementations of depthwise 2d convoltion
  1. kAnalytic
    Implicit gemm 2d convoltion algorithm.
  2. kOptimized
    An optimized algorithm and supports arbitrary stride and dilation.
  3. kFixedStrideDilation
    An optimized algorithm with fixed stride and dilation to reduce the runtime computation and do
more optimizations.

In general, the perf of kFixedStrideDilation would be better than kOptimized. However, if the filter
size, stride or dilation is large, it would encounter register spilling and may hurt the perf. If
in this case, please use kOptimized.

For kOptimized and kFixedStrideDilation, in order to fully utilize GPU hardware resources and achieve
better perf, when the output tensor size is large, splitk should be enabled to achieve better perf.

In this example, it demonstrates how to construct and run a FixedStrideDilation depthwise 2d
convolution kernel.
*/

#include <iostream>
#include <fstream>
#include <sstream>

#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm.h"
#include "cutlass/conv/kernel/default_depthwise_fprop.h"
#include "cutlass/conv/device/implicit_gemm_convolution.h"
#include "cutlass/conv/device/direct_convolution.h"

#include "cutlass/util/command_line.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/device/gemm.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/host/convolution.h"
#include "cutlass/util/tensor_view_io.h"

#include "helper.h"

// The code section below describes datatype for input, output tensors and computation between
// elements
using ElementAccumulator = cutlass::half_t;      // Data type of accumulator
using ElementComputeEpilogue = cutlass::half_t;  // Data type of epilogue computation (alpha, beta)
using ElementInputA = cutlass::half_t;           // Data type of elements in input tensor
using ElementInputB = cutlass::half_t;           // Data type of elements in input tensor
using ElementOutput = cutlass::half_t;           // Data type of elements in output tensor

using LayoutInputA = cutlass::layout::TensorNHWC;
using LayoutInputB = cutlass::layout::TensorNHWC;
using LayoutOutput = cutlass::layout::TensorNHWC;

// This code section describes whether you want to use tensor cores or regular SIMT cores on GPU SM
using MMAOp = cutlass::arch::OpClassSimt;

// This code section describes CUDA SM architecture number
using SmArch = cutlass::arch::Sm60;

// This code section describes the groups a thread block will compute
constexpr int groups_per_cta = 64;

// This code section describes the output tile <N, O, P, Q> a thread block will compute
using ThreadBlockOutputShape = cutlass::conv::TensorNHWCShape<1, 8, 8, groups_per_cta>;

// This code section describes the filter shape <R, S>
using FilterShape = cutlass::MatrixShape<3, 3>;

// Threadblock tile shape
using ThreadblockShape =
    cutlass::gemm::GemmShape<ThreadBlockOutputShape::kNHW, groups_per_cta, FilterShape::kCount>;

// This code section describes tile size a warp will computes
// WarpShape::kM = P * Q the warps would process
// WarpShape::kN = groups_per_cta that the warps would process
// WarpShape::kK = filter_size that the warps would process
using WarpShape = cutlass::gemm::GemmShape<16, groups_per_cta, FilterShape::kCount>;

// This code section describes the size of MMA op
using InstructionShape = cutlass::gemm::GemmShape<1, 1, 1>;

// This code section describes how threadblocks are scheduled on GPU
using SwizzleThreadBlock =
    cutlass::conv::threadblock::DepthwiseDirect2dConvIdentityThreadblockSwizzle<
        1,
        ThreadBlockOutputShape::kN,
        ThreadBlockOutputShape::kH,
        ThreadBlockOutputShape::kW>;

// Number of pipelines you want to use
constexpr int NumStages = 4;

// This code section describe iterator algorithm selected is kFixedStrideDilation
static cutlass::conv::IteratorAlgorithm const IteratorAlgorithm =
    cutlass::conv::IteratorAlgorithm::kFixedStrideDilation;
using StrideShape = cutlass::MatrixShape<1, 1>;
using DilationShape = cutlass::MatrixShape<1, 1>;

constexpr int kEpilogueElementsPerAccess = 128 / cutlass::sizeof_bits<ElementOutput>::value;

// This code section describes the epilogue part of the kernel, we use default value
using EpilogueOp = cutlass::epilogue::thread::LinearCombination<
    ElementOutput,               // Data type of output matrix.
    kEpilogueElementsPerAccess,  // The number of elements per vectorized.
                                 // memory access. This becomes the vector width of
                                 // math instructions in the epilogue too.
    ElementAccumulator,          // Data type of accumulator
    ElementComputeEpilogue,      // Data type for alpha/beta in linear combination
    cutlass::epilogue::thread::ScaleType::OnlyAlphaScaling>;  // Epilogue scaling operation.

using DepthwiseDirect2dConv = typename cutlass::conv::kernel::DefaultDepthwiseDirect2dConvFprop<
    ElementInputA,
    LayoutInputA,
    ElementInputB,
    LayoutInputB,
    ElementOutput,
    LayoutOutput,
    ElementAccumulator,
    MMAOp,
    SmArch,
    ThreadblockShape,
    ThreadBlockOutputShape,
    FilterShape,
    WarpShape,
    InstructionShape,
    EpilogueOp,
    SwizzleThreadBlock,
    NumStages,
    cutlass::arch::OpMultiplyAdd,
    IteratorAlgorithm,
    cutlass::conv::StrideSupport::kFixed,
    StrideShape,
    DilationShape>::Kernel;

using Direct2dConv = cutlass::conv::device::DirectConvolution<DepthwiseDirect2dConv>;

/////////////////////////////////////////////////////////////////////////////////////////////////

// Command line options parsing
struct Options {
  bool help;
  cutlass::Tensor4DCoord input_size;
  cutlass::Tensor4DCoord filter_size;
  cutlass::Tensor4DCoord padding;
  cutlass::MatrixCoord conv_stride;
  cutlass::MatrixCoord dilation;
  int groups;
  int splitk;
  bool reference_check;
  bool measure_performance;
  int iterations;
  bool save_workspace;
  ElementComputeEpilogue alpha;
  ElementComputeEpilogue beta;
  std::string tag;

  Options()
      : help(false),
        input_size(1, 128, 128, 32),
        filter_size(32, 3, 3, 1),
        groups(32),
        padding(1, 1, 1, 1),
        conv_stride(1, 1),
        dilation(1, 1),
        reference_check(false),
        measure_performance(true),
        iterations(20),
        save_workspace(false),
        alpha(1),
        beta(0),
        splitk(1) {}

  // Verify the problem size is compatible with the CUTLASS Convolution implementation.
  bool valid() {
    //
    // CUTLASS attempts to load 128b vectors of cutlass::half_t (F16) elements. Consequently,
    // all pointers, strides, and tensor extents must be divisible by 8 elements.
    //
    int const kAlignment = 8;

    if ((input_size.c() % kAlignment) || (filter_size.n() % kAlignment)) {
      // misaligned tensors
      return false;
    }

    // depthwise conv
    if (groups != input_size.c()) {
      return false;
    }

    if (filter_size.n() != groups) {
      return false;
    }

    // Invalid padding
    if ((padding.h() != filter_size.h() / 2) || (padding.w() != filter_size.w() / 2)) {
      return false;
    }

    return true;
  }

  /// Updates input and filter sizes
  void update(cutlass::Tensor4DCoord input_size, cutlass::Tensor4DCoord filter_size) {
    this->input_size = input_size;
    this->filter_size = filter_size;

    padding.n() = filter_size.h() / 2;
    padding.h() = filter_size.h() / 2;
    padding.w() = filter_size.w() / 2;
    padding.c() = filter_size.w() / 2;
  }

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
    }

    if (cmd.check_cmd_line_flag("ref-check")) {
      reference_check = true;
    }

    if (cmd.check_cmd_line_flag("perf-check")) {
      measure_performance = true;
    }

    if (cmd.check_cmd_line_flag("save-workspace")) {
      save_workspace = true;
    }

    cmd.get_cmd_line_argument("n", input_size.n());
    cmd.get_cmd_line_argument("h", input_size.h());
    cmd.get_cmd_line_argument("w", input_size.w());
    cmd.get_cmd_line_argument("c", input_size.c());

    cmd.get_cmd_line_argument("k", filter_size.n());
    cmd.get_cmd_line_argument("r", filter_size.h());
    cmd.get_cmd_line_argument("s", filter_size.w());

    cmd.get_cmd_line_argument("g", groups);

    filter_size.c() = 1;
    filter_size.n() = input_size.c();

    cmd.get_cmd_line_argument("alpha", alpha);
    cmd.get_cmd_line_argument("beta", beta);
    cmd.get_cmd_line_argument("splitk", splitk);

    cmd.get_cmd_line_argument("iterations", iterations);
    cmd.get_cmd_line_argument("tag", tag);

    int32_t padding_h = filter_size.h() / 2;
    int32_t padding_w = filter_size.w() / 2;
    padding = {padding_h, padding_h, padding_w, padding_w};
  }

  /// Prints the usage statement.
  std::ostream &print_usage(std::ostream &out) const {
    out << "41_depthwise_gemm_fprop example\n\n"
        << "  This example uses Ampere's Tensor Core operators on F16 data types to compute\n"
        << "  forward convolution on tensors of layout NHWC.\n\n"
        << "Options:\n\n"
        << "  --help               If specified, displays this usage statement.\n\n"
        << "  --n=<int>            Input tensor extent N\n"
        << "  --h=<int>            Input tensor extent H\n"
        << "  --w=<int>            Input tensor extent W\n"
        << "  --c=<int>            Input tensor extent C\n"
        << "  --k=<int>            Filter extent K\n"
        << "  --r=<int>            Filter extent R\n"
        << "  --s=<int>            Filter extent S\n\n"
        << "  --g=<int>            Groups\n\n"
        << "  --alpha=<float>      Epilogue scalar alpha\n"
        << "  --beta=<float>       Epilogue scalar beta\n\n"
        << "  --splitk=<int>       Enable splitK\n\n"
        << "  --ref-check          If set (true), reference check on the host is computed\n"
        << "  --perf-check         If set (true), performance is measured.\n"
        << "  --iterations=<int>   Number of profiling iterations to perform.\n"
        << "  --save-workspace     If set, workspace is written to a text file.\n"
        << "  --tag=<string>       String to replicate across the first column in the results "
           "table\n";

    out << "\n\nExamples:\n\n"
        << "$ ./examples/45_depthwise_simt_conv2dfprop/45_depthwise_simt_conv2dfprop  --n=32 "
           "--h=224 --w=224 --c=128 --k=128 --g=128 --r=3 --s=3\n\n"
        << "$ ./examples/45_depthwise_simt_conv2dfprop/45_depthwise_simt_conv2dfprop  --n=1 "
           "--h=224 --w=224 --c=32 --k=32 --g=32 --r=3 --s=3 --splitk=10 --ref-check\n\n";

    return out;
  }

  /// Computes the output tensor size (NPQK)
  cutlass::Tensor4DCoord output_size() const {
    return cutlass::Tensor4DCoord(
        input_size.n(),
        (input_size.h() + padding.n() + padding.h() - filter_size.h()) / conv_stride.row() + 1,
        (input_size.w() + padding.w() + padding.c() - filter_size.w()) / conv_stride.column() + 1,
        filter_size.n());
  }

  /// Compute performance in GFLOP/s
  double gflops(double runtime_s) const {
    // Number of multiply-adds = NPQK * CRS
    int64_t fmas =
        output_size().product() * int64_t(filter_size.h() * filter_size.w() * filter_size.c());

    // Two flops per multiply-add
    return 2.0 * double(fmas) / double(1.0e9) / runtime_s;
  }
};

/////////////////////////////////////////////////////////////////////////////////////////////////

struct Result {
  double runtime_ms;
  double gflops;
  cutlass::Status status;
  cutlass::Status reference_check;
  hipError_t error;

  Result()
      : runtime_ms(0),
        gflops(0),
        status(cutlass::Status::kSuccess),
        reference_check(cutlass::Status::kInvalid),
        error(hipSuccess) {}

  static std::ostream &print_header(std::ostream &out, Options const &options) {
    if (!options.tag.empty()) {
      out << "Name,";
    }

    out << "Layer,N,H,W,C,K,R,S,G,stride_h,stride_w,dilation_h,dilation_w,splitK,Runtime,GFLOPs";

    return out;
  }

  std::ostream &print(std::ostream &out, int idx, Options const &options) {
    if (!options.tag.empty()) {
      out << options.tag << ",";
    }

    cutlass::Tensor4DCoord output_size = options.output_size();
    out << "conv_" << idx << "," << options.input_size.n() << "," << options.input_size.h() << ","
        << options.input_size.w() << "," << options.input_size.c() << ","

        << options.filter_size.n() << "," << options.filter_size.h() << ","
        << options.filter_size.w() << ","

        << options.groups << "," << options.conv_stride.row() << "," << options.conv_stride.column()
        << ","

        << options.dilation.row() << "," << options.dilation.column() << ","

        << options.splitk << ","

        << runtime_ms << "," << gflops;

    return out;
  }
};

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Runs one testcase
Result profile_convolution(Options const &options) {
  Result result;

  //
  // Allocate host-device tensors using the CUTLASS Utilities.
  //

  cutlass::HostTensor<ElementInputA, LayoutInputA> tensor_a(options.input_size);
  cutlass::HostTensor<ElementInputB, LayoutInputB> tensor_b(options.filter_size);
  cutlass::HostTensor<ElementInputB, LayoutInputB> tensor_b_transpose(options.filter_size);
  cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_c(options.output_size());
  cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_d(options.output_size());
  cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_ref_d(options.output_size());

  //
  // Initialize tensors
  //

  // Fill tensor A on host with uniform-distribution random data
  cutlass::reference::host::TensorFillRandomUniform(
      tensor_a.host_view(), 1, ElementInputA(5), ElementInputA(-6), 0);

  // Fill tensor B on host with uniform-distribution random data
  cutlass::reference::host::TensorFillRandomUniform(
      tensor_b.host_view(), 1, ElementInputB(3), ElementInputB(-6), 0);

  // Fill tensor C on host with uniform-distribution random data
  cutlass::reference::host::TensorFillRandomUniform(
      tensor_c.host_view(), 1, ElementOutput(5), ElementOutput(-6), 0);

  // Fill tensor D on host with zeros
  cutlass::reference::host::TensorFill(tensor_d.host_view());

  // Fill tensor D for reference on host with zeros
  cutlass::reference::host::TensorFill(tensor_ref_d.host_view());

  // Copy data from host to GPU
  tensor_a.sync_device();
  tensor_b.sync_device();
  tensor_b_transpose.sync_device();
  tensor_c.sync_device();
  tensor_d.sync_device();
  tensor_ref_d.sync_device();

  //
  // Define arguments for CUTLASS Convolution
  //

  cutlass::conv::Mode mode = cutlass::conv::Mode::kCrossCorrelation;

  // Split P*Q into multiple CTA
  int split_k_slices = options.splitk;

  // Construct Conv2dProblemSize with user defined output size
  cutlass::conv::Conv2dProblemSize problem_size(options.input_size,
                                                options.filter_size,
                                                options.padding,
                                                options.conv_stride,
                                                options.dilation,
                                                options.output_size(),
                                                mode,
                                                split_k_slices,
                                                options.groups);

  // Construct Direc2dConv::Argument structure with conv2d
  // problem size, data pointers, and epilogue values
  typename Direct2dConv::Arguments arguments{problem_size,
                                             tensor_a.device_ref(),
                                             tensor_b.device_ref(),
                                             tensor_c.device_ref(),
                                             tensor_d.device_ref(),
                                             {options.alpha, options.beta},
                                             tensor_b_transpose.device_ref()};

  //
  // Initialize CUTLASS Convolution
  //

  Direct2dConv implicit_gemm_op;

  size_t workspace_size = implicit_gemm_op.get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  result.status = implicit_gemm_op.can_implement(arguments);
  CUTLASS_CHECK(result.status);

  result.status = implicit_gemm_op.initialize(arguments, workspace.get());
  CUTLASS_CHECK(result.status);

  //
  // Launch initialized CUTLASS kernel
  //
  result.status = implicit_gemm_op();

  CUTLASS_CHECK(result.status);

  //
  // Optional reference check
  //

  if (options.reference_check) {
    std::cout << "Verification on host...\n";

    // Compute with reference implementation
    cutlass::reference::host::Conv2dFprop<
        ElementInputA,
        LayoutInputA,
        ElementInputB,
        LayoutInputB,
        ElementOutput,
        LayoutOutput,
        ElementComputeEpilogue,
        ElementAccumulator,
        cutlass::NumericConverter<ElementOutput, ElementComputeEpilogue> >(problem_size,
                                                                           tensor_a.host_ref(),
                                                                           tensor_b.host_ref(),
                                                                           tensor_c.host_ref(),
                                                                           tensor_ref_d.host_ref(),
                                                                           options.alpha,
                                                                           options.beta);

    // Check if output from CUTLASS kernel and reference kernel are equal or not
    tensor_d.sync_host();

    bool passed =
        cutlass::reference::host::TensorEquals(tensor_d.host_view(), tensor_ref_d.host_view());

    if (!passed) {
      result.reference_check = cutlass::Status::kErrorInternal;
      std::cout << "ERROR - results miscompared.\n";
    } else {
      result.reference_check = cutlass::Status::kSuccess;
      std::cout << "Passed.\n";
    }
  } else {
    result.reference_check = cutlass::Status::kInvalid;
  }

  if (options.save_workspace) {
    std::stringstream ss;

    ss << "45_depthwise_simt_conv2dfprop" << options.input_size.n() << "x" << options.input_size.h()
       << "x" << options.input_size.w() << "x" << options.input_size.c() << "_"
       << options.filter_size.n() << "x" << options.filter_size.h() << "x"
       << options.filter_size.w() << "x" << options.filter_size.c() << ".dat";

    std::ofstream output_workspace(ss.str());

    output_workspace << "Input = \n"
                     << tensor_a.host_view() << "\n\n"
                     << "Filters = \n"
                     << tensor_b.host_view() << "\n\n";

    if (options.reference_check) {
      output_workspace << "Reference = \n" << tensor_ref_d.host_view() << "\n\n";
    }

    output_workspace << "Computed = \n" << tensor_d.host_view() << std::endl;

    std::cout << "Results written to '" << ss.str() << "'." << std::endl;
  }

  //
  // Performance measurement
  //

  if (options.measure_performance) {
    hipEvent_t events[2];

    for (auto &event : events) {
      result.error = hipEventCreate(&event);
      if (result.error != hipSuccess) {
        std::cerr << "hipEventCreate() failed: " << hipGetErrorString(result.error) << std::endl;
        return result;
      }
    }

    // Record an event at the start of a series of convolution operations.
    result.error = hipEventRecord(events[0]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Launch a sequence of implicit GEMM operations on the device
    for (int iteration = 0; iteration < options.iterations; ++iteration) {
      result.status = implicit_gemm_op();
      CUTLASS_CHECK(result.status);
    }

    // Record an event when the convolutions have been launched.
    result.error = hipEventRecord(events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Wait for work on the device to complete.
    result.error = hipEventSynchronize(events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventSynchronize() failed: " << hipGetErrorString(result.error)
                << std::endl;
      return result;
    }

    // Measure elapsed runtime
    float runtime_ms = 0;
    result.error = hipEventElapsedTime(&runtime_ms, events[0], events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "cudaEventElapsed() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Print average runtime and GFLOPs.
    result.runtime_ms = double(runtime_ms) / double(options.iterations);
    result.gflops = options.gflops(result.runtime_ms / 1000.0);

    // Cleanup
    for (auto event : events) {
      (void)hipEventDestroy(event);
    }
  }

  return result;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {
  bool notSupported = false;

  hipDeviceProp_t props;
  CUDA_CHECK(hipGetDeviceProperties(&props, 0));

  if (!(props.major >= 6)) {
    std::cerr << "Run on a machine with compute capability at least 60." << std::endl;
    notSupported = true;
  }

  if (notSupported) {
    return 0;
  }

  Options options;

  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  // Execute one problem size
  if (!options.valid()) {
    std::cerr << "Invalid problem." << std::endl;
    return -1;
  }

  Result result = profile_convolution(options);

  Result::print_header(std::cout, options) << std::endl;
  result.print(std::cout, 1, options) << std::endl;

  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
