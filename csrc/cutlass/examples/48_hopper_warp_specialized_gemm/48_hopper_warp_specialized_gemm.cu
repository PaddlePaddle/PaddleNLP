/***************************************************************************************************
 * Copyright (c) 2023 - 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*! \file
    \brief Simple Hopper GEMM example using CUTLASS 3.0 APIs for NVIDIA Hopper architecture

    This example demonstrate a simple way to instantiate and run a TF32 GEMM using the new CUTLASS 3.0
    APIs on NVIDIA Hopper architecture. New features that will be showcased in this example are as follows:

    1. NVIDIA Hopper architecture introduces a new series of tensor core instructions (GMMA) 
    which are more efficient than the Ampere tensor core instructions.

    2. NVIDIA Hopper architecture includes new Tensor Memory Accelerator (TMA) unit to transfer large 
    blocks of data efficiently between global memory and shared memory. TMA also supports asynchronous
    copies between thread blocks in a cluster. Another advantage is that TMA can load in FP32 data and
    convert them implicitly to TF32.

    3. This example uses the Warp Specialized kernel design (see /media/docs/efficient_gemm.md for details).

    Examples:

      $ ./examples/48_hopper_warp_specialized_gemm/48_hopper_warp_specialized_gemm --m=2048 --n=2048 --k=2048
*/

#include <iostream>

#include "cutlass/cutlass.h"

#include "cute/tensor.hpp"
#include "cutlass/tensor_ref.h"
#include "cutlass/epilogue/collective/default_epilogue.hpp"
#include "cutlass/epilogue/thread/linear_combination.h"
#include "cutlass/gemm/dispatch_policy.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/packed_stride.hpp"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/device/gemm.h"
#include "cutlass/util/reference/device/tensor_compare.h"
#include "cutlass/util/reference/device/tensor_fill.h"

#include "helper.h"

using namespace cute;

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM kernel configurations
/////////////////////////////////////////////////////////////////////////////////////////////////

// A matrix configuration
using         ElementA    = float;                                          // Element type for A matrix operand
using         LayoutA     = cutlass::layout::RowMajor;                      // Layout type for A matrix operand
constexpr int AlignmentA  = 128 / cutlass::sizeof_bits<ElementA>::value;    // Memory access granularity/alignment of A matrix in units of elements (up to 16 bytes)

// B matrix configuration
using         ElementB    = float;                                          // Element type for B matrix operand
using         LayoutB     = cutlass::layout::ColumnMajor;                   // Layout type for B matrix operand
constexpr int AlignmentB  = 128 / cutlass::sizeof_bits<ElementB>::value;    // Memory access granularity/alignment of B matrix in units of elements (up to 16 bytes)

// C/D matrix configuration
using         ElementC    = float;                                          // Element type for C and D matrix operands
using         LayoutC     = cutlass::layout::ColumnMajor;                   // Layout type for C and D matrix operands

// Core kernel configurations
using ElementAccumulator  = float;                                          // Element type for internal accumulation
using ArchTag             = cutlass::arch::Sm90;                            // Tag indicating the minimum SM that supports the intended feature
using OperatorClass       = cutlass::arch::OpClassTensorOp;                 // Operator class tag
using TilesShape          = Shape<_128,_128,_32>;                           // Threadblock-level tile size
using ClusterShape        = Shape<_1,_2,_1>;                                // Shape of the threadblocks in a cluster
using StageCountType = cutlass::gemm::collective::StageCountAuto;           // Stage count maximized based on the tile size
using KernelSchedule = cutlass::gemm::collective::KernelScheduleAuto;       // Kernel to launch based on the default setting in the Collective Builder 

using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
    ArchTag, OperatorClass,
    ElementA, LayoutA, AlignmentA,
    ElementB, LayoutB, AlignmentB,
    ElementAccumulator,
    TilesShape, ClusterShape,
    cutlass::gemm::collective::StageCountAuto,
    cutlass::gemm::collective::KernelScheduleAuto
  >::CollectiveOp;

using CollectiveEpilogue = cutlass::epilogue::collective::DefaultEpilogue<
    cutlass::gemm::TagToStrideC_t<LayoutC>,
    cutlass::gemm::TagToStrideC_t<LayoutC>,
    cutlass::epilogue::thread::LinearCombination<ElementC, 1, ElementAccumulator, ElementAccumulator>>;

using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
    Shape<int,int,int>, // Indicates ProblemShape
    CollectiveMainloop,
    CollectiveEpilogue
>;

using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

// Reference device GEMM implementation type
using DeviceGemmReference = cutlass::reference::device::Gemm<
  ElementA,
  LayoutA,
  ElementB,
  LayoutB,
  ElementC,
  LayoutC,
  ElementAccumulator,
  ElementAccumulator>;

using StrideA = typename Gemm::GemmKernel::StrideA;
using StrideB = typename Gemm::GemmKernel::StrideB;
using StrideC = typename Gemm::GemmKernel::StrideC;
using StrideD = typename Gemm::GemmKernel::StrideD;

//
// Data members
//

/// Initialization
StrideA stride_A;
StrideB stride_B;
StrideC stride_C;
StrideD stride_D;
uint64_t seed;

cutlass::DeviceAllocation<typename Gemm::ElementA> block_A;
cutlass::DeviceAllocation<typename Gemm::ElementB> block_B;
cutlass::DeviceAllocation<typename Gemm::ElementC> block_C;
cutlass::DeviceAllocation<typename Gemm::EpilogueOutputOp::ElementOutput> block_D;
cutlass::DeviceAllocation<typename Gemm::EpilogueOutputOp::ElementOutput> block_ref_D;

#endif // defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// Testbed utility types
/////////////////////////////////////////////////////////////////////////////////////////////////

// Command line options parsing
struct Options {

  bool help;

  float alpha, beta;
  int iterations;
  int m, n, k;

  Options():
    help(false),
    m(5120), n(4096), k(4096),
    alpha(1.f), beta(0.f),
    iterations(1000)
  { }

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
      return;
    }

    cmd.get_cmd_line_argument("m", m);
    cmd.get_cmd_line_argument("n", n);
    cmd.get_cmd_line_argument("k", k);
    cmd.get_cmd_line_argument("alpha", alpha, 1.f);
    cmd.get_cmd_line_argument("beta", beta, 0.f);
    cmd.get_cmd_line_argument("iterations", iterations);
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "48_hopper_warp_specialized_gemm\n\n"
      << "  Hopper FP32 GEMM using a Warp Specialized kernel.\n\n"
      << "Options:\n\n"
      << "  --help                      If specified, displays this usage statement\n\n"
      << "  --m=<int>                   Sets the M extent of the GEMM\n"
      << "  --n=<int>                   Sets the N extent of the GEMM\n"
      << "  --k=<int>                   Sets the K extent of the GEMM\n"
      << "  --alpha=<f32>               Epilogue scalar alpha\n"
      << "  --beta=<f32>                Epilogue scalar beta\n\n"
      << "  --iterations=<int>          Number of profiling iterations to perform.\n\n";

    out
      << "\n\nExamples:\n\n"
      << "$ " << "48_hopper_warp_specialized_gemm" << " --m=1024 --n=512 --k=1024 --alpha=2 --beta=0.707 \n\n";

    return out;
  }

  /// Compute performance in GFLOP/s
  double gflops(double runtime_s) const
  {
    // Two flops per multiply-add
    uint64_t flop = uint64_t(2) * m * n * k;
    double gflop = double(flop) / double(1.0e9);
    return gflop / runtime_s;
  }
};

/// Result structure
struct Result
{
  double avg_runtime_ms;
  double gflops;
  cutlass::Status status;
  hipError_t error;
  bool passed;

  Result(
    double avg_runtime_ms = 0,
    double gflops = 0,
    cutlass::Status status = cutlass::Status::kSuccess,
    hipError_t error = hipSuccess)
  :
    avg_runtime_ms(avg_runtime_ms), gflops(gflops), status(status), error(error), passed(false)
  {}

};

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM setup and evaluation
/////////////////////////////////////////////////////////////////////////////////////////////////

/// Helper to initialize a block of device data
template <class Element>
bool initialize_block(
  cutlass::DeviceAllocation<Element>& block,
  uint64_t seed=2023) {

  Element scope_max, scope_min;
  int bits_input = cutlass::sizeof_bits<Element>::value;

  if (bits_input == 1) {
    scope_max = 2;
    scope_min = 0;
  } else if (bits_input <= 8) {
    scope_max = 2;
    scope_min = -2;
  } else {
    scope_max = 8;
    scope_min = -8;
  }

  cutlass::reference::device::BlockFillRandomUniform(
    block.get(), block.size(), seed, scope_max, scope_min, 0);

  return true;
}

/// Initialize operands to be used in the GEMM and reference GEMM
void initialize(const Options &options) {

  stride_A = make_cute_packed_stride(StrideA{}, cute::make_shape(options.m, options.k, Int<1>{}));
  stride_B = make_cute_packed_stride(StrideB{}, cute::make_shape(options.n, options.k, Int<1>{}));
  stride_C = make_cute_packed_stride(StrideC{}, cute::make_shape(options.m, options.n, Int<1>{}));
  stride_D = make_cute_packed_stride(StrideD{}, cute::make_shape(options.m, options.n, Int<1>{}));

  block_A.reset(options.m * options.k);
  block_B.reset(options.k * options.n);
  block_C.reset(options.m * options.n);
  block_D.reset(options.m * options.n);
  block_ref_D.reset(options.m * options.n);

  initialize_block(block_A, seed + 2023);
  initialize_block(block_B, seed + 2022);
  initialize_block(block_C, seed + 2021);
}

/// Populates a Gemm::Arguments structure from the given commandline options
typename Gemm::Arguments args_from_options(const Options &options)
{
  typename Gemm::Arguments arguments{
    cutlass::gemm::GemmUniversalMode::kGemm,
    {options.m, options.n, options.k},
    block_A.get(),
    stride_A,
    block_B.get(),
    stride_B,
    {block_C.get(), stride_C, block_D.get(), stride_D, {options.alpha, options.beta}}
  };

  return arguments;
}

bool verify(const Options &options) {
  cutlass::TensorRef ref_A(block_A.get(), Gemm::LayoutA::packed({options.m, options.k}));
  cutlass::TensorRef ref_B(block_B.get(), Gemm::LayoutB::packed({options.n, options.k}));
  cutlass::TensorRef ref_C(block_C.get(), Gemm::LayoutC::packed({options.m, options.n}));
  cutlass::TensorRef ref_D(block_ref_D.get(), Gemm::LayoutD::packed({options.m, options.n}));

  //
  // Compute reference output
  //

  // Create instantiation for device reference gemm kernel
  DeviceGemmReference gemm_reference;

  // Launch device reference gemm kernel
  gemm_reference(
    {options.m, options.n, options.k},
    ElementAccumulator(options.alpha),
    ref_A,
    ref_B,
    ElementAccumulator(options.beta),
    ref_C,
    ref_D);

  // Wait for kernel to finish
  CUDA_CHECK(hipDeviceSynchronize());

  // Check if output from CUTLASS kernel and reference kernel are equal or not
  bool passed = cutlass::reference::device::BlockCompareEqual(block_ref_D.get(), block_D.get(), block_D.size());

  return passed;
}

/// Execute a given example GEMM computation
template <typename Gemm>
int run(Options &options)
{
  initialize(options);

  // Instantiate CUTLASS kernel depending on templates
  Gemm gemm;

  // Create a structure of gemm kernel arguments suitable for invoking an instance of Gemm
  auto arguments = args_from_options(options);

  // Using the arguments, query for extra workspace required for matrix multiplication computation
  size_t workspace_size = Gemm::get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  // Check if the problem size is supported or not
  CUTLASS_CHECK(gemm.can_implement(arguments));

  // Initialize CUTLASS kernel with arguments and workspace pointer
  CUTLASS_CHECK(gemm.initialize(arguments, workspace.get()));

  // Correctness / Warmup iteration
  CUTLASS_CHECK(gemm.run());

  // Check if output from CUTLASS kernel and reference kernel are equal or not
  Result result;
  result.passed = verify(options);

  std::cout << "  Disposition: " << (result.passed ? "Passed" : "Failed") << std::endl;

  if (!result.passed) {
    exit(-1);
  }

  // Run profiling loop
  if (options.iterations > 0)
  {
    GpuTimer timer;
    timer.start();
    for (int iter = 0; iter < options.iterations; ++iter) {
      CUTLASS_CHECK(gemm.run());
    }
    timer.stop();

    // Compute average runtime and GFLOPs.
    float elapsed_ms = timer.elapsed_millis();
    result.avg_runtime_ms = double(elapsed_ms) / double(options.iterations);
    result.gflops = options.gflops(result.avg_runtime_ms / 1000.0);

    std::cout << "  Problem Size: " << options.m << 'x' << options.n << 'x' << options.k << std::endl;
    std::cout << "  Avg runtime: " << result.avg_runtime_ms << " ms" << std::endl;
    std::cout << "  GFLOPS: " << result.gflops << std::endl;
  }

  return 0;
}

#endif // defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

///////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {

  // CUTLASS must be compiled with CUDA 12.0 Toolkit to run this example
  // and must have compute capability at least 90.
  if (__CUDACC_VER_MAJOR__ < 12) {
    std::cerr << "This example requires CUDA 12 or newer.\n";
    // Returning zero so this test passes on older Toolkits. Its actions are no-op.
    return 0;
  }

  hipDeviceProp_t props;
  int current_device_id;
  CUDA_CHECK(hipGetDevice(&current_device_id));
  CUDA_CHECK(hipGetDeviceProperties(&props, current_device_id));
  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (props.major < 9) {
    std::cerr
      << "This example requires a GPU of NVIDIA's Hopper Architecture or "
      << "later (compute capability 90 or greater).\n";
    return 0;
  }

  //
  // Parse options
  //

  Options options;

  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  //
  // Evaluate CUTLASS kernels
  //

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)
  run<Gemm>(options);
#endif

  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
