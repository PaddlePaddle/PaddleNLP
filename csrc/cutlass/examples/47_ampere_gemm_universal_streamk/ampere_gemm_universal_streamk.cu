#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/***************************************************************************************************
 Example contrasting the Stream-K parallel decomposition for GEMM threadblocks versus the
 "classic data-parallel" and "Split-K" decompositions.

 For more details regarding the Stream-K method, see "Stream-K: Work-centric Parallel Decomposition
 for Dense Matrix-Matrix Multiplication on the GPU" (https://arxiv.org/abs/2301.03598)

 Requires NVIDIA Ampere or newer device (SM80+).

 - To lock persistence mode, power (400W), clocks (1005MHz) for evaluation (assumes device 0 and A100)

     cutlass$ sudo nvidia-smi -pm 1 -i 0

     cutlass$ sudo nvidia-smi -i 0 -pl 400

     cutlass$ sudo nvidia-smi -i 0 -lgc 1005

 - Build and run:

     cutlass$ mkdir build

     cutlass$ cd build

     cutlass/build$ cmake .. -DCUTLASS_NVCC_ARCHS=80

     cutlass/build$ make 47_ampere_gemm_universal_streamk

     cutlass/build$ ./examples/47_ampere_gemm_universal_streamk/47_ampere_gemm_universal_streamk

        10000 timing iterations of 2048 x 2048 x 2048 matrix-matrix multiply

        Basic data-parallel GEMM
          Disposition: Passed
          Avg runtime: 0.112633 ms
          GFLOPs: 152530

        StreamK GEMM with default load-balancing
          Disposition: Passed
          Avg runtime: 0.0941929 ms
          GFLOPs: 182390
          Speedup vs Basic-DP: 1.196

        StreamK emulating basic data-parallel GEMM
          Disposition: Passed
          Avg runtime: 0.113119 ms
          GFLOPs: 151875
          Speedup vs Basic-DP: 0.996

        Basic split-K GEMM with tile-splitting factor 2
          Disposition: Passed
          Avg runtime: 0.104772 ms
          GFLOPs: 163973

        StreamK emulating Split-K GEMM with tile-splitting factor 2
          Disposition: Passed
          Avg runtime: 0.105379 ms
          GFLOPs: 163029
          Speedup vs Basic-SplitK: 0.994

 **************************************************************************************************/

#include <iostream>
#include <string>

#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm_universal.h"

#include "cutlass/util/command_line.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/device/gemm.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/tensor_view_io.h"

#include "helper.h"



/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM kernel configurations (cutlass_tensorop_h16816gemm_128x128_32x4_nn_align8)
/////////////////////////////////////////////////////////////////////////////////////////////////

// A matrix configuration
using         ElementA    = cutlass::half_t;                                // Element type for A matrix operand
using         LayoutA     = cutlass::layout::RowMajor;                      // Layout type for A matrix operand
constexpr int AlignmentA  = 128 / cutlass::sizeof_bits<ElementA>::value;    // Memory access granularity/alignment of A matrix in units of elements (up to 16 bytes)

// B matrix configuration
using         ElementB    = cutlass::half_t;                                // Element type for B matrix operand
using         LayoutB     = cutlass::layout::RowMajor;                      // Layout type for B matrix operand
constexpr int AlignmentB  = 128 / cutlass::sizeof_bits<ElementB>::value;    // Memory access granularity/alignment of B matrix in units of elements (up to 16 bytes)

// C/D matrix configuration
using         ElementC    = cutlass::half_t;                                // Element type for C and D matrix operands
using         LayoutC     = cutlass::layout::RowMajor;                      // Layout type for C and D matrix operands
constexpr int AlignmentC  = 128 / cutlass::sizeof_bits<ElementC>::value;    // Memory access granularity/alignment of C/D matrices in units of elements (up to 16 bytes)

// Multiply-accumulate blocking/pipelining details
using ElementAccumulator  = cutlass::half_t;                          // Element type for internal accumulation
using ArchTag             = cutlass::arch::Sm80;                      // Tag indicating the minimum SM that supports the intended feature
using OperatorClass       = cutlass::arch::OpClassTensorOp;           // Operator class tag
using ThreadblockShape    = cutlass::gemm::GemmShape<128, 128, 32>;   // Threadblock-level tile size (concept: GemmShape)
using WarpShape           = cutlass::gemm::GemmShape<64, 64, 32>;     // Warp-level tile size (concept: GemmShape)
using InstructionShape    = cutlass::gemm::GemmShape<16, 8, 16>;      // Instruction-level tile size (concept: GemmShape)
constexpr int NumStages   = 4;                                        // Number of global->shared pipeline stages used in the GEMM mainloop

// Epilogue output operator
using EpilogueOp = cutlass::epilogue::thread::LinearCombination<
    ElementC,               // Element type for C and D matrix operands
    AlignmentC,             // Memory access granularity of C and D matrix in units of elements
    ElementAccumulator,     // Element type from internal accumaccumulation
    ElementAccumulator>;    // Data type used to compute linear combination

// Reference device GEMM implementation type
using DeviceGemmReference = cutlass::reference::device::Gemm<
  ElementA,
  LayoutA,
  ElementB,
  LayoutB,
  ElementC,
  LayoutC,
  ElementAccumulator,
  ElementAccumulator>;

// Classic data-parallel device GEMM implementation type
using DeviceGemmBasic = cutlass::gemm::device::GemmUniversal<
    ElementA, LayoutA,
    ElementB, LayoutB,
    ElementC, LayoutC,
    ElementAccumulator,
    OperatorClass,
    ArchTag,
    ThreadblockShape,
    WarpShape,
    InstructionShape,
    EpilogueOp,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    NumStages,
    AlignmentA,
    AlignmentB>;

// StreamK device GEMM implementation type
using DeviceGemmStreamK = cutlass::gemm::device::GemmUniversal<
    ElementA, LayoutA,
    ElementB, LayoutB,
    ElementC, LayoutC,
    ElementAccumulator,
    OperatorClass,
    ArchTag,
    ThreadblockShape,
    WarpShape,
    InstructionShape,
    EpilogueOp,
    cutlass::gemm::threadblock::ThreadblockSwizzleStreamK, // <-- Only difference
    NumStages,
    AlignmentA,
    AlignmentB>;


/////////////////////////////////////////////////////////////////////////////////////////////////
/// Testbed utility types
/////////////////////////////////////////////////////////////////////////////////////////////////

/// Result structure
struct Result
{
  double avg_runtime_ms;
  double gflops;
  cutlass::Status status;
  hipError_t error;
  bool passed;

  Result(
    double avg_runtime_ms = 0,
    double gflops = 0,
    cutlass::Status status = cutlass::Status::kSuccess,
    hipError_t error = hipSuccess)
  :
    avg_runtime_ms(avg_runtime_ms), gflops(gflops), status(status), error(error), passed(true)
  {}

};


/// Command line options parsing
struct Options
{
  std::string               command_name;
  bool                      help;
  cutlass::gemm::GemmCoord  problem_size;
  float                     alpha;
  float                     beta;
  int                       split_k_factor;
  int                       avail_sms;
  bool                      reference_check;
  int                       iterations;

  cutlass::HostTensor<ElementA, LayoutA> tensor_a;
  cutlass::HostTensor<ElementB, LayoutB> tensor_b;
  cutlass::HostTensor<ElementC, LayoutC> tensor_c;
  cutlass::HostTensor<ElementC, LayoutC> tensor_d;
  cutlass::HostTensor<ElementC, LayoutC> tensor_ref_d;

  Options(std::string command_name) :
    command_name(command_name),
    help(false),
    problem_size({2048, 2048, 2048}),
    alpha(1.0f),
    beta(0.0f),
    split_k_factor(1),
    avail_sms(-1),              // Number of device SMs to use is unlimited
    reference_check(true),
    iterations(10000)
  {}

  bool valid() const
  {
    return true;
  }

  void parse(int argc, char const **args)
  {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
    }

    cmd.get_cmd_line_argument("m", problem_size.m());
    cmd.get_cmd_line_argument("n", problem_size.n());
    cmd.get_cmd_line_argument("k", problem_size.k());
    cmd.get_cmd_line_argument("alpha", alpha);
    cmd.get_cmd_line_argument("beta", beta);
    cmd.get_cmd_line_argument("split", split_k_factor);
    cmd.get_cmd_line_argument("iterations", iterations);
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const
  {
    out
      << "Performs a GEMM computation.\n"
      << "\n"
      << "Options:\n"
      << "\n"
      << "  --help                      If specified, displays this usage statement.\n\n"
      << "  --m=<int>                   GEMM M dimension\n"
      << "  --n=<int>                   GEMM N dimension\n"
      << "  --k=<int>                   GEMM K dimension\n"
      << "  --alpha=<f32>               Epilogue scalar alpha\n"
      << "  --beta=<f32>                Epilogue scalar beta\n\n"
      << "  --split=<int>               Split-K factor to emulate\n\n"
      << "  --iterations=<int>          Number of profiling iterations to perform.\n\n";

    out
      << "\n\nExamples:\n\n"
      << "$ " << command_name << " --m=1024 --n=512 --k=1024 --alpha=2 --beta=0.707 \n\n";

    return out;
  }

  /// Compute performance in GFLOP/s
  double gflops(double runtime_s) const
  {
    // Two flops per multiply-add
    return 2.0 * double(problem_size.product()) / double(1.0e9) / runtime_s;
  }
};


/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM evaluation
/////////////////////////////////////////////////////////////////////////////////////////////////

/// Populates a DeviceGemmBasic::Arguments structure from the given commandline options
typename DeviceGemmBasic::Arguments args_from_options(
    const DeviceGemmBasic &device_gemm,
    const Options &options,
    cutlass::HostTensor<ElementA, LayoutA> &tensor_a,
    cutlass::HostTensor<ElementB, LayoutB> &tensor_b,
    cutlass::HostTensor<ElementC, LayoutC> &tensor_c,
    cutlass::HostTensor<ElementC, LayoutC> &tensor_d)
{
  return typename DeviceGemmBasic::Arguments(
    cutlass::gemm::GemmUniversalMode::kGemm,  // universal mode
    options.problem_size,                     // problem_size
    options.split_k_factor,                   // batch count / splitk slices
    {                                         // epilogue parameters
      ElementAccumulator(options.alpha),
      ElementAccumulator(options.beta)
    },
    tensor_a.device_data(),                   // ptr_A
    tensor_b.device_data(),                   // ptr_B
    tensor_c.device_data(),                   // ptr_C
    tensor_d.device_data(),                   // ptr_D
    options.problem_size.mk().product(),      // batch_stride_A
    options.problem_size.nk().product(),      // batch_stride_B
    options.problem_size.mn().product(),      // batch_stride_C
    options.problem_size.mn().product(),      // batch_stride_D
    tensor_a.layout().stride(0),              // stride_a
    tensor_b.layout().stride(0),              // stride_b
    tensor_c.layout().stride(0),              // stride_c
    tensor_d.layout().stride(0));             // stride_d
}

/// Populates a DeviceGemmStreamK::Arguments structure from the given commandline options
typename DeviceGemmStreamK::Arguments args_from_options(
    const DeviceGemmStreamK &device_gemm,
    const Options &options,
    cutlass::HostTensor<ElementA, LayoutA> &tensor_a,
    cutlass::HostTensor<ElementB, LayoutB> &tensor_b,
    cutlass::HostTensor<ElementC, LayoutC> &tensor_c,
    cutlass::HostTensor<ElementC, LayoutC> &tensor_d)
{
  return typename DeviceGemmStreamK::Arguments(
    cutlass::gemm::GemmUniversalMode::kGemm,  // universal mode
    options.problem_size,                     // problem_size
    options.split_k_factor,                   // batch count / splitk slices
    {                                         // epilogue parameters
      ElementAccumulator(options.alpha),
      ElementAccumulator(options.beta)
    },
    tensor_a.device_data(),                   // ptr_A
    tensor_b.device_data(),                   // ptr_B
    tensor_c.device_data(),                   // ptr_C
    tensor_d.device_data(),                   // ptr_D
    options.problem_size.mk().product(),      // batch_stride_A
    options.problem_size.nk().product(),      // batch_stride_B
    options.problem_size.mn().product(),      // batch_stride_C
    options.problem_size.mn().product(),      // batch_stride_D
    tensor_a.layout().stride(0),              // stride_a
    tensor_b.layout().stride(0),              // stride_b
    tensor_c.layout().stride(0),              // stride_c
    tensor_d.layout().stride(0),              // stride_d
    options.avail_sms);                       // avail_sms
}


/// Execute a given example GEMM computation
template <typename DeviceGemmT>
Result run(std::string description, Options &options)
{
  // Display test description
  std::cout << std::endl << description << std::endl;

  // Zero-initialize test output matrix D
  cutlass::reference::host::TensorFill(options.tensor_d.host_view());
  options.tensor_d.sync_device();

  // Instantiate CUTLASS kernel depending on templates
  DeviceGemmT device_gemm;

  // Create a structure of gemm kernel arguments suitable for invoking an instance of DeviceGemmT
  auto arguments = args_from_options(device_gemm, options, options.tensor_a, options.tensor_b, options.tensor_c, options.tensor_d);

  // Using the arguments, query for extra workspace required for matrix multiplication computation
  size_t workspace_size = DeviceGemmT::get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  // Check the problem size is supported or not
  CUTLASS_CHECK(device_gemm.can_implement(arguments));

  // Initialize CUTLASS kernel with arguments and workspace pointer
  CUTLASS_CHECK(device_gemm.initialize(arguments, workspace.get()));

  // Correctness / Warmup iteration
  CUTLASS_CHECK(device_gemm());

  // Copy output data from CUTLASS and reference kernel to host for comparison
  options.tensor_d.sync_host();

  // Check if output from CUTLASS kernel and reference kernel are equal or not
  Result result;
  result.passed = cutlass::reference::host::TensorEquals(
    options.tensor_d.host_view(),
    options.tensor_ref_d.host_view());

  std::cout << "  Disposition: " << (result.passed ? "Passed" : "Failed") << std::endl;

  // Run profiling loop
  if (options.iterations > 0)
  {
    GpuTimer timer;
    timer.start();
    for (int iter = 0; iter < options.iterations; ++iter) {
      CUTLASS_CHECK(device_gemm());
    }
    timer.stop();

    // Compute average runtime and GFLOPs.
    float elapsed_ms = timer.elapsed_millis();
    result.avg_runtime_ms = double(elapsed_ms) / double(options.iterations);
    result.gflops = options.gflops(result.avg_runtime_ms / 1000.0);

    std::cout << "  Avg runtime: " << result.avg_runtime_ms << " ms" << std::endl;
    std::cout << "  GFLOPs: " << result.gflops << std::endl;
  }

  if (!result.passed) {
    exit(-1);
  }

  return result;
}


/// Program entrypoint
int main(int argc, const char **argv)
{
  // CUTLASS must be compiled with CUDA 11.0 Toolkit to run these examples.
  if (!(__CUDACC_VER_MAJOR__ >= 11)) {
    std::cerr << "Ampere Tensor Core operations must be compiled with CUDA 11.0 Toolkit or later." << std::endl;

    // Returning zero so this test passes on older Toolkits. Its actions are no-op.
    return 0;
  }

  // Current device must must have compute capability at least 80
  hipDeviceProp_t props;
  int current_device_id;
  CUDA_CHECK(hipGetDevice(&current_device_id));
  CUDA_CHECK(hipGetDeviceProperties(&props, current_device_id));
  if (!((props.major * 10 + props.minor) >= 80))
  {
    std::cerr << "Ampere Tensor Core operations must be run on a machine with compute capability at least 80."
              << std::endl;

    // Returning zero so this test passes on older Toolkits. Its actions are no-op.
    return 0;
  }

  // Parse commandline options
  Options options("ampere_streamk_gemm");
  options.parse(argc, argv);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  std::cout <<
    options.iterations << " timing iterations of " <<
    options.problem_size.m() << " x " <<
    options.problem_size.n() << " x " <<
    options.problem_size.k() << " matrix-matrix multiply" << std::endl;

  if (!options.valid()) {
    std::cerr << "Invalid problem." << std::endl;
    return -1;
  }


  //
  // Initialize GEMM datasets
  //

  // Initialize tensors using CUTLASS helper functions
  options.tensor_a.resize(options.problem_size.mk());       // <- Create matrix A with dimensions M x K
  options.tensor_b.resize(options.problem_size.kn());       // <- Create matrix B with dimensions K x N
  options.tensor_c.resize(options.problem_size.mn());       // <- Create matrix C with dimensions M x N
  options.tensor_d.resize(options.problem_size.mn());       // <- Create matrix D with dimensions M x N used to store output from CUTLASS kernel
  options.tensor_ref_d.resize(options.problem_size.mn());   // <- Create matrix D with dimensions M x N used to store output from reference kernel

  // Fill matrix A on host with uniform-random data [4, -4]
  cutlass::reference::host::TensorFillRandomUniform(
      options.tensor_a.host_view(),
      1,
      ElementA(2),
      ElementA(-2),
      0);

  // Fill matrix B on host with uniform-random data [4, -4]
  cutlass::reference::host::TensorFillRandomUniform(
      options.tensor_b.host_view(),
      1,
      ElementB(2),
      ElementB(-2),
      0);

  // Fill matrix C on host with uniform-random data [4, -4]
  cutlass::reference::host::TensorFillRandomUniform(
      options.tensor_c.host_view(),
      1,
      ElementC(2),
      ElementC(-2),
      0);


  //
  // Compute reference output
  //

  // Copy data from host to GPU
  options.tensor_a.sync_device();
  options.tensor_b.sync_device();
  options.tensor_c.sync_device();

  // Zero-initialize reference output matrix D
  cutlass::reference::host::TensorFill(options.tensor_ref_d.host_view());
  options.tensor_ref_d.sync_device();

  // Create instantiation for device reference gemm kernel
  DeviceGemmReference gemm_reference;

  // Launch device reference gemm kernel
  gemm_reference(
    options.problem_size,
    ElementAccumulator(options.alpha),
    options.tensor_a.device_ref(),
    options.tensor_b.device_ref(),
    ElementAccumulator(options.beta),
    options.tensor_c.device_ref(),
    options.tensor_ref_d.device_ref());

  // Wait for kernels to finish
  CUDA_CHECK(hipDeviceSynchronize());

  // Copy output data from reference kernel to host for comparison
  options.tensor_ref_d.sync_host();


  //
  // Evaluate CUTLASS kernels
  //

  // Test default operation
  if (options.split_k_factor == 1)
  {
    // Compare basic data-parallel version versus StreamK version using default load-balancing heuristics
    Result basic_dp         = run<DeviceGemmBasic>("Basic data-parallel GEMM", options);
    Result streamk_default  = run<DeviceGemmStreamK>("StreamK GEMM with default load-balancing", options);

    printf("  Speedup vs Basic-DP: %.3f\n", (basic_dp.avg_runtime_ms / streamk_default.avg_runtime_ms));

    // Show that StreamK can emulate basic data-parallel GEMM when we set the number of SMs to load-balance across = 1
    options.avail_sms       = 1;        // Set loadbalancing width to 1 SM (no load balancing)
    Result streamk_dp       = run<DeviceGemmStreamK>("StreamK emulating basic data-parallel GEMM", options);
    options.avail_sms       = -1;       // Reset loadbalancing width to unspecified SMs (i.e., the number of device SMs)

    printf("  Speedup vs Basic-DP: %.3f\n", (basic_dp.avg_runtime_ms / streamk_dp.avg_runtime_ms));

    options.split_k_factor++;     // Increment splitting factor for next evaluation

  }

  // Show that StreamK can emulate "Split-K" with a tile-splitting factor
  Result basic_splitk = run<DeviceGemmBasic>(
    std::string("Basic split-K GEMM with tile-splitting factor ") + std::to_string(options.split_k_factor),
    options);

  Result streamk_splitk = run<DeviceGemmStreamK>(
    std::string("StreamK emulating Split-K GEMM with tile-splitting factor ") + std::to_string(options.split_k_factor),
    options);

  printf("  Speedup vs Basic-SplitK: %.3f\n", (basic_splitk.avg_runtime_ms / streamk_splitk.avg_runtime_ms));

  return 0;
}
