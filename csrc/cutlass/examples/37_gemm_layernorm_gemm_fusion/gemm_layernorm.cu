#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*! \file
    \brief CUTLASS Layernorm Example.

    This workload provides a layer normalization example using a one-pass, square-sum-based
    variance calculation. Specifically, we fuse the reduction operation to find 
    local mean and local square sum mean in the epilogue of 1st GEMM. After a light 
    full reduction kernel, the mean / variance values are readily calculated for element-wise
    operations which are fused into the 2nd GEMM.

    As stated in https://en.wikipedia.org/wiki/Algorithms_for_calculating_variance#Computing_shifted_data,
    the square-sum based one-pass implementation may raise concerns on numerical stability issues. 
    That being said, though this fully fused layernorm example almost perfectly hides all the memory cost to 
    access the intermediate matrix for layernorm computation, the numerical issue might hinder a persuasive 
    usage in real-world scenarios. If that is the case, a user may turn to the stand-alone CUTLASS layernorm
    example in tools/util/include/cutlass/util/device_layernorm.h

    Examples:

      # Run a CUTLASS layernorm example with default setup , 
      # using the language of the transformer model as an example,
      (Column Major output matrix, hidden dimension = 768, valid word number = 4096, intermediate_scale = 4)
      $ ./examples/37_gemm_layernorm_gemm_fusion/37_gemm_layernorm_gemm_fusion

      # Run an attention example with hidden dimension = 512
      $ ./examples/37_gemm_layernorm_gemm_fusion/37_gemm_layernorm_gemm_fusion --hidden_dim=512

*/

#include <cmath>
#include <iostream>
#include <vector>
#include <limits>

#include "cutlass/cutlass.h"
#include "cutlass/arch/memory.h"
#include "cutlass/arch/memory_sm75.h"
#include "cutlass/gemm/device/gemm_complex.h"
#include "cutlass/epilogue/thread/scale_type.h"

#include "cutlass/util/command_line.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/device/gemm.h"
#include "cutlass/util/reference/host/gemm_complex.h"
#include "cutlass/util/reference/host/tensor_reduce.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_norm.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/host/error_metrics.h"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/fast_math.h"
/////////////////////////////////////////////////////////////////////////////////////////////////

#include "gemm_with_layernorm.h"

/////////////////////////////////////////////////////////////////////////////////////////////////

enum class Disposition {
  kPassed,
  kIncorrect,
  kNotVerified
};

/////////////////////////////////////////////////////////////////////////////////////////////////

// Command line options parsing
template<typename LayoutOutput_>
struct Options {

  using LayoutOutput = LayoutOutput_;

  static bool const kIsColumnMajorOutput = cutlass::platform::is_same<LayoutOutput, cutlass::layout::ColumnMajor>::value;

  bool help;
  cutlass::gemm::GemmCoord problem_size0;
  cutlass::gemm::GemmCoord problem_size1;
  int hidden_dim;
  int valid_word_num;
  int intermediate_scale;
  int iterations;
  unsigned seed;
  float alpha;
  float beta;
  bool verification_enabled;
  double tolerance;

  Options():
    help(false),
    iterations(20),
    seed(2022),
    hidden_dim(768),
    valid_word_num(4096),
    intermediate_scale(4),
    alpha(1),
    beta(0),
    verification_enabled(true),
    tolerance(0.01),
    problem_size1(problem_size0.m() * 4, problem_size0.n(), problem_size0.m())
  { }

  bool valid() {

    return true;
  }

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
    }

    cmd.get_cmd_line_argument("hidden_dim", hidden_dim, 768);
    cmd.get_cmd_line_argument("valid_word_num", valid_word_num, 4096);
    cmd.get_cmd_line_argument("iterations", iterations);
    cmd.get_cmd_line_argument("verify", verification_enabled);
    cmd.get_cmd_line_argument("seed", seed);
    cmd.get_cmd_line_argument("tolerance", tolerance);

    if (kIsColumnMajorOutput) {
      // column major output setup
      problem_size0.m() = hidden_dim;
      problem_size0.n() = valid_word_num;
      problem_size0.k() = hidden_dim;

      problem_size1.m() = hidden_dim * intermediate_scale;
      problem_size1.n() = valid_word_num;
      problem_size1.k() = hidden_dim;
    }else{
      // row major output setup
      problem_size0.m() = valid_word_num;
      problem_size0.n() = hidden_dim;
      problem_size0.k() = hidden_dim;

      problem_size1.m() = valid_word_num;
      problem_size1.n() = hidden_dim * intermediate_scale;
      problem_size1.k() = hidden_dim;
    }

  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "37_gemm_layernorm_gemm_fusion example\n\n"
      << "  This example uses the CUTLASS Library to compute GEMM + Layernorm for arbitrary problem sizes.\n\n"
      << "Options:\n\n"
      << "  --help                      If specified, displays this usage statement.\n\n"
      << "  --hidden_dim=<int>          Hidden dimension\n"
      << "  --valid_word_num=<int>      Valid word number\n"
      << "  --seed=<int>                Random number seed (1*)\n\n"
      << "  --iterations=<int>          Number of profiling iterations to perform (0 to disable profiling).\n\n"
      << "  --verify=<bool>             If true, performs reference calculation.\n\n"
      << "  --tolerance <float>         Error tolerance\n"
    ;

    out << "\n\nExamples:\n\n"
      << "$ ./examples/37_gemm_layernorm_gemm_fusion/37_gemm_layernorm_gemm_fusion \\\n"
      << "     --hidden_dim=768 --valid_word_num=1024 \n\n";

    return out;
  }

  /// Returns true if the environment and Toolkit support this
  bool supported(bool verbose = true) const {

    // Ampere Tensor Core operations exposed with mma.sync and ldmatrix are first available
    // in CUDA 11.0.
    //
    // CUTLASS must be compiled with CUDA 11.0 Toolkit to run these examples.
    if (!(__CUDACC_VER_MAJOR__ >= 11)) {
      if (verbose) {
        std::cerr << "Ampere Tensor Core operations must be compiled with CUDA 11.0 Toolkit or later." << std::endl;
      }
      return false;
    }

    hipDeviceProp_t props;

    hipError_t error = hipGetDeviceProperties(&props, 0);
    if (error != hipSuccess) {
      if (verbose) {
        std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;
      }
      return false;
    }

    if (!((props.major * 10 + props.minor) >= 80)) {
      if (verbose) {
        std::cerr << "Ampere Tensor Core operations must be run on a machine with compute capability at least 80."
                  << std::endl;
      }
      return false;
    }

    //
    // CUTLASS attempts to load 128b vectors of cutlass::half_t (F16) elements. Consequently,
    // all pointers, strides, and tensor extents must be divisible by 8 elements.
    //
    int const kAlignment = 8;

    if ((problem_size0.m() % kAlignment) ||
        (problem_size0.n() % kAlignment) ||
        (problem_size0.k() % kAlignment)) {
      if (verbose) {
        std::cerr << "Misaligned input in 1st GEMM." << std::endl;
      }
      // misaligned tensors for Gemm1
      return false;
    }

    if ((problem_size1.m() % kAlignment) ||
        (problem_size1.n() % kAlignment) ||
        (problem_size1.k() % kAlignment)) {
      if (verbose) {
        std::cerr << "Misaligned input in 2nd GEMM." << std::endl;
      }
      // misaligned tensors for Gemm2
      return false;
    }

    return true;
  }
};

/////////////////////////////////////////////////////////////////////////////////////////////////

template<
  typename LayoutOutput_>
struct Testbed {

  //
  // Type definitions
  //

  // User-defined data types
  using ElementInputA0 = cutlass::half_t;
  using ElementInputB0 = cutlass::half_t;
  using ElementOutput = cutlass::half_t;
  using ElementCompute = cutlass::half_t;

  using LayoutInputA0 = cutlass::layout::RowMajor;
  using LayoutInputB0 = cutlass::layout::ColumnMajor;
  using LayoutOutput = LayoutOutput_;

  static bool const kIsColumnMajorOutput = cutlass::platform::is_same<LayoutOutput, cutlass::layout::ColumnMajor>::value;
  // turn of shifted K by default
  static bool const kIsShiftedVariance = false;

  /// Linear scaling operator
  using EpilogueFunctorOp = cutlass::epilogue::thread::LinearCombination<
    ElementOutput,
    128 / cutlass::sizeof_bits<ElementOutput>::value,
    ElementCompute,
    ElementCompute
  >;

  using ThreadblockShape = cutlass::gemm::GemmShape<128, 128, 32>;
  using WarpShape        = cutlass::gemm::GemmShape<64, 64, 32>;
  using InstructionShape = cutlass::gemm::GemmShape<16, 8, 16>;
  
  static int const kStages0  = 3;
  static int const kStages1  = 4;

  using GemmLayernorm = cutlass::GemmLayernorm<
    ElementInputA0,
    LayoutInputA0,
    ElementInputB0,
    LayoutInputB0,
    ElementOutput,
    LayoutOutput,
    ElementCompute,
    EpilogueFunctorOp,
    ThreadblockShape,
    WarpShape,
    InstructionShape,
    kStages0,
    kStages1,
    kIsShiftedVariance
  >;
  
  using ElementInputA1 = typename GemmLayernorm::ElementInputA1;
  using ElementOutputC1 = typename GemmLayernorm::ElementOutputC1;
  using ElementInputScaleBias = typename GemmLayernorm::ElementInputScaleBias;
  using ElementLayernormCompute = typename GemmLayernorm::ElementLayernormCompute;

  using LayoutInputA1 = typename GemmLayernorm::LayoutInputA1;
  using LayoutOutputC0 = typename GemmLayernorm::LayoutOutputC0;
  using LayoutOutputC1 = typename GemmLayernorm::LayoutOutputC1;
  using LayoutInputScaleBias = typename GemmLayernorm::LayoutInputScaleBias;

  //
  // Data members
  //

  Options<LayoutOutput> const &options;

  cutlass::HostTensor<ElementInputA0, LayoutInputA0>                 tensor_A0;
  cutlass::HostTensor<ElementInputB0, LayoutInputB0>                 tensor_B0;
  cutlass::HostTensor<ElementOutput, LayoutOutputC0>                 tensor_C0;
  cutlass::HostTensor<ElementInputA1, LayoutInputA1>                 tensor_A1;
  cutlass::HostTensor<ElementOutputC1, LayoutOutputC1>               tensor_C1;

  cutlass::HostTensor<ElementOutput, LayoutOutputC0>                 reference_C0;
  cutlass::HostTensor<ElementOutputC1, LayoutOutputC1>               reference_C1;

  cutlass::HostTensor<ElementInputScaleBias, LayoutInputScaleBias>   tensor_Variance;
  cutlass::HostTensor<ElementInputScaleBias, LayoutInputScaleBias>   tensor_Mean;
  cutlass::HostTensor<ElementInputScaleBias, LayoutInputScaleBias>   tensor_Beta;
  cutlass::HostTensor<ElementInputScaleBias, LayoutInputScaleBias>   tensor_Gamma;

  cutlass::HostTensor<ElementInputScaleBias, LayoutInputScaleBias>   reference_Mean;
  cutlass::HostTensor<ElementInputScaleBias, LayoutInputScaleBias>   reference_Variance;

  // shifted K tensor to better ensure the numerical stability
  // According to https://en.wikipedia.org/wiki/Algorithms_for_calculating_variance
  // the closer shifted K to the actual mean, the better numerical stability we'll observe
  cutlass::HostTensor<ElementOutput, LayoutOutputC0>                 tensor_Shifted_K;

  //
  // Methods
  //

  Testbed(
    Options<LayoutOutput> const &options_
  ):
    options(options_)
  {

    tensor_A0.reset({options.problem_size0.m(), options.problem_size0.k()});
    tensor_B0.reset({options.problem_size0.k(), options.problem_size0.n()});

    tensor_C0.reset({options.problem_size0.m(), options.problem_size0.n()});

    tensor_A1.reset({options.problem_size1.m(), options.problem_size1.k()});
    tensor_C1.reset({options.problem_size1.m(), options.problem_size1.n()});

    reference_C0.reset({options.problem_size0.m(), options.problem_size0.n()});
    reference_C1.reset({options.problem_size1.m(), options.problem_size1.n()});

    int leading_dim_0 = kIsColumnMajorOutput ? options.problem_size0.n() : options.problem_size0.m();
    int leading_dim_1 = kIsColumnMajorOutput ? options.problem_size0.m() : options.problem_size0.n();

    int block_num = (leading_dim_1 + GemmLayernorm::ThreadblockShape::kM - 1) / GemmLayernorm::ThreadblockShape::kM;

    tensor_Variance.reset({block_num, leading_dim_0});
    tensor_Mean.reset({block_num, leading_dim_0});
    tensor_Shifted_K.reset({1, leading_dim_0});

    tensor_Beta.reset({1, leading_dim_1});
    tensor_Gamma.reset({1, leading_dim_1});

    reference_Mean.reset({1, leading_dim_0}, false);
    reference_Variance.reset({1, leading_dim_0}, false);
    
  }

  /// Run
  Disposition run() {

    Disposition disposition = Disposition::kNotVerified;

    //
    // Initialize the workspace
    //

    initialize();

    //
    // Launch device kernel
    //
    cutlass::Status status = cutlass::Status::kSuccess;

    status = execute_device_kernel();

    if (status != cutlass::Status::kSuccess) {
      std::cerr << "Device execution failed." << std::endl;
      return disposition;
    }

    hipError_t result = hipDeviceSynchronize();
    if (result != hipSuccess) {
      std::cerr << "Device synchronize failed with error "
        << hipGetErrorString(result) << std::endl;
      return disposition;
    }

    //
    // Compute the reference
    //
    compute_reference();

    //
    // Verify
    //

    if (options.verification_enabled) {

      bool passed = verify();

      if (passed) {
        disposition = Disposition::kPassed;
      }
      else {
        disposition = Disposition::kIncorrect;
      }
    }

    //
    // Profiling
    //
    if (options.iterations) {
      profile();
    }

    return disposition;
  }

  /// Random initialization
  void initialize() {

    cutlass::reference::host::TensorFillRandomUniform(
      tensor_A0.host_view(),
        options.seed,
        ElementInputA0(5),
        ElementInputA0(-5),
        0
      );

    cutlass::reference::host::TensorFillRandomUniform(
      tensor_B0.host_view(),
        options.seed + 1,
        ElementInputB0(5),
        ElementInputB0(-5),
        0
      );

    cutlass::reference::host::TensorFillRandomUniform(
      tensor_A1.host_view(),
        options.seed + 2,
        ElementInputA1(5),
        ElementInputA1(-5),
        0
      );

    cutlass::reference::host::TensorFillRandomUniform(
      tensor_Beta.host_view(),
        options.seed + 3,
        ElementInputScaleBias(5),
        ElementInputScaleBias(-5),
        0
      );

    cutlass::reference::host::TensorFillRandomUniform(
      tensor_Gamma.host_view(),
        options.seed + 4,
        ElementInputScaleBias(5),
        ElementInputScaleBias(-5),
        0
      );

    cutlass::reference::host::TensorFillRandomUniform(
      tensor_Shifted_K.host_view(),
        options.seed + 5,
        ElementOutput(5),
        ElementOutput(-6),
        0
      );

    tensor_A0.sync_device();
    tensor_B0.sync_device();
    tensor_A1.sync_device();
    tensor_Beta.sync_device();
    tensor_Gamma.sync_device();

  }



  cutlass::Status execute_device_kernel() {

    cutlass::Status status = cutlass::Status::kSuccess;

    //
    // Setup arguments
    //

    typename GemmLayernorm::Arguments args(
      options.problem_size0,
      options.problem_size1,
      tensor_A0.device_ref().data(),
      tensor_B0.device_ref().data(),
      tensor_C0.device_ref().data(),
      tensor_C0.device_ref().data(),
      tensor_A1.device_ref().data(),
      tensor_C1.device_ref().data(),
      tensor_A0.device_ref().stride(0),
      tensor_B0.device_ref().stride(0),
      tensor_C0.device_ref().stride(0),
      tensor_C0.device_ref().stride(0),
      tensor_A1.device_ref().stride(0),
      tensor_C1.device_ref().stride(0),
      {
        ElementCompute(options.alpha),
        ElementCompute(options.beta)
      },
      tensor_Variance.device_ref(),
      tensor_Mean.device_ref(),
      tensor_Gamma.device_ref(),
      tensor_Beta.device_ref(),
      tensor_Shifted_K.device_ref().data()
    );

    //
    // Launch
    //

    GemmLayernorm gemm_layernorm;

    // Initialize
    status = gemm_layernorm.initialize(args);
    if (status != cutlass::Status::kSuccess) {
      return status;
    }

    // Run
    status = gemm_layernorm();

    return status;
  }

  /// Reference calculation
  void compute_reference() {

    cutlass::reference::device::Gemm<
      ElementInputA0,
      LayoutInputA0,
      ElementInputB0,
      LayoutInputB0,
      ElementOutput,
      LayoutOutputC0,
      ElementCompute,
      ElementCompute
    > gemm_device0;

    cutlass::reference::device::Gemm<
      ElementInputA1,
      LayoutInputA1,
      ElementOutput,
      LayoutOutputC0,
      ElementOutputC1,
      LayoutOutputC1,
      ElementCompute,
      ElementCompute
    > gemm_device1;

    // Compute 1st GEMM
    gemm_device0(
      options.problem_size0,
      ElementCompute(options.alpha),
      tensor_A0.device_ref(),
      tensor_B0.device_ref(),
      ElementCompute(options.beta),
      tensor_C0.device_ref(),
      reference_C0.device_ref()
    );

    reference_C0.sync_host();

    tensor_Mean.sync_host();
    tensor_Variance.sync_host();
    tensor_Gamma.sync_host();
    tensor_Beta.sync_host();
    tensor_Shifted_K.sync_host();

    // Compute the sum and square sum for verification purpose
    if (kIsColumnMajorOutput) {
      for (int n = 0; n < options.problem_size0.n(); ++n) {
      
        ElementLayernormCompute sum = ElementLayernormCompute(0);
        ElementLayernormCompute square_sum = ElementLayernormCompute(0);
        for (int m = 0; m < options.problem_size0.m(); ++m) {
          sum += ElementLayernormCompute(reference_C0.at({m, n}));
          square_sum += ElementLayernormCompute(reference_C0.at({m, n})) * ElementLayernormCompute(reference_C0.at({m, n}));
        }
        
        ElementLayernormCompute mean = sum / ElementLayernormCompute(options.problem_size0.m());
        ElementLayernormCompute square_mean = square_sum / ElementLayernormCompute(options.problem_size0.m());
        ElementLayernormCompute variance = cutlass::constants::one<ElementLayernormCompute>() / cutlass::fast_sqrt(square_mean - mean * mean + ElementLayernormCompute(1e-6) ) ;

        mean = -mean * variance;

        reference_Mean.at({0, n}) = ElementInputScaleBias(mean);
        reference_Variance.at({0, n}) = ElementInputScaleBias(variance);
      }
    }else{
      for (int m = 0; m < options.problem_size0.m(); ++m) {
      
        ElementLayernormCompute sum = ElementLayernormCompute(0);
        ElementLayernormCompute square_sum = ElementLayernormCompute(0);
        for (int n = 0; n < options.problem_size0.n(); ++n) {
          sum += ElementLayernormCompute(reference_C0.at({m, n})) ;
          square_sum += ElementLayernormCompute(reference_C0.at({m, n})) * ElementLayernormCompute(reference_C0.at({m, n})) ;
        }

        ElementLayernormCompute mean = sum / ElementLayernormCompute(options.problem_size0.n());
        ElementLayernormCompute square_mean = square_sum / ElementLayernormCompute(options.problem_size0.n());
        ElementLayernormCompute variance = cutlass::constants::one<ElementLayernormCompute>() / cutlass::fast_sqrt(square_mean - mean * mean + ElementLayernormCompute(1e-6)) ;

        mean = -mean * variance;

        reference_Mean.at({0, m}) = ElementInputScaleBias(mean);
        reference_Variance.at({0, m}) = ElementInputScaleBias(variance);
      }
    }

    // Element-wise transform for OutputC0 using 1-pass layernorm algo
    if (kIsColumnMajorOutput) {
      for (int n = 0; n < options.problem_size0.n(); ++n) {

        ElementLayernormCompute sum = ElementLayernormCompute(0);
        for (int m = 0; m < options.problem_size0.m(); ++m) {
          sum += ElementLayernormCompute(reference_C0.at({m, n})) ;
        }

        ElementInputScaleBias mean = ElementInputScaleBias(sum / ElementLayernormCompute(options.problem_size0.m()));
        sum = ElementLayernormCompute(0);
        for (int m = 0; m < options.problem_size0.m(); ++m) {
          sum += ElementLayernormCompute(reference_C0.at({m, n}) - ElementLayernormCompute(mean)) * ElementLayernormCompute(reference_C0.at({m, n}) - ElementLayernormCompute(mean)) ;
        }

        ElementLayernormCompute square_mean = sum / ElementLayernormCompute(options.problem_size0.m());
        ElementInputScaleBias variance = ElementInputScaleBias(cutlass::constants::one<ElementLayernormCompute>() 
                            / cutlass::fast_sqrt(square_mean + ElementLayernormCompute(1e-6))) ;

        for (int m = 0; m < options.problem_size0.m(); ++m) {
          reference_C0.at({m, n}) = 
              ElementOutput( ( (ElementInputScaleBias(reference_C0.at({m, n})) - mean) * variance )
                * tensor_Gamma.at({0, m}) + tensor_Beta.at({0, m}));

        }

      }
    }else{

      for (int m = 0; m < options.problem_size0.m(); ++m) {

        float sum = float(0);
        for (int n = 0; n < options.problem_size0.n(); ++n) {
          sum += float(reference_C0.at({m, n})) ;
        }

        float mean = sum / float(options.problem_size0.n());
        sum = float(0);
        for (int n = 0; n < options.problem_size0.n(); ++n) {
          sum += float(reference_C0.at({m, n}) - mean) * float(reference_C0.at({m, n}) - mean) ;
        }

        float square_mean = sum / float(options.problem_size0.n());
        float variance = cutlass::constants::one<float>() / cutlass::fast_sqrt(square_mean + ElementLayernormCompute(1e-6)) ;

        for (int n = 0; n < options.problem_size0.n(); ++n) {
          reference_C0.at({m, n}) = 
              ElementOutput( ( (float(reference_C0.at({m, n})) - mean) * variance )
                * float(tensor_Gamma.at({0, n})) + float(tensor_Beta.at({0, n})));

        }

      }

    }


    // Sync host data with device after element-wise transform
    reference_C0.sync_device();

    // Compute 2nd GEMM
    gemm_device1(
      options.problem_size1,
      ElementCompute(options.alpha),
      kIsColumnMajorOutput ? tensor_A1.device_ref() : reference_C0.device_ref(),
      kIsColumnMajorOutput ? reference_C0.device_ref() :tensor_A1.device_ref(),
      ElementCompute(options.beta),
      reference_C1.device_ref(),
      reference_C1.device_ref()
    );

  }

  /// Emits all tensor values
  void emit_results() {
    std::cout << "tensor_C1 = \n" << tensor_C1.host_view() << "\n\n";
    std::cout << "Reference C1 = \n" << reference_C1.host_view() << "\n\n";
    std::cout << "Mean = \n" << tensor_Mean.host_view() << "\n\n";
    std::cout << "rsqrt(Variance) = \n" << tensor_Variance.host_view() << "\n\n";
    std::cout << "Reference Mean = \n" << reference_Mean.host_view() << "\n\n";
    std::cout << "Reference rsqrt(Variance) = \n" << reference_Variance.host_view() << "\n\n";
  }

  template<typename Element, typename Layout>
  bool verify_tensor(cutlass::HostTensor<Element, Layout> tensor, \
                       cutlass::HostTensor<Element, Layout> reference,
                       int leading_dim0, int leading_dim1, bool is_print = false) {
    float const kThreshold = float(options.tolerance);
    float const kAbsThreshold = 0.5f;
    float const kRelativeThreshold = 0.1f;
    // Adds a constant bias to avoid being divided by '0'
    float const kBias = 1e-5f;
    int counter = 0;
    for (int m = 0; m < leading_dim0; m++) {
      for (int n = 0; n < leading_dim1; ++n) {
        float diff = (float)(tensor.at({m, n}) - reference.at({m, n}));
        float rel_diff = fabs(diff) / fabs(reference.at({m, n}) + kBias);
        if (fabs(diff) > kAbsThreshold && rel_diff > kRelativeThreshold) {
          counter++;
        }
      }
    }

    float err_rate = float(counter) / (float(leading_dim0) * float(leading_dim1));
    return (err_rate < kThreshold);
  }

  /// Verifies the reference matches
  bool verify() {

    tensor_Variance.sync_host();
    tensor_Mean.sync_host();
    tensor_C1.sync_host();
    reference_C1.sync_host();

    // Verification checks - set any of these to 'true' to override the verification checks.
    bool verified_C1 = false;
    bool verified_Mean = false;
    bool verified_Variance = false;

    // Verify layernorm output
    if (!verified_C1) {
      verified_C1 = verify_tensor<ElementOutputC1, LayoutOutputC1>(tensor_C1, reference_C1, options.problem_size1.m(), options.problem_size1.n());
    }

    if (!verified_Variance) {
      verified_Variance = verify_tensor<ElementInputScaleBias, LayoutInputScaleBias>(tensor_Variance, reference_Variance, 1, options.problem_size0.n());
    }

    if (!verified_Mean) {
      verified_Mean = verify_tensor<ElementInputScaleBias, LayoutInputScaleBias>(tensor_Mean, reference_Mean, 1, options.problem_size0.n());
    }

    if (!verified_C1 || !verified_Mean || !verified_Variance) {

      // emit_results();

      std::cerr << "Verification check failed for tensor Layernorm" << std::endl;

      // Summarize which checks failed
      if (!verified_C1) {
        std::cerr << "Verification of O tensor failed\n";
      }

      if (!verified_Mean) {
        std::cerr << "Verification of Mean tensor failed\n";
      }

      if (!verified_Variance) {
        std::cerr << "Verification of Variance tensor failed\n";
      }

      return false;
    }

    return true;
  }

  /// Profiles
  bool profile() {

    //
    // Profile
    //

    cutlass::Status status = cutlass::Status::kSuccess;
    hipError_t result;
    hipEvent_t events[2];
    int const kIterations = options.iterations;

    for (hipEvent_t &evt : events) {
      result = hipEventCreate(&evt);
      if (result != hipSuccess) {
        std::cerr << "hipEventCreate failed with error " << hipGetErrorString(result) << std::endl;
        return false;
      }
    }

    result = hipEventRecord(events[0]);

    if (result != hipSuccess) {
      std::cerr << "hipEventRecord() failed with error " << hipGetErrorString(result) << std::endl;
      return false;
    }

    for (int iter = 0; iter < kIterations; ++iter) {

      status = execute_device_kernel();

      if (status != cutlass::Status::kSuccess) {
        std::cerr << "Device execution failed." << std::endl;
        return false;
      }
    }

    result = hipEventRecord(events[1]);

    if (result != hipSuccess) {
      std::cerr << "hipEventRecord() failed with error " << hipGetErrorString(result) << std::endl;
      return false;
    }

    result = hipDeviceSynchronize();

    if (result != hipSuccess) {
      std::cerr << "hipDeviceSynchronize() failed with error " << hipGetErrorString(result) << std::endl;
      return false;
    }

    float elapsed_ms = 0;
    result = hipEventElapsedTime(&elapsed_ms, events[0], events[1]);

    float elapsed_ms_per_iter = elapsed_ms / float(kIterations);

    if (result != hipSuccess) {
      std::cerr << "hipEventElapsedTime() failed with error " << hipGetErrorString(result) << std::endl;
      return false;
    }

    for (hipEvent_t &evt : events) {
      result = hipEventDestroy(evt);
      if (result != hipSuccess) {
        std::cerr << "hipEventDestroy() failed with error " << hipGetErrorString(result) << std::endl;
        return false;
      }
    }

    int64_t flops = int64_t(options.problem_size0.m()) * options.problem_size0.n() * options.problem_size0.k() * 2 \
                   + int64_t(options.problem_size1.m()) * options.problem_size1.n() * options.problem_size1.k() * 2;

    double gflops_per_second = double(flops) * kIterations / double(elapsed_ms / 1000.0f) / double(1.0e9);

    std::cout << "    1st  GEMM: "
              << options.problem_size0.m() << "-by-" << options.problem_size0.n() << "-by-" << options.problem_size0.k() << "\n"
              << "    2nd  GEMM: "
              << options.problem_size1.m() << "-by-" << options.problem_size1.n() << "-by-" << options.problem_size1.k()
              << std::endl;

    std::cout << " Runtime / iteration: " << elapsed_ms_per_iter << " ms\n" << std::endl;
    std::cout << "              GFLOPs: " << gflops_per_second << "  GFLOPs" << std::endl;

    return true;
  }
};

/////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, const char **argv) {
  
  // Define final layout
  using LayoutOutput = cutlass::layout::ColumnMajor;

  // Options parsing
  Options<LayoutOutput> options;
  options.parse(argc, argv);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  if (!options.supported()) {
    return 0;
  }

  // Run
  Testbed<LayoutOutput> testbed(options);

  Disposition disposition = testbed.run();

  std::cout << std::endl;

  switch (disposition) {
    case Disposition::kPassed:
      std::cout << "Passed" << std::endl;
      break;
    case Disposition::kIncorrect:
      std::cout << "Incorrect" << std::endl;
      break;
    case Disposition::kNotVerified:
      std::cout << "Not verified" << std::endl;
      break;
  }

  return (disposition == Disposition::kPassed ? 0 : -1);
}


/////////////////////////////////////////////////////////////////////////////////////////////////
