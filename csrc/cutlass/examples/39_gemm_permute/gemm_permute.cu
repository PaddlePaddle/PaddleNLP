/***************************************************************************************************
 * Copyright (c) 2017 - 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*! \file
    \brief GEMM Permute Example.

    This example computes batched GEMM operations with output results permuted as reshaped tensors.

    We provide layout plugin as a flexible tool for users to add any customized output tensor permute operation, 
    or any other generalized global memory writeout address computation. To add a customized layout, add new class
    in include/cutlass/layout/permute.h

    In this example, we used Tensor4DPermuteBMM0213 layout to perform Batched GEMM with permute([0, 2, 1, 3]) on BMM
    whole output tensor, and used Tensor5DPermute20314 layout to perform Normal GEMM with permute([2, 0, 3, 1, 4]) on
    output matrix. The address computations are performed in compute(col_init, row_init, stride_init, 
    BMM_batch_idx) with {col_permute, row_permute and stride_permute} as new addresses after permute op.
    (check include/cutlass/layout/permute.h)

    Tips:
    
      1) Make sure to set batch_stride_D to zero for BMM permute; Also the BMM GEMM should be in mode
      cutlass::gemm::GemmUniversalMode::kBatched instead of kArray

      2) When the last dimension is touched in permute op (for example permute([0, 2, 3, 1])), AlignmentC should 
      be set to 1. If the last dimension is untouched, one can set AlignmentC to be larger like 8 in our example.
      As a result, permute op without touching the last dimension is recommended to obtain the best performance gain.

    Examples:

      # Runs a batched GEMM with 96 batches
      $ ./examples/39_gemm_permute/39_gemm_permute --problem-count=96

      # Runs a batched GEMM with 96 batches (with GEMM-K dimension equal to 1024)
      $ ./examples/39_gemm_permute/39_gemm_permute --problem-count=96 --k=1024 --verbose=true

      # Execute batched GEMM and profile with NSight
      $ nv-nsight-cu-cli ./examples/39_gemm_permute/39_gemm_permute --m=256 --n=192 --k=256 --verbose=true --iterations=1 --reference-check=false

*/

/////////////////////////////////////////////////////////////////////////////////////////////////

#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <map>
#include <unordered_map>

#include "cutlass/cutlass.h"
#include "cutlass/gemm/gemm.h"
#include "cutlass/gemm/device/gemm_universal.h"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/device_memory.h"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/host/gemm_complex.h"
#include "cutlass/util/reference/device/gemm_complex.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/device/tensor_fill.h"
#include "cutlass/util/reference/host/tensor_norm.h"

#include "cutlass/layout/permute.h"

/// Tensor4DPermuteBMM0213 --->
/// Permute layout function for 4-D permuted tensors for BMM with BMM output tensor (dimension as [B, M, N]) reshaped
/// as [B/D1, D1, M, N]. Then perform permute([0, 2, 1, 3]) on the corresponding whole BMM output tensor.
const int D1 = 12;

/// Tensor5DPermute20314 --->
/// Permute layout function for 5-D permuted tensors with output matrix (dimension as [M, N]) reshaped
/// as [M/T1, T1, T2, T3, N/T2/T3]. Then perform permute([2, 0, 3, 1, 4]) on the corresponding output tensor.
const int T1 = 16; 
const int T2 = 3;
const int T3 = 8;

// Alignment C
const int AlignmentC = 8;

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Result structure
struct Result {

  double runtime_ms;
  double gflops;
  cutlass::Status status;
  hipError_t error;
  bool passed;

  //
  // Methods
  //

  Result(
    double runtime_ms = 0,
    double gflops = 0,
    cutlass::Status status = cutlass::Status::kSuccess,
    hipError_t error = hipSuccess
  ):
    runtime_ms(runtime_ms), gflops(gflops), status(status), error(error), passed(true) { }
};

/////////////////////////////////////////////////////////////////////////////////////////////////

// Command line options parsing
struct Options {

  bool help;
  bool error;
  bool reference_check;

  cutlass::gemm::GemmCoord problem_each;

  int batch_count;
  int iterations;
  int cuda_streams;
  bool verbose;
  float alpha;
  float beta;

  //
  // Methods
  // 

  Options():
    help(false),
    error(false),
    reference_check(true),
    batch_count(-1),
    iterations(20),
    cuda_streams(0),
    verbose(false),
    alpha(1),
    beta()
  { }

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
      return;
    }

    cmd.get_cmd_line_argument("alpha", alpha, 1.0f);
    cmd.get_cmd_line_argument("beta", beta, 0.0f);    
    cmd.get_cmd_line_argument("iterations", iterations, 20);
    cmd.get_cmd_line_argument("streams", cuda_streams, 0);
    cmd.get_cmd_line_argument("verbose", verbose, false);
    cmd.get_cmd_line_argument("reference-check", reference_check, true);

    int m, n, k;

    cmd.get_cmd_line_argument("m", m, 128);
    cmd.get_cmd_line_argument("n", n, 192);
    cmd.get_cmd_line_argument("k", k, 128);
    cmd.get_cmd_line_argument("batch-count", batch_count, 768);

    cutlass::gemm::GemmCoord problem(m, n, k);
    problem_each = problem;

    if (batch_count % D1 != 0){
      std::cerr << "\nProblem count error (problem-count = " << batch_count << "). " 
        << "problem-count needs to be divided with no remain by " << D1 << " (D1)."
        << " (Required by the Batched GEMM permute Tensor4DPermuteBMM0213)\n\n";
      error = true;
    }

    if (m % (AlignmentC * T1) != 0){
      std::cerr << "\nProblem m size error (m = " << m << "). " 
        << "m needs to be divided with no remain by " << (AlignmentC * T1) << " (AlignmentC * T1)."
        << " (Required by the normal GEMM permute Tensor5DPermute20314)\n\n";
        error = true;
    }

    if (n % (AlignmentC * (T2 * T3)) != 0){
      std::cerr << "\nProblem n size error (n = " << n << "). " 
        << "n needs to be divided with no remain by " << (AlignmentC * (T2 * T3)) << " (AlignmentC * T2 * T3)."
        << " (Required by the normal GEMM permute Tensor5DPermute20314)\n\n";
        error = true;
    }
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "39_gemm_permute\n\n"
      << " 1) This example firstly profiles the performance of a batched GEMM kernel with BMM whole output"
      << " (including output matrices for each batch) as permuted 4D Tensor."
      << " The BMM tensor output in shape of [B, M, N] is reshaped as [B/D1, D1, M, N] and then permuted with"
      << " permute([0, 2, 1, 3]) to be in shape of [B/D1, M, D1, N].\n\n"
      << " 2) This example also profiles the performance of a normal GEMM kernel with output as permuted 5D Tensor."
      << " The GEMM matrix output in shape of [M, N]  is reshaped as [M/T1, T1, T2, T3, N/T2/T3] and then permuted"
      << " with permute([2, 0, 3, 1, 4]) to be in shape of [T2, M/T1, T3, T1, N/T2/T3].\n\n"
      << " Note: D1, T1, T2, T3 are compile-time constants defined in gemm_permute.cu\n\n"
      << "Options:\n\n"
      << "  --help                      If specified, displays this usage statement.\n\n"
      << "  --batch-count=<int>         Sets the number of batches in batched GEMM (batch number for BMM). (default: --batch-count=768)\n"
      << "  --m=<int>                   Sets the M dimension for both batched GEMM and normal GEMM problems. (default: --m=128)\n"
      << "  --n=<int>                   Sets the N dimension for both batched GEMM and normal GEMM problems. (default: --n=192)\n"
      << "  --k=<int>                   Sets the K dimension for both batched GEMM and normal GEMM problems. (default: --k=128)\n"
      << "  --alpha=<f32>               Epilogue scalar alpha (real part)\n"
      << "  --beta=<f32>                Epilogue scalar beta (real part)\n\n"
      << "  --iterations=<int>          Number of profiling iterations to perform.\n"
      << "  --reference-check=<bool>    If true, performs reference check.\n"
      << "  --verbose=<bool>            If true, prints problem sizes and batching structure.\n";

    out << "\n\nExamples:\n\n"

      << "# Runs a batched GEMM with 96 batches\n"
      << "$ ./examples/39_gemm_permute/39_gemm_permute --problem-count=96\n\n"

      << "# Runs a batched GEMM with 96 batches (with GEMM-K dimension equal to 1024)\n"
      << "$ ./examples/39_gemm_permute/39_gemm_permute --problem-count=96 --k=1024 --verbose=true\n\n"

      << "# Execute batched GEMM and profile with NSight\n"
      << "$ nv-nsight-cu-cli ./examples/39_gemm_permute/39_gemm_permute --m=256 --n=192 --k=256 --verbose=true --iterations=1 --reference-check=false\n\n";

    return out;
  }

  /// Compute performance in GFLOP/s
  double gflops(double runtime_s) const {

    // Number of real-valued multiply-adds 
    int64_t fmas = int64_t();

   fmas += problem_each.product() * batch_count;
    
    // Two flops per multiply-add
    return 2.0 * double(fmas) / double(1.0e9) / runtime_s;
  }
};

///////////////////////////////////////////////////////////////////////////////////////////////////

template <typename GemmBatched, typename GemmPermute>
class Testbed {
public:

  //
  // Type definitions
  //

  using ElementA = typename GemmBatched::ElementA;
  using ElementB = typename GemmBatched::ElementB;
  using ElementC = typename GemmBatched::ElementC;
  using ElementAccumulator = typename GemmBatched::ElementAccumulator;

  using EpilogueOutputOp = typename GemmBatched::GemmKernel::Epilogue::OutputOp;
  using ElementCompute = typename EpilogueOutputOp::ElementCompute;

  using LayoutA = typename GemmBatched::LayoutA;
  using LayoutB = typename GemmBatched::LayoutB;
  using LayoutC = typename GemmBatched::LayoutC;

  using MatrixCoord = typename LayoutC::TensorCoord;

private:

  //
  // Data members
  //

  Options & options;

  /// Initialization
  cutlass::Distribution::Kind init_A;
  cutlass::Distribution::Kind init_B;
  cutlass::Distribution::Kind init_C;
  uint32_t seed;

  cutlass::DeviceAllocation<ElementA> block_A;
  cutlass::DeviceAllocation<ElementB> block_B;
  cutlass::DeviceAllocation<ElementC> block_C;
  cutlass::DeviceAllocation<ElementC> block_D;

public:

  //
  // Methods
  //

  Testbed(
    Options &options_,
    cutlass::Distribution::Kind init_A_ = cutlass::Distribution::Uniform,
    cutlass::Distribution::Kind init_B_ = cutlass::Distribution::Uniform,
    cutlass::Distribution::Kind init_C_ = cutlass::Distribution::Uniform,
    uint32_t seed_ = 3090
  ):
    options(options_), init_A(init_A_), init_B(init_B_), init_C(init_C_), seed(seed_) { }

  /// Verbose BMM info
  void print_BMM_info_() {

    // Print batched GEMM
    std::cout << "Batched GEMM with permute([0, 2, 1, 3]) on BMM whole output tensor:\n";

    auto problem = options.problem_each;
    std::cout 
      << problem.m() << "-by-" << problem.n() << "-by-" << problem.k() 
      << ", batch count: " << options.batch_count << "\n";

    std::cout << "output tensor shape: [" << options.batch_count << ", " << problem.m() << ", "
      << problem.n() <<"]\n";
    std::cout << "reshaped as: [" << options.batch_count / D1 << ", " << D1 << ", "
      << problem.m() << ", " << problem.n() <<"]\n";
    std::cout << "finally permuted as: [" << options.batch_count / D1 << ", " << problem.m() << ", "
      << D1 << ", " << problem.n() <<"]\n";

    std::cout << "----------------------------------------------------\n";

  }

  /// Verbose normal GEMM info
  void print_GEMM_info_() {

    // Print batched GEMM
    std::cout << "Normal GEMM with permute([2, 0, 3, 1, 4]):\n";

    auto problem = options.problem_each;
    std::cout 
      << problem.m() << "-by-" << problem.n() << "-by-" << problem.k() << "\n";

    std::cout << "output tensor shape: [" << problem.m() << ", " << problem.n() <<"]" << std::endl;
    std::cout << "reshaped as: [" << problem.m() / T1 << ", " << T1 << ", "
      << T2 << ", " << T3 << ", " << problem.n() / T2 / T3 <<"]" << std::endl;
    std::cout << "finally permuted as: [" << T2 << ", " << problem.m() / T1 << ", "
      << T3 << ", " << T1 << ", " << problem.n() / T2 / T3 <<"]" << std::endl;

    std::cout << "----------------------------------------------------\n";

  }

private:

  /// Helper to initialize a tensor view
  template <typename Element>
  void initialize_tensor_(
    Element *ptr,
    size_t capacity, 
    cutlass::Distribution::Kind dist_kind,
    uint32_t seed) {

    if (dist_kind == cutlass::Distribution::Uniform) {

      Element scope_max, scope_min;
      int bits_input = cutlass::sizeof_bits<Element>::value;
      int bits_output = cutlass::sizeof_bits<typename GemmBatched::ElementC>::value;

      if (bits_input == 1) {
        scope_max = 2;
        scope_min = 0;
      } else if (bits_input <= 8) {
        scope_max = 2;
        scope_min = -2;
      } else if (bits_output == 16) {
        if (cutlass::sizeof_bits<ElementAccumulator>::value <= 16) {
          scope_max = 5;
          scope_min = -5;
        }
        else {
          scope_max = 8;
          scope_min = -8;
        }
      } else {
        scope_max = 8;
        scope_min = -8;
      }

      cutlass::reference::device::BlockFillRandomUniform(
        ptr, capacity, seed, scope_max, scope_min, 0);
    } 
    else if (dist_kind == cutlass::Distribution::Gaussian) {

      cutlass::reference::device::BlockFillRandomGaussian(
        ptr, capacity, seed, Element(), Element(0.5f));
    }
    else if (dist_kind == cutlass::Distribution::Sequential) {

      // Fill with increasing elements
      cutlass::reference::device::BlockFillSequential(
        ptr, capacity, Element(1), Element());
    } 
    else {

      // Fill with all 1s
      cutlass::reference::device::BlockFillSequential(
        ptr, capacity, Element(), Element(1));
    }
  }

  /// Initializes data structures
  void initialize_(int batch_count) {

    //
    // Choose random problem sizes
    //

    // construct a few problems of random sizes
    srand(seed);

    int64_t total_elements_A = options.problem_each.m() * options.problem_each.k() * batch_count;
    int64_t total_elements_B = options.problem_each.n() * options.problem_each.k() * batch_count;
    int64_t total_elements_C = options.problem_each.m() * options.problem_each.n() * batch_count;
    int64_t total_elements_D = options.problem_each.m() * options.problem_each.n() * batch_count;

    //
    // Assign space
    //

    block_A.reset(total_elements_A);
    block_B.reset(total_elements_B);
    block_C.reset(total_elements_C);
    block_D.reset(total_elements_D);

    //
    // Initialize the problems of the workspace
    //

    initialize_tensor_(block_A.get(), total_elements_A, init_A, seed * 2021);
    initialize_tensor_(block_B.get(), total_elements_B, init_B, seed * 2022);
    initialize_tensor_(block_C.get(), total_elements_C, init_C, seed * 2023);

    cutlass::reference::device::BlockFillSequential(
      block_D.get(), total_elements_D, ElementC(), ElementC());
  }

  /// Verifies the BMM GEMM result
  bool verify_BMM_() {

    bool passed = true;

    cutlass::gemm::GemmCoord problem = options.problem_each;

    LayoutA layout_A(LayoutA::packed({problem.m(), problem.k()}).stride(0));
    LayoutB layout_B(LayoutB::packed({problem.k(), problem.n()}).stride(0));
    LayoutC layout_C(LayoutC::packed({problem.m(), problem.n()}).stride(0));
    LayoutC layout_D(LayoutC::packed({problem.m(), problem.n()}).stride(0));

    MatrixCoord extent_A{problem.m(), problem.k()};
    MatrixCoord extent_B{problem.k(), problem.n()};
    MatrixCoord extent_C{problem.m(), problem.n()};
    
    cutlass::TensorView<ElementA, LayoutA> view_A(block_A.get(), layout_A, extent_A);
    cutlass::TensorView<ElementB, LayoutB> view_B(block_B.get(), layout_B, extent_B);
    cutlass::TensorView<ElementC, LayoutC> view_C(block_C.get(), layout_C, extent_C);

    cutlass::DeviceAllocation<ElementC>    block_Ref(layout_D.capacity(extent_C) * options.batch_count);
    cutlass::TensorView<ElementC, LayoutC> view_Ref_device(block_Ref.get(), layout_D, extent_C);

    // Reference GEMM
    cutlass::reference::device::GemmComplex<
        ElementA, LayoutA,
        ElementB, LayoutB,
        ElementC, LayoutC, 
        ElementCompute, ElementAccumulator
    >(
      problem,
      options.alpha, 
      view_A,
      GemmBatched::kTransformA,
      view_B,
      GemmBatched::kTransformB,
      options.beta, 
      view_C, 
      view_Ref_device, 
      ElementAccumulator(0),
      options.batch_count,
      options.problem_each.m() * options.problem_each.k(),
      options.problem_each.n() * options.problem_each.k(),
      options.problem_each.m() * options.problem_each.n(),
      options.problem_each.m() * options.problem_each.n()
    );

    // Copy to host memory
    std::vector<ElementC> matrix_D(layout_D.capacity(extent_C) * options.batch_count);
    std::vector<ElementC> matrix_Ref(layout_D.capacity(extent_C) * options.batch_count);

    cutlass::device_memory::copy_to_host(matrix_D.data(), block_D.get(), matrix_D.size());
    cutlass::device_memory::copy_to_host(matrix_Ref.data(), block_Ref.get(), matrix_D.size());

    // Print out the results and reference in 4D Tensor
    // [options.batch_count, options.problem_each.m() * options.problem_each.n()] -> [D0, D1, D2, D3].
    // After permute Op, -> [D0, D2, D1, D3].
    int D0 = options.batch_count / D1;
    int D2 = options.problem_each.m();
    int D3 = options.problem_each.n();

    cutlass::TensorView<ElementC, cutlass::layout::TensorNHWC> view_D_Tensor(matrix_D.data(),   // if LayoutC = cutlass::layout::ColumnMajor, view_D_Tensor should be constructed differently
      cutlass::layout::TensorNHWC().packed(cutlass::Tensor4DCoord({D0, D2, D1, D3})), cutlass::Tensor4DCoord({D0, D2, D1, D3})); 

    cutlass::TensorView<ElementC, cutlass::layout::TensorNHWC> view_Ref_Tensor(matrix_Ref.data(), 
      cutlass::layout::TensorNHWC().packed(cutlass::Tensor4DCoord({D0, D1, D2, D3})), cutlass::Tensor4DCoord({D0, D1, D2, D3}));

    // Tensor Permute Op on reference tensor
    cutlass::HostTensor<ElementC, cutlass::layout::TensorNHWC> view_Ref_Permute_Tensor(cutlass::Tensor4DCoord({D0, D2, D1, D3}));
    for (int n = 0; n < D0; ++n) {
      for (int h = 0; h < D1; ++h) {
        for (int w = 0; w < D2; ++w) {
          for (int c = 0; c < D3; ++c) {
            view_Ref_Permute_Tensor.at({n, w, h, c}) = view_Ref_Tensor.at({n, h, w, c});
          }
        }
      }
    }

    // Reference check
    passed = cutlass::reference::host::TensorEquals(view_Ref_Permute_Tensor.host_view(), view_D_Tensor);

    if (!passed) {
      std::cerr << "\n***\nError - problem failed the QA check\n***\n" << std::endl;
      return passed;
    }

    std::cout << "Passed verification" << std::endl;
    return passed;
  }

  bool verify_GEMM_normal_() {

    bool passed = true;

    cutlass::gemm::GemmCoord problem = options.problem_each;

    LayoutA layout_A(LayoutA::packed({problem.m(), problem.k()}).stride(0));
    LayoutB layout_B(LayoutB::packed({problem.k(), problem.n()}).stride(0));
    LayoutC layout_C(LayoutC::packed({problem.m(), problem.n()}).stride(0));
    LayoutC layout_D(LayoutC::packed({problem.m(), problem.n()}).stride(0));

    MatrixCoord extent_A{problem.m(), problem.k()};
    MatrixCoord extent_B{problem.k(), problem.n()};
    MatrixCoord extent_C{problem.m(), problem.n()};
    
    cutlass::TensorView<ElementA, LayoutA> view_A(block_A.get(), layout_A, extent_A);
    cutlass::TensorView<ElementB, LayoutB> view_B(block_B.get(), layout_B, extent_B);
    cutlass::TensorView<ElementC, LayoutC> view_C(block_C.get(), layout_C, extent_C);

    cutlass::DeviceAllocation<ElementC>    block_Ref(layout_D.capacity(extent_C));
    cutlass::TensorView<ElementC, LayoutC> view_Ref_device(block_Ref.get(), layout_D, extent_C);

    // Reference GEMM
    cutlass::reference::device::GemmComplex<
        ElementA, LayoutA,
        ElementB, LayoutB,
        ElementC, LayoutC, 
        ElementCompute, ElementAccumulator
    >(
      problem,
      options.alpha, 
      view_A,
      GemmBatched::kTransformA,
      view_B,
      GemmBatched::kTransformB,
      options.beta, 
      view_C, 
      view_Ref_device, 
      ElementAccumulator(0)
    );

    // Copy to host memory
    std::vector<ElementC> matrix_D(layout_D.capacity(extent_C));
    std::vector<ElementC> matrix_Ref(layout_D.capacity(extent_C));

    cutlass::device_memory::copy_to_host(matrix_D.data(),   block_D.get(), matrix_D.size());
    cutlass::device_memory::copy_to_host(matrix_Ref.data(), block_Ref.get(),                matrix_D.size());

    // Print out the results and reference in 5D Tensor
    // [options.problem_each.m(),  options.problem_each.n()] -> [T0, T1, T2, T3, T4].
    // options.problem_each.m() == T0 * T1
    // options.problem_each.n() == T2 * T3 * T4
    // After permute Op, -> [T2, T0, T3, T1, T4].
    int T0 = options.problem_each.m() / T1;
    int T4 = options.problem_each.n() / T2 / T3;

    cutlass::TensorView<ElementC, cutlass::layout::TensorNDHWC> view_D_Tensor(matrix_D.data(),   // if LayoutC = cutlass::layout::ColumnMajor, view_D_Tensor should be constructed differently
      cutlass::layout::TensorNDHWC().packed(cutlass::Tensor5DCoord({T2, T0, T3, T1, T4})), cutlass::Tensor5DCoord({T2, T0, T3, T1, T4})); 
    cutlass::TensorView<ElementC, cutlass::layout::TensorNDHWC> view_Ref_Tensor(matrix_Ref.data(), 
      cutlass::layout::TensorNDHWC().packed(cutlass::Tensor5DCoord({T0, T1, T2, T3, T4})), cutlass::Tensor5DCoord({T0, T1, T2, T3, T4}));

    // Tensor Permute Op on reference tensor
    cutlass::HostTensor<ElementC, cutlass::layout::TensorNDHWC> view_Ref_Permute_Tensor(cutlass::Tensor5DCoord({T2, T0, T3, T1, T4}));
    for (int n = 0; n < T0; ++n) {
      for (int d = 0; d < T1; ++d) {
        for (int h = 0; h < T2; ++h) {
          for (int w = 0; w < T3; ++w) {
            for (int c = 0; c < T4; ++c) {
              view_Ref_Permute_Tensor.at({h, n, w, d, c}) = view_Ref_Tensor.at({n, d, h, w, c}); // permute([2,0,3,1,4])
            }
          }
        }
      }
    }

    // Reference check
    passed = cutlass::reference::host::TensorEquals(view_Ref_Permute_Tensor.host_view(), view_D_Tensor);

    if (!passed) {
      std::cerr << "\n***\nError - problem failed the QA check\n***\n" << std::endl;
      return passed;
    }

    std::cout << "Passed verification" << std::endl;
    return passed;
}

public:
  /// Executes a conventional batched GEMM kernel.
  Result profile_batched_kBatched() {

    std::cout << "\n====================================================" << std::endl;
    std::cout << "Batched GEMM (CUTLASS):\n"
      << "====================================================" << std::endl;
    
    if (options.verbose) {
      print_BMM_info_();
    }

    Result result;

    result.passed = false;

    // Initialize the problem
    initialize_(options.batch_count);

    // Configure the GEMM arguments
    typename EpilogueOutputOp::Params epilogue_op(options.alpha, options.beta);

    // Please make sure all problem_sizes are the same for kBatched mode
    auto problem = options.problem_each;

    // For regular BMM
    int64_t batch_stride_C = problem.m() * problem.n();
    // For BMM permute output ---> make sure to set batch_stride_D to zero for BMM permute op
    int64_t batch_stride_D = 0;

    // Configure GEMM arguments
    typename GemmBatched::Arguments arguments{
      cutlass::gemm::GemmUniversalMode::kBatched,
      options.problem_each,
      options.batch_count,
      epilogue_op,
      (void*)block_A.get(),
      (void*)block_B.get(),
      (void*)block_C.get(),
      (void*)block_D.get(),
      problem.m() * problem.k(),
      problem.n() * problem.k(),
      batch_stride_C,
      batch_stride_D,
      problem.k(),
      problem.n(),
      problem.n(),
      problem.n()
    };

    // Initialize the GEMM object
    GemmBatched gemm;

    result.status = gemm.initialize(arguments, nullptr);

    if (result.status != cutlass::Status::kSuccess) {
      std::cerr << "Failed to initialize CUTLASS Batched GEMM kernel." << std::endl;
      return result;
    }

    // Run the batched GEMM object
    result.status = gemm.run();

    if (result.status != cutlass::Status::kSuccess) {
      std::cerr << "Failed to run CUTLASS Batched GEMM kernel." << std::endl;
      return result;
    }

    // Wait for completion
    result.error = hipDeviceSynchronize();

    if (result.error != hipSuccess)  {
      std::cerr << "Kernel execution error: " << hipGetErrorString(result.error);
      return result;
    }

    //
    // Verify correctness
    //
    result.passed = true;

    if (options.reference_check) {
      result.passed = verify_BMM_();
    }

    //
    // Warm-up run of the batched GEMM object
    //
    result.status = gemm.run();

    if (result.status != cutlass::Status::kSuccess) {
      std::cerr << "Failed to run CUTLASS Batched GEMM kernel." << std::endl;
      return result;
    }

    //
    // Construct events
    //

    hipEvent_t events[2];

    for (auto & event : events) {
      result.error = hipEventCreate(&event);
      if (result.error != hipSuccess) {
        std::cerr << "hipEventCreate() failed: " << hipGetErrorString(result.error) << std::endl;
        return -1;
      }
    }

    // Record an event at the start of a series of GEMM operations
    result.error = hipEventRecord(events[0]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    //
    // Run profiling loop
    //

    for (int iter = 0; iter < options.iterations; ++iter) {
      gemm();
    }

    //
    // Stop profiling loop
    //

    // Record an event when the GEMM operations have been launched.
    result.error = hipEventRecord(events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Wait for work on the device to complete.
    result.error = hipEventSynchronize(events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventSynchronize() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Measure elapsed runtime
    float runtime_ms = 0;
    result.error = hipEventElapsedTime(&runtime_ms, events[0], events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "cudaEventElapsed() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Compute average runtime and GFLOPs.
    result.runtime_ms = double(runtime_ms) / double(options.iterations);
    result.gflops = options.gflops(result.runtime_ms / 1000.0);

    //
    // Cleanup
    //

    for (auto event : events) {
      (void)hipEventDestroy(event);
    }

    std::cout << "    " << 1 << " batched GEMMs launched\n";

    std::cout << std::endl;
    std::cout << "    " << "Batched Runtime: " << result.runtime_ms << " ms\n";
    std::cout << "    " << "Batched  GFLOPs: " << result.gflops << "\n";

    return result;
  }

  Result profile_GEMM_permute() {

    std::cout << "\n====================================================" << std::endl;
    std::cout << "Normal GEMM (CUTLASS):\n"
      << "====================================================" << std::endl;

    if (options.verbose) {
      print_GEMM_info_();
    }

    Result result;

    result.passed = false;

    // Initialize the problem
    initialize_(1);

    // Configure the GEMM arguments
    typename EpilogueOutputOp::Params epilogue_op(options.alpha, options.beta);

    // Please make sure all problem_sizes are the same for kBatched mode
    auto problem = options.problem_each;

    // Configure GEMM arguments
    typename GemmPermute::Arguments arguments{
      cutlass::gemm::GemmUniversalMode::kGemm,
      options.problem_each,
      1,
      epilogue_op,
      (void*)block_A.get(),
      (void*)block_B.get(),
      (void*)block_C.get(),
      (void*)block_D.get(),
      0,
      0,
      0,
      0,
      problem.k(),
      problem.n(),
      problem.n(),
      problem.n()
    };

    // Initialize the GEMM object
    GemmPermute gemm_normal;

    result.status = gemm_normal.initialize(arguments, nullptr);

    if (result.status != cutlass::Status::kSuccess) {
      std::cerr << "Failed to initialize CUTLASS Batched GEMM kernel." << std::endl;
      return result;
    }

    // Run the normal GEMM object
    result.status = gemm_normal.run();

    if (result.status != cutlass::Status::kSuccess) {
      std::cerr << "Failed to run CUTLASS Batched GEMM kernel." << std::endl;
      return result;
    }

    // Wait for completion
    result.error = hipDeviceSynchronize();

    if (result.error != hipSuccess)  {
      std::cerr << "Kernel execution error: " << hipGetErrorString(result.error);
      return result;
    }

    //
    // Verify correctness
    //
    result.passed = true;

    if (options.reference_check) {
      result.passed = verify_GEMM_normal_();
    }

    //
    // Warm-up run of the normal GEMM object
    //
    result.status = gemm_normal.run();

    if (result.status != cutlass::Status::kSuccess) {
      std::cerr << "Failed to run CUTLASS Batched GEMM kernel." << std::endl;
      return result;
    }

    //
    // Construct events
    //

    hipEvent_t events[2];

    for (auto & event : events) {
      result.error = hipEventCreate(&event);
      if (result.error != hipSuccess) {
        std::cerr << "hipEventCreate() failed: " << hipGetErrorString(result.error) << std::endl;
        return -1;
      }
    }

    // Record an event at the start of a series of GEMM operations
    result.error = hipEventRecord(events[0]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    //
    // Run profiling loop
    //

    for (int iter = 0; iter < options.iterations; ++iter) {
      gemm_normal();
    }

    //
    // Stop profiling loop
    //

    // Record an event when the GEMM operations have been launched.
    result.error = hipEventRecord(events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Wait for work on the device to complete.
    result.error = hipEventSynchronize(events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventSynchronize() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Measure elapsed runtime
    float runtime_ms = 0;
    result.error = hipEventElapsedTime(&runtime_ms, events[0], events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "cudaEventElapsed() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Compute average runtime and GFLOPs.
    result.runtime_ms = double(runtime_ms) / double(options.iterations);
    result.gflops = options.gflops(result.runtime_ms / 1000.0);

    //
    // Cleanup
    //

    for (auto event : events) {
      (void)hipEventDestroy(event);
    }

    std::cout << std::endl;
    std::cout << "    " << "Normal Runtime: " << result.runtime_ms << " ms" << std::endl;
    std::cout << "    " << "Normal  GFLOPs: " << result.gflops << "\n";

    return result;
  }
};

///////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {

  //
  // This example uses mma.sync to directly access Tensor Cores to achieve peak performance.
  //

  hipDeviceProp_t props;

  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (error != hipSuccess) {
    std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;
    return -1;
  }

  if (__CUDACC_VER_MAJOR__ < 11 || props.major < 8) {
  
    //
    // This example requires an NVIDIA Ampere-architecture GPU.
    //

    std::cout 
      << "CUTLASS's Grouped GEMM example requires a GPU of NVIDIA's Ampere Architecture or "
      << "later (compute capability 80 or greater).\n";

    return 0;
  }

  //
  // Parse options
  //

  Options options;
  
  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  if (options.error) {
    std::cerr << "Aborting execution." << std::endl;
    return -1;
  }

  //
  // Define the GEMM types
  //

  using ElementOutput = cutlass::half_t;
  using ElementAccumulator = float;

  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::RowMajor;
  using LayoutC = cutlass::layout::RowMajor;

  //
  // Define a conventional batched GEMM type
  //

  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_128x128_32x4_nt_align8
  using GemmBatched = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, LayoutA,
    cutlass::half_t, LayoutB,
    ElementOutput,   LayoutC,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 128, 32>,
    cutlass::gemm::GemmShape<64, 64, 32>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      ElementOutput, 
      AlignmentC, //128 / cutlass::sizeof_bits<ElementOutput>::value,
      ElementAccumulator, 
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    4,
    8,     /*alignmentA*/
    8,     /*alignmengB*/
    cutlass::arch::OpMultiplyAdd,
    cutlass::ComplexTransform::kNone,
    cutlass::ComplexTransform::kNone,
    false,  /*GatherA*/
    false,  /*GatherB*/
    false,  /*ScatterD*/
    cutlass::layout::Tensor4DPermuteBMM0213<D1>   /*PermuteDLayout*/
  >;

  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_128x128_32x4_nt_align8
  using GemmPermute = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, LayoutA,
    cutlass::half_t, LayoutB,
    ElementOutput,   LayoutC,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 128, 32>,
    cutlass::gemm::GemmShape<64, 64, 32>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      ElementOutput, 
      AlignmentC, //128 / cutlass::sizeof_bits<ElementOutput>::value,
      ElementAccumulator, 
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    4,
    8,     /*alignmentA*/
    8,     /*alignmengB*/
    cutlass::arch::OpMultiplyAdd,
    cutlass::ComplexTransform::kNone,
    cutlass::ComplexTransform::kNone,
    false,  /*GatherA*/
    false,  /*GatherB*/
    false,  /*ScatterD*/
    cutlass::layout::Tensor5DPermute20314<T1, T2, T3>   /*PermuteDLayout*/
  >;

  //
  // Profile it
  //

  Testbed<GemmBatched, GemmPermute> testbed(options);

  Result result;
  result = testbed.profile_batched_kBatched();
  if (!result.passed) {
    std::cout << "Profiling batched GEMM has failed.\n";
    std::cout << "\nFailed\n";
  } else {
    std::cout << "\nPassed CUTLASS batched GEMM\n";
  }

  result = testbed.profile_GEMM_permute();
  if (!result.passed) {
    std::cout << "Profiling normal GEMM has failed.\n";
    std::cout << "\nFailed\n";
  } else {
    std::cout << "\nPassed CUTLASS normal GEMM\n";
  }

  std::cout << "\n====================================================" << std::endl;
  std::cout << "Finished\n";
  std::cout << "====================================================" << std::endl;

  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
