/***************************************************************************************************
 * Copyright (c) 2023 - 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*! \file
    \brief Hopper GEMM example leveraging collective operation builders.

    This example showcases the use of CUTLASS's CollectiveBuilder to easily construct performant kernels
    targetting the NVIDIA Hopper architecture.

    Background and motivation
    -------------------------
    CUTLASS kernels are highly parameterizable via template parameters. To ease the selection of template
    parameters, CUTLASS 2 leveraged DefaultGemmConfigurations. Given a small set of parameters, such as
    the data types of operands and the compute capability of the GPU, DefaultGemmConfigurations defined sensible
    defaults for the many other parameters to the kernel (e.g., warp shape, stage count).

    However, DefaultGemmConfigurations leave multiple opportunities for improvement, which are addressed
    in CUTLASS 3:
      (1) DefaultGemmConfigurations do not allow one to use a more-performant set of parameters without
          specifying every parameter. For example, the DefaultGemmConfigurations for GEMMs targetting
          Ampere specify that three pipeline stages should be used regardless of the sizes of operands.
          If one wished to increase this value, one would also need to specify all other template parameters.
          This leaves a gap between a high-level ease-of-use interface and a lower-level detailed interface.
      (2) A new DefaultGemmConfiguration was required for each combination of operand types, GPU architecture,
          and operation type (e.g., Tensor Core or SIMT). This led to increased code size to cover each unique
          configuration and a lack of extensibility from one DefaultGemmConfiguration to another.

    Alongside these opportunities for improvement, the Hopper architecture offers new features that increase
    the number of valid configurations of a kernel. In addition to the many template parameters already available
    in CUTLASS 2 kernels, CUTLASS 3 kernels targetting Hopper also have various scheduling modes to select from that control:
      (1) how data is to be loaded (e.g., using the Hopper TMA feature or Ampere cp.async)
      (2) how work is to be divided among warps in a thread block (e.g., whether to use "warp specialization")
      (3) whether persistent thread blocks should be used
    This increased configuration space further motivates rethinking DefaultGemmConfigurations.

    Introduction to the CollectiveBuilder
    -------------------------------------
    CUTLASS 3 introduces the CollectiveBuilder to further ease the process of selecting template parameters
    for kernels targetting Hopper. Similar to the DefaultGemmConfigurations used in CUTLASS 2, the CollectiveBuilder
    takes in a small set of template parameters (e.g., the data types of operands A and B). It then automatically
    determines the data loading strategy to use depending on whether the Hopper TMA feature can be used with the provided
    parameters. If one does not indicate a particular scheduling policy or stage count to use (by using `Auto` template
    parameters), the CollectiveBuilder will also automatically select these.

    Unlike DefaultGemmConfigurations a parital specialization of the CollectiveBuilder is not needed for many
    configurations of operand types. Instead the CollectiveBuilder "builds" a configuration based on generic
    properties of the specified operands, layouts, and other parameters. For example, when the stage count
    is set to `Auto`, the CollectiveBuilder may automatically calculate the maximum number of stages that
    will fit in shared memory given the types of operands and the thread block shape, rather than simply using
    a single default value.

    Note that one does not need to use the CollectiveBuilder to declare CUTLASS 3 kernels; one can still provide
    every template parameter to the gemm::collective::CollectiveMma. Specifying every template parameter in this
    manner remains the primary API for using CUTLASS 3 kernels. The CollectiveBuilder is simply meant to be
    a convenience interface.

    Note also that, while the selections made by CollectiveBuilder attempt to maximize performance, this is not
    a guarantee. Furthermore, the behavior of the CollectiveBuilder when `Auto` parameters are provided is subject
    to change in future CUTLASS releases -- do not rely on `Auto` if you require a specific scheduling policy and/or
    stage count to be used.

    Details of this example
    -----------------------
    This example walks through the use of the CollectiveBuilder with various schedules and stage counts specified.
    This example also illustrates how CUTLASS 3 GEMMs targetting Hopper automatically support batched GEMMs by simply
    extending the problem size with an additional tensor rank.

    Example usage:
      $ ./examples/49_hopper_gemm_schedules_with_collective_builder/49_hopper_gemm_schedules_with_collective_builder \
            --m=2048 --n=2048 --k=2048 --l=2
*/

#include <iostream>

#include "cute/tensor.hpp"

#include "cutlass/cutlass.h"
#include "cutlass/tensor_ref.h"
#include "cutlass/epilogue/collective/default_epilogue.hpp"
#include "cutlass/epilogue/thread/linear_combination.h"
#include "cutlass/gemm/dispatch_policy.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/packed_stride.hpp"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/device/gemm_complex.h"
#include "cutlass/util/reference/device/tensor_compare.h"
#include "cutlass/util/reference/device/tensor_fill.h"

using namespace cute;

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Command line options parsing
struct Options {

  bool help;
  bool error;

  int m, n, k, l;
  float alpha, beta;

  Options():
    help(false),
    error(false),
    m(2048), n(2048), k(2048), l(1),
    alpha(1.f), beta(0.f)
  { }

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
      return;
    }

    cmd.get_cmd_line_argument("m", m, 2048);
    cmd.get_cmd_line_argument("n", n, 2048);
    cmd.get_cmd_line_argument("k", k, 2048);
    cmd.get_cmd_line_argument("l", l, 1);
    cmd.get_cmd_line_argument("alpha", alpha, 1.f);
    cmd.get_cmd_line_argument("beta", beta, 0.f);
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "49_hopper_gemm_schedules_with_collective_builder\n\n"
      << "  This example showcases the use of CUTLASS's collective operation builders to easily construct\n"
      << "  performant kernels targetting NVIDIA's Hopper architecture.\n\n"
      << "Options:\n\n"
      << "  --help                      If specified, displays this usage statement\n\n"
      << "  --m=<int>                   Sets the M extent of the GEMM\n"
      << "  --n=<int>                   Sets the N extent of the GEMM\n"
      << "  --k=<int>                   Sets the K extent of the GEMM\n"
      << "  --l=<int>                   Sets the L extent (batch count) of the GEMM\n"
      << "  --alpha=<f32>               Epilogue scalar alpha\n"
      << "  --beta=<f32>                Epilogue scalar beta\n\n";

    return out;
  }
};

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Helper to initialize a block of device data
template <class Element>
bool initialize_block(
  cutlass::DeviceAllocation<Element>& block,
  uint64_t seed=2023) {

  Element scope_max, scope_min;
  int bits_input = cutlass::sizeof_bits<Element>::value;

  if (bits_input == 1) {
    scope_max = 2;
    scope_min = 0;
  } else if (bits_input <= 8) {
    scope_max = 2;
    scope_min = -2;
  } else {
    scope_max = 8;
    scope_min = -8;
  }

  cutlass::reference::device::BlockFillRandomUniform(
    block.get(), block.size(), seed, scope_max, scope_min, 0);

  return true;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

// Wrapper to construct, run, and verify a GEMM. This example showcases CUTLASS's collective
// operation builders by specializing the GEMM only on the kernel schedule it will use and the
// number of pipeline stages.
//
// For either option, one can use a special `Auto` type that tells the CollectiveBuilder
// to select an appropriate value on its own. The CollectiveBuilder will attempt to select
// values that will result in the most-performant kernel, but this is not a guarantee. Furthermore,
// the behavior of the CollectiveBuilder with `Auto` types is subject to change in future releases
// -- do not rely on `Auto` if you require a specific scheduling policy.
template <
  // Type of kernel schedule to generate
  class KernelScheduleType = cutlass::gemm::collective::KernelScheduleAuto,
  // Number of pipeline stages to use
  class StageCountType = cutlass::gemm::collective::StageCountAuto
>
struct ExampleRunner {

  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::ColumnMajor;
  using LayoutD = cutlass::layout::ColumnMajor;

  static constexpr int kAlignmentA = 8;
  static constexpr int kAlignmentB = 8;
  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      cutlass::half_t, LayoutA, kAlignmentA,
      cutlass::half_t, LayoutB, kAlignmentB,
      float,
      Shape<_128,_128,_64>, Shape<_2,_1,_1>,
      StageCountType,
      KernelScheduleType
    >::CollectiveOp;

  using CollectiveEpilogue = cutlass::epilogue::collective::DefaultEpilogue<
      cutlass::gemm::TagToStrideC_t<LayoutC>,
      cutlass::gemm::TagToStrideC_t<LayoutD>,
      cutlass::epilogue::thread::LinearCombination<cutlass::half_t, 1, float, float>>;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

  using ProblemShapeType = typename Gemm::GemmKernel::ProblemShape;

  using StrideA = typename Gemm::GemmKernel::StrideA;
  using StrideB = typename Gemm::GemmKernel::StrideB;
  using StrideC = typename Gemm::GemmKernel::StrideC;
  using StrideD = typename Gemm::GemmKernel::StrideD;

  using LayoutTagA = decltype(cutlass::gemm::detail::stride_to_layout_tag_A<StrideA>());
  using LayoutTagB = decltype(cutlass::gemm::detail::stride_to_layout_tag_B<StrideB>());
  using LayoutTagC = decltype(cutlass::gemm::detail::stride_to_layout_tag_A<StrideC>());
  using LayoutTagD = decltype(cutlass::gemm::detail::stride_to_layout_tag_A<StrideD>());

  //
  // Data members
  //

  /// Initialization
  StrideA stride_A;
  StrideB stride_B;
  StrideC stride_C;
  StrideD stride_D;
  uint64_t seed = 0;

  cutlass::DeviceAllocation<typename Gemm::ElementA> block_A;
  cutlass::DeviceAllocation<typename Gemm::ElementB> block_B;
  cutlass::DeviceAllocation<typename Gemm::ElementC> block_C;
  cutlass::DeviceAllocation<typename Gemm::EpilogueOutputOp::ElementOutput> block_D;
  cutlass::DeviceAllocation<typename Gemm::EpilogueOutputOp::ElementOutput> block_ref_D;

  //
  // Methods
  //

  bool verify(const ProblemShapeType& problem_size, float alpha, float beta) {
    auto [M, N, K, L] = problem_size;

    cutlass::TensorRef ref_A(block_A.get(), Gemm::LayoutA::packed({M, K}));
    cutlass::TensorRef ref_B(block_B.get(), Gemm::LayoutB::packed({K, N}));
    cutlass::TensorRef ref_C(block_C.get(), Gemm::LayoutC::packed({M, N}));
    cutlass::TensorRef ref_D(block_ref_D.get(), Gemm::LayoutD::packed({M, N}));

    cutlass::reference::device::GemmComplex(
          {M, N, K},
          typename Gemm::EpilogueOutputOp::ElementCompute(alpha),
          ref_A,
          cutlass::ComplexTransform::kNone,
          ref_B,
          cutlass::ComplexTransform::kNone,
          typename Gemm::EpilogueOutputOp::ElementCompute(beta),
          ref_C,
          ref_D,
          typename Gemm::EpilogueOutputOp::ElementAccumulator(0.f),
          L,     // batch_count
          M * K, // batch_stride_A
          K * N, // batch_stride_B
          M * N, // batch_stride_C
          M * N  // batch_stride_D
        );

    hipError_t result = hipDeviceSynchronize();
    if (result != hipSuccess) {
      std::cerr << "Reference kernel failed. Last CUDA error: "
                << hipGetErrorString(result) << std::endl;
      return false;
    }

    // Check if output from CUTLASS kernel and reference kernel are equal or not
    bool passed = cutlass::reference::device::BlockCompareEqual(block_ref_D.get(), block_D.get(), block_D.size());

    return passed;
  }

  /// Initialize operands to be used in the GEMM and reference GEMM
  void initialize(const ProblemShapeType& problem_size) {
    auto problem_shape_MNKL = cute::append<4>(problem_size, 1);
    auto [M, N, K, L] = problem_shape_MNKL;

    stride_A = make_cute_packed_stride(StrideA{}, cute::make_shape(M, K, L));
    stride_B = make_cute_packed_stride(StrideB{}, cute::make_shape(N, K, L));
    stride_C = make_cute_packed_stride(StrideC{}, cute::make_shape(M, N, L));
    stride_D = make_cute_packed_stride(StrideD{}, cute::make_shape(M, N, L));

    block_A.reset(M * K * L);
    block_B.reset(K * N * L);
    block_C.reset(M * N * L);
    block_D.reset(M * N * L);
    block_ref_D.reset(M * N * L);

    initialize_block(block_A, seed + 2023);
    initialize_block(block_B, seed + 2022);
    initialize_block(block_C, seed + 2021);
  }

  bool run(const Options& options, const cutlass::KernelHardwareInfo& hw_info) {
    ProblemShapeType problem_size = ProblemShapeType{options.m, options.n, options.k, options.l};

    initialize(problem_size);

    typename Gemm::Arguments arguments{
      cutlass::gemm::GemmUniversalMode::kGemm,
      problem_size,
      block_A.get(),
      stride_A,
      block_B.get(),
      stride_B,
      {block_C.get(), stride_C, block_D.get(), stride_D, {options.alpha, options.beta}},
      hw_info
    };

    Gemm gemm_op;

    size_t workspace_size = Gemm::get_workspace_size(arguments);
    cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

    cutlass::Status status = gemm_op.can_implement(arguments);
    if (status != cutlass::Status::kSuccess) {
      std::cerr << "This kernel is not supported. Last CUDA error is: "
                << hipGetErrorString(hipGetLastError()) << std::endl;
      return false;
    }

    status = gemm_op.initialize(arguments, workspace.get());
    if (status != cutlass::Status::kSuccess) {
      std::cerr << "Failed to initialize the CUTLASS kernel. Last CUDA error is: "
                << hipGetErrorString(hipGetLastError()) << std::endl;
      return false;
    }

    // Run the GEMM
    status = gemm_op.run();
    if (status != cutlass::Status::kSuccess) {
      std::cerr << "Failed to launch the CUTLASS kernel. Last CUDA error is: "
                << hipGetErrorString(hipGetLastError()) << std::endl;
      return false;
    }

    hipError_t result = hipDeviceSynchronize();
    if (result != hipSuccess) {
      std::cerr << "Error running the CUTLASS kernel. Last CUDA error is: "
                << hipGetErrorString(result) << std::endl;
      return false;
    }

    // Verify that the result is correct
    bool passed = verify(problem_size, options.alpha, options.beta);
    if (!passed) {
      std::cerr << "Reference check failed" << std::endl;
    }

    return passed;
  }

};

#endif // defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Helper to print a description of the example run and its result
void print_result(const std::string& description, bool passed) {
  std::cout << description << ": " << (passed ? "Passed" : "Failed") << std::endl;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {

  hipDeviceProp_t props;

  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (error != hipSuccess) {
    std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;
    return -1;
  }

  if (__CUDACC_VER_MAJOR__ < 12 || props.major < 9) {
    std::cout
      << "This example requires a GPU of NVIDIA's Hopper Architecture or "
      << "later (compute capability 90 or greater) and CUDA 12.0 or greater.\n";
    return 0;
  }

  //
  // Parse options
  //

  Options options;

  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  if (options.error) {
    std::cerr << "Aborting execution." << std::endl;
    return -1;
  }

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

  //
  // Run examples
  //

  // The KernelHardwareInfo struct holds the number of SMs on the GPU with a given device ID. This
  // information is used by the underlying kernel.
  cutlass::KernelHardwareInfo hw_info;

  // Change device_id to another value if you are running on a machine with multiple GPUs and wish
  // to use a GPU other than that with device ID 0.
  hw_info.device_id = 0;
  hw_info.sm_count = cutlass::KernelHardwareInfo::query_device_multiprocessor_count(hw_info.device_id);

  bool passed;

  // This first example constructs a GEMM using the default schedule and stage count provided by
  // the CollectiveBuilder. The scheduling policy that is expected to be most performant will be
  // selected and the maximum number of stages that can fit in shared memory will be selected.
  //
  // This example is equivalent to declaring
  //    ExampleRunner<cutlass::gemm::collective::KernelScheduleAuto, cutlass::gemm::collective::StageCountAuto>
  // Each of the `Auto` types indicate that the CollectiveBuilder should determine the scheduling policy and
  // stage count. Note that the behavior of the CollectiveBuilder with `Auto` parameters is subject to change
  // -- do not rely on `Auto` if you require a specific scheduling policy.
  ExampleRunner<> auto_schedule_auto_stage_runner;
  passed = auto_schedule_auto_stage_runner.run(options, hw_info);
  print_result("Automatically-selected schedule and stage count", passed);

  // One can override the stage count used in the GEMM by replacing cutlass::gemm::collective::StageCountAuto
  // with the number of stages to use (5 in this case).
  ExampleRunner<cutlass::gemm::collective::KernelScheduleAuto, _5> auto_schedule_5_stage_runner;
  passed = auto_schedule_5_stage_runner.run(options, hw_info);
  print_result("Automatically-selected schedule with 5 stages", passed);

  // One can also override the scheduling policy to use. In this case, use the KernelTma scheduling
  // policy, which specifies that the Hopper TMA feature should be used.
  ExampleRunner<cutlass::gemm::KernelTma> tma_schedule_auto_stage_runner;
  passed = tma_schedule_auto_stage_runner.run(options, hw_info);
  print_result("TMA schedule with automatically-selected stage count", passed);

  // Here, we override the scheduling policy to use Hopper's TMA feature alongside the warp-specialized
  // scheduling policy.
  //
  // Note that, as of the CUTLASS 3.0 release, this is the default scheduling policy
  // used by the CollectiveBuilder, so this declaration is equivalent to ExampleRunner<> and
  // ExampleRunner<cutlass::gemm::collective::KernelScheduleAuto>. However, this default is subject to
  // change in future releases -- do not rely on `Auto` if you require a specific scheduling policy.
  ExampleRunner<cutlass::gemm::KernelTmaWarpSpecialized> ws_schedule_auto_stage_runner;
  passed = ws_schedule_auto_stage_runner.run(options, hw_info);
  print_result("Warp-specialized TMA schedule with automatically-selected stage count", passed);

  // Finally, we override the scheduling policy to use Hopper's TMA feature, alongside the warp-specialized
  // scheduling policy, leveraging persistent thread blocks.
  ExampleRunner<cutlass::gemm::KernelTmaWarpSpecializedPersistent> ws_persistent_schedule_auto_stage_runner;
  passed = ws_persistent_schedule_auto_stage_runner.run(options, hw_info);
  print_result("Persistent warp-specialized TMA schedule with automatically-selected stage count", passed);

#endif

  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
