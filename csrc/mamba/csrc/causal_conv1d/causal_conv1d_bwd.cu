#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2024, Tri Dao.
 ******************************************************************************/

#include <paddle/phi/common/data_type.h>
#include <paddle/extension.h>


#ifndef USE_ROCM
    #include <cub/block/block_load.cuh>
    #include <cub/block/block_store.cuh>
    #include <cub/block/block_reduce.cuh>
#else
    #include <hipcub/hipcub.hpp>
    namespace cub = hipcub;
#endif

#include "causal_conv1d.h"
#include "causal_conv1d_common.h"
#include "static_switch.h"

template<int kNThreads_, int kWidth_, bool kSiluAct_, bool kIsVecLoad_, typename input_t_, typename weight_t_>
struct Causal_conv1d_bwd_kernel_traits {
    using input_t = input_t_;
    using weight_t = weight_t_;
    static constexpr int kNThreads = kNThreads_;
    static constexpr int kWidth = kWidth_;
    static constexpr bool kSiluAct = kSiluAct_;
    static constexpr int kNBytes = sizeof(input_t);
    static_assert(kNBytes == 2 || kNBytes == 4);
    static constexpr int kNElts = kNBytes == 4 ? 4 : 8;
    static_assert(kWidth <= kNElts);
    // It's possible that we need to do 2 rounds of exchange if input_t is 16 bits
    // (since then we'd have 8 values of float, and each round we can exchange 4 floats).
    static constexpr int kNExchangeRounds = sizeof(float) / sizeof(input_t);
    static constexpr bool kIsVecLoad = kIsVecLoad_;
    using vec_t = typename BytesToType<kNBytes * kNElts>::Type;
    using BlockLoadT = hipcub::BlockLoad<input_t, kNThreads, kNElts, hipcub::BLOCK_LOAD_WARP_TRANSPOSE>;
    using BlockLoadVecT = hipcub::BlockLoad<vec_t, kNThreads, 1, hipcub::BLOCK_LOAD_DIRECT>;
    using BlockStoreT = hipcub::BlockStore<input_t, kNThreads, kNElts, hipcub::BLOCK_STORE_WARP_TRANSPOSE>;
    using BlockStoreVecT = hipcub::BlockStore<vec_t, kNThreads, 1, hipcub::BLOCK_STORE_DIRECT>;
    using BlockReduceFloatT = hipcub::BlockReduce<float, kNThreads>;
    static constexpr int kSmemIOSize = kIsVecLoad
        ? 0
        : custom_max({sizeof(typename BlockLoadT::TempStorage), sizeof(typename BlockStoreT::TempStorage)});
    static constexpr int kSmemExchangeSize = kNThreads * kNBytes * kNElts * (!kSiluAct ? 1 : kNExchangeRounds + 1);
    static constexpr int kSmemSize = custom_max({kSmemExchangeSize,
            int(sizeof(typename BlockReduceFloatT::TempStorage))}) + (kIsVecLoad ? 0 : kSmemIOSize);
};

template<typename Ktraits>
__global__ __launch_bounds__(Ktraits::kNThreads)
void causal_conv1d_bwd_kernel(ConvParamsBwd params) {
    constexpr int kWidth = Ktraits::kWidth;
    constexpr int kNThreads = Ktraits::kNThreads;
    constexpr bool kSiluAct = Ktraits::kSiluAct;
    static constexpr int kNElts = Ktraits::kNElts;
    constexpr int kNExchangeRounds = Ktraits::kNExchangeRounds;
    static constexpr bool kIsVecLoad = Ktraits::kIsVecLoad;
    using input_t = typename Ktraits::input_t;
    using vec_t = typename Ktraits::vec_t;
    using weight_t = typename Ktraits::weight_t;

    // Shared memory.
    extern __shared__ char smem_[];
    auto& smem_load = reinterpret_cast<typename Ktraits::BlockLoadT::TempStorage&>(smem_);
    auto& smem_load_vec = reinterpret_cast<typename Ktraits::BlockLoadVecT::TempStorage&>(smem_);
    auto& smem_store = reinterpret_cast<typename Ktraits::BlockStoreT::TempStorage&>(smem_);
    auto& smem_store_vec = reinterpret_cast<typename Ktraits::BlockStoreVecT::TempStorage&>(smem_);
    vec_t *smem_exchange = reinterpret_cast<vec_t *>(smem_ + Ktraits::kSmemIOSize);
    vec_t *smem_exchange_x = reinterpret_cast<vec_t *>(smem_ + Ktraits::kSmemIOSize) + kNThreads * kNExchangeRounds;
    auto& smem_reduce_float = *reinterpret_cast<typename Ktraits::BlockReduceFloatT::TempStorage*>(smem_ + Ktraits::kSmemIOSize);

    const int tidx = threadIdx.x;
    const int batch_id = blockIdx.x;
    const int dim_id = blockIdx.y;
    input_t *x = reinterpret_cast<input_t *>(params.x_ptr) + batch_id * params.x_batch_stride
        + dim_id * params.x_c_stride;
    weight_t *weight = reinterpret_cast<weight_t *>(params.weight_ptr) + dim_id * params.weight_c_stride;
    input_t *dout = reinterpret_cast<input_t *>(params.dout_ptr) + batch_id * params.dout_batch_stride
        + dim_id * params.dout_c_stride;
    input_t *dx = reinterpret_cast<input_t *>(params.dx_ptr) + batch_id * params.dx_batch_stride
        + dim_id * params.dx_c_stride;
    float *dweight = reinterpret_cast<float *>(params.dweight_ptr) + dim_id * params.dweight_c_stride;
    float bias_val = params.bias_ptr == nullptr ? 0.f : float(reinterpret_cast<weight_t *>(params.bias_ptr)[dim_id]);

    // Thread kNThreads - 1 will load the first elements of the next chunk so we initialize those to 0.
    if (tidx == 0) {
        if constexpr (!kSiluAct) {
            input_t zeros[kNElts] = {input_t(0)};
            smem_exchange[0] = reinterpret_cast<vec_t *>(zeros)[0];
        } else {
            float zeros[kNElts] = {input_t(0)};
            #pragma unroll
            for (int r = 0; r < kNExchangeRounds; ++r) {
                smem_exchange[r * kNThreads] = reinterpret_cast<vec_t *>(zeros)[r];
            }
        }
    }

    float weight_vals[kWidth];
    #pragma unroll
    for (int i = 0; i < kWidth; ++i) { weight_vals[i] = weight[i * params.weight_width_stride]; }

    float dweight_vals[kWidth] = {input_t(0)};
    float dbias_val = 0;

    constexpr int kChunkSize = kNThreads * kNElts;
    const int n_chunks = (params.seqlen + kChunkSize - 1) / kChunkSize;
    x += (n_chunks - 1) * kChunkSize;
    dout += (n_chunks - 1) * kChunkSize;
    dx += (n_chunks - 1) * kChunkSize;
    for (int chunk = n_chunks - 1; chunk >= 0; --chunk) {
        input_t x_vals_load[2 * kNElts] = {input_t(0)};
        input_t dout_vals_load[2 * kNElts] = {input_t(0)};
        if constexpr(kIsVecLoad) {
            typename Ktraits::BlockLoadVecT(smem_load_vec).Load(reinterpret_cast<vec_t*>(x), *reinterpret_cast<vec_t (*)[1]>(&x_vals_load[kNElts]), (params.seqlen - chunk * kChunkSize) / kNElts);
            typename Ktraits::BlockLoadVecT(smem_load_vec).Load(reinterpret_cast<vec_t*>(dout), *reinterpret_cast<vec_t (*)[1]>(&dout_vals_load[0]), (params.seqlen - chunk * kChunkSize) / kNElts);
        } else {
            __syncthreads();
            typename Ktraits::BlockLoadT(smem_load).Load(x, *reinterpret_cast<input_t (*)[kNElts]>(&x_vals_load[kNElts]), params.seqlen - chunk * kChunkSize);
            __syncthreads();
            typename Ktraits::BlockLoadT(smem_load).Load(dout, *reinterpret_cast<input_t (*)[kNElts]>(&dout_vals_load[0]), params.seqlen - chunk * kChunkSize);
        }
        float dout_vals[2 * kNElts], x_vals[2 * kNElts];
        if constexpr (!kSiluAct) {
            __syncthreads();
            // Thread 0 don't write yet, so that thread kNThreads - 1 can read
            // the first elements of the next chunk.
            if (tidx > 0) { smem_exchange[tidx] = reinterpret_cast<vec_t *>(dout_vals_load)[0]; }
            __syncthreads();
            reinterpret_cast<vec_t *>(dout_vals_load)[1] = smem_exchange[tidx < kNThreads - 1 ? tidx + 1 : 0];
            __syncthreads();
            // Now thread 0 can write the first elements of the current chunk.
            if (tidx == 0) { smem_exchange[tidx] = reinterpret_cast<vec_t *>(dout_vals_load)[0]; }
            #pragma unroll
            for (int i = 0; i < 2 * kNElts; ++i) {
                dout_vals[i] = float(dout_vals_load[i]);
                x_vals[i] = float(x_vals_load[i]);
            }
        } else {
            if (tidx == 0 && chunk > 0) {
                if constexpr(kIsVecLoad) {
                    reinterpret_cast<vec_t *>(x_vals_load)[0] = reinterpret_cast<vec_t *>(x)[-1];
                } else {
                    #pragma unroll
                    for (int i = 0; i < kNElts; ++i) {
                        if (chunk * kChunkSize + i < params.seqlen) { x_vals_load[i] = x[-kNElts + i]; }
                    }
                }
            }
            __syncthreads();
            smem_exchange_x[tidx] = reinterpret_cast<vec_t *>(x_vals_load)[1];
            __syncthreads();
            if (tidx > 0) { reinterpret_cast<vec_t *>(x_vals_load)[0] = smem_exchange_x[tidx - 1]; }
            #pragma unroll
            for (int i = 0; i < 2 * kNElts; ++i) { x_vals[i] = float(x_vals_load[i]); }
            // Recompute the output
            #pragma unroll
            for (int i = 0; i < kNElts; ++i) {
                float out_val = bias_val;
                #pragma unroll
                for (int w = 0; w < kWidth; ++w) {
                    out_val += weight_vals[w] * x_vals[kNElts + i - (kWidth - w - 1)];
                }
                float out_sigmoid_val = 1.0f / (1.0f + expf(-out_val));
                dout_vals[i] = float(dout_vals_load[i]) * out_sigmoid_val
                               * (1.0f + out_val * (1.0f - out_sigmoid_val));
            }
            // Exchange the dout_vals. It's possible that we need to do 2 rounds of exchange
            // if input_t is 16 bits (since then we'd have 8 values of float)
            __syncthreads();
            // Thread 0 don't write yet, so that thread kNThreads - 1 can read
            // the first elements of the next chunk.
            if (tidx > 0) {
                #pragma unroll
                for (int r = 0; r < kNExchangeRounds; ++r) {
                    smem_exchange[r * kNThreads + tidx] = reinterpret_cast<vec_t *>(dout_vals)[r];
                }
            }
            __syncthreads();
            #pragma unroll
            for (int r = 0; r < kNExchangeRounds; ++r) {
                reinterpret_cast<vec_t *>(dout_vals)[kNExchangeRounds + r]
                    = smem_exchange[r * kNThreads + (tidx < kNThreads - 1 ? tidx + 1 : 0)];
            }
            __syncthreads();
            // Now thread 0 can write the first elements of the current chunk.
            if (tidx == 0) {
                #pragma unroll
                for (int r = 0; r < kNExchangeRounds; ++r) {
                    smem_exchange[r * kNThreads + tidx] = reinterpret_cast<vec_t *>(dout_vals)[r];
                }
            }
        }
        dout -= kChunkSize;
        x -= kChunkSize;

        #pragma unroll
        for (int i = 0; i < kNElts; ++i) { dbias_val += dout_vals[i]; }

        float dx_vals[kNElts] = {input_t(0)};
        #pragma unroll
        for (int i = 0; i < kNElts; ++i) {
            #pragma unroll
            for (int w = 0; w < kWidth; ++w) {
                dx_vals[i] += weight_vals[w] * dout_vals[i + kWidth - w - 1];
            }
        }

        input_t dx_vals_store[kNElts];
        #pragma unroll
        for (int i = 0; i < kNElts; ++i) { dx_vals_store[i] = dx_vals[i]; }
        if constexpr(kIsVecLoad) {
            typename Ktraits::BlockStoreVecT(smem_store_vec).Store(reinterpret_cast<vec_t*>(dx), reinterpret_cast<vec_t (&)[1]>(dx_vals_store), (params.seqlen - chunk * kChunkSize) / kNElts);
        } else {
            typename Ktraits::BlockStoreT(smem_store).Store(dx, dx_vals_store, params.seqlen - chunk * kChunkSize);
        }
        dx -= kChunkSize;

        #pragma unroll
        for (int w = 0; w < kWidth; ++w) {
            #pragma unroll
            for (int i = 0; i < kNElts; ++i) {
                dweight_vals[w] += x_vals[kNElts + i] * dout_vals[i + kWidth - w - 1];
            }
        }
    }

    #pragma unroll
    for (int w = 0; w < kWidth; ++w) {
        __syncthreads();
        dweight_vals[w] = typename Ktraits::BlockReduceFloatT(smem_reduce_float).Sum(dweight_vals[w]);
        if (tidx == 0) {
            atomicAdd(&reinterpret_cast<float *>(dweight)[w * params.dweight_width_stride], dweight_vals[w]);
        }
    }
    if (params.bias_ptr != nullptr) {
        __syncthreads();
        dbias_val = typename Ktraits::BlockReduceFloatT(smem_reduce_float).Sum(dbias_val);
        if (tidx == 0) {
            atomicAdd(&reinterpret_cast<float *>(params.dbias_ptr)[dim_id], dbias_val);
        }
    }
}

template<int kNThreads, int kWidth, typename input_t, typename weight_t>
void causal_conv1d_bwd_launch(ConvParamsBwd &params, hipStream_t stream) {
    static constexpr int kNElts = sizeof(input_t) == 4 ? 4 : 8;
    BOOL_SWITCH(params.seqlen % kNElts == 0, kIsVecLoad, [&] {
        BOOL_SWITCH(params.silu_activation, kSiluAct, [&] {
            using Ktraits = Causal_conv1d_bwd_kernel_traits<kNThreads, kWidth, kSiluAct, kIsVecLoad, input_t, weight_t>;
            constexpr int kSmemSize = Ktraits::kSmemSize;
            dim3 grid(params.batch, params.dim);
            auto kernel = &causal_conv1d_bwd_kernel<Ktraits>;

            if (kSmemSize >= 48 * 1024) {
                #ifndef USE_ROCM
                hipFuncSetAttribute(reinterpret_cast<const void*>(
                    kernel), hipFuncAttributeMaxDynamicSharedMemorySize, kSmemSize);
                #else
                // There is a slight signature discrepancy in HIP and CUDA "FuncSetAttribute" function.
                hipFuncSetAttribute(
                    (void *) kernel, hipFuncAttributeMaxDynamicSharedMemorySize, kSmemSize);
                std::cerr << "Warning (causal_conv1d bwd launch): attempting to set maxDynamicSharedMemorySize on an AMD GPU which is currently a non-op (in ROCm versions <= 6.1). This might lead to undefined behavior. \n" << std::endl;
                #endif
            }


            kernel<<<grid, Ktraits::kNThreads, kSmemSize, stream>>>(params);
        });
    });
}

template<typename input_t, typename weight_t>
void causal_conv1d_bwd_cuda(ConvParamsBwd &params, hipStream_t stream) {
    if (params.width == 2) {
        causal_conv1d_bwd_launch<128, 2, input_t, weight_t>(params, stream);
    } else if (params.width == 3) {
        causal_conv1d_bwd_launch<128, 3, input_t, weight_t>(params, stream);
    } else if (params.width == 4) {
        causal_conv1d_bwd_launch<128, 4, input_t, weight_t>(params, stream);
    }
}

template<int kNThreads_, int kWidth_, int kChunkSizeL_, bool kSiluAct_, bool kIsVecLoad_, typename input_t_, typename weight_t_>
struct Causal_conv1d_channellast_bwd_kernel_traits {
    // The cache line is 128 bytes, and we try to read 16 bytes per thread.
    // So we have 8 threads per "row", so 32 or 64 elements in the channel dimension.
    // That leaves 4 columns per warp, and so 16 columns per block (assuming each block has 128
    // threads). Each each load is 16 x 32|64 elements in the L x C dimensions.
    using input_t = input_t_;
    using weight_t = weight_t_;
    static constexpr bool kSiluAct = kSiluAct_;
    static constexpr int kNThreads = kNThreads_;
    static_assert(kNThreads % 32 == 0);
    static constexpr int kNWarps = kNThreads / 32;
    static constexpr int kWidth = kWidth_;
    static constexpr int kChunkSizeL = kChunkSizeL_;
    static constexpr int kNBytes = sizeof(input_t);
    static_assert(kNBytes == 2 || kNBytes == 4);
    static constexpr int kNElts = kNBytes == 4 ? 4 : 8;
    static constexpr int kNEltsPerRow = 128 / kNBytes;
    static constexpr int kNThreadsPerRow = kNEltsPerRow / kNElts;  // Always 8 for now
    static_assert(kNThreadsPerRow * kNBytes * kNElts == 128);
    static constexpr int kNColsPerWarp = 32 / kNThreadsPerRow;  // Always 4 for now
    static_assert(kNColsPerWarp * kNThreadsPerRow == 32);
    static constexpr int kNColsPerLoad = kNColsPerWarp * kNWarps;
    static constexpr int kNLoads = kChunkSizeL / kNColsPerLoad;
    static_assert(kNLoads * kNColsPerLoad == kChunkSizeL);
    static constexpr bool kIsVecLoad = kIsVecLoad_;
    using vec_t = typename BytesToType<kNBytes * kNElts>::Type;
    // using BlockLoadT = hipcub::BlockLoad<input_t, kNThreads, kNItems, hipcub::BLOCK_LOAD_WARP_TRANSPOSE>;
    // using BlockStoreT = hipcub::BlockStore<input_t, kNThreads, kNItems, hipcub::BLOCK_STORE_WARP_TRANSPOSE>;
    // static constexpr int kSmemSize = std::max({sizeof(typename BlockLoadT::TempStorage),
    //                                            sizeof(typename BlockStoreT::TempStorage)});
    // static constexpr int kSmemSize = kChunkSizeL * kNEltsPerRow * kNBytes;
};

template<typename Ktraits, bool kHasSeqIdx, bool kHasDfinalStates>
__global__ __launch_bounds__(Ktraits::kNThreads)
void causal_conv1d_channellast_bwd_kernel(ConvParamsBwd params) {
    constexpr int kWidth = Ktraits::kWidth;
    constexpr int kNThreads = Ktraits::kNThreads;
    constexpr bool kSiluAct = Ktraits::kSiluAct;
    constexpr int kNElts = Ktraits::kNElts;
    constexpr int kNWarp = Ktraits::kNWarps;
    constexpr int kNThreadsPerC = Ktraits::kNThreadsPerRow;
    constexpr int kLPerLoad = Ktraits::kNColsPerLoad;
    constexpr int kChunkSizeL = Ktraits::kChunkSizeL;
    constexpr int kChunkSizeC = Ktraits::kNEltsPerRow;
    using input_t = typename Ktraits::input_t;
    using vec_t = typename Ktraits::vec_t;
    using weight_t = typename Ktraits::weight_t;

    // Shared memory.
    __shared__ input_t dout_smem[kChunkSizeL + kWidth - 1][kChunkSizeC + kNElts];
    __shared__ input_t x_smem[kWidth - 1 + kChunkSizeL + kWidth - 1][kChunkSizeC + kNElts];

    const int batch_id = blockIdx.x;
    const int chunk_l_id = blockIdx.y;
    const int chunk_c_id = blockIdx.z;
    const int tid = threadIdx.x;
    const int l_idx = tid / kNThreadsPerC;
    const int c_idx = tid % kNThreadsPerC;
    input_t *x = reinterpret_cast<input_t *>(params.x_ptr) + batch_id * params.x_batch_stride
        + (chunk_l_id * kChunkSizeL + l_idx) * params.x_l_stride + chunk_c_id * kChunkSizeC + c_idx * kNElts;
    weight_t *weight = reinterpret_cast<weight_t *>(params.weight_ptr)
        + chunk_c_id * kChunkSizeC * params.weight_c_stride;
    input_t *dout = reinterpret_cast<input_t *>(params.dout_ptr) + batch_id * params.dout_batch_stride
        + (chunk_l_id * kChunkSizeL + l_idx) * params.dout_l_stride + chunk_c_id * kChunkSizeC + c_idx * kNElts;
    input_t *dx = reinterpret_cast<input_t *>(params.dx_ptr) + batch_id * params.dx_batch_stride
        + (chunk_l_id * kChunkSizeL + l_idx) * params.dx_l_stride + chunk_c_id * kChunkSizeC + c_idx * kNElts;
    float *dweight = reinterpret_cast<float *>(params.dweight_ptr)
        + chunk_c_id * kChunkSizeC * params.dweight_c_stride;
    int *seq_idx = !kHasSeqIdx ? nullptr : reinterpret_cast<int *>(params.seq_idx_ptr)
        + batch_id * params.seqlen + chunk_l_id * kChunkSizeL;
    input_t *initial_states = params.initial_states_ptr == nullptr || chunk_l_id > 0 ? nullptr
        : reinterpret_cast<input_t *>(params.initial_states_ptr) + batch_id * params.initial_states_batch_stride + l_idx * params.initial_states_l_stride + chunk_c_id * kChunkSizeC + c_idx * kNElts;
    input_t *dinitial_states = params.dinitial_states_ptr == nullptr || chunk_l_id > 0 ? nullptr
        : reinterpret_cast<input_t *>(params.dinitial_states_ptr) + batch_id * params.dinitial_states_batch_stride + l_idx * params.dinitial_states_l_stride + chunk_c_id * kChunkSizeC + c_idx * kNElts;
    input_t *dfinal_states = params.dfinal_states_ptr == nullptr ? nullptr
        : reinterpret_cast<input_t *>(params.dfinal_states_ptr) + batch_id * params.dfinal_states_batch_stride + chunk_c_id * kChunkSizeC;

    #pragma unroll
    for (int l = 0; l < Ktraits::kNLoads; ++l) {
        input_t dout_vals_load[kNElts] = {input_t(0)};
        input_t x_vals_load[kNElts] = {input_t(0)};
        if (chunk_l_id * kChunkSizeL + l * kLPerLoad + l_idx < params.seqlen
            && chunk_c_id * kChunkSizeC + c_idx * kNElts < params.dim) {
            reinterpret_cast<vec_t *>(dout_vals_load)[0] = *reinterpret_cast<vec_t *>(dout + l * kLPerLoad * params.dout_l_stride);
            reinterpret_cast<vec_t *>(x_vals_load)[0] = *reinterpret_cast<vec_t *>(x + l * kLPerLoad * params.x_l_stride);
        }
        reinterpret_cast<vec_t *>(dout_smem[l * kLPerLoad + l_idx])[c_idx] = reinterpret_cast<vec_t *>(dout_vals_load)[0];
        reinterpret_cast<vec_t *>(x_smem[kWidth - 1 + l * kLPerLoad + l_idx])[c_idx] = reinterpret_cast<vec_t *>(x_vals_load)[0];
    }
    // Load the elements from the previous chunk or next chunk that are needed for convolution.
    if (l_idx < kWidth - 1) {
        input_t dout_vals_load[kNElts] = {input_t(0)};
        input_t x_vals_load[kNElts] = {input_t(0)};
        if ((chunk_l_id + 1) * kChunkSizeL + l_idx < params.seqlen
            && chunk_c_id * kChunkSizeC + c_idx * kNElts < params.dim) {
            reinterpret_cast<vec_t *>(dout_vals_load)[0] = *reinterpret_cast<vec_t *>(dout + kChunkSizeL * params.dout_l_stride);
        }
        if (chunk_l_id * kChunkSizeL + l_idx - (kWidth - 1) >= 0
            && chunk_l_id * kChunkSizeL + l_idx - (kWidth - 1) < params.seqlen
            && chunk_c_id * kChunkSizeC + c_idx * kNElts < params.dim) {
            reinterpret_cast<vec_t *>(x_vals_load)[0] = *reinterpret_cast<vec_t *>(x - (kWidth - 1) * params.x_l_stride);
        } else if (initial_states != nullptr
                   && chunk_l_id * kChunkSizeL + l_idx - (kWidth - 1) < 0
                   && chunk_c_id * kChunkSizeC + c_idx * kNElts < params.dim) {
            reinterpret_cast<vec_t *>(x_vals_load)[0] = *reinterpret_cast<vec_t *>(initial_states);
        }
        reinterpret_cast<vec_t *>(dout_smem[kChunkSizeL + l_idx])[c_idx] = reinterpret_cast<vec_t *>(dout_vals_load)[0];
        reinterpret_cast<vec_t *>(x_smem[l_idx])[c_idx] = reinterpret_cast<vec_t *>(x_vals_load)[0];
    }
    // Need to load (kWdith - 1) extra x's on the right to recompute the (kChunkSizeL + kWidth - 1) outputs
    if constexpr (kSiluAct) {
        if (l_idx < kWidth - 1) {
            input_t x_vals_load[kNElts] = {input_t(0)};
            if ((chunk_l_id + 1) * kChunkSizeL + l_idx < params.seqlen
                && chunk_c_id * kChunkSizeC + c_idx * kNElts < params.dim) {
                reinterpret_cast<vec_t *>(x_vals_load)[0] = *reinterpret_cast<vec_t *>(x + kChunkSizeL * params.x_l_stride);
            }
            reinterpret_cast<vec_t *>(x_smem[kWidth - 1 + kChunkSizeL + l_idx])[c_idx] = reinterpret_cast<vec_t *>(x_vals_load)[0];
        }
    }

    __syncthreads();

    constexpr int kLPerThread = constexpr_min(kChunkSizeL * kChunkSizeC / kNThreads, kChunkSizeL);
    static_assert(kLPerThread * kNThreads == kChunkSizeL * kChunkSizeC);
    constexpr int kNThreadsPerRow = kChunkSizeL / kLPerThread;
    static_assert(kNThreadsPerRow * kLPerThread == kChunkSizeL);
    // kChunkSizeL, kLPerThread, kNThreadsPerRow should be powers of 2 for simplicity
    static_assert((kChunkSizeL & (kChunkSizeL - 1)) == 0);
    static_assert((kLPerThread & (kLPerThread - 1)) == 0);
    static_assert((kNThreadsPerRow & (kNThreadsPerRow - 1)) == 0);
    static_assert(kNThreadsPerRow <= 32);

    const int row_idx = tid / kNThreadsPerRow;
    const int col_idx = tid % kNThreadsPerRow;

    float bias_val = params.bias_ptr == nullptr || chunk_c_id * kChunkSizeC + row_idx >= params.dim ? 0.f : float(reinterpret_cast<weight_t *>(params.bias_ptr)[chunk_c_id * kChunkSizeC + row_idx]);
    float weight_vals[kWidth] = {input_t(0)};
    if (chunk_c_id * kChunkSizeC + row_idx < params.dim) {
        #pragma unroll
        for (int w = 0; w < kWidth; ++w) {
            weight_vals[w] = weight[row_idx * params.weight_c_stride + w * params.weight_width_stride];
        }
    }
    float dout_vals[kLPerThread + kWidth - 1];
    float x_vals[kWidth - 1 + kLPerThread + kWidth - 1];
    #pragma unroll
    for (int i = 0; i < kWidth - 1 + kLPerThread; ++i) {
        dout_vals[i] = float(dout_smem[col_idx * kLPerThread + i][row_idx]);
        x_vals[i] = float(x_smem[col_idx * kLPerThread + i][row_idx]);
    }

    int seq_idx_thread[kWidth - 1 + kLPerThread + kWidth - 1];
    if constexpr (kHasSeqIdx) {
        #pragma unroll
        for (int i = 0; i < kWidth - 1 + kLPerThread + kWidth - 1; ++i) {
            const int l_idx = chunk_l_id * kChunkSizeL + col_idx * kLPerThread + i - (kWidth - 1);
            seq_idx_thread[i] = l_idx >= 0 && l_idx < params.seqlen ? seq_idx[col_idx * kLPerThread + i - (kWidth - 1)] : -1;
        }
    }

    if constexpr (kSiluAct) {  // Recompute the output
        #pragma unroll
        for (int i = kWidth - 1 + kLPerThread; i < kWidth - 1 + kLPerThread + kWidth - 1; ++i) {
            x_vals[i] = float(x_smem[col_idx * kLPerThread + i][row_idx]);
        }
        #pragma unroll
        for (int i = 0; i < kLPerThread + kWidth - 1; ++i) {
            float out_val = bias_val;
            const int seq_idx_cur = !kHasSeqIdx ? 0 : seq_idx_thread[i + kWidth - 1];
            #pragma unroll
            for (int w = 0; w < kWidth; ++w) {
                if constexpr (!kHasSeqIdx) {
                    out_val += weight_vals[w] * x_vals[i + w];
                } else {
                    out_val += seq_idx_thread[i + w] == seq_idx_cur ? weight_vals[w] * x_vals[i + w] : 0.f;
                }
            }
            float out_val_sigmoid = 1.f / (1.f + expf(-out_val));
            dout_vals[i] *= out_val_sigmoid * (1 + out_val * (1 - out_val_sigmoid));
        }
    }

    float dweight_vals[kWidth] = {input_t(0)};
    SumOp<float> sum_op;
    #pragma unroll
    for (int w = 0; w < kWidth; ++w) {
        #pragma unroll
        for (int i = 0; i < kLPerThread; ++i) {
            if constexpr (!kHasSeqIdx) {
                dweight_vals[w] += x_vals[i + w] * dout_vals[i];
            } else {
                dweight_vals[w] += seq_idx_thread[i + w] == seq_idx_thread[kWidth - 1 + i] ? x_vals[i + w] * dout_vals[i] : 0.f;
            }
        }
        dweight_vals[w] = Allreduce<kNThreadsPerRow>::run(dweight_vals[w], sum_op);
        if (col_idx == 0 && chunk_c_id * kChunkSizeC + row_idx < params.dim) {
            atomicAdd(&reinterpret_cast<float *>(dweight)[row_idx * params.dweight_c_stride + w * params.dweight_width_stride], dweight_vals[w]);
        }
    }

    if (params.bias_ptr != nullptr) {
        float dbias_val = 0.f;
        for (int i = 0; i < kLPerThread; ++i) { dbias_val += dout_vals[i]; }
        dbias_val = Allreduce<kNThreadsPerRow>::run(dbias_val, sum_op);
        if (col_idx == 0 && chunk_c_id * kChunkSizeC + row_idx < params.dim) {
            atomicAdd(&reinterpret_cast<float *>(params.dbias_ptr)[chunk_c_id * kChunkSizeC + row_idx], dbias_val);
        }
    }

    float dx_vals[kLPerThread] = {input_t(0)};
    #pragma unroll
    for (int i = 0; i < kLPerThread; ++i) {
        const int seq_idx_cur = !kHasSeqIdx ? 0 : seq_idx_thread[i + kWidth - 1];
        #pragma unroll
        for (int w = 0; w < kWidth; ++w) {
            if constexpr (!kHasSeqIdx) {
                dx_vals[i] += weight_vals[kWidth - 1 - w] * dout_vals[i + w];
            } else {
                dx_vals[i] += seq_idx_thread[kWidth - 1 + i + w] == seq_idx_cur ? weight_vals[kWidth - 1 - w] * dout_vals[i + w] : 0.f;
            }
        }
        // if (dfinal_states != nullptr) {
        if constexpr (kHasDfinalStates) {
            if (chunk_l_id * kChunkSizeL + col_idx * kLPerThread + i >= params.seqlen - kWidth + 1
                && chunk_l_id * kChunkSizeL + col_idx * kLPerThread + i < params.seqlen
                && chunk_c_id * kChunkSizeC + row_idx < params.dim) {
                dx_vals[i] += float(dfinal_states[((chunk_l_id * kChunkSizeL + col_idx * kLPerThread + i) - (params.seqlen - kWidth + 1)) * params.dfinal_states_l_stride + row_idx * params.dfinal_states_c_stride]);
            }
        }
    }

    float dxinit_vals[kWidth - 1] = {input_t(0)};
    static_assert(kLPerThread >= kWidth - 1);  // So only threads with col_idx == 0 need to handle dinitial_states
    if (dinitial_states != nullptr && col_idx == 0) {
        #pragma unroll
        for (int i = 0; i < kWidth - 1; ++i) {
            #pragma unroll
            for (int w = 0; w < kWidth; ++w) {
                dxinit_vals[i] += i + w - (kWidth - 1) >= 0 ? weight_vals[kWidth - 1 - w] * dout_vals[i + w - (kWidth - 1)] : 0.f;
            }
            // chunk_l_id must be 0 because dinitial_states != nullptr
            // if (dfinal_states != nullptr) {
            if constexpr (kHasDfinalStates) {
                if (i >= params.seqlen) {
                    dxinit_vals[i] += float(dfinal_states[(i - params.seqlen) * params.dfinal_states_l_stride + row_idx * params.dfinal_states_c_stride]);
                }
            }
        }
    }

    __syncthreads();
    #pragma unroll
    for (int i = 0; i < kLPerThread; ++i) { x_smem[kWidth - 1 + col_idx * kLPerThread + i][row_idx] = dx_vals[i]; }
    if (dinitial_states != nullptr && col_idx == 0) {
        #pragma unroll
        for (int i = 0; i < kWidth - 1; ++i) { x_smem[i][row_idx] = dxinit_vals[i]; }
    }
    __syncthreads();

    #pragma unroll
    for (int l = 0; l < Ktraits::kNLoads; ++l) {
        input_t dx_vals_store[kNElts];
        reinterpret_cast<vec_t *>(dx_vals_store)[0] = reinterpret_cast<vec_t *>(x_smem[kWidth - 1 + l * kLPerLoad + l_idx])[c_idx];
        if (chunk_l_id * kChunkSizeL + l * kLPerLoad + l_idx < params.seqlen
            && chunk_c_id * kChunkSizeC + c_idx * kNElts < params.dim) {
            *reinterpret_cast<vec_t *>(dx + l * kLPerLoad * params.dx_l_stride) = reinterpret_cast<vec_t *>(dx_vals_store)[0];
        }
    }
    if (dinitial_states != nullptr
        && l_idx < kWidth - 1
        && chunk_c_id * kChunkSizeC + c_idx * kNElts < params.dim) {
        input_t dxinit_vals_store[kNElts];
        reinterpret_cast<vec_t *>(dxinit_vals_store)[0] = reinterpret_cast<vec_t *>(x_smem[l_idx])[c_idx];
        *reinterpret_cast<vec_t *>(dinitial_states) = reinterpret_cast<vec_t *>(dxinit_vals_store)[0];
    }

}

template<int kNThreads, int kWidth, typename input_t, typename weight_t>
void causal_conv1d_channellast_bwd_launch(ConvParamsBwd &params, hipStream_t stream) {
    BOOL_SWITCH(params.silu_activation, kSiluAct, [&] {
        BOOL_SWITCH(params.seq_idx_ptr != nullptr, kHasSeqIdx, [&] {
            BOOL_SWITCH(params.dfinal_states_ptr != nullptr, kHasDfinalStates, [&] {
                BOOL_SWITCH(params.seqlen <= 128, kChunkSizeL64, [&] {
                    // kChunkSizeL = 128 is slightly faster than 64 when seqlen is larger
                    static constexpr int kChunk = kChunkSizeL64 ? 64 : 128;
                    using Ktraits = Causal_conv1d_channellast_bwd_kernel_traits<kNThreads, kWidth, kChunk, kSiluAct, true, input_t, weight_t>;
                    // constexpr int kSmemSize = Ktraits::kSmemSize;
                    constexpr int kChunkSizeL = Ktraits::kChunkSizeL;
                    constexpr int kChunkSizeC = Ktraits::kNEltsPerRow;
                    const int n_chunks_L = (params.seqlen + kChunkSizeL - 1) / kChunkSizeL;
                    const int n_chunks_C = (params.dim + kChunkSizeC - 1) / kChunkSizeC;
                    dim3 grid(params.batch, n_chunks_L, n_chunks_C);
                    dim3 block(Ktraits::kNThreads);
                    auto kernel = &causal_conv1d_channellast_bwd_kernel<Ktraits, kHasSeqIdx, kHasDfinalStates>;
                    // if (kSmemSize >= 48 * 1024) {
                    //     C10_CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(
                    //         kernel), hipFuncAttributeMaxDynamicSharedMemorySize, kSmemSize));
                    //     }
                    // kernel<<<grid, Ktraits::kNThreads, kSmemSize, stream>>>(params);
                    kernel<<<grid, Ktraits::kNThreads, 0, stream>>>(params);
                });
            });
        });
    });
}

template<typename input_t, typename weight_t>
void causal_conv1d_channellast_bwd_cuda(ConvParamsBwd &params, hipStream_t stream) {
    if (params.width == 2) {
        causal_conv1d_channellast_bwd_launch<128, 2, input_t, weight_t>(params, stream);
    } else if (params.width == 3) {
        causal_conv1d_channellast_bwd_launch<128, 3, input_t, weight_t>(params, stream);
    } else if (params.width == 4) {
        causal_conv1d_channellast_bwd_launch<128, 4, input_t, weight_t>(params, stream);
    }
}

template void causal_conv1d_bwd_cuda<float, float>(ConvParamsBwd &params, hipStream_t stream);
template void causal_conv1d_bwd_cuda<phi::dtype::float16, float>(ConvParamsBwd &params, hipStream_t stream);
template void causal_conv1d_bwd_cuda<float, phi::dtype::float16>(ConvParamsBwd &params, hipStream_t stream);
template void causal_conv1d_bwd_cuda<phi::dtype::float16, phi::dtype::float16>(ConvParamsBwd &params, hipStream_t stream);
template void causal_conv1d_channellast_bwd_cuda<float, float>(ConvParamsBwd &params, hipStream_t stream);
template void causal_conv1d_channellast_bwd_cuda<phi::dtype::float16, float>(ConvParamsBwd &params, hipStream_t stream);
template void causal_conv1d_channellast_bwd_cuda<float, phi::dtype::float16>(ConvParamsBwd &params, hipStream_t stream);
template void causal_conv1d_channellast_bwd_cuda<phi::dtype::float16, phi::dtype::float16>(ConvParamsBwd &params, hipStream_t stream);

#if defined(CUDA_BFLOAT16_AVAILABLE)
template void causal_conv1d_bwd_cuda<phi::dtype::bfloat16, float>(ConvParamsBwd &params, hipStream_t stream);
template void causal_conv1d_bwd_cuda<phi::dtype::bfloat16, phi::dtype::float16>(ConvParamsBwd &params, hipStream_t stream);
template void causal_conv1d_bwd_cuda<float, phi::dtype::bfloat16>(ConvParamsBwd &params, hipStream_t stream);
template void causal_conv1d_bwd_cuda<phi::dtype::float16, phi::dtype::bfloat16>(ConvParamsBwd &params, hipStream_t stream);
template void causal_conv1d_bwd_cuda<phi::dtype::bfloat16, phi::dtype::bfloat16>(ConvParamsBwd &params, hipStream_t stream);
template void causal_conv1d_channellast_bwd_cuda<phi::dtype::bfloat16, float>(ConvParamsBwd &params, hipStream_t stream);
template void causal_conv1d_channellast_bwd_cuda<phi::dtype::bfloat16, phi::dtype::float16>(ConvParamsBwd &params, hipStream_t stream);
template void causal_conv1d_channellast_bwd_cuda<float, phi::dtype::bfloat16>(ConvParamsBwd &params, hipStream_t stream);
template void causal_conv1d_channellast_bwd_cuda<phi::dtype::float16, phi::dtype::bfloat16>(ConvParamsBwd &params, hipStream_t stream);
template void causal_conv1d_channellast_bwd_cuda<phi::dtype::bfloat16, phi::dtype::bfloat16>(ConvParamsBwd &params, hipStream_t stream);
#endif