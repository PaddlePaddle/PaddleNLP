#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2024, Tri Dao.
 ******************************************************************************/

#include <paddle/phi/common/data_type.h>

#ifndef USE_ROCM
    #include <cub/block/block_load.cuh>
    #include <cub/block/block_store.cuh>
#else
    #include <hipcub/hipcub.hpp>
    namespace cub = hipcub;
#endif

#include "causal_conv1d.h"
#include "causal_conv1d_common.h"
#include "static_switch.h"

template<int kNThreads_, int kWidth_, bool kIsVecLoad_, typename input_t_, typename weight_t_>
struct Causal_conv1d_fwd_kernel_traits {
    using input_t = input_t_;
    using weight_t = weight_t_;
    static constexpr int kNThreads = kNThreads_;
    static constexpr int kWidth = kWidth_;
    static constexpr int kNBytes = sizeof(input_t);
    static_assert(kNBytes == 2 || kNBytes == 4);
    static constexpr int kNElts = kNBytes == 4 ? 4 : 8;
    static_assert(kWidth <= kNElts);
    static constexpr bool kIsVecLoad = kIsVecLoad_;
    using vec_t = typename BytesToType<kNBytes * kNElts>::Type;
    using BlockLoadT = hipcub::BlockLoad<input_t, kNThreads, kNElts, hipcub::BLOCK_LOAD_WARP_TRANSPOSE>;
    using BlockLoadVecT = hipcub::BlockLoad<vec_t, kNThreads, 1, hipcub::BLOCK_LOAD_DIRECT>;
    using BlockStoreT = hipcub::BlockStore<input_t, kNThreads, kNElts, hipcub::BLOCK_STORE_WARP_TRANSPOSE>;
    using BlockStoreVecT = hipcub::BlockStore<vec_t, kNThreads, 1, hipcub::BLOCK_STORE_DIRECT>;
    static constexpr int kSmemIOSize = kIsVecLoad
        ? 0
        : custom_max({sizeof(typename BlockLoadT::TempStorage), sizeof(typename BlockStoreT::TempStorage)});
    static constexpr int kSmemExchangeSize = kNThreads * kNBytes * kNElts;
    static constexpr int kSmemSize = kSmemIOSize + kSmemExchangeSize;
};

template<typename Ktraits>
__global__ __launch_bounds__(Ktraits::kNThreads)
void causal_conv1d_fwd_kernel(ConvParamsBase params) {
    constexpr int kWidth = Ktraits::kWidth;
    constexpr int kNThreads = Ktraits::kNThreads;
    constexpr int kNElts = Ktraits::kNElts;
    static constexpr bool kIsVecLoad = Ktraits::kIsVecLoad;
    using input_t = typename Ktraits::input_t;
    using vec_t = typename Ktraits::vec_t;
    using weight_t = typename Ktraits::weight_t;

    // Shared memory.
    extern __shared__ char smem_[];
    auto& smem_load = reinterpret_cast<typename Ktraits::BlockLoadT::TempStorage&>(smem_);
    auto& smem_load_vec = reinterpret_cast<typename Ktraits::BlockLoadVecT::TempStorage&>(smem_);
    auto& smem_store = reinterpret_cast<typename Ktraits::BlockStoreT::TempStorage&>(smem_);
    auto& smem_store_vec = reinterpret_cast<typename Ktraits::BlockStoreVecT::TempStorage&>(smem_);
    vec_t *smem_exchange = reinterpret_cast<vec_t *>(smem_ + Ktraits::kSmemIOSize);

    const int tidx = threadIdx.x;
    const int batch_id = blockIdx.x;
    const int channel_id = blockIdx.y;
    input_t *x = reinterpret_cast<input_t *>(params.x_ptr) + batch_id * params.x_batch_stride
        + channel_id * params.x_c_stride;
    weight_t *weight = reinterpret_cast<weight_t *>(params.weight_ptr) + channel_id * params.weight_c_stride;
    input_t *out = reinterpret_cast<input_t *>(params.out_ptr) + batch_id * params.out_batch_stride
        + channel_id * params.out_c_stride;
    float bias_val = params.bias_ptr == nullptr ? 0.f : float(reinterpret_cast<weight_t *>(params.bias_ptr)[channel_id]);

    // Thread 0 will load the last elements of the previous chunk, so we initialize those to 0.
    if (tidx == 0) {
        input_t zeros[kNElts] = {input_t(0)};
        smem_exchange[kNThreads - 1] = reinterpret_cast<vec_t *>(zeros)[0];
    }

    float weight_vals[kWidth];
    #pragma unroll
    for (int i = 0; i < kWidth; ++i) { weight_vals[i] = float(weight[i * params.weight_width_stride]); }

    constexpr int kChunkSize = kNThreads * kNElts;
    const int n_chunks = (params.seqlen + kChunkSize - 1) / kChunkSize;
    for (int chunk = 0; chunk < n_chunks; ++chunk) {
        input_t x_vals_load[2 * kNElts] = {input_t(0)};
        if constexpr(kIsVecLoad) {
            typename Ktraits::BlockLoadVecT(smem_load_vec).Load(reinterpret_cast<vec_t*>(x), *reinterpret_cast<vec_t (*)[1]>(&x_vals_load[kNElts]), (params.seqlen - chunk * kChunkSize) / kNElts);
        } else {
            __syncthreads();
            typename Ktraits::BlockLoadT(smem_load).Load(x, *reinterpret_cast<input_t (*)[kNElts]>(&x_vals_load[kNElts]), params.seqlen - chunk * kChunkSize);
        }
        x += kChunkSize;
        __syncthreads();
        // Thread kNThreads - 1 don't write yet, so that thread 0 can read
        // the last elements of the previous chunk.
        if (tidx < kNThreads - 1) { smem_exchange[tidx] = reinterpret_cast<vec_t *>(x_vals_load)[1]; }
        __syncthreads();
        reinterpret_cast<vec_t *>(x_vals_load)[0] = smem_exchange[tidx > 0 ? tidx - 1 : kNThreads - 1];
        __syncthreads();
        // Now thread kNThreads - 1 can write the last elements of the current chunk.
        if (tidx == kNThreads - 1) { smem_exchange[tidx] = reinterpret_cast<vec_t *>(x_vals_load)[1]; }

        float x_vals[2 * kNElts];
        #pragma unroll
        for (int i = 0; i < 2 * kNElts; ++i) { x_vals[i] = float(x_vals_load[i]); }

        float out_vals[kNElts];
        #pragma unroll
        for (int i = 0; i < kNElts; ++i) {
            out_vals[i] = bias_val;
            #pragma unroll
            for (int w = 0; w < kWidth; ++w) {
                out_vals[i] += weight_vals[w] * x_vals[kNElts + i - (kWidth - w - 1)];
            }
        }

        if (params.silu_activation) {
            #pragma unroll
            for (int i = 0; i < kNElts; ++i) {
                out_vals[i] = out_vals[i] / (1 + expf(-out_vals[i]));
            }
        }

        input_t out_vals_store[kNElts];
        #pragma unroll
        for (int i = 0; i < kNElts; ++i) { out_vals_store[i] = out_vals[i]; }
        if constexpr(kIsVecLoad) {
            typename Ktraits::BlockStoreVecT(smem_store_vec).Store(reinterpret_cast<vec_t*>(out), reinterpret_cast<vec_t (&)[1]>(out_vals_store), (params.seqlen - chunk * kChunkSize) / kNElts);
        } else {
            typename Ktraits::BlockStoreT(smem_store).Store(out, out_vals_store, params.seqlen - chunk * kChunkSize);
        }
        out += kChunkSize;
    }
}

template<int kNThreads, int kWidth, typename input_t, typename weight_t>
void causal_conv1d_fwd_launch(ConvParamsBase &params, hipStream_t stream) {
    static constexpr int kNElts = sizeof(input_t) == 4 ? 4 : 8;
    BOOL_SWITCH(params.seqlen % kNElts == 0, kIsVecLoad, [&] {
        using Ktraits = Causal_conv1d_fwd_kernel_traits<kNThreads, kWidth, kIsVecLoad, input_t, weight_t>;
        constexpr int kSmemSize = Ktraits::kSmemSize;
        dim3 grid(params.batch, params.dim);

        auto kernel = &causal_conv1d_fwd_kernel<Ktraits>;

        if (kSmemSize >= 48 * 1024) {
            #ifndef USE_ROCM
            hipFuncSetAttribute(reinterpret_cast<const void*>(
                kernel), hipFuncAttributeMaxDynamicSharedMemorySize, kSmemSize);
            #else
            // There is a slight signature discrepancy in HIP and CUDA "FuncSetAttribute" function.
            hipFuncSetAttribute(
                (void *) kernel, hipFuncAttributeMaxDynamicSharedMemorySize, kSmemSize);
            std::cerr << "Warning (causal_conv1d fwd launch): attempting to set maxDynamicSharedMemorySize on an AMD GPU which is currently a non-op (in ROCm versions <= 6.1). This might lead to undefined behavior. \n" << std::endl;
            #endif
        }
        kernel<<<grid, Ktraits::kNThreads, kSmemSize, stream>>>(params);

    });
}

template<typename input_t, typename weight_t>
void causal_conv1d_fwd_cuda(ConvParamsBase &params, hipStream_t stream) {
    if (params.width == 2) {
        causal_conv1d_fwd_launch<128, 2, input_t, weight_t>(params, stream);
    } else if (params.width == 3) {
        causal_conv1d_fwd_launch<128, 3, input_t, weight_t>(params, stream);
    } else if (params.width == 4) {
        causal_conv1d_fwd_launch<128, 4, input_t, weight_t>(params, stream);
    }
}

template<int kNThreads_, int kWidth_, int kChunkSizeL_, bool kIsVecLoad_, typename input_t_, typename weight_t_>
struct Causal_conv1d_channellast_fwd_kernel_traits {
    // The cache line is 128 bytes, and we try to read 16 bytes per thread.
    // So we have 8 threads per "row", so 32 or 64 elements in the channel dimension.
    // That leaves 4 columns per warp, and so 16 columns per block (assuming each block has 128
    // threads). Each each load is 16 x 32|64 elements in the L x C dimensions.
    using input_t = input_t_;
    using weight_t = weight_t_;
    static constexpr int kNThreads = kNThreads_;
    static_assert(kNThreads % 32 == 0);
    static constexpr int kNWarps = kNThreads / 32;
    static constexpr int kWidth = kWidth_;
    static constexpr int kChunkSizeL = kChunkSizeL_;
    static constexpr int kNBytes = sizeof(input_t);
    static_assert(kNBytes == 2 || kNBytes == 4);
    static constexpr int kNElts = kNBytes == 4 ? 4 : 8;
    static constexpr int kNEltsPerRow = 128 / kNBytes;
    static constexpr int kNThreadsPerRow = kNEltsPerRow / kNElts;  // Always 8 for now
    static_assert(kNThreadsPerRow * kNBytes * kNElts == 128);
    static constexpr int kNColsPerWarp = 32 / kNThreadsPerRow;  // Always 4 for now
    static_assert(kNColsPerWarp * kNThreadsPerRow == 32);
    static constexpr int kNColsPerLoad = kNColsPerWarp * kNWarps;
    static constexpr int kNLoads = kChunkSizeL / kNColsPerLoad;
    static_assert(kNLoads * kNColsPerLoad == kChunkSizeL);
    static constexpr bool kIsVecLoad = kIsVecLoad_;
    using vec_t = typename BytesToType<kNBytes * kNElts>::Type;
    // using BlockLoadT = hipcub::BlockLoad<input_t, kNThreads, kNItems, hipcub::BLOCK_LOAD_WARP_TRANSPOSE>;
    // using BlockStoreT = hipcub::BlockStore<input_t, kNThreads, kNItems, hipcub::BLOCK_STORE_WARP_TRANSPOSE>;
    // static constexpr int kSmemSize = std::max({sizeof(typename BlockLoadT::TempStorage),
    //                                            sizeof(typename BlockStoreT::TempStorage)});
    // static constexpr int kSmemSize = kChunkSizeL * kNEltsPerRow * kNBytes;
};

template<typename Ktraits, bool kHasSeqIdx>
__global__ __launch_bounds__(Ktraits::kNThreads)
void causal_conv1d_channellast_fwd_kernel(ConvParamsBase params) {
    constexpr int kWidth = Ktraits::kWidth;
    constexpr int kNThreads = Ktraits::kNThreads;
    constexpr int kNElts = Ktraits::kNElts;
    constexpr int kNWarp = Ktraits::kNWarps;
    constexpr int kNThreadsPerC = Ktraits::kNThreadsPerRow;
    constexpr int kLPerLoad = Ktraits::kNColsPerLoad;
    constexpr int kChunkSizeL = Ktraits::kChunkSizeL;
    constexpr int kChunkSizeC = Ktraits::kNEltsPerRow;
    using input_t = typename Ktraits::input_t;
    using vec_t = typename Ktraits::vec_t;
    using weight_t = typename Ktraits::weight_t;

    // Shared memory.
    __shared__ input_t x_smem[kWidth - 1 + kChunkSizeL][kChunkSizeC + kNElts];

    const int batch_id = blockIdx.x;
    const int chunk_l_id = blockIdx.y;
    const int chunk_c_id = blockIdx.z;
    const int tid = threadIdx.x;
    const int l_idx = tid / kNThreadsPerC;
    const int c_idx = tid % kNThreadsPerC;
    input_t *x = reinterpret_cast<input_t *>(params.x_ptr) + batch_id * params.x_batch_stride
        + (chunk_l_id * kChunkSizeL + l_idx) * params.x_l_stride + chunk_c_id * kChunkSizeC + c_idx * kNElts;
    weight_t *weight = reinterpret_cast<weight_t *>(params.weight_ptr)
        + chunk_c_id * kChunkSizeC * params.weight_c_stride;
    input_t *out = reinterpret_cast<input_t *>(params.out_ptr) + batch_id * params.out_batch_stride
        + (chunk_l_id * kChunkSizeL + l_idx) * params.out_l_stride + chunk_c_id * kChunkSizeC + c_idx * kNElts;
    int *seq_idx = !kHasSeqIdx ? nullptr : reinterpret_cast<int *>(params.seq_idx_ptr)
        + batch_id * params.seqlen + chunk_l_id * kChunkSizeL;
    input_t *initial_states = params.initial_states_ptr == nullptr || chunk_l_id > 0 ? nullptr
        : reinterpret_cast<input_t *>(params.initial_states_ptr) + batch_id * params.initial_states_batch_stride + l_idx * params.initial_states_l_stride + chunk_c_id * kChunkSizeC + c_idx * kNElts;
    // The last L-chunk will also have enough info to write to final states, since it also contain a few x values
    // from the previous L-chunk.
    input_t *final_states = params.final_states_ptr == nullptr || chunk_l_id < gridDim.y - 1 ? nullptr
        : reinterpret_cast<input_t *>(params.final_states_ptr) + batch_id * params.final_states_batch_stride + l_idx * params.final_states_l_stride + chunk_c_id * kChunkSizeC + c_idx * kNElts;

    #pragma unroll
    for (int l = 0; l < Ktraits::kNLoads; ++l) {
        input_t x_vals_load[kNElts] = {input_t(0)};
        if (chunk_l_id * kChunkSizeL + l * kLPerLoad + l_idx < params.seqlen
            && chunk_c_id * kChunkSizeC + c_idx * kNElts < params.dim) {
            reinterpret_cast<vec_t *>(x_vals_load)[0] = *reinterpret_cast<vec_t *>(x + l * kLPerLoad * params.x_l_stride);
        }
        reinterpret_cast<vec_t *>(x_smem[kWidth - 1 + l * kLPerLoad + l_idx])[c_idx] = reinterpret_cast<vec_t *>(x_vals_load)[0];
    }
    // Load the elements from the previous chunk that are needed for convolution.
    if (l_idx < kWidth - 1) {
        input_t x_vals_load[kNElts] = {input_t(0)};
        if (chunk_l_id * kChunkSizeL + l_idx - (kWidth - 1) >= 0
            && chunk_l_id * kChunkSizeL + l_idx - (kWidth - 1) < params.seqlen
            && chunk_c_id * kChunkSizeC + c_idx * kNElts < params.dim) {
            reinterpret_cast<vec_t *>(x_vals_load)[0] = *reinterpret_cast<vec_t *>(x - (kWidth - 1) * params.x_l_stride);
        } else if (initial_states != nullptr
                   && chunk_l_id * kChunkSizeL + l_idx - (kWidth - 1) < 0
                   && chunk_c_id * kChunkSizeC + c_idx * kNElts < params.dim) {
            reinterpret_cast<vec_t *>(x_vals_load)[0] = *reinterpret_cast<vec_t *>(initial_states);
        }
        reinterpret_cast<vec_t *>(x_smem[l_idx])[c_idx] = reinterpret_cast<vec_t *>(x_vals_load)[0];
    }

    __syncthreads();

    if (final_states != nullptr
        && l_idx < kWidth - 1
        && chunk_c_id * kChunkSizeC + c_idx * kNElts < params.dim) {
        // x_smem[0] contains element at index chunk_l_id * kChunkSizeL - (kWidth - 1)
        // So last few elements (index params.seqlen - kWidth + 1 + l_idx) are stored in x_smem[params.seqlen - kWidth + 1 + l_idx - (chunk_l_id * kChunkSizeL - kWidth + 1)][c_idx]
        *reinterpret_cast<vec_t *>(final_states) = reinterpret_cast<vec_t *>(x_smem[params.seqlen + l_idx - chunk_l_id * kChunkSizeL])[c_idx];
    }

    constexpr int kLPerThread = constexpr_min(kChunkSizeL * kChunkSizeC / kNThreads, kChunkSizeL);
    static_assert(kLPerThread * kNThreads == kChunkSizeL * kChunkSizeC);
    constexpr int kNThreadsPerRow = kChunkSizeL / kLPerThread;
    static_assert(kNThreadsPerRow * kLPerThread == kChunkSizeL);
    // kChunkSizeL, kLPerThread, kNThreadsPerRow should be powers of 2 for simplicity
    static_assert((kChunkSizeL & (kChunkSizeL - 1)) == 0);
    static_assert((kLPerThread & (kLPerThread - 1)) == 0);
    static_assert((kNThreadsPerRow & (kNThreadsPerRow - 1)) == 0);
    static_assert(kNThreadsPerRow <= 32);

    const int row_idx = tid / kNThreadsPerRow;
    const int col_idx = tid % kNThreadsPerRow;

    float bias_val = params.bias_ptr == nullptr || chunk_c_id * kChunkSizeC + row_idx >= params.dim ? 0.f : float(reinterpret_cast<weight_t *>(params.bias_ptr)[chunk_c_id * kChunkSizeC + row_idx]);
    float weight_vals[kWidth] = {input_t(0)};
    if (chunk_c_id * kChunkSizeC + row_idx < params.dim) {
        #pragma unroll
        for (int w = 0; w < kWidth; ++w) {
            weight_vals[w] = weight[row_idx * params.weight_c_stride + w * params.weight_width_stride];
        }
    }
    float x_vals[kWidth - 1 + kLPerThread];
    #pragma unroll
    for (int i = 0; i < kWidth - 1 + kLPerThread; ++i) {
        x_vals[i] = float(x_smem[col_idx * kLPerThread + i][row_idx]);
    }
    int seq_idx_thread[kWidth - 1 + kLPerThread];
    if constexpr (kHasSeqIdx) {
        #pragma unroll
        for (int i = 0; i < kWidth - 1 + kLPerThread; ++i) {
            seq_idx_thread[i] = chunk_l_id * kChunkSizeL + col_idx * kLPerThread + i - (kWidth - 1) >= 0 ? seq_idx[col_idx * kLPerThread + i - (kWidth - 1)] : -1;
        }
    }

    float out_vals[kLPerThread];
    #pragma unroll
    for (int i = 0; i < kLPerThread; ++i) {
        out_vals[i] = bias_val;
        const int seq_idx_cur = !kHasSeqIdx ? 0 : seq_idx_thread[i + kWidth - 1];
        #pragma unroll
        for (int w = 0; w < kWidth; ++w) {
            if constexpr (!kHasSeqIdx) {
                out_vals[i] += weight_vals[w] * x_vals[i + w];
            } else {
                out_vals[i] += seq_idx_thread[i + w] == seq_idx_cur ? weight_vals[w] * x_vals[i + w] : 0.f;
            }
        }
        if (params.silu_activation) {out_vals[i] = out_vals[i] / (1 + expf(-out_vals[i])); }
    }

    __syncthreads();
    #pragma unroll
    for (int i = 0; i < kLPerThread; ++i) { x_smem[col_idx * kLPerThread + i][row_idx] = out_vals[i]; }
    __syncthreads();

    #pragma unroll
    for (int l = 0; l < Ktraits::kNLoads; ++l) {
        input_t out_vals_store[kNElts];
        reinterpret_cast<vec_t *>(out_vals_store)[0] = reinterpret_cast<vec_t *>(x_smem[l * kLPerLoad + l_idx])[c_idx];
        if (chunk_l_id * kChunkSizeL + l * kLPerLoad + l_idx < params.seqlen
            && chunk_c_id * kChunkSizeC + c_idx * kNElts < params.dim) {
            *reinterpret_cast<vec_t *>(out + l * kLPerLoad * params.out_l_stride) = reinterpret_cast<vec_t *>(out_vals_store)[0];
        }
    }

}

template<int kNThreads, int kWidth, typename input_t, typename weight_t>
void causal_conv1d_channellast_fwd_launch(ConvParamsBase &params, hipStream_t stream) {
    BOOL_SWITCH(params.seq_idx_ptr != nullptr, kHasSeqIdx, [&] {
        using Ktraits = Causal_conv1d_channellast_fwd_kernel_traits<kNThreads, kWidth, 64, true, input_t, weight_t>;
        // constexpr int kSmemSize = Ktraits::kSmemSize;
        constexpr int kChunkSizeL = Ktraits::kChunkSizeL;
        constexpr int kChunkSizeC = Ktraits::kNEltsPerRow;
        const int n_chunks_L = (params.seqlen + kChunkSizeL - 1) / kChunkSizeL;
        const int n_chunks_C = (params.dim + kChunkSizeC - 1) / kChunkSizeC;
        dim3 grid(params.batch, n_chunks_L, n_chunks_C);
        dim3 block(Ktraits::kNThreads);
        auto kernel = &causal_conv1d_channellast_fwd_kernel<Ktraits, kHasSeqIdx>;
        // if (kSmemSize >= 48 * 1024) {
        //     C10_CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(
        //         kernel), hipFuncAttributeMaxDynamicSharedMemorySize, kSmemSize));
        //     }
        // kernel<<<grid, Ktraits::kNThreads, kSmemSize, stream>>>(params);
        kernel<<<grid, Ktraits::kNThreads, 0, stream>>>(params);
    });
}

template<typename input_t, typename weight_t>
void causal_conv1d_channellast_fwd_cuda(ConvParamsBase &params, hipStream_t stream) {
    if (params.width == 2) {
        causal_conv1d_channellast_fwd_launch<128, 2, input_t, weight_t>(params, stream);
    } else if (params.width == 3) {
        causal_conv1d_channellast_fwd_launch<128, 3, input_t, weight_t>(params, stream);
    } else if (params.width == 4) {
        causal_conv1d_channellast_fwd_launch<128, 4, input_t, weight_t>(params, stream);
    }
}

template void causal_conv1d_fwd_cuda<float, float>(ConvParamsBase &params, hipStream_t stream);
template void causal_conv1d_fwd_cuda<phi::dtype::float16, float>(ConvParamsBase &params, hipStream_t stream);
template void causal_conv1d_fwd_cuda<float, phi::dtype::float16>(ConvParamsBase &params, hipStream_t stream);
template void causal_conv1d_fwd_cuda<phi::dtype::float16, phi::dtype::float16>(ConvParamsBase &params, hipStream_t stream);
template void causal_conv1d_channellast_fwd_cuda<float, float>(ConvParamsBase &params, hipStream_t stream);
template void causal_conv1d_channellast_fwd_cuda<phi::dtype::float16, float>(ConvParamsBase &params, hipStream_t stream);
template void causal_conv1d_channellast_fwd_cuda<float, phi::dtype::float16>(ConvParamsBase &params, hipStream_t stream);
template void causal_conv1d_channellast_fwd_cuda<phi::dtype::float16, phi::dtype::float16>(ConvParamsBase &params, hipStream_t stream);

#if defined(CUDA_BFLOAT16_AVAILABLE)
template void causal_conv1d_fwd_cuda<phi::dtype::bfloat16, float>(ConvParamsBase &params, hipStream_t stream);
template void causal_conv1d_fwd_cuda<phi::dtype::bfloat16, phi::dtype::float16>(ConvParamsBase &params, hipStream_t stream);
template void causal_conv1d_fwd_cuda<float, phi::dtype::bfloat16>(ConvParamsBase &params, hipStream_t stream);
template void causal_conv1d_fwd_cuda<phi::dtype::float16, phi::dtype::bfloat16>(ConvParamsBase &params, hipStream_t stream);
template void causal_conv1d_fwd_cuda<phi::dtype::bfloat16, phi::dtype::bfloat16>(ConvParamsBase &params, hipStream_t stream);
template void causal_conv1d_channellast_fwd_cuda<phi::dtype::bfloat16, float>(ConvParamsBase &params, hipStream_t stream);
template void causal_conv1d_channellast_fwd_cuda<phi::dtype::bfloat16, phi::dtype::float16>(ConvParamsBase &params, hipStream_t stream);
template void causal_conv1d_channellast_fwd_cuda<float, phi::dtype::bfloat16>(ConvParamsBase &params, hipStream_t stream);
template void causal_conv1d_channellast_fwd_cuda<phi::dtype::float16, phi::dtype::bfloat16>(ConvParamsBase &params, hipStream_t stream);
template void causal_conv1d_channellast_fwd_cuda<phi::dtype::bfloat16, phi::dtype::bfloat16>(ConvParamsBase &params, hipStream_t stream);
#endif