#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2023, Tri Dao.
 ******************************************************************************/

#include <paddle/phi/common/data_type.h>

#include "causal_conv1d.h"
#include "causal_conv1d_common.h"
#include "static_switch.h"

template<int kNThreads_, int kWidth_, typename input_t_, typename weight_t_>
struct Causal_conv1d_update_kernel_traits {
    using input_t = input_t_;
    using weight_t = weight_t_;
    static constexpr int kNThreads = kNThreads_;
    static constexpr int kWidth = kWidth_;
    static constexpr int kNBytes = sizeof(input_t);
    static_assert(kNBytes == 2 || kNBytes == 4);
};

template<typename Ktraits, bool kIsCircularBuffer>
__global__ __launch_bounds__(Ktraits::kNThreads)
void causal_conv1d_update_kernel(ConvParamsBase params) {
    constexpr int kWidth = Ktraits::kWidth;
    constexpr int kNThreads = Ktraits::kNThreads;
    using input_t = typename Ktraits::input_t;
    using weight_t = typename Ktraits::weight_t;

    const int tidx = threadIdx.x;
    const int batch_id = blockIdx.x;
    const int channel_id = blockIdx.y * kNThreads + tidx;
    if (channel_id >= params.dim) return;

    input_t *x = reinterpret_cast<input_t *>(params.x_ptr) + batch_id * params.x_batch_stride
        + channel_id * params.x_c_stride;
    input_t *conv_state = reinterpret_cast<input_t *>(params.conv_state_ptr) + batch_id * params.conv_state_batch_stride
        + channel_id * params.conv_state_c_stride;
    weight_t *weight = reinterpret_cast<weight_t *>(params.weight_ptr) + channel_id * params.weight_c_stride;
    input_t *out = reinterpret_cast<input_t *>(params.out_ptr) + batch_id * params.out_batch_stride
        + channel_id * params.out_c_stride;
    float bias_val = params.bias_ptr == nullptr ? 0.f : float(reinterpret_cast<weight_t *>(params.bias_ptr)[channel_id]);

    int state_len = params.conv_state_len;
    int advance_len = params.seqlen;
    int cache_seqlen = kIsCircularBuffer ? params.cache_seqlens[batch_id] % state_len : 0;
    int update_idx = cache_seqlen - (kWidth - 1);
    update_idx = update_idx < 0 ? update_idx + state_len : update_idx;

    float weight_vals[kWidth] = {0};
    #pragma unroll
    for (int i = 0; i < kWidth; ++i) { weight_vals[i] = float(weight[i * params.weight_width_stride]); }

    float x_vals[kWidth] = {0};
    if constexpr (!kIsCircularBuffer) {
        #pragma unroll 2
        for (int i = 0; i < state_len - advance_len - (kWidth - 1); ++i) {
            conv_state[i * params.conv_state_l_stride] = conv_state[(i + advance_len) * params.conv_state_l_stride];
        }
        #pragma unroll
        for (int i = 0; i < kWidth - 1; ++i) {
            input_t state_val = conv_state[(state_len - (kWidth - 1) + i) * params.conv_state_l_stride];
            if (i < advance_len + (kWidth - 1) && state_len - advance_len - (kWidth - 1) + i >= 0) {
                conv_state[(state_len - advance_len - (kWidth - 1) + i) * params.conv_state_l_stride] = state_val;
            }
            x_vals[i] = float(state_val);
        }
    } else {
        #pragma unroll
        for (int i = 0; i < kWidth - 1; ++i, update_idx = update_idx + 1 >= state_len ? update_idx + 1 - state_len : update_idx + 1) {
            input_t state_val = conv_state[update_idx * params.conv_state_l_stride];
            x_vals[i] = float(state_val);
        }
    }
    #pragma unroll 2
    for (int i = 0; i < params.seqlen; ++i) {
        input_t x_val = x[i * params.x_l_stride];
        if constexpr (!kIsCircularBuffer) {
            if (i < advance_len && state_len - advance_len + i >= 0) {
                conv_state[(state_len - advance_len + i) * params.conv_state_l_stride] = x_val;
            }
        } else {
            conv_state[update_idx * params.conv_state_l_stride] = x_val;
            ++update_idx;
            update_idx = update_idx >= state_len ? update_idx - state_len : update_idx;
        }
        x_vals[kWidth - 1] = float(x_val);
        float out_val = bias_val;
        #pragma unroll
        for (int j = 0; j < kWidth; ++j) { out_val += weight_vals[j] * x_vals[j]; }
        if (params.silu_activation) { out_val = out_val / (1 + expf(-out_val)); }
        out[i * params.out_l_stride] = input_t(out_val);
        // Shift the input buffer by 1
        #pragma unroll
        for (int i = 0; i < kWidth - 1; ++i) { x_vals[i] = x_vals[i + 1]; }
    }
}

template<int kNThreads, int kWidth, typename input_t, typename weight_t>
void causal_conv1d_update_launch(ConvParamsBase &params, hipStream_t stream) {
    using Ktraits = Causal_conv1d_update_kernel_traits<kNThreads, kWidth, input_t, weight_t>;
    dim3 grid(params.batch, (params.dim + kNThreads - 1) / kNThreads);
    auto kernel = params.cache_seqlens == nullptr
        ? &causal_conv1d_update_kernel<Ktraits, false>
        : &causal_conv1d_update_kernel<Ktraits, true>;
    kernel<<<grid, Ktraits::kNThreads, 0, stream>>>(params);
}

template<typename input_t, typename weight_t>
void causal_conv1d_update_cuda(ConvParamsBase &params, hipStream_t stream) {
    if (params.width == 2) {
        causal_conv1d_update_launch<64, 2, input_t, weight_t>(params, stream);
    } else if (params.width == 3) {
        causal_conv1d_update_launch<64, 3, input_t, weight_t>(params, stream);
    } else if (params.width == 4) {
        causal_conv1d_update_launch<64, 4, input_t, weight_t>(params, stream);
    }
}

template void causal_conv1d_update_cuda<float, float>(ConvParamsBase &params, hipStream_t stream);
template void causal_conv1d_update_cuda<phi::dtype::float16, float>(ConvParamsBase &params, hipStream_t stream);
template void causal_conv1d_update_cuda<phi::dtype::bfloat16, float>(ConvParamsBase &params, hipStream_t stream);
template void causal_conv1d_update_cuda<float, phi::dtype::float16>(ConvParamsBase &params, hipStream_t stream);

#if defined(CUDA_BFLOAT16_AVAILABLE)
template void causal_conv1d_update_cuda<phi::dtype::float16, phi::dtype::float16>(ConvParamsBase &params, hipStream_t stream);
template void causal_conv1d_update_cuda<phi::dtype::bfloat16, phi::dtype::float16>(ConvParamsBase &params, hipStream_t stream);
template void causal_conv1d_update_cuda<float, phi::dtype::bfloat16>(ConvParamsBase &params, hipStream_t stream);
template void causal_conv1d_update_cuda<phi::dtype::float16, phi::dtype::bfloat16>(ConvParamsBase &params, hipStream_t stream);
template void causal_conv1d_update_cuda<phi::dtype::bfloat16, phi::dtype::bfloat16>(ConvParamsBase &params, hipStream_t stream);
#endif