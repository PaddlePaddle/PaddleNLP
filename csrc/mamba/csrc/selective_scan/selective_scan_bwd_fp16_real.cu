/******************************************************************************
 * Copyright (c) 2023, Tri Dao.
 ******************************************************************************/

// Split into multiple files to compile in paralell

#include "selective_scan_bwd_kernel.cuh"

template void selective_scan_bwd_cuda<phi::dtype::float16, float>(SSMParamsBwd &params, hipStream_t stream);