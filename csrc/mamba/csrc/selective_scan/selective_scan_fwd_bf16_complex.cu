/******************************************************************************
 * Copyright (c) 2023, Tri Dao.
 ******************************************************************************/

// Split into multiple files to compile in paralell

#include "selective_scan_fwd_kernel.cuh"

template void selective_scan_fwd_cuda<phi::dtype::bfloat16, complex_t>(SSMParamsBase &params, hipStream_t stream);