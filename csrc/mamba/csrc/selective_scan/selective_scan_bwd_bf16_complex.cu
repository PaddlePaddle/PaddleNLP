/******************************************************************************
 * Copyright (c) 2023, Tri Dao.
 ******************************************************************************/

// Split into multiple files to compile in paralell

#include "selective_scan_bwd_kernel.cuh"

template void selective_scan_bwd_cuda<phi::dtype::bfloat16, complex_t>(SSMParamsBwd &params, hipStream_t stream);