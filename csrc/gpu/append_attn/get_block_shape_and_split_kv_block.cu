#include "hip/hip_runtime.h"
// Copyright (c) 2024 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "helper.h"
#include "paddle/extension.h"

template <typename T>
inline __device__ __host__ T div_up(T m, T n) {
  return (m + n - 1) / n;
}

__global__ void split_q_block(const int* __restrict__ seq_lens_q,
                              const int* __restrict__ seq_lens_encoder,
                              int* __restrict__ batch_ids,
                              int* __restrict__ tile_ids_per_batch,
                              int* __restrict__ num_blocks_x,
                              const int bsz,
                              const int num_rows_per_block,
                              const int group_size) {
  if (threadIdx.x == 0) {
    int gridx = 0;
    int index = 0;
    for (uint32_t bid = 0; bid < bsz; bid++) {
      int seq_len = seq_lens_q[bid];
      if (seq_lens_encoder && seq_lens_encoder[bid] > 0) {
        seq_len = 0;
      }
      const int loop_times =
          div_up(seq_len * group_size, num_rows_per_block);
      for (uint32_t tile_id = 0; tile_id < loop_times; tile_id++) {
        batch_ids[index] = bid;
        tile_ids_per_batch[index++] = tile_id;
      }
      gridx += loop_times;
    }
    *num_blocks_x = gridx;
  }
}

__global__ void split_kv_block(const int* __restrict__ seq_lens_decoder,
                               const int* __restrict__ seq_lens_encoder,
                               int* __restrict__ batch_ids,
                               int* __restrict__ tile_ids_per_batch,
                               int* __restrict__ num_blocks_x,
                               const int bsz,
                               const int pad_len,
                               const int num_row_per_block) {
  if (threadIdx.x == 0) {
    int gridx = 0;
    int index = 0;
    for (uint32_t bid = 0; bid < bsz; bid++) {
      const int start_len = seq_lens_decoder[bid];
      int seq_len = seq_lens_encoder[bid] + start_len % pad_len;
      if (seq_lens_encoder[bid] == 0) {
        seq_len = 0;
      }
      const int loop_times = div_up(seq_len, num_row_per_block);
      for (uint32_t tile_id = 0; tile_id < loop_times; tile_id++) {
        batch_ids[index] = bid;
        tile_ids_per_batch[index++] = tile_id;
      }
      gridx += loop_times;
    }
    *num_blocks_x = gridx;
  }
}

template <int THREADBLOCK_SIZE>
__global__ void get_max_len_kv_ernel(int* max_seq_lens_out,
                                  const int* seq_lens_this_time,
                                  const int* seq_lens_decoder,
                                  const int batch_size) {
  const int tid = threadIdx.x;

  
  typedef hipcub::BlockReduce<int, THREADBLOCK_SIZE> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;

  int max_len_this_thread = 0;
  for (int i = tid; i < batch_size; i += blockDim.x) {
    if (seq_lens_decoder[i] == 0) continue;
    max_len_this_thread = max(seq_lens_this_time[i] + seq_lens_decoder[i], max_len_this_thread);
  }
  int total = BlockReduce(temp_storage).Reduce(max_len_this_thread, MaxOp<int>());
  if (tid == 0) {
    *max_seq_lens_out = total;
  }
}

std::vector<paddle::Tensor> GetBlockShapeAndSplitKVBlock(
    const paddle::Tensor& seq_lens_encoder,
    const paddle::Tensor& seq_lens_decoder,
    const paddle::Tensor& max_enc_len_this_time,
    const paddle::Tensor& seq_lens_this_time,
    const paddle::Tensor& cum_offsets,
    const int encoder_block_shape_q,
    const int decoder_block_shape_q,
    const int group_size,
    const int block_size,
    const int decoder_step_token_num) {
  auto stream = seq_lens_encoder.stream();
  int bsz = cum_offsets.shape()[0];

  // decoder
  const uint32_t decoder_max_tile_size_per_bs_q =
      div_up((decoder_step_token_num * group_size), decoder_block_shape_q);
  auto decoder_batch_ids =
      GetEmptyTensor({bsz * decoder_max_tile_size_per_bs_q},
                     paddle::DataType::INT32,
                     seq_lens_encoder.place());
  auto decoder_tile_ids_per_batch =
      GetEmptyTensor({bsz * decoder_max_tile_size_per_bs_q},
                     paddle::DataType::INT32,
                     seq_lens_encoder.place());
  auto decoder_num_blocks_x =
      GetEmptyTensor({1}, paddle::DataType::INT32, seq_lens_encoder.place());
  split_q_block<<<1, 32, 0, stream>>>(seq_lens_this_time.data<int>(),
                                      seq_lens_encoder.data<int>(),
                                      decoder_batch_ids.data<int>(),
                                      decoder_tile_ids_per_batch.data<int>(),
                                      decoder_num_blocks_x.data<int>(),
                                      bsz,
                                      decoder_block_shape_q,
                                      group_size);
  auto decoder_num_blocks_x_cpu =
      decoder_num_blocks_x.copy_to(paddle::CPUPlace(), false);
  
  auto max_len_kv =
      GetEmptyTensor({1}, paddle::DataType::INT32, seq_lens_decoder.place());
  get_max_len_kv_ernel<128><<<1, 128, 0, stream>>>(
    max_len_kv.data<int>(),
    seq_lens_this_time.data<int>(),
    seq_lens_decoder.data<int>(),
    bsz
  );
  auto max_len_kv_cpu =
      max_len_kv.copy_to(paddle::CPUPlace(), false);

  int max_enc_len_this_time_data = max_enc_len_this_time.data<int>()[0];
  if (max_enc_len_this_time_data <= 0) {
    auto encoder_batch_ids =
        paddle::full({1}, -1, paddle::DataType::INT32, paddle::GPUPlace());
    auto encoder_tile_ids_per_batch =
        paddle::full({1}, -1, paddle::DataType::INT32, paddle::GPUPlace());
    auto encoder_num_blocks_x_cpu =
        paddle::full({1}, -1, paddle::DataType::INT32, paddle::CPUPlace());
    auto kv_batch_ids =
        paddle::full({1}, -1, paddle::DataType::INT32, paddle::GPUPlace());
    auto kv_tile_ids_per_batch =
        paddle::full({1}, -1, paddle::DataType::INT32, paddle::GPUPlace());
    auto kv_num_blocks_x_cpu =
        paddle::full({1}, -1, paddle::DataType::INT32, paddle::CPUPlace());

    return {encoder_batch_ids,
            encoder_tile_ids_per_batch,
            encoder_num_blocks_x_cpu, /*cpu*/
            kv_batch_ids,
            kv_tile_ids_per_batch,
            kv_num_blocks_x_cpu, /*cpu*/
            decoder_batch_ids,
            decoder_tile_ids_per_batch,
            decoder_num_blocks_x_cpu, /*cpu*/
            max_len_kv_cpu /*cpu*/};
  }

  // encoder
  const uint32_t encoder_max_tile_size_per_bs_q = div_up(
      (max_enc_len_this_time_data * group_size), encoder_block_shape_q);
  auto encoder_batch_ids =
      GetEmptyTensor({bsz * encoder_max_tile_size_per_bs_q},
                     paddle::DataType::INT32,
                     seq_lens_encoder.place());
  auto encoder_tile_ids_per_batch =
      GetEmptyTensor({bsz * encoder_max_tile_size_per_bs_q},
                     paddle::DataType::INT32,
                     seq_lens_encoder.place());
  auto encoder_num_blocks_x =
      GetEmptyTensor({1}, paddle::DataType::INT32, seq_lens_encoder.place());
  split_q_block<<<1, 32, 0, stream>>>(seq_lens_encoder.data<int>(),
                                      nullptr,
                                      encoder_batch_ids.data<int>(),
                                      encoder_tile_ids_per_batch.data<int>(),
                                      encoder_num_blocks_x.data<int>(),
                                      bsz,
                                      encoder_block_shape_q,
                                      group_size);
  auto encoder_num_blocks_x_cpu =
      encoder_num_blocks_x.copy_to(paddle::CPUPlace(), false);

  // kv
  const uint32_t max_tile_size_per_bs_kv =
      div_up(max_enc_len_this_time_data, block_size);
  auto kv_batch_ids = GetEmptyTensor({bsz * max_tile_size_per_bs_kv},
                                     paddle::DataType::INT32,
                                     seq_lens_encoder.place());
  auto kv_tile_ids_per_batch = GetEmptyTensor({bsz * max_tile_size_per_bs_kv},
                                              paddle::DataType::INT32,
                                              seq_lens_encoder.place());
  auto kv_num_blocks_x =
      GetEmptyTensor({1}, paddle::DataType::INT32, seq_lens_encoder.place());
  split_kv_block<<<1, 32, 0, stream>>>(seq_lens_decoder.data<int>(),
                                       seq_lens_encoder.data<int>(),
                                       kv_batch_ids.data<int>(),
                                       kv_tile_ids_per_batch.data<int>(),
                                       kv_num_blocks_x.data<int>(),
                                       bsz,
                                       block_size,
                                       block_size);
  auto kv_num_blocks_x_cpu = kv_num_blocks_x.copy_to(paddle::CPUPlace(), false);
  return {encoder_batch_ids,
          encoder_tile_ids_per_batch,
          encoder_num_blocks_x_cpu, /*cpu*/
          kv_batch_ids,
          kv_tile_ids_per_batch,
          kv_num_blocks_x_cpu, /*cpu*/
          decoder_batch_ids,
          decoder_tile_ids_per_batch,
          decoder_num_blocks_x_cpu, /*cpu*/
          max_len_kv_cpu /*cpu*/};
}

std::vector<paddle::DataType> GetBlockShapeAndSplitKVBlockInferDtype(
    const paddle::DataType& seq_lens_encoder_dtype,
    const paddle::DataType& seq_lens_decoder_dtype,
    const paddle::DataType& max_enc_len_this_time_dtype,
    const paddle::DataType& seq_lens_this_time_dtype,
    const paddle::DataType& cum_offsets_dtype) {
  return {paddle::DataType::INT32,
          paddle::DataType::INT32,
          paddle::DataType::INT32,
          paddle::DataType::INT32,
          paddle::DataType::INT32,
          paddle::DataType::INT32,
          paddle::DataType::INT32,
          paddle::DataType::INT32,
          paddle::DataType::INT32,
          paddle::DataType::INT32};
}

std::vector<std::vector<int64_t>> GetBlockShapeAndSplitKVBlockInferShape(
    const std::vector<int64_t>& seq_lens_encoder_shape,
    const std::vector<int64_t>& seq_lens_decoder_shape,
    const std::vector<int64_t>& max_enc_len_this_time_shape,
    const std::vector<int64_t>& seq_lens_this_time_shape,
    const std::vector<int64_t>& cum_offsets_shape) {
  std::vector<int64_t> dynamic_shape = {-1};

  return {dynamic_shape,
          dynamic_shape,
          {1},
          dynamic_shape,
          dynamic_shape,
          {1},
          dynamic_shape,
          dynamic_shape,
          {1},
          {1}};
}

PD_BUILD_OP(get_block_shape_and_split_kv_block)
    .Inputs({"seq_lens_encoder",
             "seq_lens_decoder",
             "max_enc_len_this_time",
             "seq_lens_this_time",
             "cum_offsets"})
    .Outputs({"encoder_batch_ids",
              "encoder_tile_ids_per_batch",
              "encoder_num_blocks",
              "kv_batch_ids",
              "kv_tile_ids_per_batch",
              "kv_num_blocks",
              "decoder_batch_ids",
              "decoder_tile_ids_per_batch",
              "decoder_num_blocks",
              "max_len_kv"})
    .Attrs({"encoder_block_shape_q: int",
            "decoder_block_shape_q: int",
            "group_size: int",
            "block_size: int",
            "decoder_step_token_num: int"})
    .SetKernelFn(PD_KERNEL(GetBlockShapeAndSplitKVBlock))
    .SetInferShapeFn(PD_INFER_SHAPE(GetBlockShapeAndSplitKVBlockInferShape))
    .SetInferDtypeFn(PD_INFER_DTYPE(GetBlockShapeAndSplitKVBlockInferDtype));
