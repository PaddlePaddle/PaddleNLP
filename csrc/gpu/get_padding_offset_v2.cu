#include "hip/hip_runtime.h"
// Copyright (c) 2024 PaddlePaddle Authors. All Rights Reserved.
// 
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
// 
//     http://www.apache.org/licenses/LICENSE-2.0
// 
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/extension.h"

__global__ void RemovePaddingAndComputeOffsets(
    int64_t *output_data,
    int *padding_offset,
    int *cum_offsets_out,
    int *cu_seqlens_q,
    int *cu_seqlens_k,
    const int64_t *input_data,
    const int *seq_lens,
    const int *cum_offsets,
    const int sequence_length) {
  const int bi = blockIdx.x;  // Batch index
  const int tid = threadIdx.x;

  int cum_offset_prev = (bi == 0) ? 0 : cum_offsets[bi - 1];
  int cum_offset = cum_offsets[bi];

  if (tid == 0) {
    cum_offsets_out[bi] = cum_offset_prev;
    int cum_seq_len = (bi + 1) * sequence_length - cum_offset;
    cu_seqlens_q[bi + 1] = cum_seq_len;
    cu_seqlens_k[bi + 1] = cum_seq_len;
  }

  for (int i = tid; i < seq_lens[bi]; i += blockDim.x) {
    int src_seq_id = bi * sequence_length + i;
    int tgt_seq_id = bi * sequence_length - cum_offset_prev + i;
    output_data[tgt_seq_id] = input_data[src_seq_id];
    padding_offset[tgt_seq_id] = cum_offset_prev;
  }
}

std::vector<paddle::Tensor> GetPaddingOffsetV2(const paddle::Tensor& input_ids,
                                               const paddle::Tensor& cum_offsets,
                                               const paddle::Tensor& token_num,
                                               const paddle::Tensor& seq_len) {
    auto cu_stream = input_ids.stream();
    std::vector<int64_t> input_ids_shape = input_ids.shape();
    const int bsz = seq_len.shape()[0];
    const int sequence_length = input_ids_shape[1];
    auto cum_offsets_out = cum_offsets.copy_to(cum_offsets.place(), false);
    auto cpu_token_num = token_num.copy_to(paddle::CPUPlace(), false);

    const int token_num_data = cpu_token_num.data<int64_t>()[0];
    auto x_remove_padding = paddle::full({token_num_data}, 0, paddle::DataType::INT64, input_ids.place());
    auto padding_offset = paddle::full({token_num_data}, 0, paddle::DataType::INT32, input_ids.place());
    auto cu_seqlens_q = paddle::full({bsz + 1}, 0, paddle::DataType::INT32, input_ids.place());
    auto cu_seqlens_k = paddle::full({bsz + 1}, 0, paddle::DataType::INT32, input_ids.place());

    int blockSize = std::min((token_num_data + 32 - 1) / 32 * 32, 128);
    int gridSize = bsz;

    RemovePaddingAndComputeOffsets<<<gridSize, blockSize, 0, cu_stream>>>(
        x_remove_padding.data<int64_t>(),
        padding_offset.data<int>(),
        cum_offsets_out.data<int>(),
        cu_seqlens_q.data<int>(),
        cu_seqlens_k.data<int>(),
        input_ids.data<int64_t>(),
        seq_len.data<int>(),
        cum_offsets.data<int>(),
        sequence_length);

    return {x_remove_padding, cum_offsets_out, padding_offset, cu_seqlens_q, cu_seqlens_k};
}

std::vector<std::vector<int64_t>> GetPaddingOffsetV2InferShape(const std::vector<int64_t>& input_ids_shape,
                                                             const std::vector<int64_t>& cum_offsets_shape,
                                                             const std::vector<int64_t>& token_num_shape,
                                                             const std::vector<int64_t>& seq_len_shape) {
    int64_t bsz = seq_len_shape[0];
    int64_t seq_len = input_ids_shape[1];
    return {{-1}, {bsz}, {-1}, {bsz + 1}, {bsz + 1}};
}

std::vector<paddle::DataType> GetPaddingOffsetV2InferDtype(const paddle::DataType& input_ids_dtype,
                                                         const paddle::DataType& cum_offsets_dtype,
                                                         const paddle::DataType& token_num_dtype,
                                                         const paddle::DataType& seq_len_dtype) {
    return {input_ids_dtype, seq_len_dtype, seq_len_dtype, seq_len_dtype, seq_len_dtype};
}

PD_BUILD_OP(get_padding_offset_v2)
    .Inputs({"input_ids", "cum_offsets", "token_num", "seq_len"})
    .Outputs({"x_remove_padding", "cum_offsets_out", "padding_offset", "cu_seqlens_q", "cu_seqlens_k"})
    .SetKernelFn(PD_KERNEL(GetPaddingOffsetV2))
    .SetInferShapeFn(PD_INFER_SHAPE(GetPaddingOffsetV2InferShape))
    .SetInferDtypeFn(PD_INFER_DTYPE(GetPaddingOffsetV2InferDtype));