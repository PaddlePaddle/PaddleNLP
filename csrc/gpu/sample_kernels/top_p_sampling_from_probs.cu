// Copyright (c) 2024 PaddlePaddle Authors. All Rights Reserved.
// 
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
// 
//     http://www.apache.org/licenses/LICENSE-2.0
// 
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "sampling.cuh"
#include "helper.h"

std::vector<paddle::Tensor> top_p_sampling_from_probs(const paddle::Tensor& probs,
                                                      const paddle::Tensor& uniform_samples,
                                                      const paddle::Tensor& top_p
                                                      ) {
                                                                                                   
  std::vector<int64_t> probs_shape = probs.shape();
  unsigned int batch_size = probs_shape[0];
  unsigned int vocab_size = probs_shape[1];
  std::vector<int64_t> uniform_samples_shape = uniform_samples.shape();
  PD_CHECK(uniform_samples_shape[0], batch_size);  
  unsigned int max_top_p_rounds = uniform_samples_shape[1];
  // todo: add parameter for deterministic, now default is true
  bool deterministic = true;
  paddle::Tensor probs_input;
  paddle::Tensor uniform_samples_input;

  probs_input = paddle::experimental::cast(probs,paddle::DataType::FLOAT32);
  uniform_samples_input =paddle::experimental::cast(uniform_samples, paddle::DataType::FLOAT32);
  auto cu_stream = probs.stream();

  auto samples = paddle::full({batch_size}, 0, paddle::DataType::INT32, probs.place());
  auto success = paddle::full({batch_size}, 0, paddle::DataType::BOOL, probs.place());

  hipError_t status = sampling::TopPSamplingFromProb<float, int>(
      probs_input.data<float>(), uniform_samples_input.data<float>(),
      samples.data<int>(), success.data<bool>(),
      nullptr, batch_size, top_p.data<float>(),
      vocab_size, max_top_p_rounds, deterministic, cu_stream);
  PD_CHECK(status == hipSuccess, 
        "SamplingFromProbs failed with error code " + std::string(hipGetErrorString(status)));
  paddle::Tensor samples_output;
  samples_output =paddle::experimental::cast(samples, paddle::DataType::INT64);
  return {samples_output};
}

std::vector<std::vector<int64_t>> top_p_sampling_from_probs_InferShape(const std::vector<int64_t>& probs_shape,
                                                                       const std::vector<int64_t>& uniform_samples_shape,
                                                                       const std::vector<int64_t>& top_p_shape
                                                                       ) {
  int64_t bs = probs_shape[0];
  return {{bs, 1}};
}

std::vector<paddle::DataType> top_p_sampling_from_probs_InferDtype(const paddle::DataType& probs_dtype, 
                                                                   const paddle::DataType& uniform_samples_dtype,
                                                                   const paddle::DataType& top_p_shape)
{   
    return {probs_dtype};
}

PD_BUILD_OP(top_p_sampling_from_probs)
    .Inputs({"probs", "uniform_samples", "top_p"})
    .Outputs({"samples"})
    .SetKernelFn(PD_KERNEL(top_p_sampling_from_probs))
    .SetInferShapeFn(PD_INFER_SHAPE(top_p_sampling_from_probs_InferShape))
    .SetInferDtypeFn(PD_INFER_DTYPE(top_p_sampling_from_probs_InferDtype));




