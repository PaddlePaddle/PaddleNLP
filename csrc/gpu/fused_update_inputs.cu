#include "hip/hip_runtime.h"
// Copyright (c) 2024 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
#include "helper.h"

__device__ bool is_in_end(const int64_t id,
                          const int64_t *end_ids,
                          int length) {
  for (int i = 0; i < length; i++) {
    if (id == end_ids[i]) {
      return true;
    }
  }
  return false;
}

template <int THREADBLOCK_SIZE>
__global__ void fused_update_inputs_kernel(bool *not_need_stop,
                                           int *seq_lens_this_time,
                                           int *seq_lens_encoder,
                                           int *seq_lens_decoder,
                                           int64_t *input_ids,
                                           const int64_t *stop_nums,
                                           bool *stop_flags,
                                           const bool *is_block_step,
                                           int64_t *next_tokens,
                                           int64_t *topk_ids,
                                           const int64_t *end_ids,
                                           const int bsz,
                                           const int max_bsz,
                                           const int input_ids_stride,
                                           const int end_length) {
  int thread_idx = threadIdx.x;
  typedef hipcub::BlockReduce<int64_t, THREADBLOCK_SIZE> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;

  bool stop_flag_now = false;
  int64_t stop_flag_now_int = 0;

  if (thread_idx < max_bsz) {
    if (thread_idx < bsz) {
      // Begin merging set_value_by_flags_v2 logic
      if (stop_flags[thread_idx]) {
        if (seq_lens_decoder[thread_idx] == 0) {
          topk_ids[thread_idx] = -1;
        } else {
          topk_ids[thread_idx] = end_ids[0];
          next_tokens[thread_idx] = end_ids[0];
        }
      } else {
        next_tokens[thread_idx] = topk_ids[thread_idx];
      }

      if (is_in_end(topk_ids[thread_idx], end_ids, end_length)) {
        stop_flags[thread_idx] = true;
      }

      // Continue with update_inputs_kernel logic
      stop_flag_now = stop_flags[thread_idx];

      if (is_block_step[thread_idx]) {
        stop_flag_now_int = 0;
      } else {
        stop_flag_now_int = static_cast<int64_t>(stop_flag_now);
      }
    } else {
      stop_flag_now_int = 1;
    }
  }

  __syncthreads();

  if (thread_idx < bsz) {
    const int seq_len_this_time = seq_lens_this_time[thread_idx];
    const int seq_len_encoder = seq_lens_encoder[thread_idx];
    const int seq_len_decoder = seq_lens_decoder[thread_idx];

    // seq_lens_decoder[thread_idx] = stop_flag_now
    //     ? 0
    //     : (seq_len_decoder == 0 ? seq_len_encoder : seq_len_decoder + 1);
    seq_lens_decoder[thread_idx] =
        stop_flag_now
            ? 0
            : (seq_len_encoder > 0 ? (seq_len_encoder + seq_len_decoder)
                                   : seq_len_decoder + 1);

    seq_lens_this_time[thread_idx] = stop_flag_now ? 0 : 1;
    seq_lens_encoder[thread_idx] = 0;

    int64_t *input_ids_now = input_ids + thread_idx * input_ids_stride;
    input_ids_now[0] = next_tokens[thread_idx];
  }

  __syncthreads();

  int64_t stop_sum = BlockReduce(temp_storage).Sum(stop_flag_now_int);
  if (thread_idx == 0) {
    not_need_stop[0] = stop_sum < stop_nums[0];
  }
}

void FusedUpdateInputs(const paddle::Tensor &stop_flags,
                       const paddle::Tensor &not_need_stop,
                       const paddle::Tensor &seq_lens_this_time,
                       const paddle::Tensor &seq_lens_encoder,
                       const paddle::Tensor &seq_lens_decoder,
                       const paddle::Tensor &input_ids,
                       const paddle::Tensor &stop_nums,
                       const paddle::Tensor &next_tokens,
                       const paddle::Tensor &is_block_step,
                       const paddle::Tensor &topk_ids,
                       const paddle::Tensor &end_ids) {
  const int max_bsz = stop_flags.shape()[0];
  const int now_bsz = seq_lens_this_time.shape()[0];
  const int input_ids_stride = input_ids.shape()[1];
  const int64_t end_length = end_ids.shape()[0];

  int threads_per_block = 1024;
  int blocks_per_grid = (max_bsz + threads_per_block - 1) / threads_per_block;

  fused_update_inputs_kernel<1024>
      <<<blocks_per_grid, threads_per_block, 0, input_ids.stream()>>>(
          const_cast<bool *>(not_need_stop.data<bool>()),
          const_cast<int *>(seq_lens_this_time.data<int>()),
          const_cast<int *>(seq_lens_encoder.data<int>()),
          const_cast<int *>(seq_lens_decoder.data<int>()),
          const_cast<int64_t *>(input_ids.data<int64_t>()),
          stop_nums.data<int64_t>(),
          const_cast<bool *>(stop_flags.data<bool>()),
          is_block_step.data<bool>(),
          const_cast<int64_t *>(next_tokens.data<int64_t>()),
          const_cast<int64_t *>(topk_ids.data<int64_t>()),
          end_ids.data<int64_t>(),
          now_bsz,
          max_bsz,
          input_ids_stride,
          end_length);
}

PD_BUILD_OP(fused_update_inputs)
    .Inputs({"stop_flags",
             "not_need_stop",
             "seq_lens_this_time",
             "seq_lens_encoder",
             "seq_lens_decoder",
             "input_ids",
             "stop_nums",
             "next_tokens",
             "is_block_step",
             "topk_ids",
             "end_ids"})
    .Outputs({"not_need_stop_out",
              "seq_lens_this_time_out",
              "seq_lens_encoder_out",
              "seq_lens_decoder_out",
              "input_ids_out",
              "stop_flags_out",
              "next_tokens_out",
              "topk_ids_out"})
    .SetInplaceMap({{"not_need_stop", "not_need_stop_out"},
                    {"seq_lens_this_time", "seq_lens_this_time_out"},
                    {"seq_lens_encoder", "seq_lens_encoder_out"},
                    {"seq_lens_decoder", "seq_lens_decoder_out"},
                    {"input_ids", "input_ids_out"},
                    {"stop_flags", "stop_flags_out"},
                    {"next_tokens", "next_tokens_out"},
                    {"topk_ids", "topk_ids_out"}})
    .SetKernelFn(PD_KERNEL(FusedUpdateInputs));
