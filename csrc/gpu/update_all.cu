#include "hip/hip_runtime.h"
// Copyright (c) 2024 PaddlePaddle Authors. All Rights Reserved.
// 
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
// 
//     http://www.apache.org/licenses/LICENSE-2.0
// 
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "helper.h"

__device__ bool is_in_end_update_all(const int64_t id, const int64_t *end_ids, int length) {
    bool flag = false;
    for (int i = 0; i < length; i++) {
        if (id == end_ids[i]) {
            return true;
        }
    }
    return flag;
}

template <int THREADBLOCK_SIZE>
__global__ void update_all_kernel(
    bool *not_need_stop,
    int64_t *step_idx,
    bool *stop_flags,
    int *seq_lens_this_time,
    int *seq_lens_encoder,
    int *seq_lens_decoder,
    int64_t *next_tokens,
    int64_t *kwargs_next_tokens,
    int64_t *input_ids,
    const int64_t *end_ids, 
    const int64_t *stop_nums,
    const bool *is_block_step,
    const int64_t *max_dec_len,
    const int bsz,
    const int max_bsz,
    const int input_ids_stride,
    const int end_length) {
  int thread_idx = threadIdx.x;
  // update step_idx and stop_flags
  if (thread_idx < max_bsz) {
    bool stop_flag = stop_flags[thread_idx];
    if (!stop_flag) {
      step_idx[thread_idx] += 1;
    }
    if (step_idx[thread_idx] >= max_dec_len[thread_idx]) {
      stop_flags[thread_idx] = true;
    }
  }
  // update inputs
  if (thread_idx < bsz) {
    if (stop_flags[thread_idx]) {
      if (seq_lens_this_time[thread_idx] == 0) {
        next_tokens[thread_idx] = -1;
      } else {
        next_tokens[thread_idx] = end_ids[0];
        kwargs_next_tokens[thread_idx] = end_ids[0];
      }
    } else {
      kwargs_next_tokens[thread_idx] = next_tokens[thread_idx];
    }
    if (is_in_end_update_all(next_tokens[thread_idx], end_ids, end_length)) {
      stop_flags[thread_idx] = true;
    }
  }

  typedef hipcub::BlockReduce<int64_t, THREADBLOCK_SIZE> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;

  bool stop_flag_now = false;
  int64_t stop_flag_now_int = 0;
  if (thread_idx < max_bsz) {
    if (thread_idx < bsz) {
      stop_flag_now = stop_flags[thread_idx];
      if (is_block_step[thread_idx]) {
        stop_flag_now_int=0;
      } else {
        stop_flag_now_int = static_cast<int64_t>(stop_flag_now);
      }
    } else {
      stop_flag_now_int = 1;
    }
  }
  if (thread_idx < bsz) {
    const int seq_len_this_time = seq_lens_this_time[thread_idx];
    const int seq_len_encoder = seq_lens_encoder[thread_idx];
    const int seq_len_decoder = seq_lens_decoder[thread_idx];

    seq_lens_decoder[thread_idx] = stop_flag_now ? 0 : (seq_len_encoder > 0 ? (seq_len_encoder + seq_len_decoder) : seq_len_decoder + 1);

    seq_lens_this_time[thread_idx] = stop_flag_now ? 0 : 1;
    seq_lens_encoder[thread_idx] = 0;
    int64_t *input_ids_now = input_ids + thread_idx * input_ids_stride;
    input_ids_now[0] = next_tokens[thread_idx];
  }
  __syncthreads();
  int64_t stop_sum = BlockReduce(temp_storage).Sum(stop_flag_now_int);
  if (thread_idx == 0) {
    not_need_stop[0] = stop_sum < stop_nums[0];
  }
}

void UpdateAll(const paddle::Tensor& stop_flags,
               const paddle::Tensor& step_idx,
               const paddle::Tensor& not_need_stop, // cpu
               const paddle::Tensor& seq_lens_this_time,
               const paddle::Tensor& seq_lens_encoder,
               const paddle::Tensor& seq_lens_decoder,
               const paddle::Tensor& max_dec_len,
               const paddle::Tensor& input_ids,
               const paddle::Tensor& stop_nums,
               const paddle::Tensor& next_tokens,
               const paddle::Tensor& is_block_step,
               const paddle::Tensor& end_ids,
               const paddle::Tensor& kwargs_next_tokens) {
  const int max_bsz = stop_flags.shape()[0];
  const int now_bsz = seq_lens_this_time.shape()[0];
  const int input_ids_stride = input_ids.shape()[1];
  const int end_length = end_ids.shape()[0];
  auto not_need_stop_gpu = not_need_stop.copy_to(stop_flags.place(), false);
  update_all_kernel<1024><<<1, 1024, 0, input_ids.stream()>>>(
    const_cast<bool*>(not_need_stop_gpu.data<bool>()),
    const_cast<int64_t*>(step_idx.data<int64_t>()),
    const_cast<bool*>(stop_flags.data<bool>()),
    const_cast<int*>(seq_lens_this_time.data<int>()),
    const_cast<int*>(seq_lens_encoder.data<int>()),
    const_cast<int*>(seq_lens_decoder.data<int>()),
    const_cast<int64_t*>(next_tokens.data<int64_t>()),
    const_cast<int64_t*>(kwargs_next_tokens.data<int64_t>()),
    const_cast<int64_t*>(input_ids.data<int64_t>()),
    end_ids.data<int64_t>(),
    stop_nums.data<int64_t>(),
    is_block_step.data<bool>(),
    max_dec_len.data<int64_t>(),
    now_bsz,
    max_bsz,
    input_ids_stride,
    end_length
  );
  auto not_need_stop_cpu = not_need_stop_gpu.copy_to(not_need_stop.place(), false);
  bool *not_need_stop_data = const_cast<bool*>(not_need_stop.data<bool>());
  not_need_stop_data[0] = not_need_stop_cpu.data<bool>()[0];
}

PD_BUILD_OP(update_all)
    .Inputs({"stop_flags", 
             "step_idx",
             "not_need_stop", 
             "seq_lens_this_time", 
             "seq_lens_encoder", 
             "seq_lens_decoder",
             "max_dec_len",
             "input_ids",
             "stop_nums",
             "next_tokens",
             "is_block_step",
             "end_ids",
             "kwargs_next_tokens"})
    .Outputs({"not_need_stop_out",
              "seq_lens_this_time_out",
              "seq_lens_encoder_out",
              "seq_lens_decoder_out",
              "input_ids_out",
              "next_tokens_out",
              "kwargs_next_tokens_out",
              "step_idx_out"})
    .SetInplaceMap({{"not_need_stop", "not_need_stop_out"},
                    {"seq_lens_this_time", "seq_lens_this_time_out"},
                    {"seq_lens_encoder", "seq_lens_encoder_out"},
                    {"seq_lens_decoder", "seq_lens_decoder_out"},
                    {"input_ids", "input_ids_out"},
                    {"next_tokens", "next_tokens_out"},
                    {"kwargs_next_tokens", "kwargs_next_tokens_out"},
                    {"step_idx", "step_idx_out"}})
    .SetKernelFn(PD_KERNEL(UpdateAll));