#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <algorithm>
#include <iterator>
#include <random>
#include <sstream>
#include <vector>

// TODO(guosheng): `HOST` conflict exists in float.h of paddle and mpi.h of mpi
#include "fusion_gpt_op.h"
#include "pd_traits.h"
#ifdef HOST
#undef HOST
#endif
#include "fastertransformer/cuda/hipcub/hipcub.hpp"
#include "fastertransformer/gpt.h"
#include "fastertransformer/utils/common.h"

#ifdef BUILD_GPT  // consistent with FasterTransformer
#include <map>
#include <memory>
#include <mutex>

static std::mutex mpi_global_mutex;
static std::once_flag once_flag_init_mpi;

void MPIExit() {
  std::unique_lock<std::mutex> global_lock(mpi_global_mutex);
  MPICHECK(MPI_Finalize());
}

void InitMPIOnce() {
  // Initialize MPI environment
  std::call_once(once_flag_init_mpi, []() {
    MPICHECK(MPI_Init(nullptr, nullptr));
    if (std::atexit(MPIExit)) {
      throw std::runtime_error("Fail to register the MPI exit handler");
    }
  });
}

void InitNCCLComm(ncclUniqueId& tensor_para_nccl_uid,
                  ncclUniqueId& layer_para_nccl_uid,
                  ncclComm_t& tensor_para_nccl_comm,
                  ncclComm_t& layer_para_nccl_comm,
                  int rank,
                  int tensor_para_size,
                  int layer_para_size,
                  int tensor_para_rank,
                  int layer_para_rank) {
  // assume gpu_num = n * k,
  // tensor parallelism group size is n
  // layer parallelism group size is k

  if (tensor_para_rank == 0) {
    // get the uid of each tensor parallelism group
    // here, 0, 1, ..., n-1 are in group 0,
    //       n, ..., 2n - 1 are in group 1.
    NCCLCHECK(ncclGetUniqueId(&tensor_para_nccl_uid));
    for (int i = 1; i < tensor_para_size; i++) {
      printf("[INFO] rank %d sends tensor_para_nccl_uid to rank %d \n",
             rank,
             rank + i);
      MPICHECK(MPI_Send(&tensor_para_nccl_uid,
                        sizeof(tensor_para_nccl_uid),
                        MPI_BYTE,
                        rank + i,
                        0,
                        MPI_COMM_WORLD));
    }
  } else {
    MPI_Status status;
    printf("[INFO] rank %d receives tensor_para_nccl_uid from rank %d \n",
           rank,
           rank - tensor_para_rank);
    MPICHECK(MPI_Recv(&tensor_para_nccl_uid,
                      sizeof(tensor_para_nccl_uid),
                      MPI_BYTE,
                      rank - tensor_para_rank,
                      0,
                      MPI_COMM_WORLD,
                      &status));
  }

  if (layer_para_rank == 0) {
    // get the uid of each layer parallelism group
    // 0, k, 2k, are in group 0
    // 1, k+1, 2k+1 are in group 1
    NCCLCHECK(ncclGetUniqueId(&layer_para_nccl_uid));
    for (int i = 1; i < layer_para_size; i++) {
      printf("[INFO] rank %d sends layer_para_nccl_uid to rank %d \n",
             rank,
             rank + i * tensor_para_size);
      MPICHECK(MPI_Send(&layer_para_nccl_uid,
                        sizeof(layer_para_nccl_uid),
                        MPI_BYTE,
                        rank + i * tensor_para_size,
                        0,
                        MPI_COMM_WORLD));
    }
  } else {
    MPI_Status status;
    printf("[INFO] rank %d receives layer_para_nccl_uid from rank %d \n",
           rank,
           rank % tensor_para_size);
    MPICHECK(MPI_Recv(&layer_para_nccl_uid,
                      sizeof(layer_para_nccl_uid),
                      MPI_BYTE,
                      rank % tensor_para_size,
                      0,
                      MPI_COMM_WORLD,
                      &status));
  }

  NCCLCHECK(ncclCommInitRank(&tensor_para_nccl_comm,
                             tensor_para_size,
                             tensor_para_nccl_uid,
                             tensor_para_rank));
  NCCLCHECK(ncclCommInitRank(&layer_para_nccl_comm,
                             layer_para_size,
                             layer_para_nccl_uid,
                             layer_para_rank));
}

struct ModelParaDesc {
  TensorParallelParam tensor_parallel_param;
  LayerParallelParam layer_parallel_param;
  ncclComm_t tensor_para_nccl_comm, layer_para_nccl_comm;
  std::mt19937_64 gen;
  std::uniform_int_distribution<> dist{0, std::numeric_limits<int>::max()};


  ModelParaDesc(int head_num,
                int size_per_head,
                int layer_num,
                int tensor_para_size,
                int layer_para_size,
                int layer_para_batch_size) {
    int rank;
    MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &rank));
    const int local_head_num = head_num / tensor_para_size;
    const int local_hidden_units = local_head_num * size_per_head;
    const int layers_per_group = layer_num / layer_para_size;
    assert(layer_num % layer_para_size == 0);
    const int tensor_para_rank = rank % tensor_para_size;
    const int layer_para_rank = rank / tensor_para_size;
    ncclUniqueId tensor_para_nccl_uid, layer_para_nccl_uid;
    InitNCCLComm(tensor_para_nccl_uid,
                 layer_para_nccl_uid,
                 tensor_para_nccl_comm,
                 layer_para_nccl_comm,
                 rank,
                 tensor_para_size,
                 layer_para_size,
                 tensor_para_rank,
                 layer_para_rank);
    tensor_parallel_param.rank = tensor_para_rank;
    tensor_parallel_param.world_size = tensor_para_size;
    tensor_parallel_param.local_head_num_ = local_head_num;
    tensor_parallel_param.local_hidden_units_ = local_hidden_units;
    tensor_parallel_param.nccl_comm = tensor_para_nccl_comm;
    layer_parallel_param.rank = layer_para_rank;
    layer_parallel_param.world_size = layer_para_size;
    layer_parallel_param.layers_per_group = layers_per_group;
    layer_parallel_param.local_batch_size = layer_para_batch_size;
    layer_parallel_param.nccl_comm = layer_para_nccl_comm;
    // fix the seed to prevent the seed of different gpu are differnet in Tensor
    // Parallel
    size_t meta_seed =
        *(reinterpret_cast<size_t*>(tensor_para_nccl_uid.internal));
    gen = std::mt19937_64(meta_seed);
  }

  ~ModelParaDesc() {
    if (tensor_para_nccl_comm) ncclCommDestroy(tensor_para_nccl_comm);
    if (layer_para_nccl_comm) ncclCommDestroy(layer_para_nccl_comm);
  }
};

// Make model parallel settings init only once for one model by using a global
// dict mapping parameters representing different models to corresponding
// settings. Note: `paddle::Tensor` for custom_op is re-created every step and
// we use pointers as keys. Maybe using weakref as keys is better.
static std::unordered_map<void*, std::unique_ptr<ModelParaDesc>>
    model_para_infos;
struct ModelParaDescFactory {
  static ModelParaDesc* CreateModelParaDesc(int head_num,
                                            int size_per_head,
                                            int layer_num,
                                            int tensor_para_size,
                                            int layer_para_size,
                                            int layer_para_batch_size,
                                            void* param_ptr = nullptr) {
    InitMPIOnce();
    auto it = model_para_infos.find(param_ptr);
    if (it != model_para_infos.end()) {
      return it->second.get();
    } else {
      model_para_infos.emplace(param_ptr,
                               std::unique_ptr<ModelParaDesc>(
                                   new ModelParaDesc(head_num,
                                                     size_per_head,
                                                     layer_num,
                                                     tensor_para_size,
                                                     layer_para_size,
                                                     layer_para_batch_size)));
      return model_para_infos[param_ptr].get();
    }
  }
};
#endif

template <paddle::DataType D>
std::vector<paddle::Tensor> gpt2_kernel(
    const paddle::Tensor& input,
    const paddle::Tensor& attn_mask,
    const paddle::Tensor& start_length,
    const paddle::Tensor& word_emb,
    const std::vector<paddle::Tensor>& self_ln_weight,
    const std::vector<paddle::Tensor>& self_ln_bias,
    const std::vector<paddle::Tensor>& self_q_weight,
    const std::vector<paddle::Tensor>& self_q_bias,
    const std::vector<paddle::Tensor>& self_k_weight,
    const std::vector<paddle::Tensor>& self_k_bias,
    const std::vector<paddle::Tensor>& self_v_weight,
    const std::vector<paddle::Tensor>& self_v_bias,
    const std::vector<paddle::Tensor>& self_out_weight,
    const std::vector<paddle::Tensor>& self_out_bias,
    const std::vector<paddle::Tensor>& ffn_ln_weight,
    const std::vector<paddle::Tensor>& ffn_ln_bias,
    const std::vector<paddle::Tensor>& ffn_inter_weight,
    const std::vector<paddle::Tensor>& ffn_inter_bias,
    const std::vector<paddle::Tensor>& ffn_out_weight,
    const std::vector<paddle::Tensor>& ffn_out_bias,
    const paddle::Tensor& decoder_ln_weight,
    const paddle::Tensor& decoder_ln_bias,
    const paddle::Tensor& positional_embedding_weight,
    const paddle::Tensor& emb_weight,
    paddle::Tensor& output_ids,
    const int& topk,
    const float& topp,
    const int& max_len,
    const int& n_head,
    const int& size_per_head,
    const int& num_layer,
    const int& bos_id,
    const int& eos_id,
    const float& temperature,
    hipblasHandle_t cublas_handle_,
    hipblasLtHandle_t cublaslt_handle_,
    hipStream_t stream,
    const int& tensor_para_size = 1,
    const int& layer_para_size = 1,
    const int& layer_para_batch_size = 1) {
  auto input_dims = input.shape();
  int batch_size_ = input_dims[0];
  int start_len = input_dims[1];
  const int vocab_size = word_emb.shape()[0];

  typedef PDTraits<D> traits_;
  typedef typename traits_::DataType DataType_;
  typedef typename traits_::data_t data_t_;

  DecodingInitParam<DataType_> decoding_params;
  decoding_params.cublas_handle = cublas_handle_;
  decoding_params.cublaslt_handle = cublaslt_handle_;

  decoding_params.output_ids = output_ids.mutable_data<int>(word_emb.place());

  typedef DecoderTransformerTraits<traits_::OpType> DecodingTraits_;
  decoding_params.stream = stream;
  fastertransformer::Allocator<AllocatorType::PD> allocator_(stream);

  const int hidden_unit = size_per_head * n_head;

#ifdef BUILD_GPT
  auto* model_para_desc = ModelParaDescFactory::CreateModelParaDesc(
      n_head,
      size_per_head,
      num_layer,
      tensor_para_size,
      layer_para_size,
      layer_para_batch_size,
      const_cast<data_t_*>(word_emb.data<data_t_>()));
  auto& tensor_parallel_param = model_para_desc->tensor_parallel_param;
  auto& layer_parallel_param = model_para_desc->layer_parallel_param;
  auto seed = model_para_desc->dist(model_para_desc->gen);
#else
  TensorParallelParam tensor_parallel_param;
  LayerParallelParam layer_parallel_param;
  tensor_parallel_param.rank = 0;
  tensor_parallel_param.world_size = 1;
  tensor_parallel_param.local_head_num_ = n_head;
  tensor_parallel_param.local_hidden_units_ = hidden_unit;

  layer_parallel_param.rank = 0;
  layer_parallel_param.world_size = 1;
  layer_parallel_param.layers_per_group = num_layer;
  layer_parallel_param.local_batch_size = batch_size_;
  int seed = -1;
#endif

  DecodingGpt<DecodingTraits_::OpType>* gpt_decoding;

  decoding_params.request_batch_size = batch_size_;
  decoding_params.max_input_len = start_len;
  decoding_params.request_input_len = start_len;
  decoding_params.request_output_len = max_len - start_len;

  decoding_params.d_start_ids = const_cast<int *>(input.data<int>());
  decoding_params.d_attn_mask =
      reinterpret_cast<DataType_*>(const_cast<data_t_ *>(attn_mask.data<data_t_>()));
  decoding_params.d_start_lengths = start_length.data<int>();

  gpt_decoding =
      new DecodingGpt<DecodingTraits_::OpType>(allocator_,
                                               batch_size_,
                                               max_len,
                                               n_head,
                                               size_per_head,
                                               vocab_size,
                                               num_layer,
                                               bos_id,
                                               eos_id,
                                               topk,
                                               topp,
                                               temperature,
                                               tensor_para_size,
                                               layer_para_size,
                                               true, /*is_fuse_QKV*/
                                               1.0,  /*repetition_penalty*/
                                               seed);

  gpt_decoding->set_tensor_parallel_param(tensor_parallel_param);
  gpt_decoding->set_layer_parallel_param(layer_parallel_param);

  DecoderInitParam<DataType_>* params =
      new DecoderInitParam<DataType_>[num_layer];

  for (int i = 0; i < self_ln_weight.size(); ++i) {
    // Allow python passing weights of all layers or only passing the
    // corresponding layers to save memory.
    int layer_idx = self_ln_weight.size() != num_layer
                        ? layer_parallel_param.rank *
                                  layer_parallel_param.layers_per_group +
                              i
                        : i;

    params[layer_idx].stream = stream;
    params[layer_idx].cublas_handle = cublas_handle_;
    params[layer_idx].cublaslt_handle = cublaslt_handle_;

    params[layer_idx].request_batch_size = batch_size_;
    params[layer_idx].request_max_mem_seq_len = start_len;

    params[layer_idx].self_layernorm.gamma =
        reinterpret_cast<const DataType_*>(self_ln_weight[i].data<data_t_>());
    params[layer_idx].self_layernorm.beta =
        reinterpret_cast<const DataType_*>(self_ln_bias[i].data<data_t_>());

    params[layer_idx].self_attention.query_weight.kernel =
        reinterpret_cast<const DataType_*>(self_q_weight[i].data<data_t_>());
    params[layer_idx].self_attention.query_weight.bias =
        reinterpret_cast<const DataType_*>(self_q_bias[i].data<data_t_>());
    // For `is_fuse_QKV == true`, ignore weight and bias of key and value to
    // remove requirements on python passing weights to save memory.
    // params[layer_idx].self_attention.key_weight.kernel =
    //     reinterpret_cast<const DataType_*>(self_k_weight[i].data<data_t_>());
    // params[layer_idx].self_attention.key_weight.bias =
    //     reinterpret_cast<const DataType_*>(self_k_bias[i].data<data_t_>());
    // params[layer_idx].self_attention.value_weight.kernel =
    //     reinterpret_cast<const DataType_*>(self_v_weight[i].data<data_t_>());
    // params[layer_idx].self_attention.value_weight.bias =
    //     reinterpret_cast<const DataType_*>(self_v_bias[i].data<data_t_>());

    params[layer_idx].self_attention.attention_output_weight.kernel =
        reinterpret_cast<const DataType_*>(self_out_weight[i].data<data_t_>());
    params[layer_idx].self_attention.attention_output_weight.bias =
        reinterpret_cast<const DataType_*>(self_out_bias[i].data<data_t_>());

    params[layer_idx].ffn_layernorm.gamma =
        reinterpret_cast<const DataType_*>(ffn_ln_weight[i].data<data_t_>());
    params[layer_idx].ffn_layernorm.beta =
        reinterpret_cast<const DataType_*>(ffn_ln_bias[i].data<data_t_>());

    params[layer_idx].ffn.intermediate_weight.kernel =
        reinterpret_cast<const DataType_*>(ffn_inter_weight[i].data<data_t_>());
    params[layer_idx].ffn.intermediate_weight.bias =
        reinterpret_cast<const DataType_*>(ffn_inter_bias[i].data<data_t_>());
    params[layer_idx].ffn.output_weight.kernel =
        reinterpret_cast<const DataType_*>(ffn_out_weight[i].data<data_t_>());
    params[layer_idx].ffn.output_weight.bias =
        reinterpret_cast<const DataType_*>(ffn_out_bias[i].data<data_t_>());
  }

  decoding_params.layernorm.gamma =
      reinterpret_cast<const DataType_*>(decoder_ln_weight.data<data_t_>());
  decoding_params.layernorm.beta =
      reinterpret_cast<const DataType_*>(decoder_ln_bias.data<data_t_>());
  decoding_params.embedding_table =
      reinterpret_cast<const DataType_*>(word_emb.data<data_t_>());
  decoding_params.embedding_kernel =
      reinterpret_cast<const DataType_*>(emb_weight.data<data_t_>());
  decoding_params.position_encoding_table = reinterpret_cast<const DataType_*>(
      positional_embedding_weight.data<data_t_>());

  gpt_decoding->forward_context(params, decoding_params);
  gpt_decoding->forward(params, decoding_params);

  delete gpt_decoding;
  delete[] params;

  return {output_ids};
}

std::vector<paddle::Tensor> GPT2CUDAForward(
    const paddle::Tensor& input,
    const paddle::Tensor& attn_mask,
    const paddle::Tensor& start_length,
    const paddle::Tensor& word_embedding,
    const std::vector<paddle::Tensor>& self_ln_weight,
    const std::vector<paddle::Tensor>& self_ln_bias,
    const std::vector<paddle::Tensor>& self_q_weight,
    const std::vector<paddle::Tensor>& self_q_bias,
    const std::vector<paddle::Tensor>& self_k_weight,
    const std::vector<paddle::Tensor>& self_k_bias,
    const std::vector<paddle::Tensor>& self_v_weight,
    const std::vector<paddle::Tensor>& self_v_bias,
    const std::vector<paddle::Tensor>& self_out_weight,
    const std::vector<paddle::Tensor>& self_out_bias,
    const std::vector<paddle::Tensor>& ffn_ln_weight,
    const std::vector<paddle::Tensor>& ffn_ln_bias,
    const std::vector<paddle::Tensor>& ffn_inter_weight,
    const std::vector<paddle::Tensor>& ffn_inter_bias,
    const std::vector<paddle::Tensor>& ffn_out_weight,
    const std::vector<paddle::Tensor>& ffn_out_bias,
    const paddle::Tensor& decoder_ln_weight,
    const paddle::Tensor& decoder_ln_bias,
    const paddle::Tensor& positional_embedding_weight,
    const paddle::Tensor& emb_weight,
    paddle::Tensor& output_ids,
    const int& topk,
    const float& topp,
    const int& max_len,
    const int& n_head,
    const int& size_per_head,
    const int& num_layer,
    const int& bos_id,
    const int& eos_id,
    const float& temperature,
    const bool& use_fp16 = false,
    const int& tensor_para_size = 1,
    const int& layer_para_size = 1,
    const int& layer_para_batch_size = 1) {
  auto stream = word_embedding.stream();
  // TODO(guosheng): use the global cublas handle
  hipblasHandle_t cublas_handle_;
  hipblasCreate(&cublas_handle_);
  hipblasLtHandle_t cublaslt_handle_;
  hipblasLtCreate(&cublaslt_handle_);
  hipblasSetStream(cublas_handle_, stream);

  std::vector<paddle::Tensor> ret;

  if (use_fp16) {
    ret = gpt2_kernel<paddle::DataType::FLOAT16>(input,
                                                 attn_mask,
                                                 start_length,
                                                 word_embedding,
                                                 self_ln_weight,
                                                 self_ln_bias,
                                                 self_q_weight,
                                                 self_q_bias,
                                                 self_k_weight,
                                                 self_k_bias,
                                                 self_v_weight,
                                                 self_v_bias,
                                                 self_out_weight,
                                                 self_out_bias,
                                                 ffn_ln_weight,
                                                 ffn_ln_bias,
                                                 ffn_inter_weight,
                                                 ffn_inter_bias,
                                                 ffn_out_weight,
                                                 ffn_out_bias,
                                                 decoder_ln_weight,
                                                 decoder_ln_bias,
                                                 positional_embedding_weight,
                                                 emb_weight,
                                                 output_ids,
                                                 topk,
                                                 topp,
                                                 max_len,
                                                 n_head,
                                                 size_per_head,
                                                 num_layer,
                                                 bos_id,
                                                 eos_id,
                                                 temperature,
                                                 cublas_handle_,
                                                 cublaslt_handle_,
                                                 stream,
                                                 tensor_para_size,
                                                 layer_para_size,
                                                 layer_para_batch_size);
  } else {
    ret = gpt2_kernel<paddle::DataType::FLOAT32>(input,
                                                 attn_mask,
                                                 start_length,
                                                 word_embedding,
                                                 self_ln_weight,
                                                 self_ln_bias,
                                                 self_q_weight,
                                                 self_q_bias,
                                                 self_k_weight,
                                                 self_k_bias,
                                                 self_v_weight,
                                                 self_v_bias,
                                                 self_out_weight,
                                                 self_out_bias,
                                                 ffn_ln_weight,
                                                 ffn_ln_bias,
                                                 ffn_inter_weight,
                                                 ffn_inter_bias,
                                                 ffn_out_weight,
                                                 ffn_out_bias,
                                                 decoder_ln_weight,
                                                 decoder_ln_bias,
                                                 positional_embedding_weight,
                                                 emb_weight,
                                                 output_ids,
                                                 topk,
                                                 topp,
                                                 max_len,
                                                 n_head,
                                                 size_per_head,
                                                 num_layer,
                                                 bos_id,
                                                 eos_id,
                                                 temperature,
                                                 cublas_handle_,
                                                 cublaslt_handle_,
                                                 stream,
                                                 tensor_para_size,
                                                 layer_para_size,
                                                 layer_para_batch_size);
  }

  hipblasDestroy(cublas_handle_);
  hipblasLtDestroy(cublaslt_handle_);
  return ret;
}
