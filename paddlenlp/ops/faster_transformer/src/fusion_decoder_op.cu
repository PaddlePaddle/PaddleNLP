/* Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <algorithm>
#include <iterator>
#include <random>
#include <sstream>
#include <vector>

#include "fastertransformer/cuda/hipcub/hipcub.hpp"
#include "fusion_decoder_op.h"
#include "pd_traits.h"


template <paddle::DataType D>
std::vector<paddle::Tensor> decoder_kernel(
    const paddle::Tensor& from_tensor_input,
    const paddle::Tensor& memory_tensor_input,
    const paddle::Tensor& mem_seq_len_input,
    const paddle::Tensor& self_ln_weight,
    const paddle::Tensor& self_ln_bias,
    const paddle::Tensor& self_q_weight,
    const paddle::Tensor& self_q_bias,
    const paddle::Tensor& self_k_weight,
    const paddle::Tensor& self_k_bias,
    const paddle::Tensor& self_v_weight,
    const paddle::Tensor& self_v_bias,
    const paddle::Tensor& self_out_weight,
    const paddle::Tensor& self_out_bias,
    const paddle::Tensor& cross_ln_weight,
    const paddle::Tensor& cross_ln_bias,
    const paddle::Tensor& cross_q_weight,
    const paddle::Tensor& cross_q_bias,
    const paddle::Tensor& cross_k_weight,
    const paddle::Tensor& cross_k_bias,
    const paddle::Tensor& cross_v_weight,
    const paddle::Tensor& cross_v_bias,
    const paddle::Tensor& cross_out_weight,
    const paddle::Tensor& cross_out_bias,
    const paddle::Tensor& ffn_ln_weight,
    const paddle::Tensor& ffn_ln_bias,
    const paddle::Tensor& ffn_inter_weight,
    const paddle::Tensor& ffn_inter_bias,
    const paddle::Tensor& ffn_out_weight,
    const paddle::Tensor& ffn_out_bias,
    const paddle::Tensor& old_self_cache,
    const paddle::Tensor& old_mem_cache,
    paddle::Tensor& decoder_output_tensor,
    paddle::Tensor& new_self_cache,
    paddle::Tensor& new_mem_cache,
    int n_head,
    int size_per_head,
    hipblasHandle_t cublas_handle_,
    hipStream_t stream) {
  auto input_dims = memory_tensor_input.shape();
  const int batch_size_ = static_cast<int>(input_dims[0]);
  const int max_seq_len_ = static_cast<int>(input_dims[1]);
  const int memory_hidden_dim_ = static_cast<int>(input_dims[2]);

  typedef PDTraits<D> traits_;
  typedef typename traits_::DataType DataType_;
  typedef typename traits_::data_t data_t_;
  typedef DecoderTransformerTraits<traits_::OpType> DecoderTraits_;
  OpenDecoder<DecoderTraits_::OpType>* decoder_;
  decoder_ = new OpenDecoder<DecoderTraits_::OpType>(batch_size_,
                                                     max_seq_len_,
                                                     n_head,
                                                     size_per_head,
                                                     memory_hidden_dim_,
                                                     true,
                                                     ActivationType::RELU);

  DataType_* decoder_output = reinterpret_cast<DataType_*>(
      decoder_output_tensor.mutable_data<data_t_>());
  DataType_* self_cache =
      reinterpret_cast<DataType_*>(old_self_cache.data<data_t_>());
  DataType_* memory_cache =
      reinterpret_cast<DataType_*>(old_mem_cache.data<data_t_>());
  const DataType_* from_tensor =
      reinterpret_cast<const DataType_*>(from_tensor_input.data<data_t_>());
  const DataType_* memory_tensor =
      reinterpret_cast<const DataType_*>(memory_tensor_input.data<data_t_>());
  const int* memory_sequence_length = mem_seq_len_input.data<int>();

  DecoderInitParam<DataType_> params;
  params.cublas_handle = cublas_handle_;
  params.stream = stream;
  fastertransformer::Allocator<AllocatorType::PD> allocator_(stream);

  params.self_layernorm.gamma =
      reinterpret_cast<const DataType_*>(self_ln_weight.data<data_t_>());
  params.self_layernorm.beta =
      reinterpret_cast<const DataType_*>(self_ln_bias.data<data_t_>());
  params.self_attention.query_weight.kernel =
      reinterpret_cast<const DataType_*>(self_q_weight.data<data_t_>());
  params.self_attention.query_weight.bias =
      reinterpret_cast<const DataType_*>(self_q_bias.data<data_t_>());
  params.self_attention.key_weight.kernel =
      reinterpret_cast<const DataType_*>(self_k_weight.data<data_t_>());
  params.self_attention.key_weight.bias =
      reinterpret_cast<const DataType_*>(self_k_bias.data<data_t_>());
  params.self_attention.value_weight.kernel =
      reinterpret_cast<const DataType_*>(self_v_weight.data<data_t_>());
  params.self_attention.value_weight.bias =
      reinterpret_cast<const DataType_*>(self_v_bias.data<data_t_>());
  params.self_attention.attention_output_weight.kernel =
      reinterpret_cast<const DataType_*>(self_out_weight.data<data_t_>());
  params.self_attention.attention_output_weight.bias =
      reinterpret_cast<const DataType_*>(self_out_bias.data<data_t_>());
  params.cross_layernorm.gamma =
      reinterpret_cast<const DataType_*>(cross_ln_weight.data<data_t_>());
  params.cross_layernorm.beta =
      reinterpret_cast<const DataType_*>(cross_ln_bias.data<data_t_>());
  params.cross_attention.query_weight.kernel =
      reinterpret_cast<const DataType_*>(cross_q_weight.data<data_t_>());
  params.cross_attention.query_weight.bias =
      reinterpret_cast<const DataType_*>(cross_q_bias.data<data_t_>());
  params.cross_attention.key_weight.kernel =
      reinterpret_cast<const DataType_*>(cross_k_weight.data<data_t_>());
  params.cross_attention.key_weight.bias =
      reinterpret_cast<const DataType_*>(cross_k_bias.data<data_t_>());
  params.cross_attention.value_weight.kernel =
      reinterpret_cast<const DataType_*>(cross_v_weight.data<data_t_>());
  params.cross_attention.value_weight.bias =
      reinterpret_cast<const DataType_*>(cross_v_bias.data<data_t_>());
  params.cross_attention.attention_output_weight.kernel =
      reinterpret_cast<const DataType_*>(cross_out_weight.data<data_t_>());
  params.cross_attention.attention_output_weight.bias =
      reinterpret_cast<const DataType_*>(cross_out_bias.data<data_t_>());
  params.ffn_layernorm.gamma =
      reinterpret_cast<const DataType_*>(ffn_ln_weight.data<data_t_>());
  params.ffn_layernorm.beta =
      reinterpret_cast<const DataType_*>(ffn_ln_bias.data<data_t_>());
  params.ffn.intermediate_weight.kernel =
      reinterpret_cast<const DataType_*>(ffn_inter_weight.data<data_t_>());
  params.ffn.intermediate_weight.bias =
      reinterpret_cast<const DataType_*>(ffn_inter_bias.data<data_t_>());
  params.ffn.output_weight.kernel =
      reinterpret_cast<const DataType_*>(ffn_out_weight.data<data_t_>());
  params.ffn.output_weight.bias =
      reinterpret_cast<const DataType_*>(ffn_out_bias.data<data_t_>());

  const int step = static_cast<int>(old_self_cache.shape()[1]);
  const int hidden_units = n_head * size_per_head;
  DataType_* K_cache = self_cache;
  DataType_* V_cache = self_cache + batch_size_ * step * hidden_units;
  DataType_* K_mem_cache = memory_cache;
  DataType_* V_mem_cache =
      memory_cache + batch_size_ * max_seq_len_ * hidden_units;

  const int decoder_buffer_size =
      decoder_->getWorkspaceSize() * sizeof(DataType_);
  DataType_* decoder_buffer =
      (DataType_*)allocator_.malloc(decoder_buffer_size);

  decoder_->initialize(params, decoder_buffer);
  decoder_->forward(from_tensor,
                    memory_tensor,
                    K_cache,
                    V_cache,
                    K_mem_cache,
                    V_mem_cache,
                    memory_sequence_length,
                    decoder_output,
                    step,
                    true);
  allocator_.free(decoder_buffer);
  delete decoder_;
  return {decoder_output_tensor, new_self_cache, new_mem_cache};
}

std::vector<paddle::Tensor> DecoderCUDAForward(
    const paddle::Tensor& from_tensor,
    const paddle::Tensor& memory_tensor,
    const paddle::Tensor& mem_seq_len,
    const paddle::Tensor& self_ln_weight,
    const paddle::Tensor& self_ln_bias,
    const paddle::Tensor& self_q_weight,
    const paddle::Tensor& self_q_bias,
    const paddle::Tensor& self_k_weight,
    const paddle::Tensor& self_k_bias,
    const paddle::Tensor& self_v_weight,
    const paddle::Tensor& self_v_bias,
    const paddle::Tensor& self_out_weight,
    const paddle::Tensor& self_out_bias,
    const paddle::Tensor& cross_ln_weight,
    const paddle::Tensor& cross_ln_bias,
    const paddle::Tensor& cross_q_weight,
    const paddle::Tensor& cross_q_bias,
    const paddle::Tensor& cross_k_weight,
    const paddle::Tensor& cross_k_bias,
    const paddle::Tensor& cross_v_weight,
    const paddle::Tensor& cross_v_bias,
    const paddle::Tensor& cross_out_weight,
    const paddle::Tensor& cross_out_bias,
    const paddle::Tensor& ffn_ln_weight,
    const paddle::Tensor& ffn_ln_bias,
    const paddle::Tensor& ffn_inter_weight,
    const paddle::Tensor& ffn_inter_bias,
    const paddle::Tensor& ffn_out_weight,
    const paddle::Tensor& ffn_out_bias,
    const paddle::Tensor& old_self_cache,
    const paddle::Tensor& old_mem_cache,
    paddle::Tensor& decoder_output,
    paddle::Tensor& new_self_cache,
    paddle::Tensor& new_mem_cache,
    int n_head,
    int size_per_head) {
  auto stream = memory_tensor.stream();
  hipblasHandle_t cublas_handle_;
  hipblasCreate(&cublas_handle_);
  hipblasSetStream(cublas_handle_, stream);

  std::vector<paddle::Tensor> ret;

  switch (memory_tensor.type()) {
    case paddle::DataType::FLOAT16: {
      ret = decoder_kernel<paddle::DataType::FLOAT16>(from_tensor,
                                                      memory_tensor,
                                                      mem_seq_len,
                                                      self_ln_weight,
                                                      self_ln_bias,
                                                      self_q_weight,
                                                      self_q_bias,
                                                      self_k_weight,
                                                      self_k_bias,
                                                      self_v_weight,
                                                      self_v_bias,
                                                      self_out_weight,
                                                      self_out_bias,
                                                      cross_ln_weight,
                                                      cross_ln_bias,
                                                      cross_q_weight,
                                                      cross_q_bias,
                                                      cross_k_weight,
                                                      cross_k_bias,
                                                      cross_v_weight,
                                                      cross_v_bias,
                                                      cross_out_weight,
                                                      cross_out_bias,
                                                      ffn_ln_weight,
                                                      ffn_ln_bias,
                                                      ffn_inter_weight,
                                                      ffn_inter_bias,
                                                      ffn_out_weight,
                                                      ffn_out_bias,
                                                      old_self_cache,
                                                      old_mem_cache,
                                                      decoder_output,
                                                      new_self_cache,
                                                      new_mem_cache,
                                                      n_head,
                                                      size_per_head,
                                                      cublas_handle_,
                                                      stream);
      break;
    }
    case paddle::DataType::FLOAT32: {
      ret = decoder_kernel<paddle::DataType::FLOAT32>(from_tensor,
                                                      memory_tensor,
                                                      mem_seq_len,
                                                      self_ln_weight,
                                                      self_ln_bias,
                                                      self_q_weight,
                                                      self_q_bias,
                                                      self_k_weight,
                                                      self_k_bias,
                                                      self_v_weight,
                                                      self_v_bias,
                                                      self_out_weight,
                                                      self_out_bias,
                                                      cross_ln_weight,
                                                      cross_ln_bias,
                                                      cross_q_weight,
                                                      cross_q_bias,
                                                      cross_k_weight,
                                                      cross_k_bias,
                                                      cross_v_weight,
                                                      cross_v_bias,
                                                      cross_out_weight,
                                                      cross_out_bias,
                                                      ffn_ln_weight,
                                                      ffn_ln_bias,
                                                      ffn_inter_weight,
                                                      ffn_inter_bias,
                                                      ffn_out_weight,
                                                      ffn_out_bias,
                                                      old_self_cache,
                                                      old_mem_cache,
                                                      decoder_output,
                                                      new_self_cache,
                                                      new_mem_cache,
                                                      n_head,
                                                      size_per_head,
                                                      cublas_handle_,
                                                      stream);
      break;
    }
    default: {
      PD_THROW(
          "NOT supported data type. "
          "Only float16 and float32 are supported. ");
      break;
    }
  }
  hipblasDestroy(cublas_handle_);
  return ret;
}
