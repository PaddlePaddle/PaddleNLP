#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

namespace fastertransformer {

template <typename T>
__global__ void update_logits_without_bias_kernel(T* logits,
                                                  const int end_id,
                                                  const int n) {
  int bid = blockIdx.x;
  int offset = bid * n;

  float max_val = -1 * FLT_MAX;
  __shared__ float s_max_val;
  __shared__ float s_sum_val;

  for (int tid = threadIdx.x; tid < n; tid += blockDim.x) {
    max_val = max(max_val, logits[offset + tid]);
  }

  max_val = blockReduceMax<float>((float)max_val);
  if (threadIdx.x == 0) s_max_val = max_val;
  __syncthreads();

  float sum_val = 0.0f;
  for (int tid = threadIdx.x; tid < n; tid += blockDim.x) {
    logits[offset + tid] = __expf((float)logits[offset + tid] - s_max_val);
    sum_val += (float)logits[offset + tid];
  }

  sum_val = blockReduceSum<float>(sum_val);
  if (threadIdx.x == 0) s_sum_val = sum_val;
  __syncthreads();

  for (int tid = threadIdx.x; tid < n; tid += blockDim.x) {
    logits[offset + tid] = logf((float)logits[offset + tid] / s_sum_val);
  }
}

void update_logits_without_bias(float* logits,
                                const int end_id,
                                const int m,
                                const int n,
                                hipStream_t stream) {
  dim3 grid(m);
  dim3 block(min(n, 1024));
  /*n is the vocab_size, e.g., 30000, 7000.... vocab_size is usually very big.
   */
  update_logits_without_bias_kernel<float><<<grid, block, 0, stream>>>(
      logits, end_id, n);
}
}
