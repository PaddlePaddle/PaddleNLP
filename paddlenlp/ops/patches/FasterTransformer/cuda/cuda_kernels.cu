#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
 * Copyright (c) 2020, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

namespace fastertransformer {

template <typename T, bool ALIVE>
__global__ void update_logits_kernel(T* logits,
                                     const T* bias,
                                     const int end_id,
                                     const bool* finished,
                                     const int n) {
  int bid = blockIdx.x;
  bool finish = ALIVE ? false : finished[bid];
  int offset = bid * n;

  float max_val = -1 * FLT_MAX;
  __shared__ float s_max_val;
  __shared__ float s_sum_val;

  for (int tid = threadIdx.x; tid < n; tid += blockDim.x) {
    if (finish)
      logits[offset + tid] = (tid == end_id) ? FLT_MAX : -1 * FLT_MAX;
    else
      logits[offset + tid] += bias[tid];
    max_val = max(max_val, logits[offset + tid]);
  }

  max_val = blockReduceMax<float>((float)max_val);
  if (threadIdx.x == 0) s_max_val = max_val;
  __syncthreads();

  float sum_val = 0.0f;
  for (int tid = threadIdx.x; tid < n; tid += blockDim.x) {
    logits[offset + tid] = __expf((float)logits[offset + tid] - s_max_val);
    sum_val += (float)logits[offset + tid];
  }

  sum_val = blockReduceSum<float>(sum_val);
  if (threadIdx.x == 0) s_sum_val = sum_val;
  __syncthreads();

  for (int tid = threadIdx.x; tid < n; tid += blockDim.x) {
    logits[offset + tid] = logf((float)logits[offset + tid] / s_sum_val);
  }
}

void update_logits_v2(float* logits,
                      const float* bias,
                      const int end_id,
                      const bool* finished,
                      const int m,
                      const int n,
                      hipStream_t stream) {
  dim3 grid(m);
  dim3 block(min(n, 1024));
  /*n is the vocab_size, e.g., 30000, 7000.... vocab_size is usually very big.
   */
  update_logits_kernel<float, true><<<grid, block, 0, stream>>>(
      logits, bias, end_id, finished, n);
}

template <typename T>
__global__ void add_bias_relu_encoder(T* out, const T* bias, int m, int n) {
  for (int id = blockIdx.x * blockDim.x + threadIdx.x; id < m * n;
       id += blockDim.x * gridDim.x) {
    T reg_bias = __ldg(&bias[id % n]);
    T val = out[id] + reg_bias;
    out[id] = (T)(val > 0.0f ? val : 0.0f);
  }
}

template <>
__global__ void add_bias_relu_encoder(half* out,
                                      const half* bias,
                                      int m,
                                      int n) {
  half2* out_ptr = (half2*)out;
  const half2* bias_ptr = (half2*)bias;

  for (int id = blockIdx.x * blockDim.x + threadIdx.x; id < m * n;
       id += blockDim.x * gridDim.x) {
    half2 reg_bias = __ldg(&bias_ptr[id % n]);
    half2 val = out_ptr[id] + reg_bias;
    val.x = val.x > (half)0.0f ? val.x : (half)0.0f;
    val.y = val.y > (half)0.0f ? val.y : (half)0.0f;
    out_ptr[id] = val;
  }
}

template void add_bias_act_kernelLauncher<float>(float* out,
                                                 const float* bias,
                                                 int m,
                                                 int n,
                                                 hipStream_t stream,
                                                 bool is_gelu);

template void add_bias_act_kernelLauncher<half>(half* out,
                                                const half* bias,
                                                int m,
                                                int n,
                                                hipStream_t stream,
                                                bool is_gelu);

template <typename T>
void add_bias_act_kernelLauncher(
    T* out, const T* bias, int m, int n, hipStream_t stream, bool is_gelu) {
  dim3 grid(ceil(m / 4.));
  dim3 block(n / 4);
  assert(block.x <= 1024);
  if (is_gelu)
    add_bias_act<T><<<grid, block, 0, stream>>>(out, bias, m, n);
  else
    add_bias_relu_encoder<T><<<grid, block, 0, stream>>>(out, bias, m, n);
}

}  // namespace fastertransformer
