#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
 * Copyright (c) 2020, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

namespace fastertransformer {

template <typename T, bool ALIVE = false>
__global__ void init_kernel_v2(bool* finished,
                               bool* alive_finished,
                               int* sequence_length,
                               int* word_ids,
                               T* cum_log_probs,
                               const int sentence_id,
                               const int beam_width,
                               const int batch_size) {
  const bool IS_FP16 = std::is_same<T, half>::value;
  const T MAX_T_VAL = (IS_FP16) ? HALF_FLT_MAX : 1e20f;
  for (int index = blockIdx.x * blockDim.x + threadIdx.x;
       index < batch_size * beam_width;
       index += blockDim.x * gridDim.x) {
    finished[index] = false;
    if (index < batch_size * beam_width / 2) {
      alive_finished[index] = false;
    }
    sequence_length[index] = 0;
    if (ALIVE) {
      if (index < batch_size * beam_width / 2) word_ids[index] = sentence_id;
      cum_log_probs[index] =
          (index % beam_width == beam_width / 2) ? (T)0.0f : -MAX_T_VAL;
    } else {
      word_ids[index] = sentence_id;
      cum_log_probs[index] = (index % beam_width == 0) ? (T)0.0f : -MAX_T_VAL;
    }
  }
}

template <typename T>
void init_kernelLauncher_v2(bool* finished,
                            bool* alive_finished,
                            int* sequence_length,
                            int* word_ids,
                            T* cum_log_probs,
                            const int sentence_id,
                            const int batch_size,
                            const int beam_width,
                            hipStream_t stream) {
  dim3 grid((int)ceil(batch_size * beam_width * 1.0 / 256));
  dim3 block(256);

  init_kernel_v2<T, true><<<grid, block, 0, stream>>>(finished,
                                                      alive_finished,
                                                      sequence_length,
                                                      word_ids,
                                                      cum_log_probs,
                                                      sentence_id,
                                                      beam_width,
                                                      batch_size);
}

// TODO Add half2 implementation
template <typename T>
__global__ void embedding_position_lookups_fix_kernel(
    T* from_tensor,
    const T* embedding_table,
    const T* pos_table,
    const int* word_ids,
    const int local_batch_size,
    const int batch_size,
    const int hidden_units,
    int step,
    int ite,
    int max_input_len,
    const int* start_lengths) {
  int timestep = step - 1;
  // if the input is padded in the batch, indices of the word_id and the
  // pos_table also should be shifted forward by the length of the padding.
  int len_padding =
      max_input_len - start_lengths[local_batch_size * ite + blockIdx.x];
  int idx_word_id = (step == max_input_len) ? timestep - len_padding : timestep;
  int idx_pos_table = timestep - len_padding;

  int* word_ids_buf =
      (int*)word_ids + idx_word_id * batch_size + local_batch_size * ite;
  T* from_tensor_buf = from_tensor + blockIdx.x * hidden_units;
  for (int index = threadIdx.x; index < hidden_units; index += blockDim.x) {
    from_tensor_buf[index] =
        embedding_table[word_ids_buf[blockIdx.x] * hidden_units + index] +
        pos_table[idx_pos_table * hidden_units + index];
  }
}

template <typename T>
void embedding_position_lookups_fix_kernel_launcher(T* from_tensor,
                                                    const T* embedding_table,
                                                    const T* pos_table,
                                                    const int* word_ids,
                                                    const int local_batch_size,
                                                    const int batch_size,
                                                    const int hidden_units,
                                                    int step,
                                                    int ite,
                                                    int max_input_len,
                                                    const int* start_lengths,
                                                    hipStream_t stream) {
  dim3 grid(min(local_batch_size, 65536));
  dim3 block(min(hidden_units, 1024));
  embedding_position_lookups_fix_kernel<T>
      <<<grid, block, 0, stream>>>(from_tensor,
                                   embedding_table,
                                   pos_table,
                                   word_ids,
                                   local_batch_size,
                                   batch_size,
                                   hidden_units,
                                   step,
                                   ite,
                                   max_input_len,
                                   start_lengths);
}

template <typename T>
__global__ void embedding_position_lookups_bart_kernel(
    T* from_tensor,
    const T* embedding_table,
    const T* position_encoding,
    const int* word_ids,
    const int batch_size,
    const int hidden_units) {
  // 1. lookup from embedding table
  // 2. add the position encoding
  for (int index = blockIdx.x * blockDim.x + threadIdx.x;
       index < batch_size * hidden_units;
       index += blockDim.x * gridDim.x) {
    const int row_index = index / hidden_units;
    const int col_index = index % hidden_units;
    from_tensor[index] =
        embedding_table[word_ids[row_index] * hidden_units + col_index] +
        position_encoding[col_index];
  }
}

template <typename T>
void embedding_position_lookups_bart_kernel_launcher(T* from_tensor,
                                                     const T* embedding_table,
                                                     const T* position_encoding,
                                                     const int* word_ids,
                                                     const int batch_size,
                                                     const int hidden_units,
                                                     hipStream_t stream) {
  dim3 grid(min(batch_size, 65536));
  dim3 block(min(hidden_units, 1024));
  embedding_position_lookups_bart_kernel<T><<<grid, block, 0, stream>>>(
      from_tensor,
      embedding_table,
      position_encoding,
      word_ids,
      batch_size,
      hidden_units);
}

template <typename T>
__global__ void update_with_force_decoding_kernel(const int* trg_word,
                                                  const int* trg_length,
                                                  bool* finished,
                                                  int* word_ids,
                                                  int* sequence_length,
                                                  int* parent_ids_buf,
                                                  int* parent_ids,
                                                  int* output_ids,
                                                  T* scores,
                                                  bool keep_alive_beam,
                                                  const int batch_size,
                                                  const int beam_width,
                                                  const int max_trg_len,
                                                  const int step) {
  int bid = blockIdx.x;   // batch_size
  int tid = threadIdx.x;  // beam_width

  const T MAX_T_VAL = (sizeof(T) == 2) ? HALF_FLT_MAX : 1e20f;
  if (step <= trg_length[bid]) {
    finished[bid * beam_width + tid] = false;

    int word_id = trg_word[bid * max_trg_len + step - 1];

    if (keep_alive_beam) {
      if (tid >= beam_width / 2) {
        word_ids[bid * beam_width / 2 + tid - beam_width / 2] = word_id;
      }
    } else {
      word_ids[bid * beam_width + tid] = word_id;
    }

    output_ids[bid * beam_width + tid] = word_id;
    if (sequence_length) {
      sequence_length[bid * beam_width + tid]++;
    }

    if (parent_ids && scores) {
      if (keep_alive_beam) {
        parent_ids[bid * beam_width + tid] = bid * beam_width + beam_width / 2;
        if (tid >= beam_width / 2) {
          parent_ids_buf[bid * beam_width / 2 + tid - beam_width / 2] =
              bid * beam_width / 2;
        }

        if (tid == beam_width / 2) {
          scores[bid * beam_width + tid] = 0;
        } else {
          scores[bid * beam_width + tid] = -MAX_T_VAL;
        }
      } else {
        parent_ids[bid * beam_width + tid] = bid * beam_width;

        if (tid == 0) {
          scores[bid * beam_width + tid] = 0;
        } else {
          scores[bid * beam_width + tid] = -MAX_T_VAL;
        }
      }
    }
  }
}

template <typename T>
void update_with_force_decodingLauncher(const int* trg_word,
                                        const int* trg_length,
                                        bool* finished,
                                        int* word_ids,
                                        int* sequence_length,
                                        int* parent_ids_buf,
                                        int* parent_ids,
                                        int* output_ids,
                                        T* scores,
                                        bool keep_alive_beam,
                                        const int batch_size,
                                        const int beam_width,
                                        const int max_trg_len,
                                        const int step,
                                        hipStream_t stream) {
  if (trg_word == nullptr) {
    return;
  }

  update_with_force_decoding_kernel<<<batch_size, beam_width, 0, stream>>>(
      trg_word,
      trg_length,
      finished,
      word_ids,
      sequence_length,
      parent_ids_buf,
      parent_ids,
      output_ids,
      scores,
      keep_alive_beam,
      batch_size,
      beam_width,
      max_trg_len,
      step);
}

template <typename T>
void update_KV_cache_kernelLauncher_v2(T** key_cache,
                                       T** value_cache,
                                       const int* beam_ids,
                                       const bool* finished,
                                       const int batch_size,
                                       const int beam_width,
                                       const int head_num,
                                       const int size_per_head,
                                       const int step,
                                       const int decoder_max_seq_len,
                                       const int cache_size,
                                       const int decoder_layers,
                                       hipStream_t stream,
                                       const int memory_max_seq_len) {
  int src_id = step & 0x1;
  int tgt_id = 1 - src_id;
  int tmp_len = (memory_max_seq_len != -1) ? step + memory_max_seq_len : step;

  if (decoder_max_seq_len < 0) {
    int hidden_dim = head_num * size_per_head;
    dim3 grid(decoder_layers * batch_size * beam_width * tmp_len);
    dim3 block(min(1024, hidden_dim));
    block.x = block.x / (4 / sizeof(T));

    update_KV_cache_kernel<<<grid, block, 0, stream>>>(key_cache[src_id],
                                                       key_cache[tgt_id],
                                                       value_cache[src_id],
                                                       value_cache[tgt_id],
                                                       beam_ids,
                                                       finished,
                                                       batch_size,
                                                       beam_width,
                                                       hidden_dim,
                                                       cache_size,
                                                       tmp_len,
                                                       decoder_layers);
  } else {
    dim3 grid(batch_size * beam_width, head_num, decoder_layers);
    constexpr int block_sz = 128;
    int tmp_decoder_max_seq_len =
        (memory_max_seq_len != -1) ? (decoder_max_seq_len + memory_max_seq_len)
                                   : decoder_max_seq_len;

    update_KV_batch_major_cache_kernel<<<grid, block_sz, 0, stream>>>(
        key_cache[src_id],
        key_cache[tgt_id],
        value_cache[src_id],
        value_cache[tgt_id],
        beam_ids,
        finished,
        batch_size,
        beam_width,
        size_per_head,
        cache_size,
        tmp_len,
        tmp_decoder_max_seq_len,
        decoder_layers);
  }
}

template <typename T>
__global__ void apply_logits_mask_kernel(int vocab_size_padded,
                                         int vocab_size,
                                         int beam_width,
                                         T* log_probs,
                                         const bool* finished,
                                         const T* logits_mask = nullptr,
                                         const bool min_penalty = false,
                                         const int end_id = -1,
                                         const T* bias = nullptr) {
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int bbid = blockIdx.y;  // batch_size * beam_size: index
  const T MAX_T_VAL = (sizeof(T) == 2) ? HALF_FLT_MAX : 1e20f;

  bool finish = (finished != nullptr) ? finished[bbid] : false;

  if (!finish) {
    for (int i = tid + bid * blockDim.x; i < vocab_size;
          i += blockDim.x * gridDim.x) {
      if (min_penalty && i == end_id) {
        log_probs[i + bbid * vocab_size_padded] += -MAX_T_VAL;
      } else if (logits_mask) {
        log_probs[i + bbid * vocab_size_padded] += logits_mask[i];
      } else if (bias) {
        log_probs[i + bbid * vocab_size_padded] += bias[i];
      } else {
        continue;
      }
    }
  }
}

template <typename T>
void apply_logits_mask_kernelLauncher(T* log_probs,
                                      const bool* finished,
                                      int batch_size,
                                      int beam_width,
                                      int vocab_size_padded,
                                      int vocab_size,
                                      hipStream_t stream,
                                      const T* logits_mask,
                                      const bool min_penalty,
                                      const int end_id,
                                      const T* bias) {
  if (logits_mask == nullptr && !min_penalty && bias == nullptr) return;

  dim3 block(256);
  dim3 grid((vocab_size_padded + block.x - 1) / block.x,
            beam_width * batch_size);

  apply_logits_mask_kernel<T><<<grid, block, 0, stream>>>(vocab_size_padded,
                                                          vocab_size,
                                                          beam_width,
                                                          log_probs,
                                                          finished,
                                                          logits_mask,
                                                          min_penalty,
                                                          end_id,
                                                          bias);
}


  template <typename T> __launch_bounds__(1024, 1)
  __global__ void gptj_start_id_embedding_lookups_kernel(T* from_tensor,
                                                             int* output_ids,
                                                             const T* embedding_table,
                                                             const int* word_ids,
                                                             const int length,
                                                             const int max_length,
                                                             const int batch_size,
                                                             const int hidden_units)
  { 
      for(int index = blockIdx.x * blockDim.x + threadIdx.x; index < batch_size * length * hidden_units; index += blockDim.x * gridDim.x)
      {
          // transpose the word_ids [batch, length] (part of [batch, max_length]) to output_ids [length, batch]
          if(index < batch_size * max_length)
          {
            const int seq_id = index % max_length;
            const int batch_id = index / max_length;
            if(seq_id < length)
              output_ids[seq_id * batch_size + batch_id] = word_ids[index];
            // output_ids[index] = word_ids[index];
          }
        
          // embedding lookup from word ids [batch, length] (part of [batch, max_length]) and [vocab, hidden] to generate embedding [batch, length, hidden]
          const int word_index = index / hidden_units;
          const int word_index_row = word_index / length;
          const int word_index_col = word_index % length;
          const int real_word_index = word_index_row * max_length + word_index_col;
          const int col_index = index % hidden_units;
          from_tensor[index] = embedding_table[word_ids[real_word_index] * hidden_units + col_index];
      }
  }


  template <typename T>
  void gptj_start_id_embedding_lookups_kernel_launcher(T* from_tensor,
                                                           int *output_ids,
                                                           const T* embedding_table, 
                                                           const int* word_ids,
                                                           const int length,
                                                           const int max_length,
                                                           const int batch_size,
                                                           const int hidden_units, 
                                                           hipStream_t stream)
  {
      dim3 grid(min(batch_size * length, 65536));
      dim3 block(min(hidden_units, 1024));
      gptj_start_id_embedding_lookups_kernel<T><<<grid, block, 0, stream>>>(from_tensor,
                                                                                output_ids,
                                                                                embedding_table,
                                                                                word_ids,
                                                                                length,
                                                                                max_length,
                                                                                batch_size,
                                                                                hidden_units);
  }


  // TODO Add half2 implementation
template <typename T>
__global__ void gptj_embedding_lookups_kernel(
    T* from_tensor,
    const T* embedding_table,
    const int* word_ids,
    const int local_batch_size,
    const int batch_size,
    const int hidden_units,
    int step,
    int ite,
    int max_input_len,
    const int* start_lengths) {
  int timestep = step - 1;
  // if the input is padded in the batch, indices of the word_id 
  // should be shifted forward by the length of the padding.
  int len_padding =
      max_input_len - start_lengths[local_batch_size * ite + blockIdx.x];
  int idx_word_id = (step == max_input_len) ? timestep - len_padding : timestep;

  int* word_ids_buf =
      (int*)word_ids + idx_word_id * batch_size + local_batch_size * ite;
  T* from_tensor_buf = from_tensor + blockIdx.x * hidden_units;
  for (int index = threadIdx.x; index < hidden_units; index += blockDim.x) {
    from_tensor_buf[index] =
        embedding_table[word_ids_buf[blockIdx.x] * hidden_units + index];
  }
}

template <typename T>
void gpj_embedding_lookups_kernel_launcher(T* from_tensor,
                                                    const T* embedding_table,
                                                    const int* word_ids,
                                                    const int local_batch_size,
                                                    const int batch_size,
                                                    const int hidden_units,
                                                    int step,
                                                    int ite,
                                                    int max_input_len,
                                                    const int* start_lengths,
                                                    hipStream_t stream) {
  dim3 grid(min(local_batch_size, 65536));
  dim3 block(min(hidden_units, 1024));
  gptj_embedding_lookups_kernel<T>
      <<<grid, block, 0, stream>>>(from_tensor,
                                   embedding_table,
                                   word_ids,
                                   local_batch_size,
                                   batch_size,
                                   hidden_units,
                                   step,
                                   ite,
                                   max_input_len,
                                   start_lengths);
}

template void init_kernelLauncher_v2(bool* finished,
                                     bool* alive_finished,
                                     int* sequence_length,
                                     int* word_ids,
                                     float* cum_log_probs,
                                     const int sentence_id,
                                     const int batch_size,
                                     const int beam_width,
                                     hipStream_t stream);

template void init_kernelLauncher_v2(bool* finished,
                                     bool* alive_finished,
                                     int* sequence_length,
                                     int* word_ids,
                                     half* cum_log_probs,
                                     const int sentence_id,
                                     const int batch_size,
                                     const int beam_width,
                                     hipStream_t stream);

template void embedding_position_lookups_fix_kernel_launcher(
    float* from_tensor,
    const float* embedding_table,
    const float* pos_table,
    const int* word_ids,
    const int local_batch_size,
    const int batch_size,
    const int hidden_units,
    int step,
    int ite,
    int max_input_len,
    const int* start_lengths,
    hipStream_t stream);

template void embedding_position_lookups_fix_kernel_launcher(
    half* from_tensor,
    const half* embedding_table,
    const half* pos_table,
    const int* word_ids,
    const int local_batch_size,
    const int batch_size,
    const int hidden_units,
    int step,
    int ite,
    int max_input_len,
    const int* start_lengths,
    hipStream_t stream);

template void embedding_position_lookups_bart_kernel_launcher(
    float* from_tensor,
    const float* embedding_table,
    const float* position_encoding,
    const int* word_ids,
    const int batch_size,
    const int hidden_units,
    hipStream_t stream);

template void embedding_position_lookups_bart_kernel_launcher(
    half* from_tensor,
    const half* embedding_table,
    const half* position_encoding,
    const int* word_ids,
    const int batch_size,
    const int hidden_units,
    hipStream_t stream);

template void update_with_force_decodingLauncher(const int* trg_word,
                                                 const int* trg_length,
                                                 bool* finished,
                                                 int* word_ids,
                                                 int* sequence_length,
                                                 int* parent_ids_buf,
                                                 int* parent_ids,
                                                 int* output_ids,
                                                 float* scores,
                                                 bool keep_alive_beam,
                                                 const int batch_size,
                                                 const int beam_width,
                                                 const int max_trg_len,
                                                 const int step,
                                                 hipStream_t stream);

template void update_with_force_decodingLauncher(const int* trg_word,
                                                 const int* trg_length,
                                                 bool* finished,
                                                 int* word_ids,
                                                 int* sequence_length,
                                                 int* parent_ids_buf,
                                                 int* parent_ids,
                                                 int* output_ids,
                                                 half* scores,
                                                 bool keep_alive_beam,
                                                 const int batch_size,
                                                 const int beam_width,
                                                 const int max_trg_len,
                                                 const int step,
                                                 hipStream_t stream);

template void update_KV_cache_kernelLauncher_v2(float** key_cache,
                                                float** value_cache,
                                                const int* beam_ids,
                                                const bool* finished,
                                                const int batch_size,
                                                const int beam_width,
                                                const int head_num,
                                                const int size_per_head,
                                                const int step,
                                                const int decoder_max_seq_len,
                                                const int cache_size,
                                                const int decoder_layers,
                                                hipStream_t stream,
                                                const int memory_max_seq_len);

template void update_KV_cache_kernelLauncher_v2(half** key_cache,
                                                half** value_cache,
                                                const int* beam_ids,
                                                const bool* finished,
                                                const int batch_size,
                                                const int beam_width,
                                                const int head_num,
                                                const int size_per_head,
                                                const int step,
                                                const int decoder_max_seq_len,
                                                const int cache_size,
                                                const int decoder_layers,
                                                hipStream_t stream,
                                                const int memory_max_seq_len);

template void apply_logits_mask_kernelLauncher(
    float* log_probs,
    const bool* finished,
    int batch_size,
    int beam_width,
    int vocab_size_padded,
    int vocab_size,
    hipStream_t stream,
    const float* logits_mask,
    const bool min_penalty,
    const int end_id,
    const float* bias);

template void apply_logits_mask_kernelLauncher(
    half* log_probs,
    const bool* finished,
    int batch_size,
    int beam_width,
    int vocab_size_padded,
    int vocab_size,
    hipStream_t stream,
    const half* logits_mask,
    const bool min_penalty,
    const int end_id,
    const half* bias);

  template
  void gptj_start_id_embedding_lookups_kernel_launcher(float* from_tensor,
                                                           int* output_ids,
                                                           const float* embedding_table,
                                                           const int* word_ids,
                                                           const int length,
                                                           const int max_length,
                                                           const int batch_size,
                                                           const int hidden_units, 
                                                           hipStream_t stream);

  template
  void gptj_start_id_embedding_lookups_kernel_launcher(half* from_tensor,
                                                           int* output_ids,
                                                           const half* embedding_table,
                                                           const int* word_ids,
                                                           const int length,
                                                           const int max_length,
                                                           const int batch_size,
                                                           const int hidden_units, 
                                                           hipStream_t stream);
  
  template void gpj_embedding_lookups_kernel_launcher(
    float* from_tensor,
    const float* embedding_table,
    const int* word_ids,
    const int local_batch_size,
    const int batch_size,
    const int hidden_units,
    int step,
    int ite,
    int max_input_len,
    const int* start_lengths,
    hipStream_t stream);

template void gpj_embedding_lookups_kernel_launcher(
    half* from_tensor,
    const half* embedding_table,
    const int* word_ids,
    const int local_batch_size,
    const int batch_size,
    const int hidden_units,
    int step,
    int ite,
    int max_input_len,
    const int* start_lengths,
    hipStream_t stream);

}  // end of name space fastertransformer
