#include "hip/hip_runtime.h"
/*
* Copyright (c) 2019-2021, NVIDIA CORPORATION.  All rights reserved.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/
/**
* Open sourced multi-head attention
**/

#include "fastertransformer/utils/allocator.h"
#include "fastertransformer/cuda/multi_head_attention.h"
#include "fastertransformer/cuda/open_attention.h"
#include "fastertransformer/cuda/attention_kernels.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <cmath>

namespace fastertransformer{
namespace cuda{

/**
* Multi-head attetion open sourced
*/
#define FINAL_MASK 0xffffffff

template <typename T>
__inline__ __device__
T warpReduceSum(T val)
{
  #pragma unroll
  for(int mask = 16; mask > 0; mask >>= 1)
    val += __shfl_xor_sync(FINAL_MASK, val, mask, 32);
  return val;
}

/* Calculate the sum of all elements in a block */
template <typename T>
  __inline__ __device__
T blockReduceSum(T val)
{
  static __shared__ T shared[32]; 
  int lane = threadIdx.x & 0x1f; 
  int wid = threadIdx.x >> 5;  

  val = warpReduceSum<T>(val);

  if(lane == 0)
    shared[wid] = val;

  __syncthreads();

  val = (threadIdx.x < (blockDim.x >> 5 )) ? shared[lane] : (T)(0.0f);
  val = warpReduceSum<T>(val);
                              
  return val;
}

template <typename T>
  __inline__ __device__
T warpReduceMax(T val)
{
  #pragma unroll
  for(int mask = 16; mask > 0; mask >>= 1)
    val = max(val, __shfl_xor_sync(FINAL_MASK, val, mask, 32));
  return val;
}

/* Calculate the maximum of all elements in a block */
template <typename T>
  __inline__ __device__
T blockReduceMax(T val)
{
  static __shared__ T shared[32]; 
  int lane = threadIdx.x & 0x1f; // in-warp idx
  int wid = threadIdx.x >> 5;  // warp idx

  val = warpReduceMax(val); // get maxx in each warp

  if(lane == 0) // record in-warp maxx by warp Idx
    shared[wid] = val;

  __syncthreads();


  val = (threadIdx.x < (blockDim.x >> 5 )) ? shared[lane] : -1e20f;
  val = warpReduceMax(val);

  return val;
}

  __inline__ __device__
int target_index(int id1, int id2, int id3, int id4, int dim_1, int dim_2, int dim_3, int dim_4)
{
  return id1 * (dim_2 * dim_3 * dim_4) + id3 * (dim_2 * dim_4) + id2 * dim_4 + id4;
}

//build a mapping for fullData to removePaddingData
//grid((valid_word_num+63)/64)
//block(64)
__global__ void mappingRemovePaddingData(int *mapping, const int* sequence_id_offset, const int valid_word_num){
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx < valid_word_num)
    mapping[idx + __ldg(sequence_id_offset + idx)] = idx;
}

void mappingRemovePaddingData_kernelLauncher(const int batch_size, const int seq_len, 
                                             const int valid_word_num, int *mapping, 
                                             const int* sequence_id_offset, hipStream_t stream)
{
  hipMemsetAsync(mapping, -1, batch_size * seq_len * sizeof(int), stream);
  mappingRemovePaddingData<<<dim3((valid_word_num + 63)/64), dim3(64), 0, stream>>>(mapping, sequence_id_offset, valid_word_num);
}

//add_QK_bias_transform for batch int8 hipblasLtMatmul & per axis quantization for weight
//1.add QK bias
//2.transform each Q K CUBLASLT_ORDER_COL32 matrixes into a series of sub-matrix (with CUBLASLT_ORDER_COL32/CUBLASLT_ORDER_COL4_4R2_8C layout)
//  Q, K are CUBLASLT_ORDER_COL32 matrixes of m = batch_size * seq_len, n = head_num * size_per_head
//  q_buf_ is of batchCount = batch_size * head_num, m = seq_len, n = size_per_head, CUBLASLT_ORDER_COL32
//  k_buf_ is of batchCount = batch_size * head_num, m = seq_len, n = size_per_head, CUBLASLT_ORDER_COL4_4R2_8C
//only for int32 input & int8 output
//seq_len, size_per_head must be a multiple of 32
//grid.x = batch_size * seq_len * 2;
//block.x = head_num * size_per_head / 4;
//using char4
template <typename T>
__global__
void add_QK_bias_transform(int8_t *q_buf_, int8_t *k_buf_, const int32_t* Q, const T* bias_Q, 
                           const int32_t* K, const T* bias_K, const int m, const int batch_size, 
                           const int seq_len, const int head_num, const int size_per_head, int stride, 
                           const float * q_weight_amax, const float *q_input_deQFactor_div127_ptr, const float * k_weight_amax, 
                           const float *k_input_deQFactor_div127_ptr, const float *q_output_scale_ptr, const float *k_output_scale_ptr,
                           bool use_ORDER_COL32_2R_4R4)
{
  const int32_t* data_ptr;
  char4* buf_ptr4;
  const T* bias_ptr;
  const float* weight_amax;
  int qk_id = blockIdx.x / m;

  data_ptr = qk_id == 0 ? Q : K;
  buf_ptr4 = qk_id == 0 ? (char4*)q_buf_ : (char4*)k_buf_;
  bias_ptr = qk_id == 0 ? bias_Q : bias_K;
  const float input_deQFactor_div127 = qk_id == 0 ? __ldg(q_input_deQFactor_div127_ptr) : __ldg(k_input_deQFactor_div127_ptr);
  weight_amax = qk_id == 0 ? q_weight_amax : k_weight_amax;
  const float output_scale = qk_id == 0 ? __ldg(q_output_scale_ptr) : __ldg(k_output_scale_ptr);

  int threadIdx4 = threadIdx.x << 2;
  int batch_id = (blockIdx.x % m) / seq_len;
  int head_id = threadIdx4 / size_per_head;
  int id_in_head = threadIdx4 % size_per_head;
  int word_id = blockIdx.x % seq_len;

  int data_id = (((threadIdx4 >> 5) << 5)*m + ((blockIdx.x%m) << 5) + (threadIdx4&31));

  float scale;
  float tmp;
  char4 tmp4;
  scale = static_cast<float>(__ldg(data_ptr+data_id)) * __ldg(weight_amax+threadIdx4) * input_deQFactor_div127;
  tmp = static_cast<float>(__ldg(bias_ptr+threadIdx4)) + scale;
  tmp4.x = float_to_int8_rn(tmp*output_scale);

  data_id = data_id+1;
  threadIdx4 = threadIdx4+1;
  scale = static_cast<float>(__ldg(data_ptr+data_id)) * __ldg(weight_amax+threadIdx4)* input_deQFactor_div127;
  tmp = static_cast<float>(__ldg(bias_ptr+threadIdx4)) + scale;
  tmp4.y = float_to_int8_rn(tmp*output_scale);

  data_id = data_id+1;
  threadIdx4 = threadIdx4+1;
  scale = static_cast<float>(__ldg(data_ptr+data_id)) * __ldg(weight_amax+threadIdx4) * input_deQFactor_div127;
  tmp = static_cast<float>(__ldg(bias_ptr+threadIdx4)) + scale;
  tmp4.z = float_to_int8_rn(tmp*output_scale);

  data_id = data_id+1;
  threadIdx4 = threadIdx4+1;
  scale = static_cast<float>(__ldg(data_ptr+data_id)) * __ldg(weight_amax+threadIdx4) * input_deQFactor_div127;
  tmp = static_cast<float>(__ldg(bias_ptr+threadIdx4)) + scale;
  tmp4.w = float_to_int8_rn(tmp*output_scale);


  //row_id, col_id of sub-matrix (m = seq_len, n = size_per_head), column-major

  int row_id = word_id;
  int col_id = id_in_head;
  //new (row, rol) of LtTrans COL32/COL4 sub-matrix, leading dim = (COL32_ * seq_len)
  int new_col = col_id >> 5;
  int new_row;  
  if (use_ORDER_COL32_2R_4R4)
  {
    int row_in_tile = row_id & 31;
    int col_in_tile = col_id & 31; 
    new_row = (qk_id != 1) ?
              //COL32
              ((row_id << 5) + (col_id&31))
            :
               //COL32_2R_4R4
               (
               ((row_id >> 5) << 10) +
               //(((row%8)/2*4+row/8)*2+row%2)*32+col
               (((((((row_in_tile&7)>>1)<<2)+(row_in_tile>>3))<<1)+(row_in_tile&1))<<5)+col_in_tile
               )
               ;
  }
  else
  {
    new_row = (qk_id != 1) ?
              //COL32
              ((row_id << 5) + (col_id&31))
            :
              //COL4
              ////row_id/8 is the number of tile of (8 rows 32 columns) -- column-major
              ////row_id%2 is even row, otherwise odd row
              ////col_id%COL32_/8 is the number tile of (8 rows 8 columns)
              (
              ((((row_id >> 3) << 3) + ((row_id&1) << 2) + ((col_id&31) >> 3)) << 5) +
              ////col_id%8 >= 4 is the right half of (8 rows 8 columns) tile
              ////(row_id%8/2) is (the row id of alternating 4 rows) - 1
              (((((col_id&7) >= 4)?4:0) + ((row_id&7) >> 1)) << 2) +
              ////col_id%4 is the id of 4 cols
              (col_id&3)
              )
              ;
  }

  buf_ptr4[(((batch_id*head_num + head_id) * stride + (new_col << 5)*seq_len + new_row) >> 2)] = tmp4;
}

template <typename T>
void add_QK_bias_transform_kernelLauncher(int8_t *q_buf, int8_t *k_buf, const int32_t* Q, const T* bias_Q, 
                                          const int32_t* K, const T* bias_K, const int batch_size, 
                                          const int seq_len, const int head_num, const int size_per_head, 
                                          const float * q_weight_amax, const float *q_input_deQFactor_div127_ptr, 
                                          const float * k_weight_amax, const float *k_input_deQFactor_div127_ptr, 
                                          const float *q_output_scale_ptr, const float *k_output_scale_ptr,
                                          bool use_ORDER_COL32_2R_4R4, hipStream_t stream)
{
  add_QK_bias_transform<<<dim3(batch_size*seq_len*2), dim3((head_num * size_per_head)/4), 0, stream>>>(
    q_buf, k_buf, Q, bias_Q, K, bias_K, 
    batch_size * seq_len, batch_size, seq_len, head_num, size_per_head, seq_len*size_per_head, 
    q_weight_amax, q_input_deQFactor_div127_ptr, k_weight_amax, k_input_deQFactor_div127_ptr, 
    q_output_scale_ptr, k_output_scale_ptr, use_ORDER_COL32_2R_4R4);
}

template
void add_QK_bias_transform_kernelLauncher(int8_t *q_buf, int8_t *k_buf, const int32_t* Q, const float* bias_Q, 
                                          const int32_t* K, const float* bias_K, const int batch_size, 
                                          const int seq_len, const int head_num, const int size_per_head, 
                                          const float * q_weight_amax, const float *q_input_deQFactor_div127_ptr, 
                                          const float * k_weight_amax, const float *k_input_deQFactor_div127_ptr, 
                                          const float *q_output_scale_ptr, const float *k_output_scale_ptr,
                                          bool use_ORDER_COL32_2R_4R4, hipStream_t stream);

template
void add_QK_bias_transform_kernelLauncher(int8_t *q_buf, int8_t *k_buf, const int32_t* Q, const half* bias_Q, 
                                          const int32_t* K, const half* bias_K, const int batch_size, 
                                          const int seq_len, const int head_num, const int size_per_head, 
                                          const float * q_weight_amax, const float *q_input_deQFactor_div127_ptr, 
                                          const float * k_weight_amax, const float *k_input_deQFactor_div127_ptr, 
                                          const float *q_output_scale_ptr, const float *k_output_scale_ptr,
                                          bool use_ORDER_COL32_2R_4R4, hipStream_t stream);

//add_QK_bias_padding_transform for batch int8 hipblasLtMatmul & per tensor quantization for weight
//1.add QK bias
//2.padding seq_len in k_buf_ to a multiple of 32 named seq_len_padded
//3.transform each Q K CUBLASLT_ORDER_COL32 matrixes into a series of sub-matrix (with CUBLASLT_ORDER_COL32/CUBLASLT_ORDER_COL4_4R2_8C layout)
//  Q, K are CUBLASLT_ORDER_COL32 matrixes of m = batch_size * seq_len, n = head_num * size_per_head
//  q_buf_ is of batchCount = batch_size * head_num, m = seq_len, n = size_per_head, CUBLASLT_ORDER_COL32
//  k_buf_ is of batchCount = batch_size * head_num, m = seq_len_padded, n = size_per_head, CUBLASLT_ORDER_COL4_4R2_8C
//only for int8 IO
//size_per_head must be a multiple of 32
//grid.x = batch_size * seq_len * 2;
//block.x = head_num * size_per_head / 4;
//using char4
template <typename T>
__global__
void add_QK_bias_transform_varlen(int8_t *q_buf_, int8_t *k_buf_, const int8_t* Q, const T* bias_Q, 
                           const int8_t* K, const T* bias_K, const int m, const int batch_size, 
                           const int seq_len, const int head_num, const int size_per_head, 
                           const int seq_len_padded, const int stride_q, const int stride_k,
                           const float *q_input_deQFactor_ptr, const float *k_input_deQFactor_ptr, 
                           const float *q_output_scale_ptr, const float *k_output_scale_ptr,
                           bool use_ORDER_COL32_2R_4R4)
{
  const char4* data_ptr;
  char4* buf_ptr4;
  const T* bias_ptr;
  int qk_id = blockIdx.x / m;

  data_ptr = qk_id == 0 ? (const char4*)Q : (const char4*)K;
  buf_ptr4 = qk_id == 0 ? (char4*)q_buf_ : (char4*)k_buf_;
  bias_ptr = qk_id == 0 ? bias_Q : bias_K;
  const float input_deQFactor = qk_id == 0 ? __ldg(q_input_deQFactor_ptr) : __ldg(k_input_deQFactor_ptr);
  const float output_scale = qk_id == 0 ? __ldg(q_output_scale_ptr) : __ldg(k_output_scale_ptr);

  int threadIdx4 = threadIdx.x << 2;
  int batch_id = (blockIdx.x % m) / seq_len;
  int head_id = threadIdx4 / size_per_head;
  int id_in_head = threadIdx4 % size_per_head;
  int word_id = blockIdx.x % seq_len;

  int data_id = (((threadIdx4 >> 5) << 5)*m + ((blockIdx.x%m) << 5) + (threadIdx4&31)) >> 2;

  float scale;
  float tmp;
  char4 tmp4 = __ldg(data_ptr+data_id);
  scale = static_cast<float>(tmp4.x) * input_deQFactor;
  tmp = static_cast<float>(__ldg(bias_ptr+threadIdx4)) + scale;
  tmp4.x = float_to_int8_rn(tmp*output_scale);

  threadIdx4 = threadIdx4+1;
  scale = static_cast<float>(tmp4.y) * input_deQFactor;
  tmp = static_cast<float>(__ldg(bias_ptr+threadIdx4)) + scale;
  tmp4.y = float_to_int8_rn(tmp*output_scale);

  threadIdx4 = threadIdx4+1;
  scale = static_cast<float>(tmp4.z) * input_deQFactor;
  tmp = static_cast<float>(__ldg(bias_ptr+threadIdx4)) + scale;
  tmp4.z = float_to_int8_rn(tmp*output_scale);

  threadIdx4 = threadIdx4+1;
  scale = static_cast<float>(tmp4.w) * input_deQFactor;;
  tmp = static_cast<float>(__ldg(bias_ptr+threadIdx4)) + scale;
  tmp4.w = float_to_int8_rn(tmp*output_scale);


  //row_id, col_id of sub-matrix (m = seq_len/seq_len_padded, n = size_per_head), column-major

  int row_id = word_id;
  int col_id = id_in_head;
  //new (row, rol) of LtTrans COL32/COL4 sub-matrix, leading dim = (COL32_ * seq_len / COL32_ * seq_len_padded)
  int new_col = col_id >> 5;
  int new_row;  
  if (use_ORDER_COL32_2R_4R4)
  {
    int row_in_tile = row_id & 31;
    int col_in_tile = col_id & 31; 
    new_row = (qk_id != 1) ?
              //COL32
              ((row_id << 5) + (col_id&31))
            :
               //COL32_2R_4R4
               (
               ((row_id >> 5) << 10) +
               //(((row%8)/2*4+row/8)*2+row%2)*32+col
               (((((((row_in_tile&7)>>1)<<2)+(row_in_tile>>3))<<1)+(row_in_tile&1))<<5)+col_in_tile
               )
               ;
  }
  else
  {
    new_row = (qk_id != 1) ?
              //COL32
              ((row_id << 5) + (col_id&31))
            :
              //COL4
              ////row_id/8 is the number of tile of (8 rows 32 columns) -- column-major
              ////row_id%2 is even row, otherwise odd row
              ////col_id%COL32_/8 is the number tile of (8 rows 8 columns)
              (
              ((((row_id >> 3) << 3) + ((row_id&1) << 2) + ((col_id&31) >> 3)) << 5) +
              ////col_id%8 >= 4 is the right half of (8 rows 8 columns) tile
              ////(row_id%8/2) is (the row id of alternating 4 rows) - 1
              (((((col_id&7) >= 4)?4:0) + ((row_id&7) >> 1)) << 2) +
              ////col_id%4 is the id of 4 cols
              (col_id&3)
              )
              ;
  }

  const int act_seq_len = (qk_id == 0) ? seq_len : seq_len_padded;
  const int stride = (qk_id == 0) ? stride_q : stride_k;
  buf_ptr4[(((batch_id*head_num + head_id) * stride + (new_col << 5)*act_seq_len + new_row) >> 2)] = tmp4;
}   

//add_QK_bias_transform for batch int8 hipblasLtMatmul & per axis quantization for weight
//1.add QK bias
//2.transform each Q K CUBLASLT_ORDER_COL32 matrixes into a series of sub-matrix (with CUBLASLT_ORDER_COL32/CUBLASLT_ORDER_COL4_4R2_8C layout)
//  Q, K are CUBLASLT_ORDER_COL32 matrixes of m = batch_size * seq_len, n = head_num * size_per_head
//  q_buf_ is of batchCount = batch_size * head_num, m = seq_len, n = size_per_head, CUBLASLT_ORDER_COL32
//  k_buf_ is of batchCount = batch_size * head_num, m = seq_len, n = size_per_head, CUBLASLT_ORDER_COL4_4R2_8C
//only for int8 IO
//seq_len, size_per_head must be a multiple of 32
//grid.x = batch_size * seq_len * 2;
//block.x = head_num * size_per_head / 4;
//using char4
template <typename T>
__global__
void add_QK_bias_transform(int8_t *q_buf_, int8_t *k_buf_, const int8_t* Q, const T* bias_Q, 
                           const int8_t* K, const T* bias_K, const int m, const int batch_size, 
                           const int seq_len, const int head_num, const int size_per_head, int stride, 
                           const float *q_input_deQFactor_ptr, const float *k_input_deQFactor_ptr, const float *q_output_scale_ptr, const float *k_output_scale_ptr,
                           bool use_ORDER_COL32_2R_4R4)
{
  const char4* data_ptr;
  char4* buf_ptr4;
  const T* bias_ptr;
  int qk_id = blockIdx.x / m;

  data_ptr = qk_id == 0 ? (const char4*)Q : (const char4*)K;
  buf_ptr4 = qk_id == 0 ? (char4*)q_buf_ : (char4*)k_buf_;
  bias_ptr = qk_id == 0 ? bias_Q : bias_K;
  const float input_deQFactor = qk_id == 0 ? __ldg(q_input_deQFactor_ptr) : __ldg(k_input_deQFactor_ptr);
  const float output_scale = qk_id == 0 ? __ldg(q_output_scale_ptr) : __ldg(k_output_scale_ptr);

  int threadIdx4 = threadIdx.x << 2;
  int batch_id = (blockIdx.x % m) / seq_len;
  int head_id = threadIdx4 / size_per_head;
  int id_in_head = threadIdx4 % size_per_head;
  int word_id = blockIdx.x % seq_len;

  int data_id = (((threadIdx4 >> 5) << 5)*m + ((blockIdx.x%m) << 5) + (threadIdx4&31)) >> 2;

  float scale;
  float tmp;
  char4 tmp4 = __ldg(data_ptr+data_id);
  scale = static_cast<float>(tmp4.x) * input_deQFactor;
  tmp = static_cast<float>(__ldg(bias_ptr+threadIdx4)) + scale;
  tmp4.x = float_to_int8_rn(tmp*output_scale);

  threadIdx4 = threadIdx4+1;
  scale = static_cast<float>(tmp4.y) * input_deQFactor;
  tmp = static_cast<float>(__ldg(bias_ptr+threadIdx4)) + scale;
  tmp4.y = float_to_int8_rn(tmp*output_scale);

  threadIdx4 = threadIdx4+1;
  scale = static_cast<float>(tmp4.z) * input_deQFactor;
  tmp = static_cast<float>(__ldg(bias_ptr+threadIdx4)) + scale;
  tmp4.z = float_to_int8_rn(tmp*output_scale);

  threadIdx4 = threadIdx4+1;
  scale = static_cast<float>(tmp4.w) * input_deQFactor;;
  tmp = static_cast<float>(__ldg(bias_ptr+threadIdx4)) + scale;
  tmp4.w = float_to_int8_rn(tmp*output_scale);


  //row_id, col_id of sub-matrix (m = seq_len, n = size_per_head), column-major

  int row_id = word_id;
  int col_id = id_in_head;
  //new (row, rol) of LtTrans COL32/COL4 sub-matrix, leading dim = (COL32_ * seq_len)
  int new_col = col_id >> 5;
  int new_row;  
  if (use_ORDER_COL32_2R_4R4)
  {
    int row_in_tile = row_id & 31;
    int col_in_tile = col_id & 31; 
    new_row = (qk_id != 1) ?
              //COL32
              ((row_id << 5) + (col_id&31))
            :
               //COL32_2R_4R4
               (
               ((row_id >> 5) << 10) +
               //(((row%8)/2*4+row/8)*2+row%2)*32+col
               (((((((row_in_tile&7)>>1)<<2)+(row_in_tile>>3))<<1)+(row_in_tile&1))<<5)+col_in_tile
               )
               ;
  }
  else
  {
    new_row = (qk_id != 1) ?
              //COL32
              ((row_id << 5) + (col_id&31))
            :
              //COL4
              ////row_id/8 is the number of tile of (8 rows 32 columns) -- column-major
              ////row_id%2 is even row, otherwise odd row
              ////col_id%COL32_/8 is the number tile of (8 rows 8 columns)
              (
              ((((row_id >> 3) << 3) + ((row_id&1) << 2) + ((col_id&31) >> 3)) << 5) +
              ////col_id%8 >= 4 is the right half of (8 rows 8 columns) tile
              ////(row_id%8/2) is (the row id of alternating 4 rows) - 1
              (((((col_id&7) >= 4)?4:0) + ((row_id&7) >> 1)) << 2) +
              ////col_id%4 is the id of 4 cols
              (col_id&3)
              )
              ;
  }

  buf_ptr4[(((batch_id*head_num + head_id) * stride + (new_col << 5)*seq_len + new_row) >> 2)] = tmp4;
}   


template <typename T>
void add_QK_bias_transform_kernelLauncher(int8_t *q_buf, int8_t *k_buf, const int8_t* Q, const T* bias_Q, 
                                          const int8_t* K, const T* bias_K, const int batch_size, 
                                          const int seq_len, const int head_num, const int size_per_head, 
                                          const float *q_input_deQFactor_ptr, const float *k_input_deQFactor_ptr, 
                                          const float *q_output_scale_ptr, const float *k_output_scale_ptr,
                                          bool use_ORDER_COL32_2R_4R4, hipStream_t stream) 
{
  assert(size_per_head % 32 == 0);
  if (seq_len % 32 == 0)
  {
    add_QK_bias_transform_varlen<<<dim3(batch_size*seq_len*2), dim3((head_num*size_per_head)/4), 0, stream>>>(
      q_buf, k_buf, Q, bias_Q, K, bias_K, 
      batch_size * seq_len, batch_size, seq_len, head_num, size_per_head, 
      seq_len, seq_len*size_per_head, seq_len*size_per_head,
      q_input_deQFactor_ptr, k_input_deQFactor_ptr, q_output_scale_ptr, k_output_scale_ptr,
      use_ORDER_COL32_2R_4R4);
  }
  else
  {
    int seq_len_padded = (seq_len + 31)/32*32;
    //The padding words will not be considered in softmax, so we don't need memset for k_buf_ 
    //hipMemsetAsync(k_buf, 0, batch_size * head_num * seq_len_padded * size_per_head * sizeof(int8_t), stream);
    add_QK_bias_transform_varlen<<<dim3(batch_size*seq_len*2), dim3((head_num*size_per_head)/4), 0, stream>>>(
      q_buf, k_buf, Q, bias_Q, K, bias_K, 
      batch_size * seq_len, batch_size, seq_len, head_num, size_per_head, 
      seq_len_padded, seq_len*size_per_head, seq_len_padded*size_per_head,
      q_input_deQFactor_ptr, k_input_deQFactor_ptr, q_output_scale_ptr, k_output_scale_ptr,
      use_ORDER_COL32_2R_4R4);
  }
}

template
void add_QK_bias_transform_kernelLauncher(int8_t *q_buf, int8_t *k_buf, const int8_t* Q, const float* bias_Q, 
                                          const int8_t* K, const float* bias_K, const int batch_size, 
                                          const int seq_len, const int head_num, const int size_per_head, 
                                          const float *q_input_deQFactor_ptr, const float *k_input_deQFactor_ptr, 
                                          const float *q_output_scale_ptr, const float *k_output_scale_ptr,
                                          bool use_ORDER_COL32_2R_4R4, hipStream_t stream);
                                          
template
void add_QK_bias_transform_kernelLauncher(int8_t *q_buf, int8_t *k_buf, const int8_t* Q, const half* bias_Q, 
                                          const int8_t* K, const half* bias_K, const int batch_size, 
                                          const int seq_len, const int head_num, const int size_per_head, 
                                          const float *q_input_deQFactor_ptr, const float *k_input_deQFactor_ptr, 
                                          const float *q_output_scale_ptr, const float *k_output_scale_ptr,
                                          bool use_ORDER_COL32_2R_4R4, hipStream_t stream);

//add_QK_bias_transform & rebuild padding for batch int8 hipblasLtMatmul & per axis quantization for weight
//1.add QK bias
//2.transform each Q K CUBLASLT_ORDER_COL32 matrixes into a series of sub-matrix (with CUBLASLT_ORDER_COL32/CUBLASLT_ORDER_COL4_4R2_8C layout)
//  Q, K are CUBLASLT_ORDER_COL32 matrixes of m = valid_word_num, n = head_num * size_per_head
//  q_buf_ is of batchCount = batch_size * head_num, m = seq_len, n = size_per_head, CUBLASLT_ORDER_COL32
//  k_buf_ is of batchCount = batch_size * head_num, m = seq_len, n = size_per_head, CUBLASLT_ORDER_COL4_4R2_8C or CUBLASLT_ORDER_COL32_2R_4R4
//only for int32 input & int8 output
//seq_len, size_per_head must be a multiple of 32
//grid.x = valid_word_num * 2;
//block.x = head_num * size_per_head / 4;
//using char4
template <typename T>
__global__
void add_QK_bias_transform_rebuild_padding(int8_t *q_buf_, int8_t *k_buf_, const int32_t* Q, const T* bias_Q, 
                                           const int32_t* K, const T* bias_K, const int* sequence_id_offset, 
                                           const int valid_word_num, const int m, const int batch_size, const int seq_len, 
                                           const int head_num, const int size_per_head, int stride, const float * q_weight_amax, 
                                           const float *q_input_deQFactor_div127_ptr, const float * k_weight_amax, 
                                           const float *k_input_deQFactor_div127_ptr, const float *q_output_scale_ptr, const float *k_output_scale_ptr,
                                           bool use_ORDER_COL32_2R_4R4)
{
  const int32_t* data_ptr;
  char4* buf_ptr4;
  const T* bias_ptr;
  const float* weight_amax;
  int qk_id = blockIdx.x / valid_word_num;

  data_ptr = qk_id == 0 ? Q : K;
  buf_ptr4 = qk_id == 0 ? (char4*)q_buf_ : (char4*)k_buf_;
  bias_ptr = qk_id == 0 ? bias_Q : bias_K;
  
  int threadIdx4 = threadIdx.x << 2;
  int m_full_idx = blockIdx.x % valid_word_num;
  m_full_idx = (valid_word_num != m) ? (m_full_idx + __ldg(sequence_id_offset+m_full_idx)) : m_full_idx;
  int batch_id = m_full_idx / seq_len;
  int head_id = threadIdx4 / size_per_head;
  int id_in_head = threadIdx4 % size_per_head;
  int word_id = m_full_idx % seq_len;
  
  const float input_deQFactor_div127 = qk_id == 0 ? __ldg(q_input_deQFactor_div127_ptr) : __ldg(k_input_deQFactor_div127_ptr);
  weight_amax = qk_id == 0 ? q_weight_amax : k_weight_amax;
  const float output_scale = qk_id == 0 ? __ldg(q_output_scale_ptr) : __ldg(k_output_scale_ptr);

  int data_id = (((threadIdx4 >> 5) << 5)*valid_word_num + ((blockIdx.x%valid_word_num) << 5) + (threadIdx4&31));
    
  float scale;
  float tmp;
  char4 tmp4;
  scale = static_cast<float>(__ldg(data_ptr+data_id)) * __ldg(weight_amax+threadIdx4) * input_deQFactor_div127;
  tmp = static_cast<float>(__ldg(bias_ptr+threadIdx4)) + scale;
  tmp4.x = float_to_int8_rn(tmp*output_scale);

  data_id = data_id+1;
  threadIdx4 = threadIdx4+1;
  scale = static_cast<float>(__ldg(data_ptr+data_id)) * __ldg(weight_amax+threadIdx4)* input_deQFactor_div127;
  tmp = static_cast<float>(__ldg(bias_ptr+threadIdx4)) + scale;
  tmp4.y = float_to_int8_rn(tmp*output_scale);

  data_id = data_id+1;
  threadIdx4 = threadIdx4+1;
  scale = static_cast<float>(__ldg(data_ptr+data_id)) * __ldg(weight_amax+threadIdx4) * input_deQFactor_div127;
  tmp = static_cast<float>(__ldg(bias_ptr+threadIdx4)) + scale;
  tmp4.z = float_to_int8_rn(tmp*output_scale);

  data_id = data_id+1;
  threadIdx4 = threadIdx4+1;
  scale = static_cast<float>(__ldg(data_ptr+data_id)) * __ldg(weight_amax+threadIdx4) * input_deQFactor_div127;
  tmp = static_cast<float>(__ldg(bias_ptr+threadIdx4)) + scale;
  tmp4.w = float_to_int8_rn(tmp*output_scale);

  //row_id, col_id of sub-matrix (m = seq_len, n = size_per_head), column-major
  int row_id = word_id;
  int col_id = id_in_head;
  //new (row, rol) of LtTrans COL32/COL4 sub-matrix, leading dim = (COL32_ * seq_len)
  int new_col = col_id >> 5;
  int new_row; 
  if (use_ORDER_COL32_2R_4R4)
  {
    int row_in_tile = row_id & 31;
    int col_in_tile = col_id & 31; 
    new_row = (qk_id != 1) ?
              //COL32
              ((row_id << 5) + (col_id&31))
            :
              //COL32_2R_4R4
              (
              ((row_id >> 5) << 10) +
              //(((row%8)/2*4+row/8)*2+row%2)*32+col
              (((((((row_in_tile&7)>>1)<<2)+(row_in_tile>>3))<<1)+(row_in_tile&1))<<5)+col_in_tile
              )
              ;
  }
  else
  {
    new_row = (qk_id != 1) ?
              //COL32
              ((row_id << 5) + (col_id&31))
            :
              //COL4
              ////row_id/8 is the number of tile of (8 rows 32 columns) -- column-major
              ////row_id%2 is even row, otherwise odd row
              ////col_id%COL32_/8 is the number tile of (8 rows 8 columns)
              (
              ((((row_id >> 3) << 3) + ((row_id&1) << 2) + ((col_id&31) >> 3)) << 5) +
              ////col_id%8 >= 4 is the right half of (8 rows 8 columns) tile
              ////(row_id%8/2) is (the row id of alternating 4 rows) - 1
              (((((col_id&7) >= 4)?4:0) + ((row_id&7) >> 1)) << 2) +
              ////col_id%4 is the id of 4 cols
              (col_id&3)
              )
              ;
  }

  buf_ptr4[(((batch_id*head_num + head_id) * stride + (new_col << 5)*seq_len + new_row) >> 2)] = tmp4;
}

template <typename T>
void add_QK_bias_transform_rebuild_padding_kernelLauncher(int8_t *q_buf, int8_t *k_buf, 
                                                          const int32_t* Q, const T* bias_Q, 
                                                          const int32_t* K, const T* bias_K, 
                                                          const int* sequence_id_offset, const int valid_word_num, 
                                                          const int batch_size, const int seq_len, 
                                                          const int head_num, const int size_per_head, 
                                                          const float * q_weight_amax, 
                                                          const float *q_input_deQFactor_div127_ptr, 
                                                          const float * k_weight_amax, 
                                                          const float *k_input_deQFactor_div127_ptr, 
                                                          const float *q_output_scale_ptr, const float *k_output_scale_ptr,
                                                          bool use_ORDER_COL32_2R_4R4, hipStream_t stream)
{
  add_QK_bias_transform_rebuild_padding<<<dim3(valid_word_num*2), dim3((head_num*size_per_head)/4), 0, stream>>>(
    q_buf, k_buf, Q, bias_Q, K, bias_K, 
    sequence_id_offset, valid_word_num, 
    batch_size*seq_len, batch_size, seq_len, 
    head_num, size_per_head, seq_len*size_per_head, 
    q_weight_amax, q_input_deQFactor_div127_ptr, 
    k_weight_amax, k_input_deQFactor_div127_ptr, 
    q_output_scale_ptr, k_output_scale_ptr,
    use_ORDER_COL32_2R_4R4);
}  

template
void add_QK_bias_transform_rebuild_padding_kernelLauncher(int8_t *q_buf, int8_t *k_buf, 
                                                          const int32_t* Q, const float* bias_Q, 
                                                          const int32_t* K, const float* bias_K, 
                                                          const int* sequence_id_offset, const int valid_word_num, 
                                                          const int batch_size, const int seq_len, 
                                                          const int head_num, const int size_per_head, 
                                                          const float * q_weight_amax, 
                                                          const float *q_input_deQFactor_div127_ptr, 
                                                          const float * k_weight_amax, 
                                                          const float *k_input_deQFactor_div127_ptr, 
                                                          const float *q_output_scale_ptr, const float *k_output_scale_ptr,
                                                          bool use_ORDER_COL32_2R_4R4, hipStream_t stream);
                                                          
template
void add_QK_bias_transform_rebuild_padding_kernelLauncher(int8_t *q_buf, int8_t *k_buf, 
                                                          const int32_t* Q, const half* bias_Q, 
                                                          const int32_t* K, const half* bias_K, 
                                                          const int* sequence_id_offset, const int valid_word_num, 
                                                          const int batch_size, const int seq_len, 
                                                          const int head_num, const int size_per_head, 
                                                          const float * q_weight_amax, 
                                                          const float *q_input_deQFactor_div127_ptr, 
                                                          const float * k_weight_amax, 
                                                          const float *k_input_deQFactor_div127_ptr, 
                                                          const float *q_output_scale_ptr, const float *k_output_scale_ptr,
                                                          bool use_ORDER_COL32_2R_4R4, hipStream_t stream);  

//add_QK_bias_transform & rebuild padding for batch int8 hipblasLtMatmul & per tensor quantization for weight
//1.add QK bias
//2.transform each Q K CUBLASLT_ORDER_COL32 matrixes into a series of sub-matrix (with CUBLASLT_ORDER_COL32/CUBLASLT_ORDER_COL4_4R2_8C layout)
//  Q, K are CUBLASLT_ORDER_COL32 matrixes of m = valid_word_num, n = head_num * size_per_head
//  q_buf_ is of batchCount = batch_size * head_num, m = seq_len, n = size_per_head, CUBLASLT_ORDER_COL32
//  k_buf_ is of batchCount = batch_size * head_num, m = seq_len, n = size_per_head, CUBLASLT_ORDER_COL4_4R2_8C or CUBLASLT_ORDER_COL32_2R_4R4
//only for int8 IO
//seq_len, size_per_head must be a multiple of 32
//grid.x = valid_word_num * 2;
//block.x = head_num * size_per_head / 4;
//using char4
template <typename T>
__global__
void add_QK_bias_transform_rebuild_padding(int8_t *q_buf_, int8_t *k_buf_, const int8_t* Q, const T* bias_Q, 
                                           const int8_t* K, const T* bias_K, const int* sequence_id_offset, 
                                           const int valid_word_num, const int m, const int batch_size, const int seq_len, 
                                           const int head_num, const int size_per_head, int stride,  
                                           const float *q_deQFactor_ptr,  const float *k_deQFactor_ptr, 
                                           const float *q_output_scale_ptr, const float *k_output_scale_ptr,
                                           bool use_ORDER_COL32_2R_4R4)
{
  const char4* data_ptr;
  char4* buf_ptr4;
  const T* bias_ptr;
  int qk_id = blockIdx.x / valid_word_num;

  data_ptr = qk_id == 0 ? (const char4*)Q : (const char4*)K;
  buf_ptr4 = qk_id == 0 ? (char4*)q_buf_ : (char4*)k_buf_;
  bias_ptr = qk_id == 0 ? bias_Q : bias_K;
  
  int threadIdx4 = threadIdx.x << 2;
  int m_full_idx = blockIdx.x % valid_word_num;
  m_full_idx = (valid_word_num != m) ? (m_full_idx + __ldg(sequence_id_offset+m_full_idx)) : m_full_idx;
  int batch_id = m_full_idx / seq_len;
  int head_id = threadIdx4 / size_per_head;
  int id_in_head = threadIdx4 % size_per_head;
  int word_id = m_full_idx % seq_len;
  
  const float deQFactor = qk_id == 0 ? __ldg(q_deQFactor_ptr) : __ldg(k_deQFactor_ptr);
  const float output_scale = qk_id == 0 ? __ldg(q_output_scale_ptr) : __ldg(k_output_scale_ptr);

  int data_id = (((threadIdx4 >> 5) << 5)*valid_word_num + ((blockIdx.x%valid_word_num) << 5) + (threadIdx4&31)) >> 2;
    
  float scale;
  float tmp;
  char4 tmp4;
  
  tmp4 = __ldg(data_ptr+data_id);
  
  scale = static_cast<float>(tmp4.x) * deQFactor;
  tmp = static_cast<float>(__ldg(bias_ptr+threadIdx4)) + scale;
  tmp4.x = float_to_int8_rn(tmp*output_scale);

  threadIdx4 = threadIdx4+1;
  scale = static_cast<float>(tmp4.y) * deQFactor;
  tmp = static_cast<float>(__ldg(bias_ptr+threadIdx4)) + scale;
  tmp4.y = float_to_int8_rn(tmp*output_scale);

  threadIdx4 = threadIdx4+1;
  scale = static_cast<float>(tmp4.z) * deQFactor;
  tmp = static_cast<float>(__ldg(bias_ptr+threadIdx4)) + scale;
  tmp4.z = float_to_int8_rn(tmp*output_scale);

  threadIdx4 = threadIdx4+1;
  scale = static_cast<float>(tmp4.w) * deQFactor;
  tmp = static_cast<float>(__ldg(bias_ptr+threadIdx4)) + scale;
  tmp4.w = float_to_int8_rn(tmp*output_scale);

  //row_id, col_id of sub-matrix (m = seq_len, n = size_per_head), column-major
  int row_id = word_id;
  int col_id = id_in_head;
  //new (row, rol) of LtTrans COL32/COL4 sub-matrix, leading dim = (COL32_ * seq_len)
  int new_col = col_id >> 5;
  int new_row; 
  if (use_ORDER_COL32_2R_4R4)
  {
    int row_in_tile = row_id & 31;
    int col_in_tile = col_id & 31; 
    new_row = (qk_id != 1) ?
              //COL32
              ((row_id << 5) + (col_id&31))
            :
              //COL32_2R_4R4
              (
              ((row_id >> 5) << 10) +
              //(((row%8)/2*4+row/8)*2+row%2)*32+col
              (((((((row_in_tile&7)>>1)<<2)+(row_in_tile>>3))<<1)+(row_in_tile&1))<<5)+col_in_tile
              )
              ;
  }
  else
  {
    new_row = (qk_id != 1) ?
              //COL32
              ((row_id << 5) + (col_id&31))
            :
              //COL4
              ////row_id/8 is the number of tile of (8 rows 32 columns) -- column-major
              ////row_id%2 is even row, otherwise odd row
              ////col_id%COL32_/8 is the number tile of (8 rows 8 columns)
              (
              ((((row_id >> 3) << 3) + ((row_id&1) << 2) + ((col_id&31) >> 3)) << 5) +
              ////col_id%8 >= 4 is the right half of (8 rows 8 columns) tile
              ////(row_id%8/2) is (the row id of alternating 4 rows) - 1
              (((((col_id&7) >= 4)?4:0) + ((row_id&7) >> 1)) << 2) +
              ////col_id%4 is the id of 4 cols
              (col_id&3)
              )
              ;
  }

  buf_ptr4[(((batch_id*head_num + head_id) * stride + (new_col << 5)*seq_len + new_row) >> 2)] = tmp4;
}


//add_QK_bias_transform & rebuild padding for batch int8 hipblasLtMatmul & per tensor quantization for weight
//1.add QK bias
//2.transform each Q K CUBLASLT_ORDER_COL32 matrixes into a series of sub-matrix (with CUBLASLT_ORDER_COL32/CUBLASLT_ORDER_COL4_4R2_8C layout)
//  Q, K are CUBLASLT_ORDER_COL32 matrixes of m = valid_word_num, n = head_num * size_per_head
//  q_buf_ is of batchCount = batch_size * head_num, m = seq_len, n = size_per_head, CUBLASLT_ORDER_COL32
//  seq_len_padded = (seq_len + 31)/32*32;
//  k_buf_ is of batchCount = batch_size * head_num, m = seq_len_padded, n = size_per_head, CUBLASLT_ORDER_COL4_4R2_8C or CUBLASLT_ORDER_COL32_2R_4R4
//only for int8 IO
//seq_len, size_per_head must be a multiple of 32
//grid.x = valid_word_num * 2;
//block.x = head_num * size_per_head / 4;
//using char4
template <typename T>
__global__
void add_QK_bias_transform_rebuild_padding_varlen(int8_t *q_buf_, int8_t *k_buf_, const int8_t* Q, const T* bias_Q,
                                                  const int8_t* K, const T* bias_K, const int* sequence_id_offset,
                                                  const int valid_word_num, const int m, const int batch_size, 
                                                  const int seq_len, const int seq_len_padded, const int head_num,  
                                                  const int size_per_head, int stride_q, int stride_k,
                                                  const float *q_deQFactor_ptr,  const float *k_deQFactor_ptr,
                                                  const float *q_output_scale_ptr, const float *k_output_scale_ptr,
                                                  bool use_ORDER_COL32_2R_4R4)
{
  const char4* data_ptr;
  char4* buf_ptr4;
  const T* bias_ptr;
  int qk_id = blockIdx.x / valid_word_num;

  data_ptr = qk_id == 0 ? (const char4*)Q : (const char4*)K;
  buf_ptr4 = qk_id == 0 ? (char4*)q_buf_ : (char4*)k_buf_;
  bias_ptr = qk_id == 0 ? bias_Q : bias_K;

  int threadIdx4 = threadIdx.x << 2;
  int m_full_idx = blockIdx.x % valid_word_num;
  m_full_idx = (valid_word_num != m) ? (m_full_idx + __ldg(sequence_id_offset+m_full_idx)) : m_full_idx;
  int batch_id = m_full_idx / seq_len;
  int head_id = threadIdx4 / size_per_head;
  int id_in_head = threadIdx4 % size_per_head;
  int word_id = m_full_idx % seq_len;

  const float deQFactor = qk_id == 0 ? __ldg(q_deQFactor_ptr) : __ldg(k_deQFactor_ptr);
  const float output_scale = qk_id == 0 ? __ldg(q_output_scale_ptr) : __ldg(k_output_scale_ptr);

  int data_id = (((threadIdx4 >> 5) << 5)*valid_word_num + ((blockIdx.x%valid_word_num) << 5) + (threadIdx4&31)) >> 2;

  float scale;
  float tmp;
  char4 tmp4;

  tmp4 = __ldg(data_ptr+data_id);

  scale = static_cast<float>(tmp4.x) * deQFactor;
  tmp = static_cast<float>(__ldg(bias_ptr+threadIdx4)) + scale;
  tmp4.x = float_to_int8_rn(tmp*output_scale);

  threadIdx4 = threadIdx4+1;
  scale = static_cast<float>(tmp4.y) * deQFactor;
  tmp = static_cast<float>(__ldg(bias_ptr+threadIdx4)) + scale;
  tmp4.y = float_to_int8_rn(tmp*output_scale);

  threadIdx4 = threadIdx4+1;
  scale = static_cast<float>(tmp4.z) * deQFactor;
  tmp = static_cast<float>(__ldg(bias_ptr+threadIdx4)) + scale;
  tmp4.z = float_to_int8_rn(tmp*output_scale);

  threadIdx4 = threadIdx4+1;
  scale = static_cast<float>(tmp4.w) * deQFactor;
  tmp = static_cast<float>(__ldg(bias_ptr+threadIdx4)) + scale;
  tmp4.w = float_to_int8_rn(tmp*output_scale);

  //row_id, col_id of sub-matrix (m = seq_len or seq_len_padded, n = size_per_head), column-major
  int row_id = word_id;
  int col_id = id_in_head;
  //new (row, rol) of LtTrans COL32/COL4 sub-matrix, leading dim = (COL32_ * seq_len) or (COL32_ * seq_len_padded)
  int new_col = col_id >> 5;
  int new_row;
  if (use_ORDER_COL32_2R_4R4)
  {
    int row_in_tile = row_id & 31;
    int col_in_tile = col_id & 31;
    new_row = (qk_id != 1) ?
              //COL32
              ((row_id << 5) + (col_id&31))
            :
              //COL32_2R_4R4
              (
              ((row_id >> 5) << 10) +
              //(((row%8)/2*4+row/8)*2+row%2)*32+col
              (((((((row_in_tile&7)>>1)<<2)+(row_in_tile>>3))<<1)+(row_in_tile&1))<<5)+col_in_tile
              )
              ;
  }
  else
  {
    new_row = (qk_id != 1) ?
              //COL32
              ((row_id << 5) + (col_id&31))
            :
              //COL4
              ////row_id/8 is the number of tile of (8 rows 32 columns) -- column-major
              ////row_id%2 is even row, otherwise odd row
              ////col_id%COL32_/8 is the number tile of (8 rows 8 columns)
              (
              ((((row_id >> 3) << 3) + ((row_id&1) << 2) + ((col_id&31) >> 3)) << 5) +
              ////col_id%8 >= 4 is the right half of (8 rows 8 columns) tile
              ////(row_id%8/2) is (the row id of alternating 4 rows) - 1
              (((((col_id&7) >= 4)?4:0) + ((row_id&7) >> 1)) << 2) +
              ////col_id%4 is the id of 4 cols
              (col_id&3)
              )
              ;
  }

  const int stride = (qk_id != 1) ? stride_q : stride_k;
  const int len = (qk_id != 1) ? seq_len : seq_len_padded;
  buf_ptr4[(((batch_id*head_num + head_id) * stride + (new_col << 5)*len + new_row) >> 2)] = tmp4;
}

template <typename T>
void add_QK_bias_transform_rebuild_padding_kernelLauncher(int8_t *q_buf, int8_t *k_buf, const int8_t* Q, const T* bias_Q, 
                                                          const int8_t* K, const T* bias_K, const int* sequence_id_offset, 
                                                          const int valid_word_num, 
                                                          const int batch_size, const int seq_len, 
                                                          const int head_num, const int size_per_head,  
                                                          const float *q_deQFactor_ptr,  const float *k_deQFactor_ptr, 
                                                          const float *q_output_scale_ptr, const float *k_output_scale_ptr,
                                                          bool use_ORDER_COL32_2R_4R4, hipStream_t stream)
{
  int seq_len_padded = (seq_len + 31)/32*32;
  add_QK_bias_transform_rebuild_padding_varlen<<<dim3(valid_word_num*2), dim3((head_num*size_per_head)/4), 0, stream>>>(
    q_buf, k_buf, Q, bias_Q, K, bias_K, 
    sequence_id_offset, valid_word_num, 
    batch_size * seq_len, batch_size, 
    seq_len, seq_len_padded, head_num, size_per_head, 
    seq_len*size_per_head, seq_len_padded*size_per_head,
    q_deQFactor_ptr, k_deQFactor_ptr, 
    q_output_scale_ptr, k_output_scale_ptr, 
    use_ORDER_COL32_2R_4R4);
}

template
void add_QK_bias_transform_rebuild_padding_kernelLauncher(int8_t *q_buf, int8_t *k_buf, 
                                                          const int8_t* Q, const float* bias_Q, 
                                                          const int8_t* K, const float* bias_K, 
                                                          const int* sequence_id_offset, const int valid_word_num, 
                                                          const int batch_size, const int seq_len, 
                                                          const int head_num, const int size_per_head,  
                                                          const float *q_deQFactor_ptr,  const float *k_deQFactor_ptr, 
                                                          const float *q_output_scale_ptr, const float *k_output_scale_ptr,
                                                          bool use_ORDER_COL32_2R_4R4, hipStream_t stream);
                                                          
template
void add_QK_bias_transform_rebuild_padding_kernelLauncher(int8_t *q_buf, int8_t *k_buf, 
                                                          const int8_t* Q, const half* bias_Q, 
                                                          const int8_t* K, const half* bias_K, 
                                                          const int* sequence_id_offset, const int valid_word_num, 
                                                          const int batch_size, const int seq_len, 
                                                          const int head_num, const int size_per_head,  
                                                          const float *q_deQFactor_ptr,  const float *k_deQFactor_ptr, 
                                                          const float *q_output_scale_ptr, const float *k_output_scale_ptr,
                                                          bool use_ORDER_COL32_2R_4R4, hipStream_t stream);

//input matrix a matrix of m = batch_size*seq_len , n = head_num*size_per_head, CUBLASLT_ORDER_COL32
//output matrixes are a series of sub-matrixes with size of m = size_per_head, n = seq_len , CUBLASLT_ORDER_COL4_4R2_8C or CUBLASLT_ORDER_COL32_2R_4R4
//only for int32_t Input int8_t Output
//seq_len, size_per_head must be a multiple of 32
//grid = (size_per_head/32, seq_len/32, batch_size*head_num)
//block = (8, 32);
//using char4
//per axis quantization for weight
template <typename T>
__global__
void add_V_bias_transform(int8_t *v_buf_, const int32_t *V, const T *V_bias, const int batch_size, const int seq_len, 
                          const int head_num, const int size_per_head, int stride, const float* weight_amax, 
                          const float *input_deQFactor_div127_ptr, const float *out_scale_ptr, bool use_ORDER_COL32_2R_4R4)
{
  const float input_deQFactor_div127 = __ldg(input_deQFactor_div127_ptr);
  const float out_scale = __ldg(out_scale_ptr);
  __shared__ int8_t shm[32][33];
  const int32_t* data_ptr = V;
  char4* buf_ptr4 = (char4*) v_buf_;
  const T* bias_ptr = V_bias;

  int threadIdx4 = threadIdx.x << 2;

  //for src of (seq_len, size_per_head)
  int batch_id = blockIdx.z/head_num;
  int head_id = blockIdx.z%head_num;
  int word_id = (blockIdx.y << 5) + threadIdx.y;
  int id_in_size = (blockIdx.x << 5) + threadIdx4;

  //for V layout (batch_size*seq_len, head_num*size_per_head)
  int col = head_id*size_per_head + id_in_size;
  int row = batch_id*seq_len + word_id;
  int inIdx = (((col >> 5) << 5)*batch_size*seq_len + ((row << 5) + (col&31)));
  //for shm row-major
  int sh_col = threadIdx4;
  int sh_row = threadIdx.y;
  
  float tmp;
  float scale;

  //const half2* bias_ptr2 = (const half2*)bias_ptr;
  //half2 tmp2;

  //tmp2 = __ldg(&bias_ptr2[col >> 1]);
  
  scale = __ldg(data_ptr + inIdx) * __ldg(weight_amax + col) * input_deQFactor_div127;
  tmp = scale + static_cast<float>(__ldg(bias_ptr + col));//(tmp2.x);
  shm[sh_row][sh_col] = float_to_int8_rn(tmp*out_scale);
  
  scale = __ldg(data_ptr + inIdx + 1) * __ldg(weight_amax + col + 1) * input_deQFactor_div127;
  tmp = scale + static_cast<float>(__ldg(bias_ptr+col+1));//(tmp2.y);
  shm[sh_row][sh_col+1] = float_to_int8_rn(tmp*out_scale);
  
  //tmp2 = __ldg(&bias_ptr2[(col >> 1) + 1]);

  scale = __ldg(data_ptr+inIdx+2) * __ldg(weight_amax+col+2) * input_deQFactor_div127;
  tmp = scale + static_cast<float>(__ldg(bias_ptr+col+2));//(tmp2.x);
  shm[sh_row][sh_col+2] = float_to_int8_rn(tmp*out_scale);
  
  scale = __ldg(data_ptr+inIdx + 3) * __ldg(weight_amax+col+3) * input_deQFactor_div127;
  tmp = scale + static_cast<float>(__ldg(bias_ptr+col+3));//(tmp2.y);
  shm[sh_row][sh_col+3] = float_to_int8_rn(tmp*out_scale);

  __syncthreads();

  //for dst of (size_per_head, seq_len)
  word_id = (blockIdx.y << 5) + threadIdx4;
  id_in_size = (blockIdx.x << 5) + threadIdx.y;
  col = (word_id >> 5);

  if (use_ORDER_COL32_2R_4R4)
  {
    int row_in_tile = id_in_size & 31;
    int col_in_tile = word_id & 31; 
    row = (
          //COL32_2R_4R4
          ((id_in_size >> 5) << 10) +
          //(((row%8)/2*4+row/8)*2+row%2)*32+col
          (((((((row_in_tile&7)>>1)<<2)+(row_in_tile>>3))<<1)+(row_in_tile&1))<<5)+col_in_tile
          );
  }
  else
  { 
    row = (
          //COL4
          ////id_in_size/8 is the number of tile of (8 rows 32 columns) -- column-major
          ////id_in_size%2 is even row, otherwise odd row
          ////word_id%COL32_/8 is the number tile of (8 rows 8 columns)
          ((((id_in_size >> 3) << 3) + ((id_in_size&1) << 2) + ((word_id&31) >> 3)) << 5) +
          ////word_id%8 >= 4 is the right half of (8 rows 8 columns) tile
          ////(id_in_size%8/2) is (the row id of alternating 4 rows) - 1
          (((((word_id&7) >= 4)?4:0) + ((id_in_size&7) >> 1)) << 2) +
          ////word_id%4 is the id of 4 cols
          (word_id&3)
          );
  }

        
  char4 dataTmp;
  dataTmp.x = shm[sh_col][sh_row];
  dataTmp.y = shm[sh_col+1][sh_row];
  dataTmp.z = shm[sh_col+2][sh_row];
  dataTmp.w = shm[sh_col+3][sh_row];
  buf_ptr4[(blockIdx.z*stride + (col << 5)*size_per_head + row) >> 2] = dataTmp;
}

template <>
__global__
void add_V_bias_transform(int8_t *v_buf_, const int32_t *V, const half *V_bias, const int batch_size, const int seq_len, 
                          const int head_num, const int size_per_head, int stride, const float* weight_amax, 
                          const float *input_deQFactor_div127_ptr, const float *out_scale_ptr, bool use_ORDER_COL32_2R_4R4)
{
  const float input_deQFactor_div127 = __ldg(input_deQFactor_div127_ptr);
  const float out_scale = __ldg(out_scale_ptr);
  __shared__ int8_t shm[32][33];
  const int32_t* data_ptr = V;
  char4* buf_ptr4 = (char4*) v_buf_;

  int threadIdx4 = threadIdx.x << 2;

  //for src of (seq_len, size_per_head)
  int batch_id = blockIdx.z/head_num;
  int head_id = blockIdx.z%head_num;
  
  int blockIdy32 = (blockIdx.y << 5);
  int blockIdx32 = (blockIdx.x << 5);
  int word_id = blockIdy32 + threadIdx.y;
  int id_in_size = blockIdx32 + threadIdx4;

  //for V layout (batch_size*seq_len, head_num*size_per_head)
  int col = head_id*size_per_head + id_in_size;
  int row = batch_id*seq_len + word_id;
  int inIdx = ((col & 0xffffffe0)*batch_size*seq_len + ((row << 5) + (col&31)));
  //for shm row-major
  int sh_col = threadIdx4;
  int sh_row = threadIdx.y;
  
  int col_2 = col >> 1;
  float scale;

  const half2* bias_ptr2 = (const half2*)V_bias;
  half2 tmp2;

  tmp2 = __ldg(bias_ptr2+col_2);
  
  scale = __ldg(data_ptr+inIdx) * __ldg(weight_amax+col) * input_deQFactor_div127;
  scale = scale + static_cast<float>(tmp2.x);
  shm[sh_row][sh_col] = float_to_int8_rn(scale*out_scale);
  
  scale = __ldg(data_ptr+inIdx+1) * __ldg(weight_amax+col+1) * input_deQFactor_div127;
  scale = scale + static_cast<float>(tmp2.y);
  shm[sh_row][sh_col+1] = float_to_int8_rn(scale*out_scale);
  
  tmp2 = __ldg(bias_ptr2 + col_2 + 1);

  scale = __ldg(data_ptr + inIdx + 2) * __ldg(weight_amax + col + 2) * input_deQFactor_div127;
  scale = scale + static_cast<float>(tmp2.x);
  shm[sh_row][sh_col+2] = float_to_int8_rn(scale*out_scale);
  
  scale = __ldg(data_ptr + inIdx + 3) * __ldg(weight_amax + col + 3) * input_deQFactor_div127;
  scale = scale + static_cast<float>(tmp2.y);
  shm[sh_row][sh_col+3] = float_to_int8_rn(scale*out_scale);

  __syncthreads();

  //for dst of (size_per_head, seq_len)
  word_id = blockIdy32 + threadIdx4;
  id_in_size = blockIdx32 + threadIdx.y;
  col = (word_id >> 5);

  if (use_ORDER_COL32_2R_4R4)
  {
    int row_in_tile = id_in_size & 31;
    int col_in_tile = word_id & 31; 
    row = (
          //COL32_2R_4R4
          ((id_in_size >> 5) << 10) +
          //(((row%8)/2*4+row/8)*2+row%2)*32+col
          (((((((row_in_tile&7)>>1)<<2)+(row_in_tile>>3))<<1)+(row_in_tile&1))<<5)+col_in_tile
          );
  }
  else
  { 
    row = (
          //COL4
          ////id_in_size/8 is the number of tile of (8 rows 32 columns) -- column-major
          ////id_in_size%2 is even row, otherwise odd row
          ////word_id%COL32_/8 is the number tile of (8 rows 8 columns)
          (((id_in_size & 0xfffffff8) + ((id_in_size&1) << 2) + ((word_id&31) >> 3)) << 5) +
          ////word_id%8 >= 4 is the right half of (8 rows 8 columns) tile
          ////(id_in_size%8/2) is (the row id of alternating 4 rows) - 1
          (((((word_id&7) >= 4)?4:0) + ((id_in_size&7) >> 1)) << 2) +
          ////word_id%4 is the id of 4 cols
          (word_id&3)
          );
  }
        
  char4 dataTmp;
  dataTmp.x = shm[sh_col][sh_row];
  dataTmp.y = shm[sh_col+1][sh_row];
  dataTmp.z = shm[sh_col+2][sh_row];
  dataTmp.w = shm[sh_col+3][sh_row];
  buf_ptr4[(blockIdx.z*stride + (col << 5)*size_per_head + row) >> 2] = dataTmp;
}

template <typename T>
void add_V_bias_transform_kernelLauncher(int8_t *v_buf, const int32_t *V, const T *V_bias, 
                                         const int batch_size, const int seq_len, 
                                         const int head_num, const int size_per_head, 
                                         const float* weight_amax, 
                                         const float *input_deQFactor_div127_ptr, const float *out_scale_ptr, 
                                         bool use_ORDER_COL32_2R_4R4, hipStream_t stream)
{
  add_V_bias_transform<<<dim3(size_per_head/32, seq_len/32, batch_size*head_num), dim3(8, 32), 0, stream>>>(v_buf, V, V_bias, batch_size, seq_len, head_num, size_per_head, seq_len*size_per_head, weight_amax, input_deQFactor_div127_ptr, out_scale_ptr, use_ORDER_COL32_2R_4R4);
}

template 
void add_V_bias_transform_kernelLauncher(int8_t *v_buf, const int32_t *V, const float *V_bias, 
                                         const int batch_size, const int seq_len, 
                                         const int head_num, const int size_per_head, 
                                         const float* weight_amax, 
                                         const float *input_deQFactor_div127_ptr, const float *out_scale_ptr, 
                                         bool use_ORDER_COL32_2R_4R4, hipStream_t stream);

template 
void add_V_bias_transform_kernelLauncher(int8_t *v_buf, const int32_t *V, const half *V_bias, 
                                         const int batch_size, const int seq_len, 
                                         const int head_num, const int size_per_head, 
                                         const float* weight_amax, 
                                         const float *input_deQFactor_div127_ptr, const float *out_scale_ptr, 
                                         bool use_ORDER_COL32_2R_4R4, hipStream_t stream);

//input matrix a matrix of m = batch_size*seq_len , n = head_num*size_per_head, CUBLASLT_ORDER_COL32
//seq_len_padded = (seq_len+31)/32*32
//output matrixes are a series of sub-matrixes with size of m = size_per_head, n = seq_len_padded , CUBLASLT_ORDER_COL4_4R2_8C or CUBLASLT_ORDER_COL32_2R_4R4
//only for int8_t IO
//size_per_head must be a multiple of 32
//grid = (size_per_head/32, seq_len_padded/32, batch_size*head_num)
//block = (8, 32);
//using char4
//per tensor quantization for weight
template <typename T>
__global__
void add_V_bias_transform_varlen(int8_t *v_buf_, const int8_t *V, const T *V_bias, const int batch_size, const int seq_len, 
                          const int head_num, const int size_per_head, const int seq_len_padded, int stride,
                          const float *input_deQFactor_ptr, const float *out_scale_ptr, bool use_ORDER_COL32_2R_4R4)
{
  const float input_deQFactor = __ldg(input_deQFactor_ptr);
  const float out_scale = __ldg(out_scale_ptr);
  __shared__ int8_t shm[32][33];
  const char4* data_ptr = (const char4*)V;
  char4* buf_ptr4 = (char4*) v_buf_;
  const T* bias_ptr = V_bias;

  int threadIdx4 = threadIdx.x << 2;

  //for src of (seq_len, size_per_head)
  int batch_id = blockIdx.z/head_num;
  int head_id = blockIdx.z%head_num;
  int word_id = (blockIdx.y << 5) + threadIdx.y;
  int id_in_size = (blockIdx.x << 5) + threadIdx4;

  int col, row;
  //for shm row-major
  int sh_col = threadIdx4;
  int sh_row = threadIdx.y;
  char4 dataTmp;
  if (word_id < seq_len)
  {
    //for V layout (batch_size*seq_len, head_num*size_per_head)
    col = head_id*size_per_head + id_in_size;
    row = batch_id*seq_len + word_id;
    int inIdx = (((col >> 5) << 5)*batch_size*seq_len + ((row << 5) + (col&31))) >> 2;
  
    float tmp;
    float scale;
  
    dataTmp = __ldg(data_ptr + inIdx);
  
    scale = dataTmp.x * input_deQFactor;
    tmp = scale + static_cast<float>(__ldg(bias_ptr + col));//(tmp2.x);
    shm[sh_row][sh_col] = float_to_int8_rn(tmp*out_scale);
  
    scale = dataTmp.y * input_deQFactor;
    tmp = scale + static_cast<float>(__ldg(bias_ptr+col+1));//(tmp2.y);
    shm[sh_row][sh_col+1] = float_to_int8_rn(tmp*out_scale);

    scale = dataTmp.z * input_deQFactor;
    tmp = scale + static_cast<float>(__ldg(bias_ptr+col+2));//(tmp2.x);
    shm[sh_row][sh_col+2] = float_to_int8_rn(tmp*out_scale);
  
    scale = dataTmp.w * input_deQFactor;
    tmp = scale + static_cast<float>(__ldg(bias_ptr+col+3));//(tmp2.y);
    shm[sh_row][sh_col+3] = float_to_int8_rn(tmp*out_scale);
  }
  else
  {
    shm[sh_row][sh_col] = shm[sh_row][sh_col+1] = shm[sh_row][sh_col+2] = shm[sh_row][sh_col+3] = 0;
  }

  __syncthreads();

  //for dst of (size_per_head, seq_len_padded)
  word_id = (blockIdx.y << 5) + threadIdx4;
  id_in_size = (blockIdx.x << 5) + threadIdx.y;
  col = (word_id >> 5);

  if (use_ORDER_COL32_2R_4R4)
  {
    int row_in_tile = id_in_size & 31;
    int col_in_tile = word_id & 31; 
    row = (
          //COL32_2R_4R4
          ((id_in_size >> 5) << 10) +
          //(((row%8)/2*4+row/8)*2+row%2)*32+col
          (((((((row_in_tile&7)>>1)<<2)+(row_in_tile>>3))<<1)+(row_in_tile&1))<<5)+col_in_tile
          );
  }
  else
  { 
    row = (
          //COL4
          ////id_in_size/8 is the number of tile of (8 rows 32 columns) -- column-major
          ////id_in_size%2 is even row, otherwise odd row
          ////word_id%COL32_/8 is the number tile of (8 rows 8 columns)
          ((((id_in_size >> 3) << 3) + ((id_in_size&1) << 2) + ((word_id&31) >> 3)) << 5) +
          ////word_id%8 >= 4 is the right half of (8 rows 8 columns) tile
          ////(id_in_size%8/2) is (the row id of alternating 4 rows) - 1
          (((((word_id&7) >= 4)?4:0) + ((id_in_size&7) >> 1)) << 2) +
          ////word_id%4 is the id of 4 cols
          (word_id&3)
          );
  }

  dataTmp.x = shm[sh_col][sh_row];
  dataTmp.y = shm[sh_col+1][sh_row];
  dataTmp.z = shm[sh_col+2][sh_row];
  dataTmp.w = shm[sh_col+3][sh_row];
  buf_ptr4[(blockIdx.z*stride + (col << 5)*size_per_head + row) >> 2] = dataTmp;
}

template <typename T>
void add_V_bias_transform_kernelLauncher(int8_t *v_buf, const int8_t *V, const T *V_bias, const int batch_size, 
                                         const int seq_len, const int head_num, const int size_per_head,
                                         const float *input_deQFactor_ptr, const float *out_scale_ptr, 
                                         bool use_ORDER_COL32_2R_4R4, hipStream_t stream)
{
  assert(size_per_head % 32 == 0);
  if (seq_len % 32 == 0)
  {
    add_V_bias_transform_varlen<<<dim3(size_per_head/32, seq_len/32, batch_size*head_num), dim3(8, 32), 0, stream>>>(
      v_buf, V, V_bias, 
      batch_size, seq_len, head_num, size_per_head, 
      seq_len, seq_len*size_per_head,
      input_deQFactor_ptr, out_scale_ptr, use_ORDER_COL32_2R_4R4);
  }
  else
  {
    const int seq_len_padded = (seq_len + 31)/32*32;
    add_V_bias_transform_varlen<<<dim3(size_per_head/32, seq_len_padded/32, batch_size*head_num), dim3(8, 32), 0, stream>>>(
      v_buf, V, V_bias, 
      batch_size, seq_len, head_num, size_per_head, 
      seq_len_padded, seq_len_padded*size_per_head,
      input_deQFactor_ptr, out_scale_ptr, use_ORDER_COL32_2R_4R4);
  }
}                

template
void add_V_bias_transform_kernelLauncher(int8_t *v_buf, const int8_t *V, const float *V_bias, const int batch_size, 
                                         const int seq_len, const int head_num, const int size_per_head,
                                         const float *input_deQFactor_ptr, const float *out_scale_ptr, 
                                         bool use_ORDER_COL32_2R_4R4, hipStream_t stream); 

template
void add_V_bias_transform_kernelLauncher(int8_t *v_buf, const int8_t *V, const half *V_bias, const int batch_size, 
                                         const int seq_len, const int head_num, const int size_per_head,
                                         const float *input_deQFactor_ptr, const float *out_scale_ptr, 
                                         bool use_ORDER_COL32_2R_4R4, hipStream_t stream);                                          

//add bias into V & rebuild padding 
//input matrix a matrix of m = valid_word_num, n = head_num*size_per_head, CUBLASLT_ORDER_COL32
//output matrixes are a series of sub-matrixes with size of m = size_per_head, n = seq_len , CUBLASLT_ORDER_COL4_4R2_8C or CUBLASLT_ORDER_COL32_2R_4R4
//only for int32_t Input int8_t Output
//seq_len, size_per_head must be a multiple of 32
//grid = (size_per_head/32, seq_len/32, batch_size*head_num)
//block = (8, 32);
//using char4
//per axis quantization for weight
template <typename T>
__global__
void add_V_bias_transform_rebuild_padding(int8_t *v_buf_, const int32_t *V, const T *V_bias, const int* sequence_id_map, const int valid_word_num, 
                                          const int batch_size, const int seq_len, const int head_num, const int size_per_head, int stride, 
                                          const float* weight_amax, const float *input_deQFactor_div127_ptr, const float *out_scale_ptr, bool use_ORDER_COL32_2R_4R4)
{
  __shared__ int8_t shm[32][33];
  const int32_t* data_ptr = V;
  char4* buf_ptr4 = (char4*) v_buf_;
  const T* bias_ptr = V_bias;

  int threadIdx4 = threadIdx.x << 2;

  //for src of (seq_len, size_per_head)
  int batch_id = blockIdx.z/head_num;
  int head_id = blockIdx.z%head_num;
  int word_id = (blockIdx.y << 5) + threadIdx.y;
  int id_in_size = (blockIdx.x << 5) + threadIdx4;

  //for shm row-major
  int sh_col = threadIdx4;
  int sh_row = threadIdx.y;
  
  //for V layout (batch_size*seq_len, head_num*size_per_head)
  int col;
  int row = __ldg(sequence_id_map + batch_id*seq_len + word_id);
  
  if (row != -1){
    col = head_id*size_per_head + id_in_size;  
    int inIdx = ((col & 0xffffffe0)*valid_word_num + ((row << 5) + (col&31)));
  
    float tmp;
    float scale;
  
    const float input_deQFactor_div127 = __ldg(input_deQFactor_div127_ptr);
    const float out_scale = __ldg(out_scale_ptr);
  
    scale = __ldg(data_ptr + inIdx) * __ldg(weight_amax + col) * input_deQFactor_div127;
    tmp = scale + static_cast<float>(__ldg(bias_ptr + col));
    shm[sh_row][sh_col] = float_to_int8_rn(tmp*out_scale);
  
    scale = __ldg(data_ptr + inIdx + 1) * __ldg(weight_amax + col + 1) * input_deQFactor_div127;
    tmp = scale + static_cast<float>(__ldg(bias_ptr+col+1));
    shm[sh_row][sh_col+1] = float_to_int8_rn(tmp*out_scale);

    scale = __ldg(data_ptr+inIdx+2) * __ldg(weight_amax+col+2) * input_deQFactor_div127;
    tmp = scale + static_cast<float>(__ldg(bias_ptr+col+2));
    shm[sh_row][sh_col+2] = float_to_int8_rn(tmp*out_scale);
  
    scale = __ldg(data_ptr+inIdx + 3) * __ldg(weight_amax+col+3) * input_deQFactor_div127;
    tmp = scale + static_cast<float>(__ldg(bias_ptr+col+3));
    shm[sh_row][sh_col+3] = float_to_int8_rn(tmp*out_scale);
  }
  else{
    shm[sh_row][sh_col] = shm[sh_row][sh_col + 1] = shm[sh_row][sh_col + 2] = shm[sh_row][sh_col + 3] = 0;
  }
  __syncthreads();

  char4 dataTmp;
  dataTmp.x = shm[sh_col][sh_row];
  dataTmp.y = shm[sh_col+1][sh_row];
  dataTmp.z = shm[sh_col+2][sh_row];
  dataTmp.w = shm[sh_col+3][sh_row];

  //for dst of (size_per_head, seq_len)
  word_id = (blockIdx.y << 5) + threadIdx4;
  id_in_size = (blockIdx.x << 5) + threadIdx.y;
  col = (word_id >> 5);
  
  if (use_ORDER_COL32_2R_4R4)
  {
    int row_in_tile = id_in_size & 31;
    int col_in_tile = word_id & 31; 
    row = (
          //COL32_2R_4R4
          ((id_in_size >> 5) << 10) +
          //(((row%8)/2*4+row/8)*2+row%2)*32+col
          (((((((row_in_tile&7)>>1)<<2)+(row_in_tile>>3))<<1)+(row_in_tile&1))<<5)+col_in_tile
          ); 
  }
  else
  {
    row = (
          //COL4
          ////id_in_size/8 is the number of tile of (8 rows 32 columns) -- column-major
          ////id_in_size%2 is even row, otherwise odd row
          ////word_id%COL32_/8 is the number tile of (8 rows 8 columns)
          (((id_in_size & 0xfffffff8) + ((id_in_size&1) << 2) + ((word_id&31) >> 3)) << 5) +
          ////word_id%8 >= 4 is the right half of (8 rows 8 columns) tile
          ////(id_in_size%8/2) is (the row id of alternating 4 rows) - 1
          (((((word_id&7) >= 4)?4:0) + ((id_in_size&7) >> 1)) << 2) +
          ////word_id%4 is the id of 4 cols
          (word_id&3)
          );
  }
        
  buf_ptr4[(blockIdx.z*stride + (col << 5)*size_per_head + row) >> 2] = dataTmp;
}

template <>
__global__
void add_V_bias_transform_rebuild_padding(int8_t *v_buf_, const int32_t *V, const half *V_bias, const int* sequence_id_map, const int valid_word_num, 
                                          const int batch_size, const int seq_len, const int head_num, const int size_per_head, int stride, 
                                          const float* weight_amax, const float *input_deQFactor_div127_ptr, const float *out_scale_ptr, bool use_ORDER_COL32_2R_4R4)
{
  __shared__ int8_t shm[32][33];
  const int32_t* data_ptr = V;
  char4* buf_ptr4 = (char4*) v_buf_;

  int threadIdx4 = threadIdx.x << 2;

  //for src of (seq_len, size_per_head)
  int batch_id = blockIdx.z/head_num;
  int head_id = blockIdx.z%head_num;
  
  int blockIdy32 = (blockIdx.y << 5);
  int blockIdx32 = (blockIdx.x << 5);
  int word_id = blockIdy32 + threadIdx.y;
  int id_in_size = blockIdx32 + threadIdx4;

  
  //for shm row-major
  int sh_col = threadIdx4;
  int sh_row = threadIdx.y;
  
  //for V layout (batch_size*seq_len, head_num*size_per_head)
  int col;
  int row = __ldg(sequence_id_map + batch_id*seq_len + word_id);
  
  if (row >= 0){
    const float input_deQFactor_div127 = __ldg(input_deQFactor_div127_ptr);
    const float out_scale = __ldg(out_scale_ptr);
    col = head_id*size_per_head + id_in_size;
    int inIdx = ((col & 0xffffffe0)*valid_word_num + ((row << 5) + (col&31)));
    int col_2 = col >> 1;
    float scale;

    const half2* bias_ptr2 = (const half2*)V_bias;
    half2 tmp2;

    tmp2 = __ldg(bias_ptr2+col_2);
  
    scale = __ldg(data_ptr+inIdx) * __ldg(weight_amax+col) * input_deQFactor_div127;
    scale = scale + static_cast<float>(tmp2.x);
    shm[sh_row][sh_col] = float_to_int8_rn(scale*out_scale);
  
    scale = __ldg(data_ptr+inIdx+1) * __ldg(weight_amax+col+1) * input_deQFactor_div127;
    scale = scale + static_cast<float>(tmp2.y);
    shm[sh_row][sh_col+1] = float_to_int8_rn(scale*out_scale);
  
    tmp2 = __ldg(bias_ptr2 + col_2 + 1);

    scale = __ldg(data_ptr + inIdx + 2) * __ldg(weight_amax + col + 2) * input_deQFactor_div127;
    scale = scale + static_cast<float>(tmp2.x);
    shm[sh_row][sh_col+2] = float_to_int8_rn(scale*out_scale);
  
    scale = __ldg(data_ptr + inIdx + 3) * __ldg(weight_amax + col + 3) * input_deQFactor_div127;
    scale = scale + static_cast<float>(tmp2.y);
    shm[sh_row][sh_col+3] = float_to_int8_rn(scale*out_scale);
  }
  else{
    shm[sh_row][sh_col] = shm[sh_row][sh_col + 1] = shm[sh_row][sh_col + 2] = shm[sh_row][sh_col + 3] = 0;
  }
  __syncthreads();

  char4 dataTmp;
  dataTmp.x = shm[sh_col][sh_row];
  dataTmp.y = shm[sh_col+1][sh_row];
  dataTmp.z = shm[sh_col+2][sh_row];
  dataTmp.w = shm[sh_col+3][sh_row];

  //for dst of (size_per_head, seq_len)
  word_id = blockIdy32 + threadIdx4;
  id_in_size = blockIdx32 + threadIdx.y;
  col = (word_id >> 5);
  
  if (use_ORDER_COL32_2R_4R4)
  {
    int row_in_tile = id_in_size & 31;
    int col_in_tile = word_id & 31; 
    row = (
          //COL32_2R_4R4
          ((id_in_size >> 5) << 10) +
          //(((row%8)/2*4+row/8)*2+row%2)*32+col
          (((((((row_in_tile&7)>>1)<<2)+(row_in_tile>>3))<<1)+(row_in_tile&1))<<5)+col_in_tile
          ); 
  }
  else
  {
    row = (
          //COL4
          ////id_in_size/8 is the number of tile of (8 rows 32 columns) -- column-major
          ////id_in_size%2 is even row, otherwise odd row
          ////word_id%COL32_/8 is the number tile of (8 rows 8 columns)
          (((id_in_size & 0xfffffff8) + ((id_in_size&1) << 2) + ((word_id&31) >> 3)) << 5) +
          ////word_id%8 >= 4 is the right half of (8 rows 8 columns) tile
          ////(id_in_size%8/2) is (the row id of alternating 4 rows) - 1
          (((((word_id&7) >= 4)?4:0) + ((id_in_size&7) >> 1)) << 2) +
          ////word_id%4 is the id of 4 cols
          (word_id&3)
          );
  }
        
  buf_ptr4[(blockIdx.z*stride + (col << 5)*size_per_head + row) >> 2] = dataTmp;
}

template <typename T>
void add_V_bias_transform_rebuild_padding_kernelLauncher(int8_t *v_buf, const int32_t *V, const T *V_bias, 
                                                         const int* sequence_id_map, const int valid_word_num, 
                                                         const int batch_size, const int seq_len, 
                                                         const int head_num, const int size_per_head, 
                                                         const float* weight_amax, 
                                                         const float *input_deQFactor_div127_ptr, 
                                                         const float *out_scale_ptr, 
                                                         bool use_ORDER_COL32_2R_4R4, hipStream_t stream)
{
  add_V_bias_transform_rebuild_padding<<<dim3(size_per_head/32, seq_len/32, batch_size*head_num), dim3(8, 32), 0, stream>>>(
    v_buf, V, V_bias, 
    sequence_id_map, valid_word_num, 
    batch_size, seq_len, 
    head_num, size_per_head, 
    seq_len*size_per_head, 
    weight_amax, input_deQFactor_div127_ptr,
    out_scale_ptr, use_ORDER_COL32_2R_4R4);
}     

template 
void add_V_bias_transform_rebuild_padding_kernelLauncher(int8_t *v_buf, const int32_t *V, const float *V_bias, 
                                                         const int* sequence_id_map, const int valid_word_num, 
                                                         const int batch_size, const int seq_len, 
                                                         const int head_num, const int size_per_head, 
                                                         const float* weight_amax, 
                                                         const float *input_deQFactor_div127_ptr, 
                                                         const float *out_scale_ptr, 
                                                         bool use_ORDER_COL32_2R_4R4, hipStream_t stream);
                                                         
template 
void add_V_bias_transform_rebuild_padding_kernelLauncher(int8_t *v_buf, const int32_t *V, const half *V_bias, 
                                                         const int* sequence_id_map, const int valid_word_num, 
                                                         const int batch_size, const int seq_len, 
                                                         const int head_num, const int size_per_head, 
                                                         const float* weight_amax, 
                                                         const float *input_deQFactor_div127_ptr, 
                                                         const float *out_scale_ptr, 
                                                         bool use_ORDER_COL32_2R_4R4, hipStream_t stream);

//add bias into V & rebuild padding 
//input matrix a matrix of m = valid_word_num, n = head_num*size_per_head, CUBLASLT_ORDER_COL32
//output matrixes are a series of sub-matrixes with size of m = size_per_head, n = seq_len , CUBLASLT_ORDER_COL4_4R2_8C or CUBLASLT_ORDER_COL32_2R_4R4
//only for int8_t IO
//seq_len, size_per_head must be a multiple of 32
//grid = (size_per_head/32, seq_len/32, batch_size*head_num)
//block = (8, 32);
//using char4
//per tensor quantization for weight
template <typename T>
__global__
void add_V_bias_transform_rebuild_padding(int8_t *v_buf_, const int8_t *V, const T *V_bias, const int* sequence_id_map, const int valid_word_num, 
                                          const int batch_size, const int seq_len, const int head_num, const int size_per_head, int stride, 
                                          const float *deQFactor_ptr, const float *out_scale_ptr, bool use_ORDER_COL32_2R_4R4)
{
  __shared__ int8_t shm[32][33];
  const char4* data_ptr = (const char4*)V;
  char4* buf_ptr4 = (char4*) v_buf_;
  const T* bias_ptr = V_bias;

  int threadIdx4 = threadIdx.x << 2;

  //for src of (seq_len, size_per_head)
  int batch_id = blockIdx.z/head_num;
  int head_id = blockIdx.z%head_num;
  int word_id = (blockIdx.y << 5) + threadIdx.y;
  int id_in_size = (blockIdx.x << 5) + threadIdx4;

  //for shm row-major
  int sh_col = threadIdx4;
  int sh_row = threadIdx.y;
  
  //for V layout (batch_size*seq_len, head_num*size_per_head)
  int col;
  int row = __ldg(sequence_id_map + batch_id*seq_len + word_id);
  
  if (row != -1){
    col = head_id*size_per_head + id_in_size;  
    int inIdx = ((col & 0xffffffe0)*valid_word_num + ((row << 5) + (col&31))) >> 2;
  
    float tmp;
    float scale;
  
    const float deQFactor = __ldg(deQFactor_ptr);
    const float out_scale = __ldg(out_scale_ptr);
  
    char4 dataTmp = __ldg(data_ptr + inIdx);
  
    scale = dataTmp.x * deQFactor;
    tmp = scale + static_cast<float>(__ldg(bias_ptr + col));
    shm[sh_row][sh_col] = float_to_int8_rn(tmp*out_scale);
  
    scale = dataTmp.y * deQFactor;
    tmp = scale + static_cast<float>(__ldg(bias_ptr+col+1));
    shm[sh_row][sh_col+1] = float_to_int8_rn(tmp*out_scale);

    scale = dataTmp.z * deQFactor;
    tmp = scale + static_cast<float>(__ldg(bias_ptr+col+2));
    shm[sh_row][sh_col+2] = float_to_int8_rn(tmp*out_scale);
  
    scale = dataTmp.w * deQFactor;
    tmp = scale + static_cast<float>(__ldg(bias_ptr+col+3));
    shm[sh_row][sh_col+3] = float_to_int8_rn(tmp*out_scale);
  }
  else{
    shm[sh_row][sh_col] = shm[sh_row][sh_col + 1] = shm[sh_row][sh_col + 2] = shm[sh_row][sh_col + 3] = 0;
  }
  __syncthreads();

  char4 dataTmp;  
  dataTmp.x = shm[sh_col][sh_row];
  dataTmp.y = shm[sh_col+1][sh_row];
  dataTmp.z = shm[sh_col+2][sh_row];
  dataTmp.w = shm[sh_col+3][sh_row];

  //for dst of (size_per_head, seq_len)
  word_id = (blockIdx.y << 5) + threadIdx4;
  id_in_size = (blockIdx.x << 5) + threadIdx.y;
  col = (word_id >> 5);
  
  if (use_ORDER_COL32_2R_4R4)
  {
    int row_in_tile = id_in_size & 31;
    int col_in_tile = word_id & 31; 
    row = (
          //COL32_2R_4R4
          ((id_in_size >> 5) << 10) +
          //(((row%8)/2*4+row/8)*2+row%2)*32+col
          (((((((row_in_tile&7)>>1)<<2)+(row_in_tile>>3))<<1)+(row_in_tile&1))<<5)+col_in_tile
          ); 
  }
  else
  {
    row = (
          //COL4
          ////id_in_size/8 is the number of tile of (8 rows 32 columns) -- column-major
          ////id_in_size%2 is even row, otherwise odd row
          ////word_id%COL32_/8 is the number tile of (8 rows 8 columns)
          (((id_in_size & 0xfffffff8) + ((id_in_size&1) << 2) + ((word_id&31) >> 3)) << 5) +
          ////word_id%8 >= 4 is the right half of (8 rows 8 columns) tile
          ////(id_in_size%8/2) is (the row id of alternating 4 rows) - 1
          (((((word_id&7) >= 4)?4:0) + ((id_in_size&7) >> 1)) << 2) +
          ////word_id%4 is the id of 4 cols
          (word_id&3)
          );
  }
        
  buf_ptr4[(blockIdx.z*stride + (col << 5)*size_per_head + row) >> 2] = dataTmp;
}

//add bias into V & rebuild padding
//input matrix a matrix of m = valid_word_num, n = head_num*size_per_head, CUBLASLT_ORDER_COL32
//output matrixes are a series of sub-matrixes with size of m = size_per_head, n = seq_len_padded , CUBLASLT_ORDER_COL4_4R2_8C or CUBLASLT_ORDER_COL32_2R_4R4
//only for int8_t IO
//seq_len, size_per_head must be a multiple of 32
//grid = (size_per_head/32, seq_len_padded/32, batch_size*head_num)
//block = (8, 32);
//using char4
//per tensor quantization for weight
template <typename T>
__global__
void add_V_bias_transform_rebuild_padding_varlen(int8_t *v_buf_, const int8_t *V, const T *V_bias, const int* sequence_id_map, const int valid_word_num,
                                                 const int batch_size, const int seq_len, const int seq_len_padded, 
                                                 const int head_num, const int size_per_head, int stride,
                                                 const float *deQFactor_ptr, const float *out_scale_ptr, bool use_ORDER_COL32_2R_4R4)
{
  __shared__ int8_t shm[32][33];
  const char4* data_ptr = (const char4*)V;
  char4* buf_ptr4 = (char4*) v_buf_;
  const T* bias_ptr = V_bias;

  int threadIdx4 = threadIdx.x << 2;

  //for src of (seq_len, size_per_head)
  int batch_id = blockIdx.z/head_num;
  int head_id = blockIdx.z%head_num;
  int word_id = (blockIdx.y << 5) + threadIdx.y;
  int id_in_size = (blockIdx.x << 5) + threadIdx4;

  //for shm row-major
  int sh_col = threadIdx4;
  int sh_row = threadIdx.y;

  //for V layout (batch_size*seq_len, head_num*size_per_head)
  int col;
  int row = word_id < seq_len ? __ldg(sequence_id_map + batch_id*seq_len + word_id) : -1;

  if (row != -1){
    col = head_id*size_per_head + id_in_size;
    int inIdx = ((col & 0xffffffe0)*valid_word_num + ((row << 5) + (col&31))) >> 2;

    float tmp;
    float scale;

    const float deQFactor = __ldg(deQFactor_ptr);
    const float out_scale = __ldg(out_scale_ptr);

    char4 dataTmp = __ldg(data_ptr + inIdx);

    scale = dataTmp.x * deQFactor;
    tmp = scale + static_cast<float>(__ldg(bias_ptr + col));
    shm[sh_row][sh_col] = float_to_int8_rn(tmp*out_scale);

    scale = dataTmp.y * deQFactor;
    tmp = scale + static_cast<float>(__ldg(bias_ptr+col+1));
    shm[sh_row][sh_col+1] = float_to_int8_rn(tmp*out_scale);

    scale = dataTmp.z * deQFactor;
    tmp = scale + static_cast<float>(__ldg(bias_ptr+col+2));
    shm[sh_row][sh_col+2] = float_to_int8_rn(tmp*out_scale);

    scale = dataTmp.w * deQFactor;
    tmp = scale + static_cast<float>(__ldg(bias_ptr+col+3));
    shm[sh_row][sh_col+3] = float_to_int8_rn(tmp*out_scale);
  }
  else{
    shm[sh_row][sh_col] = shm[sh_row][sh_col + 1] = shm[sh_row][sh_col + 2] = shm[sh_row][sh_col + 3] = 0;
  }
  __syncthreads();

  char4 dataTmp;
  dataTmp.x = shm[sh_col][sh_row];
  dataTmp.y = shm[sh_col+1][sh_row];
  dataTmp.z = shm[sh_col+2][sh_row];
  dataTmp.w = shm[sh_col+3][sh_row];

  //for dst of (size_per_head, seq_len_padded)
  word_id = (blockIdx.y << 5) + threadIdx4;
  id_in_size = (blockIdx.x << 5) + threadIdx.y;
  col = (word_id >> 5);

  if (use_ORDER_COL32_2R_4R4)
  {
    int row_in_tile = id_in_size & 31;
    int col_in_tile = word_id & 31;
    row = (
          //COL32_2R_4R4
          ((id_in_size >> 5) << 10) +
          //(((row%8)/2*4+row/8)*2+row%2)*32+col
          (((((((row_in_tile&7)>>1)<<2)+(row_in_tile>>3))<<1)+(row_in_tile&1))<<5)+col_in_tile
          );
  }
  else
  {
    row = (
          //COL4
          ////id_in_size/8 is the number of tile of (8 rows 32 columns) -- column-major
          ////id_in_size%2 is even row, otherwise odd row
          ////word_id%COL32_/8 is the number tile of (8 rows 8 columns)
          (((id_in_size & 0xfffffff8) + ((id_in_size&1) << 2) + ((word_id&31) >> 3)) << 5) +
          ////word_id%8 >= 4 is the right half of (8 rows 8 columns) tile
          ////(id_in_size%8/2) is (the row id of alternating 4 rows) - 1
          (((((word_id&7) >= 4)?4:0) + ((id_in_size&7) >> 1)) << 2) +
          ////word_id%4 is the id of 4 cols
          (word_id&3)
          );
  }

  buf_ptr4[(blockIdx.z*stride + (col << 5)*size_per_head + row) >> 2] = dataTmp;
}


template <typename T>
void add_V_bias_transform_rebuild_padding_kernelLauncher(int8_t *v_buf, const int8_t *V, const T *V_bias, 
                                                         const int* sequence_id_map, const int valid_word_num, 
                                                         const int batch_size, const int seq_len, 
                                                         const int head_num, const int size_per_head,
                                                         const float *deQFactor_ptr, const float *out_scale_ptr, 
                                                         bool use_ORDER_COL32_2R_4R4, hipStream_t stream)
{
  int seq_len_padded = (seq_len + 31)/32*32;
  add_V_bias_transform_rebuild_padding_varlen<<<dim3(size_per_head/32, seq_len_padded/32, batch_size*head_num), dim3(8, 32), 0, stream>>>(
    v_buf, V, V_bias, sequence_id_map, valid_word_num, 
    batch_size, seq_len, seq_len_padded, head_num, size_per_head, seq_len_padded*size_per_head, 
    deQFactor_ptr, out_scale_ptr, use_ORDER_COL32_2R_4R4);
}           

template
void add_V_bias_transform_rebuild_padding_kernelLauncher(int8_t *v_buf, const int8_t *V, const float *V_bias, 
                                                         const int* sequence_id_map, const int valid_word_num, 
                                                         const int batch_size, const int seq_len, 
                                                         const int head_num, const int size_per_head,
                                                         const float *deQFactor_ptr, const float *out_scale_ptr, 
                                                         bool use_ORDER_COL32_2R_4R4, hipStream_t stream);
                                                         
template
void add_V_bias_transform_rebuild_padding_kernelLauncher(int8_t *v_buf, const int8_t *V, const half *V_bias, 
                                                         const int* sequence_id_map, const int valid_word_num, 
                                                         const int batch_size, const int seq_len, 
                                                         const int head_num, const int size_per_head,
                                                         const float *deQFactor_ptr, const float *out_scale_ptr, 
                                                         bool use_ORDER_COL32_2R_4R4, hipStream_t stream); 

__global__
void trt_add_QKV_bias(half2* Q, const half2* bias_Q, half2* K, const half2* bias_K, half2* V, const half2* bias_V, 
  half2* q_buf_, half2* k_buf_, half2* v_buf_, 
  const int valid_word_num, const int head_num, const int size_per_head)
{
  // Add bias, and then transpose from 
  // [3, valid_word_num, head, size] -> [valid_word_num, head, 3, size]
  
  // const int seq_id = blockIdx.x % valid_word_num;
  // const int qkv_id = (blockIdx.x - seq_id) / valid_word_num;
  const int seq_id = blockIdx.x;
  const int size_id = threadIdx.x % size_per_head;
  const int head_id = (threadIdx.x - size_id) / size_per_head;

  const int target_offset = blockIdx.x * head_num * 3 * size_per_head + head_id * 3 * size_per_head;

  q_buf_[ target_offset + 
          0 * size_per_head +
          size_id] = Q[ seq_id * blockDim.x + threadIdx.x] + bias_Q[threadIdx.x];

  q_buf_[ target_offset + 
          1 * size_per_head +
          size_id] = K[ seq_id * blockDim.x + threadIdx.x] + bias_K[threadIdx.x];

  q_buf_[ target_offset + 
          2 * size_per_head +
          size_id] = V[ seq_id * blockDim.x + threadIdx.x] + bias_V[threadIdx.x];
}

template<OperationType OpType_>
void OpenMultiHeadAttention<OpType_>::trt_add_QKV_bias_kernelLauncher(
  const DataType_* bias_Q,
  const DataType_* bias_K,
  const DataType_* bias_V)
{
  dim3 grid;
  dim3 block;

  grid.x = param_.valid_word_num;
  block.x = head_num_ * size_per_head_ / 2;

  assert(block.x <= 1024);

  trt_add_QKV_bias<<<grid, block, 0, param_.stream>>>((half2*)query_buf_, (const half2*)bias_Q, 
                                                      (half2*)key_buf_, (const half2*)bias_K, 
                                                      (half2*)value_buf_, (const half2*)bias_V, 
                                                      (half2*)q_buf_, (half2*)k_buf_, (half2*)v_buf_,
                                                      param_.valid_word_num, 
                                                      head_num_, size_per_head_ / 2);
}

// add bias and then transform from 
// 3 * ([valid_word_num, head*size] + CUBLASLT_ORDER_COL32) -> [valid_word_num, head, 3, size] + row-major
// input is INT32 && per axis quantization for weight
// output is INT8 && per tensor quantization
// grid((head*size + 31)/32, (valid_word_num + 31)/32, 3)
// block(8, 32)
// size should be a multiple of 4
//using char4 as output, int4 as input
template <typename T>
__global__
void trt_add_QKV_bias_COL32_int32IInt8O(char4* output, const int4* QKV,
                                        const T* bias_Q, const T* bias_K, const T* bias_V, 
                                        const float *input_deQFactor_div127_ptr,
                                        const float *q_weight_amax,  
                                        const float *k_weight_amax,
                                        const float *v_weight_amax,
                                        const float qkv_output_scale, const int valid_word_num, const int head_num, 
                                        const int size_per_head, const int head_num_x_size_per_head)
{
  const int qkv_id = blockIdx.z;
  const int seq_id = (blockIdx.y << 5) + threadIdx.y;
  const int threadIdx4 = threadIdx.x << 2;
  int hidden_id = (blockIdx.x << 5) + threadIdx4;
  const int size_id = hidden_id % size_per_head;
  const int head_id = hidden_id / size_per_head;
  
  const bool qual = (seq_id < valid_word_num) && (hidden_id < head_num_x_size_per_head);
  if (qual)
  {
    const float* weight_amax = qkv_id == 0 ? q_weight_amax : (qkv_id == 1 ? k_weight_amax : v_weight_amax);
    const float input_deQFactor_div127 = __ldg(input_deQFactor_div127_ptr); 
  
    const T* bias_ptr = (qkv_id == 0) ? bias_Q : ((qkv_id == 1) ? bias_K : bias_V);
  
    const int input_id = (qkv_id * valid_word_num * head_num_x_size_per_head + ((hidden_id & 0xffffffe0)*valid_word_num + (seq_id << 5) + (hidden_id&31))) >> 2;
    
    char4 tmp;
    const int4 tmp_int4 = __ldg(QKV+input_id);
    
    tmp.x = float_to_int8_rn((static_cast<float>(tmp_int4.x) * __ldg(weight_amax+hidden_id) * input_deQFactor_div127 + static_cast<float>(__ldg(bias_ptr + hidden_id))) * qkv_output_scale);
    
    hidden_id += 1;
    tmp.y = float_to_int8_rn((static_cast<float>(tmp_int4.y) * __ldg(weight_amax+hidden_id) * input_deQFactor_div127 + static_cast<float>(__ldg(bias_ptr + hidden_id))) * qkv_output_scale);
    
    hidden_id += 1;
    tmp.z = float_to_int8_rn((static_cast<float>(tmp_int4.z) * __ldg(weight_amax+hidden_id) * input_deQFactor_div127 + static_cast<float>(__ldg(bias_ptr + hidden_id))) * qkv_output_scale);
    
    hidden_id += 1;
    tmp.w = float_to_int8_rn((static_cast<float>(tmp_int4.w) * __ldg(weight_amax+hidden_id) * input_deQFactor_div127 + static_cast<float>(__ldg(bias_ptr + hidden_id))) * qkv_output_scale);

    //const int output_id = (seq_id * 3 * head_num_x_size_per_head + head_id * 3 * size_per_head + qkv_id * size_per_head + size_id) >> 2;
    const int output_id = ((seq_id * head_num_x_size_per_head + head_id * size_per_head) * 3 + qkv_id * size_per_head + size_id) >> 2;    
    
    output[output_id] = tmp;
  }
}

template<OperationType OpType_>
void OpenMultiHeadAttention<OpType_>::trt_add_QKV_bias_COL32_int32Iint8O_kernelLauncher(
  int8_t* output,
  const int32_t* Q,
  const DataType_* bias_Q,
  const DataType_* bias_K,
  const DataType_* bias_V,
  const float *input_deQFactor_div127_ptr,
  const float * q_weight_amax,  
  const float * k_weight_amax,
  const float * v_weight_amax,
  const float qkv_output_scale)
{
  int head_num_x_size_per_head = head_num_*size_per_head_;
  dim3 grid((head_num_x_size_per_head + 31)/32, (param_.valid_word_num + 31)/32, 3);
  dim3 block(8, 32);

  assert(size_per_head_ % 4 == 0);

  trt_add_QKV_bias_COL32_int32IInt8O<<<grid, block, 0, param_.stream>>>((char4*)output, (const int4*)Q,
                                                                   bias_Q, bias_K, bias_V,
                                                                   input_deQFactor_div127_ptr,
                                                                   q_weight_amax, 
                                                                   k_weight_amax,
                                                                   v_weight_amax,
                                                                   qkv_output_scale, param_.valid_word_num, 
                                                                   head_num_, size_per_head_, head_num_x_size_per_head);
}

// Add bias, and then transform from 
// 3 * ([valid_word_num, head*size] + CUBLASLT_ORDER_COL32) -> [valid_word_num, head, 3, size] + row-major
// grid((head*size + 31)/32, (valid_word_num + 31)/32, 3)
// block(8, 32)
// size should be a multiple of 4
template <typename T>
__global__
void trt_add_QKV_bias_COL32_int8IO(char4* output, const char4* QKV,  
                                   const T* bias_Q, const T* bias_K, const T* bias_V, 
                                   const float *q_input_deQFactor_ptr, const float *k_input_deQFactor_ptr, 
                                   const float *v_input_deQFactor_ptr, const float qkv_output_scale,
                                   const int valid_word_num, const int head_num, const int size_per_head,
                                   const int head_num_x_size_per_head)
{
  const int qkv_id = blockIdx.z;
  const int seq_id = (blockIdx.y << 5) + threadIdx.y;
  const int threadIdx4 = threadIdx.x << 2;
  const int hidden_id = (blockIdx.x << 5) + threadIdx4;
  const int size_id = hidden_id % size_per_head;
  const int head_id = hidden_id / size_per_head;
  
  const bool qual = (seq_id < valid_word_num) && (hidden_id < head_num_x_size_per_head);
  if (qual)
  {
    const float *input_deQFactor_ptr = (qkv_id == 0) ? q_input_deQFactor_ptr : ((qkv_id == 1) ? k_input_deQFactor_ptr : v_input_deQFactor_ptr);
    const float input_deQFactor = __ldg(input_deQFactor_ptr);

    const T* bias_ptr = (qkv_id == 0) ? bias_Q : ((qkv_id == 1) ? bias_K : bias_V);
  
    const int input_id = (qkv_id * valid_word_num * head_num_x_size_per_head + ((hidden_id & 0xffffffe0)*valid_word_num + (seq_id << 5) + (hidden_id&31))) >> 2;
    
    char4 tmp = __ldg(QKV+input_id);
    
    tmp.x = float_to_int8_rn((static_cast<float>(tmp.x) * input_deQFactor + static_cast<float>(__ldg(bias_ptr + hidden_id))) * qkv_output_scale);
    
    tmp.y = float_to_int8_rn((static_cast<float>(tmp.y) * input_deQFactor + static_cast<float>(__ldg(bias_ptr + hidden_id + 1))) * qkv_output_scale);
    
    tmp.z = float_to_int8_rn((static_cast<float>(tmp.z) * input_deQFactor + static_cast<float>(__ldg(bias_ptr + hidden_id + 2))) * qkv_output_scale);
    
    tmp.w = float_to_int8_rn((static_cast<float>(tmp.w) * input_deQFactor + static_cast<float>(__ldg(bias_ptr + hidden_id + 3))) * qkv_output_scale);

    //const int output_id = (seq_id * 3 * head_num_x_size_per_head + head_id * 3 * size_per_head + qkv_id * size_per_head + size_id) >> 2;
    const int output_id = ((seq_id * head_num_x_size_per_head + head_id * size_per_head) * 3 + qkv_id * size_per_head + size_id) >> 2;    
    
    output[output_id] = tmp;
  }
}

template<OperationType OpType_>
void OpenMultiHeadAttention<OpType_>::trt_add_QKV_bias_COL32_int8IO_kernelLauncher(
  int8_t* output,
  const int8_t* Q,
  const DataType_* bias_Q,
  const DataType_* bias_K,
  const DataType_* bias_V,
  const float *q_input_deQFactor_ptr, 
  const float *k_input_deQFactor_ptr, 
  const float *v_input_deQFactor_ptr, 
  const float qkv_output_scale)
{
  int head_num_x_size_per_head = head_num_*size_per_head_;
  dim3 grid((head_num_x_size_per_head + 31)/32, (param_.valid_word_num + 31)/32, 3);
  dim3 block(8, 32);

  assert(size_per_head_ % 4 == 0);

  trt_add_QKV_bias_COL32_int8IO<<<grid, block, 0, param_.stream>>>((char4*)output, (const char4*)Q,
                                                                   bias_Q, bias_K, bias_V,
                                                                   q_input_deQFactor_ptr, k_input_deQFactor_ptr, v_input_deQFactor_ptr,
                                                                   qkv_output_scale, param_.valid_word_num, 
                                                                   head_num_, size_per_head_, head_num_x_size_per_head);
}

template<OperationType OpType_>
void OpenMultiHeadAttention<OpType_>::int8_fused_multiHeadAttr_kernelLauncher(const void* Q, 
                                                                              const float *q_deQFactor_ptr, const float *k_deQFactor_ptr, const float *v_deQFactor_ptr, 
                                                                              const float mScaleQkv, const int S)
{
#ifdef WITH_INT8
  if (int8_mode_ == 1)
  {
    trt_add_QKV_bias_COL32_int32Iint8O_kernelLauncher((int8_t*)q_buf_, 
                                                      (const int32_t*)Q,
                                                      param_.self_attention.query_weight.bias,
                                                      param_.self_attention.key_weight.bias,
                                                      param_.self_attention.value_weight.bias,
                                                      param_.amaxList+2, query_weight_amax_list, 
                                                      key_weight_amax_list, value_weight_amax_list, 
                                                      1.0f/mScaleQkv);
  }
  else if (int8_mode_ == 2)
  {
    trt_add_QKV_bias_COL32_int8IO_kernelLauncher((int8_t*)q_buf_,
                                                 (const int8_t*)Q,
                                                 param_.self_attention.query_weight.bias,
                                                 param_.self_attention.key_weight.bias,
                                                 param_.self_attention.value_weight.bias,
                                                 q_deQFactor_ptr, k_deQFactor_ptr, v_deQFactor_ptr,
                                                 1.0f/mScaleQkv
                                                );
  }

  const int B = param_.trt_seqlen_size - 1;
  dispatcher_int8->setup(S, B); 
  dispatcher_int8->run((int8_t*)q_buf_, nullptr, param_.trt_seqlen_offset, trt_attn_workspace_, (int8_t*)transpose_dst_int_buf_, param_.stream);   
    
  //transpose_dst_int_buf_ is [batch*seqlen, hidden_dim] row-major
  rowMajorToCOL32_kernelLauncher((int8_t*)(param_.attr_out), (const int8_t*)transpose_dst_int_buf_, param_.valid_word_num, head_num_*size_per_head_, param_.stream);     
#else
  printf("[ERROR] PaddleNLP does not support INT8. \n");
  exit(-1);
#endif
}


template<OperationType OpType_>
void OpenMultiHeadAttention<OpType_>::fused_multiHeadAttr_kernelLauncher(const int S)
{

  trt_add_QKV_bias_kernelLauncher(param_.self_attention.query_weight.bias,
                                  param_.self_attention.key_weight.bias,
                                  param_.self_attention.value_weight.bias);


  const int B = param_.trt_seqlen_size - 1;
  dispatcher_fp16->setup(S, B);
  dispatcher_fp16->run(q_buf_, nullptr, param_.trt_seqlen_offset, trt_attn_workspace_, param_.attr_out, param_.stream);
}

template<typename T>
__global__
void add_QKV_bias(T* Q, const T* bias_Q, T* K, const T* bias_K, T* V, const T* bias_V, T* q_buf_, T* k_buf_, T* v_buf_, 
  const int batch_size, const int seq_len, const int head_num, const int size_per_head, const int word_per_block)
{

  T* data_ptr;
  T* buf_ptr;
  const T* bias_ptr;
  
  int m = batch_size * seq_len;
  int n = head_num * size_per_head;

  int qkv_id = blockIdx.x * word_per_block / m;
  int row_offset = (blockIdx.x * word_per_block % m) * n;

  if(qkv_id == 0)
  {
    data_ptr = Q + row_offset;
    buf_ptr = q_buf_;
    bias_ptr = bias_Q;
  }
  else if(qkv_id == 1)
  {
    data_ptr = K + row_offset;
    buf_ptr = k_buf_;
    bias_ptr = bias_K;
  }
  else
  {
    data_ptr = V + row_offset;
    buf_ptr = v_buf_;
    bias_ptr = bias_V;
  }

  int batch_id = (blockIdx.x * word_per_block % m) / seq_len;
  int head_id = threadIdx.x / size_per_head;
  int id_in_head = threadIdx.x % size_per_head;
  int word_start_id = (blockIdx.x * word_per_block) % seq_len;

  T bias = __ldg(&bias_ptr[threadIdx.x]);

  for(int i = word_start_id; i < word_start_id + word_per_block; ++i)
  {
    T tmp = data_ptr[threadIdx.x] + bias;

    int target_id = batch_id * (seq_len * head_num * size_per_head) + head_id * seq_len * size_per_head + 
      i * size_per_head + id_in_head;

    buf_ptr[target_id] = tmp;
    data_ptr += n;
  }
}

template <>
__global__
void add_QKV_bias(half* Q, const half* bias_Q, half* K, const half* bias_K, half* V, const half* bias_V, 
  half* q_buf_, half* k_buf_, half* v_buf_, 
  const int batch_size, const int seq_len, const int head_num, const int size_per_head, const int word_per_block)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int batch_id = tid / (head_num * seq_len * size_per_head);
  int seq_id = (tid % (head_num * seq_len * size_per_head)) / (head_num * size_per_head);
  int head_id = (tid % (head_num * size_per_head)) / size_per_head;
  int id = tid % size_per_head;
  int target_id = target_index(batch_id, seq_id, head_id, id, batch_size, seq_len, head_num, size_per_head);

  int bias_id = threadIdx.x;

  half2* src_ptr = (half2*)Q;
  half2* dst_ptr = (half2*)q_buf_;
  const half2* bias_ptr = (const half2*)bias_Q;

  dst_ptr[target_id] = __hadd2(src_ptr[tid],  __ldg(&bias_ptr[bias_id]));

  src_ptr = (half2*)K;
  dst_ptr = (half2*)k_buf_;
  bias_ptr = (const half2*)bias_K;
  dst_ptr[target_id] = __hadd2(src_ptr[tid],  __ldg(&bias_ptr[bias_id]));

  src_ptr = (half2*)V;
  dst_ptr = (half2*)v_buf_;
  bias_ptr = (const half2*)bias_V;
  dst_ptr[target_id] = __hadd2(src_ptr[tid],  __ldg(&bias_ptr[bias_id]));
}

template<typename T>
__global__
void add_QKV_bias_rebuild_padding(T* Q, const T* bias_Q, T* K, const T* bias_K, T* V, const T* bias_V, T* q_buf_, T* k_buf_, T* v_buf_, 
  const int batch_size, const int seq_len, const int head_num, const int size_per_head, const int* mask_offset)
{
  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  const int bdim = blockDim.x;

  const int tgt_batch_id = (bid + mask_offset[bid]) / seq_len;
  const int tgt_seq_id = (bid + mask_offset[bid]) % seq_len;
  const int tgt_head_id = tid / size_per_head;
  const int tgt_hidden_id = tid % size_per_head;

  const int src_id = bid * bdim + tid;
  const int tgt_id = tgt_batch_id * head_num * seq_len * size_per_head + \
                    tgt_head_id * seq_len * size_per_head + \
                    tgt_seq_id * size_per_head + \
                    tgt_hidden_id;
  
  q_buf_[tgt_id] = Q[src_id] + bias_Q[tid];
  k_buf_[tgt_id] = K[src_id] + bias_K[tid];
  v_buf_[tgt_id] = V[src_id] + bias_V[tid];
}

template<typename T>
void add_QKV_bias_rebuild_padding_kernelLauncher(T* Q, const T* bias_Q, T* K, const T* bias_K, T* V, const T* bias_V, T* q_buf, T* k_buf, T* v_buf, 
  const int batch_size, const int seq_len, const int head_num, const int size_per_head, const int valid_word_num, 
  const int* mask_offset, hipStream_t stream)
{
  const int k = head_num*size_per_head;
    
  if(std::is_same<T, float>::value)
  {
    add_QKV_bias_rebuild_padding<<<valid_word_num, k, 0, stream>>>(Q, bias_Q, K, bias_K, 
      V, bias_V, q_buf, k_buf, v_buf, 
      batch_size, seq_len, head_num, size_per_head, mask_offset);                                                 
  }
  else
  {
    add_QKV_bias_rebuild_padding<<<valid_word_num, k / 2, 0, stream>>>((half2*)Q, (const half2*)bias_Q, 
      (half2*)K, (const half2*)bias_K, (half2*)V, (const half2*)bias_V, 
      (half2*)q_buf, (half2*)k_buf, (half2*)v_buf,
       batch_size, seq_len, head_num, size_per_head / 2, mask_offset);
  }  
}

template
void add_QKV_bias_rebuild_padding_kernelLauncher(float* Q, const float* bias_Q, float* K, const float* bias_K, float* V, const float* bias_V, float* q_buf, float* k_buf, float* v_buf, const int batch_size, const int seq_len, const int head_num, const int size_per_head, const int valid_word_num, const int* mask_offset, hipStream_t stream);

template
void add_QKV_bias_rebuild_padding_kernelLauncher(half* Q, const half* bias_Q, half* K, const half* bias_K, half* V, const half* bias_V, half* q_buf, half* k_buf, half* v_buf, const int batch_size, const int seq_len, const int head_num, const int size_per_head, const int valid_word_num, const int* mask_offset, hipStream_t stream);

template <typename T>
__global__
void softmax_kernel(T* qk_buf_, const T* attr_mask, const int batch_size, const int head_num, const int seq_len, 
  const T scalar)
{
    int batch_id = blockIdx.x / head_num;
    int qk_offset = blockIdx.x * seq_len * seq_len;
    int mask_offset = batch_id * seq_len * seq_len;

    __shared__ float s_sum, s_max;

    for(int i = 0; i < seq_len; ++i)
    {
      float qk = threadIdx.x < seq_len ? (float)qk_buf_[threadIdx.x + qk_offset] : 0.0f;
      float mask_val = threadIdx.x < seq_len ? (float)attr_mask[threadIdx.x + mask_offset] : 0.0f;
      
      mask_val = (1.0f - mask_val) * -10000.0f;

      float tmp = threadIdx.x < seq_len ? (float)(qk * (float)scalar + mask_val): -1e20f;

      float max_val = blockReduceMax<float>(tmp);

      if(threadIdx.x == 0)
        s_max = max_val;
      __syncthreads();

      qk = threadIdx.x < seq_len ? __expf(tmp - s_max) : 0.0f;

      float sum_val = blockReduceSum<float>(qk);

      if(threadIdx.x == 0)
      {
        s_sum = sum_val + 1e-6f;
      }
      __syncthreads();

      if(threadIdx.x < seq_len)
        qk_buf_[threadIdx.x + qk_offset] = (T)(qk / s_sum);

      qk_offset += seq_len;
      mask_offset += seq_len;
    }
}


template <typename T>
__global__
void softmax_kernel_v2(T* qk_buf_, const T* attr_mask, const int batch_size, const int head_num, 
  const int seq_len, const float scalar)
{
    int batch_id = blockIdx.x / head_num / seq_len;
    int seq_id = blockIdx.x % seq_len;
    int qk_offset = blockIdx.x * seq_len;
    int mask_offset = batch_id * seq_len * seq_len + seq_id * seq_len;

    __shared__ float s_sum, s_max;

    float qk = threadIdx.x < seq_len ? (float)qk_buf_[threadIdx.x + qk_offset] : 0.0f;
    float mask_val = threadIdx.x < seq_len ? (float)attr_mask[threadIdx.x + mask_offset] : 0.0f;
      
    mask_val = (1.0f - mask_val) * -10000.0f;

    float tmp = threadIdx.x < seq_len ? (float)(qk * (float)scalar + mask_val) : -1e20f;
    float max_val = blockReduceMax<float>(tmp);
    if(threadIdx.x == 0)
      s_max = max_val;
    __syncthreads();

    float qk_tmp = threadIdx.x < seq_len ? __expf((float)(tmp - s_max)) : 0.0f;
    float sum_val = blockReduceSum<float>(qk_tmp);

    if(threadIdx.x == 0)
    {
      s_sum = sum_val + 1e-6f;
    }
    __syncthreads();

    if(threadIdx.x < seq_len)
      qk_buf_[threadIdx.x + qk_offset] = (T)(qk_tmp / s_sum);
}

//grid = (seq_len/word_per_thread, batch_size, head_num)
//block.x = max(32, (seq_len + 31)/32*32)
template <typename T>
__global__
void softmax_kernel_v3(T* qk_buf_, const T* attr_mask, const int batch_size, const int head_num, const int seq_len, const T scalar)
{
    
  bool qual = threadIdx.x < seq_len;
  for (int seq_id = blockIdx.x ; seq_id < seq_len ; seq_id += gridDim.x){
    float tmp = -1e20f;
    int qk_offset;
    __shared__ float s_mean, s_max;
    if (qual){
      qk_offset = ((blockIdx.y*head_num + blockIdx.z)*seq_len + seq_id) *seq_len + threadIdx.x;
      int mask_offset = (blockIdx.y * seq_len + seq_id) * seq_len + threadIdx.x;

      float qk = static_cast<float>(qk_buf_[qk_offset]);
      float mask_val = static_cast<float>(__ldg(&attr_mask[mask_offset]));

      mask_val = (1.0f - mask_val) * -10000.0f;

      tmp = qk * static_cast<float>(scalar) + mask_val;
    }

    float max_val = blockReduceMax<float>(tmp);
    if (threadIdx.x == 0){
      s_max = max_val;
    }
    __syncthreads();
    
    float qk_tmp = qual ? __expf(tmp - s_max) : 0.0f;
    float sum_val = blockReduceSum<float>(qk_tmp);
    if (threadIdx.x == 0){
      s_mean = sum_val + 1e-6f;
      s_mean = __fdividef(1.0f, s_mean);
    }
    __syncthreads();
    
    if(qual)
      qk_buf_[qk_offset] = (T)(qk_tmp * s_mean);
  }
}  


//grid = (seq_len/word_per_thread, batch_size, head_num)
//block.x = max(32, (seq_len/2 + 31)/32*32)
//seq_len % 2 == 0
template <>
__global__
void softmax_kernel_v3(half* qk_buf_, const half* attr_mask, 
                      const int batch_size, const int head_num, 
                      const int seq_len, const half scalar)
{
  int threadIdx2 = threadIdx.x << 1;
  bool qual = threadIdx2 < seq_len;
  half2* qk_buf_half2Ptr = (half2*) qk_buf_;
  const half2* attr_mask_half2Ptr = (const half2*) attr_mask;
  __shared__ float s_mean, s_max;
  for (int seq_id = blockIdx.x ; seq_id < seq_len ; seq_id += gridDim.x){
    int qk_offset;
    half2 tmp = __float2half2_rn(0.0f);

    float max_val = -1e20f;
    half2 qk;
    if (qual){ 
      qk_offset = ((((blockIdx.y*head_num + blockIdx.z)*seq_len + seq_id) *seq_len) >> 1) + threadIdx.x;
      int mask_offset = (((blockIdx.y * seq_len + seq_id) * seq_len) >> 1) + threadIdx.x;

      qk = qk_buf_half2Ptr[qk_offset];
      half2 mask_val = __ldg(&attr_mask_half2Ptr[mask_offset]);
      half2 mask_val_tmp = __hmul2(__hsub2(__float2half2_rn(1.0f), mask_val), __float2half2_rn(-10000.0f));
      tmp = __hadd2(__hmul2(__half2half2(scalar), qk), mask_val_tmp);
      max_val = fmax((float)tmp.x, (float)tmp.y);
    }

    max_val = blockDim.x <= 32 ? warpReduceMax(max_val) : blockReduceMax<float>(max_val);

    if (threadIdx.x == 0){
      s_max = max_val;
    }
    __syncthreads();
    
    if (qual){
      tmp = h2exp(__hsub2(tmp, __float2half2_rn(s_max)));
    }
    float sum_val = blockDim.x <= 32 ? warpReduceSum((float)(tmp.x + tmp.y)) : blockReduceSum<float>((float)(tmp.x + tmp.y));

    if (threadIdx.x == 0){
      s_mean = sum_val + 1e-6f;
      s_mean = __fdividef(1.0f, s_mean);
    }
    __syncthreads();

    if(qual){
      qk = __hmul2(tmp, __float2half2_rn(s_mean));
      qk_buf_half2Ptr[qk_offset] = qk;
    }
  }
}

//grid = (seq_len/word_per_thread, batch_size, head_num)
//block.x = max(32, (seq_len + 31)/32*32)
//for seq_len not larger than 32
template <typename T>
__global__
void softmax_kernel_v3_LE32(T* qk_buf_, const T* attr_mask, const int batch_size, const int head_num, const int seq_len, const T scalar)
{
  bool qual = threadIdx.x < seq_len;
  for (int seq_id = blockIdx.x ; seq_id < seq_len ; seq_id += gridDim.x){
    int qk_offset;
    __shared__ float s_mean, s_max;
    float tmp = -1e20f;
    if (qual){
      qk_offset = ((blockIdx.y*head_num + blockIdx.z)*seq_len + seq_id) *seq_len + threadIdx.x;
      int mask_offset = (blockIdx.y * seq_len + seq_id) * seq_len + threadIdx.x;

      float qk = static_cast<float>(qk_buf_[qk_offset]);
      float mask_val = static_cast<float>(__ldg(&attr_mask[mask_offset]));

      mask_val = (1.0f - mask_val) * -10000.0f;

      tmp = static_cast<float>(qk) * static_cast<float>(scalar) + mask_val;
    }
    float max_val = warpReduceMax<float>(tmp);

    if (threadIdx.x == 0){
      s_max = max_val;
    }
    __syncthreads();

    tmp = qual ? __expf(tmp - s_max) : 0.0f;
    float sum_val = warpReduceSum<float>(tmp);

    if (threadIdx.x == 0){
      s_mean = sum_val + 1e-6f;
      s_mean = __fdividef(1.0f, s_mean);
    }
    __syncthreads();

    if(qual)
      qk_buf_[qk_offset] = (T)(tmp * s_mean);
  }
}

//input are a series of sub-matrixes of m = seq_len, n = seq_len, CUBLASLT_ORDER_COL32
//grid = (seq_len, batch_size, head_num)
//block.x = max(32, (seq_len/4 + 31)/32*32)
//for int32_t I; int8 O;
template <typename T>
__global__
void softmax_COL32(int8_t* output, const int32_t* input, const T* attr_mask, const int batch_size, 
                   const int head_num, const int seq_len, const float scalar1a, const float *scalar1b, 
                   const float *scalar1c, const float *amax_ptr, const int head_num_x_seq_len, const int seq_len_x_seq_len)
{
  const float amax = __ldg(amax_ptr);
  const float scalar1 = scalar1a * __ldg(scalar1b) * __ldg(scalar1c);
  int mask_id;
  int threadIdx4 = threadIdx.x << 2;

  char4* buf4Ptr = (char4 *)output;

  bool qual = threadIdx4 < seq_len;
  for (int seq_id = blockIdx.x ; seq_id < seq_len ; seq_id += gridDim.x){
    char4 tmp4 = {0, 0, 0, 0};
    int inIdx = (blockIdx.y * head_num + blockIdx.z) * (seq_len_x_seq_len) +
                (threadIdx4 & 0xffffffe0) * seq_len +
                (seq_id << 5) + (threadIdx4 & 31);
                
    //set softmax of padding word to 0
    float mask_in_seq = static_cast<float>(__ldg(attr_mask+(blockIdx.y*seq_len_x_seq_len + seq_id)));
    if (mask_in_seq < 0.1f){
      if (qual)
        buf4Ptr[inIdx >> 2] = tmp4;
      continue;
    }  

    float4 floatTmp4 = {0.0f, 0.0f, 0.0f, 0.0f};    

    if (qual){
      floatTmp4.x = static_cast<float>(__ldg(input + inIdx)) * scalar1;
      floatTmp4.y = static_cast<float>(__ldg(input+inIdx+1)) * scalar1;
      floatTmp4.z = static_cast<float>(__ldg(input+inIdx+2)) * scalar1;
      floatTmp4.w = static_cast<float>(__ldg(input+inIdx+3)) * scalar1;
    }

    float mask_val, max_val;
    max_val = -1e20f;

    __shared__ float s_max, s_sum;

    if (qual){
      mask_id = threadIdx4 + blockIdx.y * seq_len_x_seq_len + seq_id * seq_len;
      //for x
      mask_val = (1.0f - static_cast<float>(__ldg(attr_mask+mask_id))) * -10000.0f;
      floatTmp4.x = floatTmp4.x + mask_val;
      max_val = fmaxf(max_val, floatTmp4.x);

      //for y
      mask_val = (1.0f - static_cast<float>(__ldg(attr_mask+mask_id+1))) * -10000.0f;
      floatTmp4.y = floatTmp4.y + mask_val;
      max_val = fmaxf(max_val, floatTmp4.y);

      //for z
      mask_val = (1.0f - static_cast<float>(__ldg(attr_mask+mask_id+2))) * -10000.0f;
      floatTmp4.z = floatTmp4.z + mask_val;
      max_val = fmaxf(max_val, floatTmp4.z);

      //for w
      mask_val = (1.0f - static_cast<float>(__ldg(attr_mask+mask_id+3))) * -10000.0f;
      floatTmp4.w = floatTmp4.w + mask_val;
      max_val = fmaxf(max_val, floatTmp4.w);
    }

    max_val = blockDim.x <= 32 ? warpReduceMax(max_val) : blockReduceMax<float>(max_val);

    if (threadIdx.x == 0){
      s_max = max_val;
    }
    __syncthreads();

    float sum_val = 0.0f;

    if (qual){
      floatTmp4.x = __expf(floatTmp4.x - s_max);
      sum_val += floatTmp4.x;
      floatTmp4.y = __expf(floatTmp4.y - s_max);
      sum_val += floatTmp4.y;
      floatTmp4.z = __expf(floatTmp4.z - s_max);
      sum_val += floatTmp4.z;
      floatTmp4.w = __expf(floatTmp4.w - s_max);
      sum_val += floatTmp4.w;
    }
    
    sum_val = blockDim.x <= 32 ? warpReduceSum(sum_val) : blockReduceSum<float>(sum_val);

    if (threadIdx.x == 0){
      s_sum = __fdividef(127.0f, (sum_val + 1e-6f));
      s_sum = __fdividef(s_sum, amax);
    }
    __syncthreads();

    if (qual){

      tmp4.x = float_to_int8_rn(floatTmp4.x*s_sum);
      tmp4.y = float_to_int8_rn(floatTmp4.y*s_sum);
      tmp4.z = float_to_int8_rn(floatTmp4.z*s_sum);
      tmp4.w = float_to_int8_rn(floatTmp4.w*s_sum);

      buf4Ptr[inIdx >> 2] = tmp4;
    }
  }
}

//input are a series of sub-matrixes of m = seq_len, n = seq_len_padded, CUBLASLT_ORDER_COL32
//seq_len_padded = (seq_len+31)/32*32
//grid = (seq_len, batch_size, head_num)
//block.x = max(32, (seq_len_padded/4 + 31)/32*32)
//for int8_t IO;
template <typename T>
__global__
void softmax_COL32_varlen(int8_t* output, const int8_t* input, const T* attr_mask, const int batch_size, 
                   const int head_num, const int seq_len, const int seq_len_padded, 
                   const float scalar1a, const float *scalar1b, const float *amax_ptr, 
                   const int seq_len_x_seq_len, const int seq_len_x_seq_len_padded)
{
  const float amax = __ldg(amax_ptr);
  const float scalar1 = scalar1a * __ldg(scalar1b);
  int mask_id;
  int threadIdx4 = threadIdx.x << 2;

  char4* buf4Ptr = (char4 *)output;
  const char4* inBuf4Ptr = (const char4*)input;

  const bool qual = threadIdx4 < seq_len;
  const bool qual_padded = threadIdx4 < seq_len_padded;
  for (int seq_id = blockIdx.x ; seq_id < seq_len ; seq_id += gridDim.x){
  
    char4 tmp4 = {0, 0, 0, 0};
    int inIdx = ((blockIdx.y * head_num + blockIdx.z) * (seq_len_x_seq_len_padded) +
                (threadIdx4 & 0xffffffe0) * seq_len +
                (seq_id << 5) + (threadIdx4 & 31)) >> 2;
  
    //set softmax of padding word in rows to 0
    const float mask_in_seq = static_cast<float>(__ldg(attr_mask+(blockIdx.y*seq_len_x_seq_len + seq_id)));
    if (mask_in_seq < 0.1f){
      if (qual_padded)
        buf4Ptr[inIdx] = tmp4;
      continue;
    }
    
    //set softmax of padding word in cols to 0
    float4 floatTmp4 = {0.0f, 0.0f, 0.0f, 0.0f};
    if (qual){
      tmp4 = __ldg(inBuf4Ptr + inIdx);
      floatTmp4.x = static_cast<float>(tmp4.x) * scalar1;
      floatTmp4.y = static_cast<float>(tmp4.y) * scalar1;
      floatTmp4.z = static_cast<float>(tmp4.z) * scalar1;
      floatTmp4.w = static_cast<float>(tmp4.w) * scalar1;
    }

    float mask_val, max_val;
    max_val = -1e20f;

    __shared__ float s_max, s_sum;

    if (qual){
      mask_id = threadIdx4 + blockIdx.y * seq_len_x_seq_len + seq_id * seq_len;
      //for x
      mask_val = (1.0f - static_cast<float>(__ldg(attr_mask+mask_id))) * -10000.0f;
      floatTmp4.x = floatTmp4.x + mask_val;
      max_val = fmaxf(max_val, floatTmp4.x);

      //for y
      mask_val = (1.0f - static_cast<float>(__ldg(attr_mask+mask_id+1))) * -10000.0f;
      floatTmp4.y = floatTmp4.y + mask_val;
      max_val = fmaxf(max_val, floatTmp4.y);

      //for z
      mask_val = (1.0f - static_cast<float>(__ldg(attr_mask+mask_id+2))) * -10000.0f;
      floatTmp4.z = floatTmp4.z + mask_val;
      max_val = fmaxf(max_val, floatTmp4.z);

      //for w
      mask_val = (1.0f - static_cast<float>(__ldg(attr_mask+mask_id+3))) * -10000.0f;
      floatTmp4.w = floatTmp4.w + mask_val;
      max_val = fmaxf(max_val, floatTmp4.w);
    }

    max_val = blockDim.x <= 32 ? warpReduceMax(max_val) : blockReduceMax<float>(max_val);

    if (threadIdx.x == 0){
      s_max = max_val;
    }
    __syncthreads();

    float sum_val = 0.0f;

    if (qual){
      floatTmp4.x = __expf(floatTmp4.x - s_max);
      sum_val += floatTmp4.x;
      floatTmp4.y = __expf(floatTmp4.y - s_max);
      sum_val += floatTmp4.y;
      floatTmp4.z = __expf(floatTmp4.z - s_max);
      sum_val += floatTmp4.z;
      floatTmp4.w = __expf(floatTmp4.w - s_max);
      sum_val += floatTmp4.w;
    }
    
    sum_val = blockDim.x <= 32 ? warpReduceSum(sum_val) : blockReduceSum<float>(sum_val);

    if (threadIdx.x == 0){
      s_sum = __fdividef(127.0f, (sum_val + 1e-6f));
      s_sum = __fdividef(s_sum, amax);
    }
    __syncthreads();

    if (qual_padded){

      tmp4.x = qual ? float_to_int8_rn(floatTmp4.x*s_sum) : static_cast<int8_t>(0);
      tmp4.y = qual ? float_to_int8_rn(floatTmp4.y*s_sum) : static_cast<int8_t>(0);
      tmp4.z = qual ? float_to_int8_rn(floatTmp4.z*s_sum) : static_cast<int8_t>(0);
      tmp4.w = qual ? float_to_int8_rn(floatTmp4.w*s_sum) : static_cast<int8_t>(0);

      buf4Ptr[inIdx] = tmp4;
    }
  }
}

//input are a series of sub-matrixes of m = seq_len, n = seq_len_padded, CUBLASLT_ORDER_COL32
//seq_len_padded = (seq_len+31)/32*32
//grid = (seq_len, batch_size, head_num)
//block.x = max(32, (seq_len_padded + 31)/32*32)
//for int8_t IO, I/O with int8_t element;
template <typename T>
__global__
void softmax_COL32_perElement_varlen(int8_t* output, const int8_t* input, const T* attr_mask, const int batch_size,
                                     const int head_num, const int seq_len, const int seq_len_padded,
                                     const float scalar1a, const float *scalar1b, const float *amax_ptr,
                                     const int seq_len_x_seq_len, const int seq_len_x_seq_len_padded)
{
  const float amax = __ldg(amax_ptr);
  const float scalar1 = scalar1a * __ldg(scalar1b);
  int mask_id;
  const int tidx = threadIdx.x;

  const bool qual = tidx < seq_len;
  const bool qual_padded = tidx < seq_len_padded;
  for (int seq_id = blockIdx.x ; seq_id < seq_len ; seq_id += gridDim.x){

    int8_t tmp = 0;
    int inIdx = ((blockIdx.y * head_num + blockIdx.z) * (seq_len_x_seq_len_padded) +
                (tidx & 0xffffffe0) * seq_len +
                (seq_id << 5) + (tidx & 31));

    //set softmax of padding word in rows to 0
    const float mask_in_seq = static_cast<float>(__ldg(attr_mask+(blockIdx.y*seq_len_x_seq_len + seq_id)));
    if (mask_in_seq < 0.1f){
      if (qual_padded)
        output[inIdx] = tmp;
      continue;
    }

    //set softmax of padding word in cols to 0
    float floatTmp = qual ? (static_cast<float>(__ldg(input + inIdx)) * scalar1) : 0.0f;

    float mask_val, max_val;
    max_val = -1e20f;

    __shared__ float s_max, s_sum;

    if (qual){
      mask_id = tidx + blockIdx.y * seq_len_x_seq_len + seq_id * seq_len;
      mask_val = (1.0f - static_cast<float>(__ldg(attr_mask+mask_id))) * -10000.0f;
      floatTmp = floatTmp + mask_val;
    }

    max_val = blockDim.x <= 32 ? warpReduceMax(floatTmp) : blockReduceMax<float>(floatTmp);

    if (threadIdx.x == 0){
      s_max = max_val;
    }
    __syncthreads();

    float sum_val = 0.0f;

    floatTmp = qual ? __expf(floatTmp - s_max) : floatTmp;

    sum_val = blockDim.x <= 32 ? warpReduceSum(floatTmp) : blockReduceSum<float>(floatTmp);

    if (threadIdx.x == 0){
      s_sum = __fdividef(127.0f, (sum_val + 1e-6f));
      s_sum = __fdividef(s_sum, amax);
    }
    __syncthreads();

    if (qual_padded){
      tmp = qual ? float_to_int8_rn(floatTmp*s_sum) : static_cast<int8_t>(0);
      output[inIdx] = tmp;
    }
  }
}


//input are a series of sub-matrixes of m = seq_len, n = seq_len, CUBLASLT_ORDER_COL32
//grid = (seq_len, batch_size, head_num)
//block.x = (seq_len + 31)/32
//for int32_t I; int8 O;
//for seq_len <= 32
template <typename T>
__global__
void softmax_COL32_LE32(int8_t* output, const int32_t* input, const T* attr_mask, const int batch_size, 
                        const int head_num, const int seq_len, const float scalar1a, const float *scalar1b, 
                        const float *scalar1c, const float *amax_ptr, const int head_num_x_seq_len, const int seq_len_x_seq_len)
{
  const float amax = __ldg(amax_ptr);
  const float scalar1 = scalar1a * __ldg(scalar1b) * __ldg(scalar1c);
  int mask_id;
  int threadIdxx = threadIdx.x;
  bool qual = threadIdxx < seq_len;
  for (int seq_id = blockIdx.x ; seq_id < seq_len ; seq_id += gridDim.x){
    int inIdx = (blockIdx.y * head_num + blockIdx.z) * (seq_len_x_seq_len) +
                (threadIdxx & 0xffffffe0) * seq_len +
                (seq_id << 5) + (threadIdxx & 31);
  
    //set softmax of padding word to 0
    float mask_in_seq = static_cast<float>(__ldg(attr_mask+(blockIdx.y*seq_len_x_seq_len + seq_id)));
    if (mask_in_seq < 0.1f){
      if (qual)
        output[inIdx] = 0;
      continue;
    }

    float floatTmp = qual ? static_cast<float>(__ldg(input + inIdx)) * scalar1 : 0.0f;

    float mask_val, max_val;

    __shared__ float s_max, s_sum;

    mask_id = qual ? threadIdxx + blockIdx.y * seq_len_x_seq_len + seq_id * seq_len : 0;
    mask_val = qual ? (1.0f - static_cast<float>(__ldg(attr_mask+mask_id))) * -10000.0f : 0.0f;
    floatTmp = qual ? floatTmp + mask_val : 0.0f;
    max_val = qual ? floatTmp : -1e20f;

    max_val = blockDim.x <= 32 ? warpReduceMax(max_val) : blockReduceMax<float>(max_val);

    if (threadIdx.x == 0){
      s_max = max_val;
    }
    __syncthreads();

    floatTmp = qual ? __expf(floatTmp - s_max) : 0.0f;
    
    float sum_val = blockDim.x <= 32 ? warpReduceSum(floatTmp) : blockReduceSum<float>(floatTmp);

    if (threadIdx.x == 0){
      s_sum = __fdividef(127.0f, (sum_val + 1e-6f));
      s_sum = __fdividef(s_sum, amax);
    }
    __syncthreads();

    
    if (qual){
      output[inIdx] = float_to_int8_rn(floatTmp*s_sum);
    }
  }
}

//input are a series of sub-matrixes of m = seq_len, n = seq_len_padded, CUBLASLT_ORDER_COL32
//seq_len_padded = (seq_len+31)/32*32
//attr_mask is [batch_size, seq_len, seq_len]
//grid = (seq_len, batch_size, head_num)
//block.x = seq_len_padded
//for int8_t IO;
//for seq_len_padded == 32
template <typename T>
__global__
void softmax_COL32_LE32_varlen(int8_t* output, const int8_t* input, const T* attr_mask, const int batch_size, 
                        const int head_num, const int seq_len, const int seq_len_padded,
                        const float scalar1a, const float *scalar1b, const float *amax_ptr, 
                        const int seq_len_x_seq_len, const int seq_len_x_seq_len_padded)
{
  const float amax = __ldg(amax_ptr);
  const float scalar1 = scalar1a * __ldg(scalar1b);
  int mask_id;
  int threadIdxx = threadIdx.x;
  const bool qual = threadIdxx < seq_len;
  const bool qual_padded = threadIdxx < seq_len_padded;
  for (int seq_id = blockIdx.x ; seq_id < seq_len ; seq_id += gridDim.x){
    int inIdx = (blockIdx.y * head_num + blockIdx.z) * (seq_len_x_seq_len_padded) +
                (threadIdxx & 0xffffffe0) * seq_len +
                (seq_id << 5) + (threadIdxx & 31);

    //set softmax of padding word in rows to 0
    float mask_in_seq = static_cast<float>(__ldg(attr_mask+(blockIdx.y*seq_len_x_seq_len + seq_id)));
    if (mask_in_seq < 0.1f){
      if (qual_padded)
        output[inIdx] = 0;
      continue;
    }

    float mask_val, max_val;
    __shared__ float s_max, s_sum;

    //set softmax of padding word in cols to 0
    float floatTmp = qual ? static_cast<float>(__ldg(input + inIdx)) * scalar1 : 0.0f;
    mask_id = qual ? threadIdxx + blockIdx.y * seq_len_x_seq_len + seq_id * seq_len : 0;
    mask_val = qual ? (1.0f - static_cast<float>(__ldg(attr_mask+mask_id))) * -10000.0f : 0.0f;
    floatTmp = qual ? floatTmp + mask_val : 0.0f;
    max_val = qual ? floatTmp : -1e20f;

    max_val = warpReduceMax(max_val);

    if (threadIdx.x == 0){
      s_max = max_val;
    }
    __syncthreads();

    floatTmp = qual ? __expf(floatTmp - s_max) : 0.0f;
    
    float sum_val = blockDim.x <= 32 ? warpReduceSum(floatTmp) : blockReduceSum<float>(floatTmp);

    if (threadIdx.x == 0){
      s_sum = __fdividef(127.0f, (sum_val + 1e-6f));
      s_sum = __fdividef(s_sum, amax);
    }
    __syncthreads();

    
    if (qual_padded){
      output[inIdx] = qual ? float_to_int8_rn(floatTmp*s_sum) : static_cast<int8_t>(0);
    }
  }
}

//input are a series of sub-matrixes of m = seq_len, n = seq_len, CUBLASLT_ORDER_COL32
//grid = (seq_len, batch_size, head_num)
//block.x = max(32, (seq_len/2 + 31)/32*32)
//for int32_t I; int8 O;
//for seq_len in (32, 64]
template <typename T>
__global__
void softmax_COL32_LE64(int8_t* output, const int32_t* input, const T* attr_mask, const int batch_size, 
                        const int head_num, const int seq_len, const float scalar1a, const float *scalar1b, 
                        const float *scalar1c, const float *amax_ptr, const int head_num_x_seq_len, const int seq_len_x_seq_len)
{
  const float amax = __ldg(amax_ptr);
  const float scalar1 = scalar1a * __ldg(scalar1b) * __ldg(scalar1c);
  int mask_id;
  int threadIdx2 = threadIdx.x << 1;

  char2* buf2Ptr = (char2 *)output;

  bool qual = threadIdx2 < seq_len;
  for (int seq_id = blockIdx.x ; seq_id < seq_len ; seq_id += gridDim.x){
    char2 tmp2 = {0, 0};
    int inIdx = (blockIdx.y * head_num + blockIdx.z) * (seq_len_x_seq_len) +
                (threadIdx2 & 0xffffffe0) * seq_len +
                (seq_id << 5) + (threadIdx2 & 31);

    //set softmax of padding word to 0
    float mask_in_seq = static_cast<float>(__ldg(attr_mask+(blockIdx.y*seq_len_x_seq_len + seq_id)));
    if (mask_in_seq < 0.1f){
      if (qual)
        buf2Ptr[inIdx >> 1] = tmp2;
      continue;
    }

    float2 floatTmp2 = {0.0f, 0.0f};
    if (qual){
      floatTmp2.x = static_cast<float>(__ldg(input + inIdx)) * scalar1;
      floatTmp2.y = static_cast<float>(__ldg(input + inIdx + 1)) * scalar1;
    }

    float mask_val, max_val;
    max_val = -1e20f;

    __shared__ float s_max, s_sum;

    if (qual){
      mask_id = threadIdx2 + blockIdx.y * seq_len_x_seq_len + seq_id * seq_len;
      //for x
      mask_val = (1.0f - static_cast<float>(__ldg(attr_mask+mask_id))) * -10000.0f;
      floatTmp2.x = floatTmp2.x + mask_val;

      //for y
      mask_val = (1.0f - static_cast<float>(__ldg(attr_mask+mask_id+1))) * -10000.0f;
      floatTmp2.y = floatTmp2.y + mask_val;
            
      max_val = fmaxf(floatTmp2.x, floatTmp2.y);
    }

    max_val = blockDim.x <= 32 ? warpReduceMax(max_val) : blockReduceMax<float>(max_val);

    if (threadIdx.x == 0){
      s_max = max_val;
    }
    __syncthreads();

    float sum_val = 0.0f;

    if (qual){
      floatTmp2.x = __expf(floatTmp2.x - s_max);
      sum_val += floatTmp2.x;
      floatTmp2.y = __expf(floatTmp2.y - s_max);
      sum_val += floatTmp2.y;
    }
    
    sum_val = blockDim.x <= 32 ? warpReduceSum(sum_val) : blockReduceSum<float>(sum_val);

    if (threadIdx.x == 0){
      s_sum = __fdividef(127.0f, (sum_val + 1e-6f));
      s_sum = __fdividef(s_sum, amax);
    }
    __syncthreads();

    if (qual){
      tmp2.x = float_to_int8_rn(floatTmp2.x*s_sum);
      tmp2.y = float_to_int8_rn(floatTmp2.y*s_sum);
      buf2Ptr[inIdx >> 1] = tmp2;
    }
  }
}

//input are a series of sub-matrixes of m = seq_len, n = seq_len_padded, CUBLASLT_ORDER_COL32
//seq_len_padded = (seq_len+31)/32*32
//grid = (seq_len, batch_size, head_num)
//block.x = 32
//for int8_t IO
//for seq_len in (32, 64]
template <typename T>
__global__
void softmax_COL32_LE64_varlen(int8_t* output, const int8_t* input, const T* attr_mask, const int batch_size, 
                        const int head_num, const int seq_len, const int seq_len_padded, 
                        const float scalar1a, const float *scalar1b, const float *amax_ptr, 
                        const int seq_len_x_seq_len, const int seq_len_x_seq_len_padded)
{
  const float amax = __ldg(amax_ptr);
  const float scalar1 = scalar1a * __ldg(scalar1b);
  int mask_id;
  int threadIdx2 = threadIdx.x << 1;

  char2* buf2Ptr = (char2 *)output;
  const char2* inBuf2Ptr = (const char2 *)input;

  const bool qual = threadIdx2 < seq_len;
  const bool qual_padded = threadIdx2 < seq_len_padded;
  for (int seq_id = blockIdx.x ; seq_id < seq_len ; seq_id += gridDim.x){
    char2 tmp2 = {0, 0};
    int inIdx = ((blockIdx.y * head_num + blockIdx.z) * (seq_len_x_seq_len_padded) +
                (threadIdx2 & 0xffffffe0) * seq_len +
                (seq_id << 5) + (threadIdx2 & 31)) >> 1;

    //set softmax of padding word in rows to 0
    float mask_in_seq = static_cast<float>(__ldg(attr_mask+(blockIdx.y*seq_len_x_seq_len + seq_id)));
    if (mask_in_seq < 0.1f){
      if (qual_padded)
        buf2Ptr[inIdx] = tmp2;
      continue;
    }

    //set softmax of padding word in cols to 0
    float2 floatTmp2 = {0.0f, 0.0f};
    if (qual){
      tmp2 = __ldg(inBuf2Ptr + inIdx);
      floatTmp2.x = static_cast<float>(tmp2.x) * scalar1;
      floatTmp2.y = static_cast<float>(tmp2.y) * scalar1;
    }

    float mask_val, max_val;
    max_val = -1e20f;

    __shared__ float s_max, s_sum;

    if (qual){
      mask_id = threadIdx2 + blockIdx.y * seq_len_x_seq_len + seq_id * seq_len;
      //for x
      mask_val = (1.0f - static_cast<float>(__ldg(attr_mask+mask_id))) * -10000.0f;
      floatTmp2.x = floatTmp2.x + mask_val;

      //for y
      mask_val = (1.0f - static_cast<float>(__ldg(attr_mask+mask_id+1))) * -10000.0f;
      floatTmp2.y = floatTmp2.y + mask_val;
            
      max_val = fmaxf(floatTmp2.x, floatTmp2.y);
    }

    max_val = warpReduceMax(max_val);

    if (threadIdx.x == 0){
      s_max = max_val;
    }
    __syncthreads();

    float sum_val = 0.0f;

    if (qual){
      floatTmp2.x = __expf(floatTmp2.x - s_max);
      sum_val += floatTmp2.x;
      floatTmp2.y = __expf(floatTmp2.y - s_max);
      sum_val += floatTmp2.y;
    }
    
    sum_val = warpReduceSum(sum_val);

    if (threadIdx.x == 0){
      s_sum = __fdividef(127.0f, (sum_val + 1e-6f));
      s_sum = __fdividef(s_sum, amax);
    }
    __syncthreads();

    if (qual_padded){
      tmp2.x = qual ? float_to_int8_rn(floatTmp2.x*s_sum) : static_cast<int8_t>(0);
      tmp2.y = qual ? float_to_int8_rn(floatTmp2.y*s_sum) : static_cast<int8_t>(0);
      buf2Ptr[inIdx] = tmp2;
    }
  }
}


template <typename T>
void softmax_COL32_kernelLauncher(int8_t* output, const int32_t* input, const T* attr_mask, 
                                  const int batch_size, const int head_num, const int seq_len, 
                                  const float scalar1a, const float *scalar1b, const float *scalar1c, 
                                  const float *amax_ptr, hipStream_t stream)
{
  dim3 grid, block;
  grid.x = seq_len;
  grid.y = batch_size;
  grid.z = head_num;

  if (seq_len <= 32){
    if (batch_size * head_num > 960)
      grid.x = ceil(float(seq_len)/32.0f);
    block.x = (seq_len + 31)/32*32;
    softmax_COL32_LE32<<<grid, block, 0, stream>>>(output, input, attr_mask, batch_size, head_num, 
                                                   seq_len, scalar1a, scalar1b, scalar1c, 
                                                   amax_ptr, seq_len*head_num, seq_len*seq_len);
  }
  else if (seq_len <= 64){
    assert(seq_len % 2 == 0);
    block.x = (seq_len/2 + 31)/32*32;
    if (batch_size * head_num > 960)
      grid.x = ceil(float(seq_len)/32.0f);
    softmax_COL32_LE64<<<grid, block, 0, stream>>>(output, input, attr_mask, batch_size, head_num, 
                                                   seq_len, scalar1a, scalar1b, scalar1c, 
                                                   amax_ptr, seq_len*head_num, seq_len*seq_len);
  }
  else
  {
    assert(seq_len % 4 == 0);
    block.x = (seq_len/4 + 31)/32*32;
    softmax_COL32<<<grid, block, 0, stream>>>(output, input, attr_mask, batch_size, head_num, 
                                              seq_len, scalar1a, scalar1b, scalar1c, 
                                              amax_ptr, seq_len*head_num, seq_len*seq_len);
  }
}

template
void softmax_COL32_kernelLauncher(int8_t* output, const int32_t* input, const float* attr_mask, 
                                  const int batch_size, const int head_num, const int seq_len, 
                                  const float scalar1a, const float *scalar1b, const float *scalar1c, 
                                  const float *amax_ptr, hipStream_t stream);
                                  
template
void softmax_COL32_kernelLauncher(int8_t* output, const int32_t* input, const half* attr_mask, 
                                  const int batch_size, const int head_num, const int seq_len, 
                                  const float scalar1a, const float *scalar1b, const float *scalar1c, 
                                  const float *amax_ptr, hipStream_t stream);

template <typename T>
void softmax_COL32_kernelLauncher(int8_t* output, const int8_t* input, const T* attr_mask, 
                                  const int batch_size, const int head_num, const int seq_len, 
                                  const float scalar1a, const float *scalar1b, const float *amax_ptr, 
                                  hipStream_t stream)
{
  dim3 grid, block;
  grid.x = seq_len;
  grid.y = batch_size;
  grid.z = head_num;
  const int seq_len_padded = (seq_len + 31)/32*32;

  if (seq_len <= 32){
    if (batch_size * head_num > 960)
      grid.x = ceil(float(seq_len)/32.0f);
    block.x = seq_len_padded;
    softmax_COL32_LE32_varlen<<<grid, block, 0, stream>>>(output, input, attr_mask, batch_size, head_num, 
                                                   seq_len, seq_len_padded, scalar1a, scalar1b, amax_ptr,
                                                   seq_len*seq_len, seq_len*seq_len_padded);
  }
  else if (seq_len <= 64 && (seq_len % 2 == 0)){
    block.x = 32;
    if (batch_size * head_num > 960)
      grid.x = ceil(float(seq_len)/32.0f);
    softmax_COL32_LE64_varlen<<<grid, block, 0, stream>>>(output, input, attr_mask, batch_size, head_num, 
                                                   seq_len, seq_len_padded, scalar1a, scalar1b, amax_ptr,
                                                   seq_len*seq_len, seq_len*seq_len_padded);
  }
  else if (seq_len > 64 && (seq_len % 4 == 0))
  {
    block.x = (seq_len_padded/4 + 31)/32*32;
    softmax_COL32_varlen<<<grid, block, 0, stream>>>(output, input, attr_mask, batch_size, head_num, 
                                              seq_len, seq_len_padded, scalar1a, scalar1b, amax_ptr,
                                              seq_len*seq_len, seq_len*seq_len_padded);
  }
  else
  {
    block.x = (seq_len_padded + 31)/32*32;
    softmax_COL32_perElement_varlen<<<grid, block, 0, stream>>>(output, input, attr_mask, batch_size, head_num,
                                              seq_len, seq_len_padded, scalar1a, scalar1b, amax_ptr,
                                              seq_len*seq_len, seq_len*seq_len_padded);
  }
}

template
void softmax_COL32_kernelLauncher(int8_t* output, const int8_t* input, const float* attr_mask, 
                                  const int batch_size, const int head_num, const int seq_len, 
                                  const float scalar1a, const float *scalar1b, 
                                  const float *amax_ptr, hipStream_t stream);
                                  
template
void softmax_COL32_kernelLauncher(int8_t* output, const int8_t* input, const half* attr_mask, 
                                  const int batch_size, const int head_num, const int seq_len, 
                                  const float scalar1a, const float *scalar1b, 
                                  const float *amax_ptr, hipStream_t stream);                                  


template<typename T>
__global__
void transpose(T* src, T* dst, const int batch_size, const int seq_len, const int head_num, const int size_per_head)
{
  int batch_id = blockIdx.x / (head_num * seq_len);
  int seq_id = blockIdx.x % seq_len;
  int head_id = (blockIdx.x % (head_num * seq_len))/ seq_len;
  dst[batch_id * (head_num * seq_len * size_per_head) + seq_id * head_num * size_per_head
    + head_id * size_per_head + threadIdx.x] = src[blockIdx.x * size_per_head + threadIdx.x];
}

template<>
  __global__
void transpose(half* src, half* dst,
    const int batch_size, const int seq_len, const int head_num, const int size_per_head)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  int batch_id = tid / (head_num * seq_len * size_per_head);
  int head_id = (tid % (head_num * seq_len * size_per_head)) / (seq_len * size_per_head);
  int seq_id = (tid % (seq_len * size_per_head)) / size_per_head;
  int id = tid % size_per_head;

  int target_id = target_index(batch_id, head_id, seq_id, id, batch_size, head_num, seq_len, size_per_head);
  half2* src_ptr = (half2*)src;
  half2* dst_ptr = (half2*)dst;

  dst_ptr[target_id] = src_ptr[tid];
}

template<typename T>
void transpose_kernelLauncher(T* src, T* dst, const int batch_size, const int seq_len, const int head_num, const int size_per_head, hipStream_t stream)
{
  dim3 grid, block;
  if (std::is_same<T, float>::value)
  {
    const int seq_per_block = 1;
    grid.x = batch_size * head_num * seq_len / seq_per_block;
    block.x = seq_per_block * size_per_head;
    transpose<<<grid, block, 0, stream>>>(src, dst, batch_size, seq_len, head_num, size_per_head);
  }
  else
  {
    const int seq_per_block = 4;
    grid.x = batch_size * head_num * seq_len / seq_per_block;
    block.x = seq_per_block * size_per_head / 2;
    assert(grid.x * seq_per_block == batch_size * head_num * seq_len);
    transpose<<<grid, block, 0, stream>>>(src, dst, batch_size, seq_len, head_num, size_per_head / 2);
  }
}

template
void transpose_kernelLauncher(float* src, float* dst, const int batch_size, const int seq_len, const int head_num, const int size_per_head, hipStream_t stream);

template
void transpose_kernelLauncher(half* src, half* dst, const int batch_size, const int seq_len, const int head_num, const int size_per_head, hipStream_t stream);

template<typename T>
__global__
void transpose_rebuild_padding(T* src, T* dst, const int batch_size, const int seq_len, const int head_num, const int size_per_head,
  const int* mask_offset)
{
  // TODO: optimize this kernel? 
  // do remove_sequence_length_padding
  const int tid = threadIdx.x; // batch * seq_len or valid_word_num
  const int bid = blockIdx.x; // head_num * size_per_head

  const int src_batch_id = (bid + mask_offset[bid]) / seq_len;
  const int src_seq_id = (bid + mask_offset[bid]) % seq_len;

  const int dst_seq_id = bid;

  const int head_id = tid / size_per_head;
  const int hidden_id = tid % size_per_head;
  dst[dst_seq_id * head_num * size_per_head + tid] = src[ src_batch_id * head_num * seq_len * size_per_head +
    head_id * seq_len * size_per_head + src_seq_id * size_per_head + hidden_id];
}

template<typename T>
void transpose_rebuild_padding_kernelLauncher(T* src, T* dst, const int valid_word_num,
                                              const int batch_size, const int seq_len, 
                                              const int head_num, const int size_per_head, 
                                              const int* mask_offset, hipStream_t stream)
{
  int k = head_num * size_per_head;
  if (std::is_same<T, float>::value)
  {
    transpose_rebuild_padding<<<valid_word_num, k, 0, stream>>>(src, dst, 
            batch_size, seq_len, head_num, size_per_head, mask_offset);
  }
  else
  {
    transpose_rebuild_padding<half2><<<valid_word_num, k / 2, 0, stream>>>(
            (half2*)src, (half2*)dst, 
            batch_size, seq_len, head_num, size_per_head / 2, mask_offset);
  }  
}

template
void transpose_rebuild_padding_kernelLauncher(float* src, float* dst, const int valid_word_num,
                                              const int batch_size, const int seq_len, 
                                              const int head_num, const int size_per_head, 
                                              const int* mask_offset, hipStream_t stream);
                                              
template
void transpose_rebuild_padding_kernelLauncher(half* src, half* dst, const int valid_word_num,
                                              const int batch_size, const int seq_len, 
                                              const int head_num, const int size_per_head, 
                                              const int* mask_offset, hipStream_t stream);

template<typename T>
__global__ void rebuild_sequence_length_padding(const T* src, T* tgt,
                                            const int* mask_offset,
                                            const int n)
{
  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  const int tgt_seq_id = bid + mask_offset[bid];
  const int src_seq_id = bid;

  for(int i = tid; i < n; i += blockDim.x)
  {
    tgt[tgt_seq_id * n + i] = src[src_seq_id * n + i];
  }
}

template void OpenMultiHeadAttention<OperationType::FP32>::trt_add_QKV_bias_kernelLauncher(
  const float* bias_Q,
  const float* bias_K,
  const float* bias_V);

template void OpenMultiHeadAttention<OperationType::FP16>::trt_add_QKV_bias_kernelLauncher(
  const half* bias_Q,
  const half* bias_K,
  const half* bias_V);

template void OpenMultiHeadAttention<OperationType::FP32>::trt_add_QKV_bias_COL32_int8IO_kernelLauncher(
  int8_t* output,
  const int8_t* Q,
  const float* bias_Q,
  const float* bias_K,
  const float* bias_V,
  const float *q_input_deQFactor_ptr, 
  const float *k_input_deQFactor_ptr, 
  const float *v_input_deQFactor_ptr, 
  const float qkv_output_scale);

template void OpenMultiHeadAttention<OperationType::FP16>::trt_add_QKV_bias_COL32_int8IO_kernelLauncher(
  int8_t* output,
  const int8_t* Q,
  const half* bias_Q,
  const half* bias_K,
  const half* bias_V,
  const float *q_input_deQFactor_ptr, 
  const float *k_input_deQFactor_ptr, 
  const float *v_input_deQFactor_ptr, 
  const float qkv_output_scale);

template void OpenMultiHeadAttention<OperationType::FP32>::trt_add_QKV_bias_COL32_int32Iint8O_kernelLauncher(
  int8_t* output,
  const int32_t* Q,
  const float* bias_Q,
  const float* bias_K,
  const float* bias_V,
  const float *input_deQFactor_div127_ptr,
  const float * q_weight_amax,
  const float * k_weight_amax,
  const float * v_weight_amax,
  const float qkv_output_scale);

template void OpenMultiHeadAttention<OperationType::FP16>::trt_add_QKV_bias_COL32_int32Iint8O_kernelLauncher(
  int8_t* output,
  const int32_t* Q,
  const half* bias_Q,
  const half* bias_K,
  const half* bias_V,
  const float *input_deQFactor_div127_ptr,
  const float * q_weight_amax,
  const float * k_weight_amax,
  const float * v_weight_amax,
  const float qkv_output_scale);

template void OpenMultiHeadAttention<OperationType::FP32>::fused_multiHeadAttr_kernelLauncher(const int S);
template void OpenMultiHeadAttention<OperationType::FP16>::fused_multiHeadAttr_kernelLauncher(const int S);

template void OpenMultiHeadAttention<OperationType::FP32>::int8_fused_multiHeadAttr_kernelLauncher(
  const void* Q, 
  const float *q_deQFactor_ptr, const float *k_deQFactor_ptr, const float *v_deQFactor_ptr, 
  const float mScaleQkv, const int S);
template void OpenMultiHeadAttention<OperationType::FP16>::int8_fused_multiHeadAttr_kernelLauncher(
  const void* Q, 
  const float *q_deQFactor_ptr, const float *k_deQFactor_ptr, const float *v_deQFactor_ptr, 
  const float mScaleQkv, const int S);

__global__
void trt_add_QKV_bias_2(const half2* Q, const half2* bias_Q, 
                        const half2* K, const half2* bias_K, 
                        const half2* V, const half2* bias_V, 
                        half2* qkv_buf_,  
                        const int valid_word_num, 
                        const int head_num, const int size_per_head)
{
  // Add bias, and then transpose from 
  // [3, valid_word_num, head, size] -> [valid_word_num, head, 3, size]

  const int seq_id = blockIdx.x;
  const int size_id = threadIdx.x % size_per_head;
  const int head_id = (threadIdx.x - size_id) / size_per_head;

  const int target_offset = blockIdx.x * head_num * 3 * size_per_head + head_id * 3 * size_per_head;

  qkv_buf_[ target_offset + 
          0 * size_per_head +
          size_id] = Q[ seq_id * blockDim.x + threadIdx.x] + bias_Q[threadIdx.x];

  qkv_buf_[ target_offset + 
          1 * size_per_head +
          size_id] = K[ seq_id * blockDim.x + threadIdx.x] + bias_K[threadIdx.x];

  qkv_buf_[ target_offset + 
          2 * size_per_head +
          size_id] = V[ seq_id * blockDim.x + threadIdx.x] + bias_V[threadIdx.x];
}

void trt_add_QKV_bias_transpose_debug_kernelLauncher(
  const half* query_buf, const half* bias_Q,
  const half* key_buf, const half* bias_K,
  const half* value_buf, const half* bias_V,
  half* context_buf, 
  const int valid_word_num, 
  const int head_num, const int size_per_head,
  hipStream_t stream)
{
  dim3 grid;
  dim3 block;
  
  grid.x = 3 * valid_word_num;
  block.x = head_num * size_per_head / 2;
  
  assert(block.x <= 1024);

  trt_add_QKV_bias_2<<<grid, block, 0, stream>>>( (const half2*)query_buf, (const half2*)bias_Q, 
                                                  (const half2*)key_buf, (const half2*)bias_K, 
                                                  (const half2*)value_buf, (const half2*)bias_V, 
                                                  (half2*)context_buf, 
                                                  valid_word_num, 
                                                  head_num, size_per_head / 2);
}


}//namespace cuda
}//namespace fastertransformer

