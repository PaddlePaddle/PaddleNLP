#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
 * Copyright (c) 2020-2021, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

namespace fastertransformer {

template <typename T>
__global__ void transpose_cache_batch_major(T* k_dst,
                                            T* v_dst,
                                            const T* k_src,
                                            const T* v_src,
                                            const int* memory_seq_len,
                                            const int head_num,
                                            const int size_per_head,
                                            const int memory_max_seq_len,
                                            const int cache_max_len) {
  const int hidden_dim = head_num * size_per_head;
  const int x = (sizeof(T) == 4) ? 4 : 8;
  const int size_per_head_split = size_per_head / x;
  const int batch_id = blockIdx.x;
  const int seq_id = blockIdx.y;

  for (int id = threadIdx.x; id < head_num * size_per_head_split * x;
       id += blockDim.x) {
    int tmp_id = id;
    int x_id = tmp_id % x;
    tmp_id /= x;
    int size_id = tmp_id % size_per_head_split;
    tmp_id /= size_per_head_split;
    int head_id = tmp_id % head_num;

    int src_seq_id =
        (seq_id < memory_seq_len[batch_id])
            ? (seq_id + memory_max_seq_len - memory_seq_len[batch_id])
            : (seq_id - memory_seq_len[batch_id]);

    // key: [B, head_num, L, size_per_head / x, x] ->
    // [B, head_num, size_per_head / x, L, x]
    k_dst[batch_id * hidden_dim * cache_max_len +
          head_id * size_per_head * cache_max_len +
          size_id * cache_max_len * x + seq_id * x + x_id] =
        k_src[batch_id * hidden_dim * memory_max_seq_len +
              head_id * size_per_head * memory_max_seq_len +
              src_seq_id * size_per_head + size_id * x + x_id];

    // value: [B, head_num, L, size_per_head/x, x] ->
    // [B, head_num, L, size_per_head/x, x]
    v_dst[batch_id * hidden_dim * cache_max_len +
          head_id * size_per_head * cache_max_len + seq_id * size_per_head +
          size_id * x + x_id] =
        v_src[batch_id * hidden_dim * memory_max_seq_len +
              head_id * size_per_head * memory_max_seq_len +
              src_seq_id * size_per_head + size_id * x + x_id];
  }
}

template <typename T>
void transpose_cache_batch_major_kernelLauncher(T* k_dst,
                                                T* v_dst,
                                                const T* k_src,
                                                const T* v_src,
                                                const int* memory_seq_len,
                                                const int local_batch_size,
                                                const int memory_max_seq_len,
                                                const int cache_max_len,
                                                const int size_per_head,
                                                const int local_head_num,
                                                hipStream_t stream) {
  constexpr int block_sz = 128;
  dim3 grid(local_batch_size, memory_max_seq_len);

  transpose_cache_batch_major<<<grid, block_sz, 0, stream>>>(k_dst,
                                                             v_dst,
                                                             k_src,
                                                             v_src,
                                                             memory_seq_len,
                                                             local_head_num,
                                                             size_per_head,
                                                             memory_max_seq_len,
                                                             cache_max_len);
}

template <typename T>
void transpose_general_kernelLauncher(T* dst,
                                      T* src,
                                      const int batch_size,
                                      const int seq_len,
                                      const int head_num,
                                      const int size_per_head,
                                      hipStream_t stream) {
  dim3 grid, block;
  int grid_size = batch_size * head_num * seq_len;
  if (sizeof(T) == 2) {
    int seq_per_block = grid_size % 4 == 0 ? 4 : 1;
    grid.x = grid_size / seq_per_block;
    block.x = seq_per_block * size_per_head / 2;
    transpose<T><<<grid, block, 0, stream>>>(
        src, dst, batch_size, seq_len, head_num, size_per_head / 2);
  } else {
    const int seq_per_block = 1;
    grid.x = grid_size / seq_per_block;
    block.x = seq_per_block * size_per_head;
    transpose<T><<<grid, block, 0, stream>>>(
        src, dst, batch_size, seq_len, head_num, size_per_head);
  }
}

template void transpose_cache_batch_major_kernelLauncher(
    float* k_dst,
    float* v_dst,
    const float* k_src,
    const float* v_src,
    const int* memory_seq_len,
    const int local_batch_size,
    const int memory_max_seq_len,
    const int cache_max_len,
    const int size_per_head,
    const int local_head_num,
    hipStream_t stream);

template void transpose_cache_batch_major_kernelLauncher(
    half* k_dst,
    half* v_dst,
    const half* k_src,
    const half* v_src,
    const int* memory_seq_len,
    const int local_batch_size,
    const int memory_max_seq_len,
    const int cache_max_len,
    const int size_per_head,
    const int local_head_num,
    hipStream_t stream);

template void transpose_general_kernelLauncher(float* dst,
                                               float* src,
                                               const int batch_size,
                                               const int seq_len,
                                               const int head_num,
                                               const int size_per_head,
                                               hipStream_t stream);

template void transpose_general_kernelLauncher(half* dst,
                                               half* src,
                                               const int batch_size,
                                               const int seq_len,
                                               const int head_num,
                                               const int size_per_head,
                                               hipStream_t stream);
}
