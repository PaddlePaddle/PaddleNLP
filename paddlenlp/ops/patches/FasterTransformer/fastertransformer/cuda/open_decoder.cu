#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
 * Copyright (c) 2020-2021, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

namespace fastertransformer {

template <typename T>
__global__ void transpose_cache_batch_major(T* k_dst,
                                            T* v_dst,
                                            const T* k_src,
                                            const T* v_src,
                                            const int* memory_seq_len,
                                            const int head_num,
                                            const int size_per_head,
                                            const int memory_max_seq_len,
                                            const int cache_max_len) {
  const int hidden_dim = head_num * size_per_head;
  const int x = (sizeof(T) == 4) ? 4 : 8;
  const int size_per_head_split = size_per_head / x;
  const int batch_id = blockIdx.x;
  const int seq_id = blockIdx.y;

  for (int id = threadIdx.x; id < head_num * size_per_head_split * x;
       id += blockDim.x) {
    int tmp_id = id;
    int x_id = tmp_id % x;
    tmp_id /= x;
    int size_id = tmp_id % size_per_head_split;
    tmp_id /= size_per_head_split;
    int head_id = tmp_id % head_num;

    int src_seq_id =
        (seq_id < memory_seq_len[batch_id])
            ? (seq_id + memory_max_seq_len - memory_seq_len[batch_id])
            : (seq_id - memory_seq_len[batch_id]);

    // key: [B, head_num, L, size_per_head / x, x] ->
    // [B, head_num, size_per_head / x, L, x]
    k_dst[batch_id * hidden_dim * cache_max_len +
          head_id * size_per_head * cache_max_len +
          size_id * cache_max_len * x + seq_id * x + x_id] =
        k_src[batch_id * hidden_dim * memory_max_seq_len +
              head_id * size_per_head * memory_max_seq_len +
              src_seq_id * size_per_head + size_id * x + x_id];

    // value: [B, head_num, L, size_per_head/x, x] ->
    // [B, head_num, L, size_per_head/x, x]
    v_dst[batch_id * hidden_dim * cache_max_len +
          head_id * size_per_head * cache_max_len + seq_id * size_per_head +
          size_id * x + x_id] =
        v_src[batch_id * hidden_dim * memory_max_seq_len +
              head_id * size_per_head * memory_max_seq_len +
              src_seq_id * size_per_head + size_id * x + x_id];
  }
}

template <typename T>
void transpose_cache_batch_major_kernelLauncher(T* k_dst,
                                                T* v_dst,
                                                const T* k_src,
                                                const T* v_src,
                                                const int* memory_seq_len,
                                                const int local_batch_size,
                                                const int memory_max_seq_len,
                                                const int cache_max_len,
                                                const int size_per_head,
                                                const int local_head_num,
                                                hipStream_t stream) {
  constexpr int block_sz = 128;
  dim3 grid(local_batch_size, memory_max_seq_len);

  transpose_cache_batch_major<<<grid, block_sz, 0, stream>>>(k_dst,
                                                             v_dst,
                                                             k_src,
                                                             v_src,
                                                             memory_seq_len,
                                                             local_head_num,
                                                             size_per_head,
                                                             memory_max_seq_len,
                                                             cache_max_len);
}

template <typename T>
void transpose_general_kernelLauncher(T* dst,
                                      T* src,
                                      const int batch_size,
                                      const int seq_len,
                                      const int head_num,
                                      const int size_per_head,
                                      hipStream_t stream) {
  dim3 grid, block;
  int grid_size = batch_size * head_num * seq_len;
  if (sizeof(T) == 2) {
    int seq_per_block = grid_size % 4 == 0 ? 4 : 1;
    grid.x = grid_size / seq_per_block;
    block.x = seq_per_block * size_per_head / 2;
    transpose<T><<<grid, block, 0, stream>>>(
        src, dst, batch_size, seq_len, head_num, size_per_head / 2);
  } else {
    const int seq_per_block = 1;
    grid.x = grid_size / seq_per_block;
    block.x = seq_per_block * size_per_head;
    transpose<T><<<grid, block, 0, stream>>>(
        src, dst, batch_size, seq_len, head_num, size_per_head);
  }
}

template void transpose_cache_batch_major_kernelLauncher(
    float* k_dst,
    float* v_dst,
    const float* k_src,
    const float* v_src,
    const int* memory_seq_len,
    const int local_batch_size,
    const int memory_max_seq_len,
    const int cache_max_len,
    const int size_per_head,
    const int local_head_num,
    hipStream_t stream);

template void transpose_cache_batch_major_kernelLauncher(
    half* k_dst,
    half* v_dst,
    const half* k_src,
    const half* v_src,
    const int* memory_seq_len,
    const int local_batch_size,
    const int memory_max_seq_len,
    const int cache_max_len,
    const int size_per_head,
    const int local_head_num,
    hipStream_t stream);

template void transpose_general_kernelLauncher(float* dst,
                                               float* src,
                                               const int batch_size,
                                               const int seq_len,
                                               const int head_num,
                                               const int size_per_head,
                                               hipStream_t stream);

template void transpose_general_kernelLauncher(half* dst,
                                               half* src,
                                               const int batch_size,
                                               const int seq_len,
                                               const int head_num,
                                               const int size_per_head,
                                               hipStream_t stream);



template <typename T>
void fusedQKV_masked_attention_dispatch_v2(
  const T* qkv_buf, const T* qkv_bias,
  T* key_cache, T* value_cache,
  T* context_buf, const bool* finished, int max_batch_size, int inference_batch_size, 
  int head_num, int size_per_head, const int step, const int max_seq_len, 
  const int max_input_len, const int* input_lengths, const int rotary_embedding_dim, hipStream_t stream)
{
  using DataType = typename std::conditional<sizeof(T) == 4, float, uint16_t>::type;
  // Prepare the parameters.
  Masked_multihead_attention_params<DataType> params;
  memset(&params, 0, sizeof(params));
  int hidden_units = head_num * size_per_head;
  if (qkv_bias != nullptr) {
      params.q_bias = reinterpret_cast<const DataType*>(qkv_bias);
      params.k_bias = reinterpret_cast<const DataType*>(qkv_bias) + hidden_units;
      params.v_bias = reinterpret_cast<const DataType*>(qkv_bias) + 2 * hidden_units;
  }
  else {
     // gptj/codegen no bias
      params.q_bias = nullptr;
      params.k_bias = nullptr;
      params.v_bias = nullptr;
  }

  // Set the output buffer.
  params.out = reinterpret_cast<DataType *>(context_buf);

  // Set the input buffers.
  params.q = reinterpret_cast<const DataType *>(qkv_buf);
  params.k = reinterpret_cast<const DataType *>(qkv_buf) + hidden_units;
  params.v = reinterpret_cast<const DataType *>(qkv_buf) + 2 * hidden_units;
  params.stride = 3 * hidden_units;
  params.finished = const_cast<bool*>(finished);

  params.k_cache = reinterpret_cast<DataType *>(key_cache);
  params.v_cache = reinterpret_cast<DataType *>(value_cache);
  params.batch_size = inference_batch_size;
  params.seq_length = max_seq_len;
  params.timestep = step-1;
  params.num_heads = head_num;
  params.hidden_size_per_head = size_per_head;
  // GptJ: rotary_embedding
  params.rotary_embedding_dim = rotary_embedding_dim;
  params.inv_sqrt_dh = 1.F / sqrtf((float) params.hidden_size_per_head);

  params.is_mask = true;
  params.input_lengths = input_lengths;
  params.max_input_len = max_input_len;

  masked_multihead_attention(params, stream);
}

template void fusedQKV_masked_attention_dispatch_v2(
  const float* qkv_buf, 
  const float* qkv_bias,
  float* key_cache, 
  float* value_cache,
  float* context_buf, 
  const bool* finished, 
  int max_batch_size, 
  int inference_batch_size, 
  int head_num, 
  int size_per_head, 
  const int step, 
  const int max_seq_len,
  const int max_input_len, 
  const int* input_lengths,
  const int rotary_embedding_dim,
  hipStream_t stream);
  
template void fusedQKV_masked_attention_dispatch_v2(
  const half* qkv_buf, 
  const half* qkv_bias,
  half* key_cache, 
  half* value_cache,
  half* context_buf, 
  const bool* finished, 
  int max_batch_size, 
  int inference_batch_size, 
  int head_num, 
  int size_per_head,
  const int step, 
  const int max_seq_len,
  const int max_input_len, 
  const int* input_lengths,
  const int rotary_embedding_dim,
  hipStream_t stream);

}
